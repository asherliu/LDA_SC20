#include "WTDense.cuh"

WTD::WTD(int argNumOfWordD, int argWordLength) {
	numOfWordD = argNumOfWordD;
	wordLength = argWordLength;
	WTDenseLength = argNumOfWordD*K;

	/*WTDense = new int[WTDenseLength];
	WTDenseCopy = new int[WTDenseLength];*/

	//hipHostMalloc((void**)&WTDense, WTDenseLength * sizeof(int));
	//hipHostMalloc((void**)&WTDenseCopy, WTDenseLength * sizeof(int));

	//memset(WTDenseCopy, 0, WTDenseLength * sizeof(int));
	/*WTRowSumDense = new int[K];*/
}

void WTD::CPUMemSet() {

	memset(WTDense, 0, WTDenseLength * sizeof(int));
	memset(WTDenseCopy, 0, WTDenseLength * sizeof(int));
	//memset(WTRowSumDense, 0, K * sizeof(int));

}


void WTD::GPUMemAllocate() {

	hipMalloc((void**)&deviceWTDense, (WTDenseLength) * sizeof(int));
	hipMalloc((void**)&deviceWTDenseCopy, (WTDenseLength) * sizeof(int));
	//hipMalloc((void**)&deviceWTRowSumDense, (K) * sizeof(int));

	WTMemory = (2*WTDenseLength + K ) / 1000000000.0 * sizeof(int);
	printf("WT memory usage(Sparse Part):%f GB\n", WTMemory);
	
}

void WTD::GPUMemInit()
{
	hipMemset(deviceWTDense, 0, (WTDenseLength) * sizeof(int));
	hipMemset(deviceWTDenseCopy, 0, (WTDenseLength) * sizeof(int));
	//hipMemset(deviceWTRowSumDense, 0, (K) * sizeof(int));
}

void WTD::GPUMemCopy(hipStream_t& stream)
{
	hipMemcpyAsync(deviceWTDense, deviceWTDenseCopy, (WTDenseLength) * sizeof(int), hipMemcpyDeviceToDevice, stream);
	
}

void WTD::GPUMemset(hipStream_t& stream)
{
	hipMemsetAsync(deviceWTDenseCopy, 0, (WTDenseLength) * sizeof(int), stream);
	//hipMemcpyAsync(deviceWTDenseCopy, WTDenseCopy, (WTDenseLength) * sizeof(int), hipMemcpyHostToDevice,stream);
	//hipMemset(deviceWTRowSumDense, 0, (K) * sizeof(int));
}


void WTD::WTDenGPU2CPU()
{

	hipMemcpy(WTDense, deviceWTDense, (WTDenseLength) * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(WTDenseCopy, deviceWTDenseCopy, (WTDenseLength) * sizeof(int), hipMemcpyDeviceToHost);
	

}
void WTD::WTDenCPU2Disk(string argFilePrefix) {

	ofstream WTDen((argFilePrefix + string("/WTDen.txt")).c_str(), ios::binary);
	for (int i = 0; i < WTDenseLength; i++) {
		WTDen << WTDense[i] << "\n";
	}
	WTDen.close();
	ofstream WTDenCopy((argFilePrefix + string("/WTDenCopy.txt")).c_str(), ios::binary);
	for (int i = 0; i < WTDenseLength; i++) {
		WTDenCopy << WTDenseCopy[i] << "\n";
	}
	WTDen.close();

}
