#include "hip/hip_runtime.h"

#include "WT.cuh"
WTAll::WTAll(int argmaxWTLength, int argWordLength, int argNumChunks, int argMaxChunkWTLength, int argNumOfWordS) {
	maxWTLength = argmaxWTLength;
	wordLength = argWordLength;
	numChunks = argNumChunks;
	maxChunkWTLength = argMaxChunkWTLength;
	numOfWordS = argNumOfWordS;
	/*WTLengthVec = new int[numChunks];
	WTRowSum = new int[K];

	NZWTCount = new int[numOfWordS];
	WTIndex = new unsigned short int[maxWTLength];
	WTValue = new unsigned short int[maxWTLength];
	WTCount = new int[wordLength];
	WTOffset = new int[wordLength];*/
	
	
	hipHostMalloc((void**)&WTLengthVec, numChunks * sizeof(int));
	hipHostMalloc((void**)&WTRowSum, K * sizeof(int));
	hipHostMalloc((void**)&NZWTCount, numOfWordS * sizeof(int));
	hipHostMalloc((void**)&WTIndex, maxWTLength * sizeof(unsigned short int));
	hipHostMalloc((void**)&WTValue, maxWTLength * sizeof(unsigned short int));
	hipHostMalloc((void**)&WTCount, wordLength * sizeof(int));
	hipHostMalloc((void**)&WTOffset, wordLength * sizeof(int));


	//////-----chunkWT-----for test--------
	//chunkNZWTCount = new int[numOfWordS];
	//chunkWTIndex = new int[maxChunkWTLength];
	//chunkWTValue = new int[maxChunkWTLength];
	//////-----chunkWT-----for test--------
	hipHostMalloc((void**)&chunkNZWTCount, numOfWordS * sizeof(int));
	hipHostMalloc((void**)&chunkWTIndex, maxChunkWTLength * sizeof(int));
	hipHostMalloc((void**)&chunkWTValue, maxChunkWTLength * sizeof(int));

	memset(chunkNZWTCount, 0, numOfWordS * sizeof(int));
	memset(chunkWTIndex, 0, maxChunkWTLength * sizeof(unsigned short int));
	memset(chunkWTValue, 0, maxChunkWTLength * sizeof(unsigned short int));

	memset(NZWTCount, 0, numOfWordS * sizeof(int));
	memset(WTIndex, 0, maxWTLength * sizeof(unsigned short int));
	memset(WTValue, 0, maxWTLength * sizeof(unsigned short int));
	memset(WTRowSum, 0, K * sizeof(int));
}




void WTAll::CPUMemSet() {

	memset(NZWTCount, 0, numOfWordS * sizeof(int));
	memset(WTIndex, 0, maxWTLength * sizeof(unsigned short int));
	memset(WTValue, 0, maxWTLength * sizeof(unsigned short int));
	memset(WTCount, 0, wordLength * sizeof(int));
	memset(WTOffset, 0, wordLength * sizeof(int));
	memset(WTRowSum, 0, K * sizeof(int));

}

void WTAll::GPUMemAllocate() {

	hipMalloc((void**)&deviceNZWTCount, (numOfWordS) * sizeof(int));
	hipMalloc((void**)&deviceWTIndex, (maxWTLength) * sizeof(unsigned short int));
	hipMalloc((void**)&deviceWTValue, (maxWTLength) * sizeof(unsigned short int));
	hipMalloc((void**)&deviceWTCount, (wordLength) * sizeof(int));
	hipMalloc((void**)&deviceWTOffset, (wordLength) * sizeof(int));

	hipMalloc((void**)&deviceWTRowSum, (K) * sizeof(int));
	hipMalloc((void**)&deviceBlockCount, (1) * sizeof(int));
	hipMalloc((void**)&deviceWarpCount, (1) * sizeof(int));

	for (int i = 0; i < numStreams; i++) {
		hipMalloc((void**)&deviceChunkWTCount[i], (numOfWordS) * sizeof(int));
		hipMalloc((void**)&deviceChunkWTOffset[i], (numOfWordS) * sizeof(int));
		hipMalloc((void**)&deviceChunkNZWTCount[i], (numOfWordS) * sizeof(int));
		hipMalloc((void**)&deviceChunkWTIndex[i], (maxChunkWTLength) * sizeof(unsigned short int));
		hipMalloc((void**)&deviceChunkWTValue[i], (maxChunkWTLength) * sizeof(unsigned short int));
	}
	


	WTMemory = (6 * wordLength + 2 * maxWTLength + K + 2 * maxChunkWTLength) /1000000000.0 * sizeof(int);
	printf("WT memory usage(Sparse):%f GB\n", WTMemory);
	WTMemory = K /1000000000.0 * wordLength * sizeof(int);
	printf("WT memory usage(Dense):%f GB\n", WTMemory);

}
void WTAll::GPUMemset(hipStream_t& stream)
{
	hipMemsetAsync(deviceNZWTCount, 0, (numOfWordS) * sizeof(int), stream);
	hipMemsetAsync(deviceWTIndex, 0, (maxWTLength) * sizeof(unsigned short int), stream);
	hipMemsetAsync(deviceWTValue, 0, (maxWTLength) * sizeof(unsigned short int), stream);
	hipMemsetAsync(deviceWTRowSum, 0, (K) * sizeof(int), stream);

//	hipMemcpyAsync(deviceNZWTCount, NZWTCount, (numOfWordS) * sizeof(int), hipMemcpyHostToDevice, stream);
//
//	hipMemcpyAsync(deviceWTIndex, WTIndex, (maxWTLength) * sizeof(unsigned short int), hipMemcpyHostToDevice, stream);
//	hipMemcpyAsync(deviceWTValue, WTValue, (maxWTLength) * sizeof(unsigned short int), hipMemcpyHostToDevice, stream);
//	hipMemcpyAsync(deviceWTRowSum, WTRowSum, (K) * sizeof(int), hipMemcpyHostToDevice, stream);




}

void WTAll::chunkGPUMemset(int argStreamId, hipStream_t& stream)
{
	hipMemsetAsync(deviceChunkNZWTCount[argStreamId], 0, (numOfWordS) * sizeof(int), stream);
	hipMemsetAsync(deviceChunkWTIndex[argStreamId], 0, (maxChunkWTLength) * sizeof(unsigned short int), stream);
	hipMemsetAsync(deviceChunkWTValue[argStreamId], 0, (maxChunkWTLength) * sizeof(unsigned short int), stream);
//	hipMemcpyAsync(deviceChunkNZWTCount[argStreamId], chunkNZWTCount, (numOfWordS) * sizeof(int), hipMemcpyHostToDevice,stream);
//	hipMemcpyAsync(deviceChunkWTIndex[argStreamId], chunkWTIndex, (maxChunkWTLength) * sizeof(unsigned short int), hipMemcpyHostToDevice, stream);
//	hipMemcpyAsync(deviceChunkWTValue[argStreamId], chunkWTValue, (maxChunkWTLength) * sizeof(unsigned short int), hipMemcpyHostToDevice,stream);



	//hipMemset(deviceWTRowSum, 0, (K) * sizeof(int));

}


void WTAll::loadWTLength(string argFilePrefix) {

	ifstream WTLength((argFilePrefix + string("/WTLength.txt")).c_str(), ios::binary);//store max Doc and DT length	
	for (int chunkId = 0; chunkId < numChunks; chunkId++) {
		WTLength >> WTLengthVec[chunkId];

	}
	WTLength.close();
}

void WTAll::loadWTCountOffset(string argFilePrefix) {

	//--------load chunkWTCountOffset--------------
	for (int chunkId = 0; chunkId < numChunks; chunkId++) {

		WTChunkData chunkWTData(chunkId, wordLength, maxChunkWTLength, WTLengthVec[chunkId], numOfWordS);
		chunkWTData.CPUMemSet();
		chunkWTData.loadWTCountOffset(argFilePrefix);
		WTChunkVec.push_back(chunkWTData);
	}
	//--------load chunkWTCountOffset--------------



	//--------load WTCountOffset--------------

	ifstream WTCountOffset((argFilePrefix + string("/WTCountOffset.txt")).c_str(), ios::binary);//store Word offset of TL
	blockCount = 0;
	for (int i = 0; i < wordLength; i++)
	{
		WTCountOffset >> WTCount[i] >> WTOffset[i];

		if (i >= wordLength - numOfWordS) {
			if (WTCount[i] > 32) {
				blockCount++;
			}
		}
		
	}
	WTCountOffset.close();
	warpCount = numOfWordS - blockCount;
	printf("WT Count and Offset loaded!...\n");

	//--------load WTCountOffset--------------

}


void WTAll::blockWarpCountCPU2GPU() {

	hipMemcpy(deviceBlockCount, &blockCount, (1) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(deviceWarpCount, &warpCount, (1) * sizeof(int), hipMemcpyHostToDevice);

}

void WTAll::CPU2GPUCountOffset(hipStream_t& stream) {

	hipMemcpyAsync(deviceWTCount, WTCount, (wordLength) * sizeof(int), hipMemcpyHostToDevice, stream);
	hipMemcpyAsync(deviceWTOffset, WTOffset, (wordLength) * sizeof(int), hipMemcpyHostToDevice, stream);

}

void WTAll::WTCPU2GPU(hipStream_t& stream) {

	hipMemcpyAsync(deviceNZWTCount, NZWTCount, (numOfWordS) * sizeof(int), hipMemcpyHostToDevice, stream);
	hipMemcpyAsync(deviceWTIndex, WTIndex, (maxWTLength) * sizeof(unsigned short int), hipMemcpyHostToDevice, stream);
	hipMemcpyAsync(deviceWTValue, WTValue, (maxWTLength) * sizeof(unsigned short int), hipMemcpyHostToDevice, stream);
	
}

void WTAll::WTGPU2CPU() {

	hipMemcpy(NZWTCount, deviceNZWTCount, (numOfWordS) * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(WTIndex, deviceWTIndex, (maxWTLength) * sizeof(unsigned short int), hipMemcpyDeviceToHost);
	hipMemcpy(WTValue, deviceWTValue, (maxWTLength) * sizeof(unsigned short int), hipMemcpyDeviceToHost);
	hipMemcpy(WTRowSum, deviceWTRowSum, (K) * sizeof(int), hipMemcpyDeviceToHost);
}


void WTAll::chunkCPU2GPUCountOffset(int argChunkId, int argStreamId, hipStream_t& stream) {

	/*int chunkId = argChunkId;*/
	
	hipMemcpyAsync(deviceChunkWTCount[argStreamId], WTChunkVec[argChunkId].WTCount, (numOfWordS) * sizeof(int), hipMemcpyHostToDevice, stream);
	hipMemcpyAsync(deviceChunkWTOffset[argStreamId], WTChunkVec[argChunkId].WTOffset, (numOfWordS) * sizeof(int), hipMemcpyHostToDevice, stream);

}

void WTAll::chunkWTCPU2GPU(int argChunkId, int argStreamId, hipStream_t& stream) {

	/*int chunkId = argChunkId;*/

	hipMemcpyAsync(deviceChunkNZWTCount[argStreamId], WTChunkVec[argChunkId].NZWTCount, (numOfWordS) * sizeof(int), hipMemcpyHostToDevice,stream);
	hipMemcpyAsync(deviceChunkWTIndex[argStreamId], WTChunkVec[argChunkId].WTIndex, (WTLengthVec[argChunkId]) * sizeof(unsigned short int), hipMemcpyHostToDevice,stream);
	hipMemcpyAsync(deviceChunkWTValue[argStreamId], WTChunkVec[argChunkId].WTValue, (WTLengthVec[argChunkId]) * sizeof(unsigned short int), hipMemcpyHostToDevice,stream);

}

void WTAll::chunkWTGPU2CPU(int argChunkId, int argStreamId, hipStream_t& stream) {

	/*int chunkId = argChunkId;*/

	hipMemcpyAsync(WTChunkVec[argChunkId].NZWTCount, deviceChunkNZWTCount[argStreamId], (numOfWordS) * sizeof(int), hipMemcpyDeviceToHost, stream);
	hipMemcpyAsync(WTChunkVec[argChunkId].WTIndex, deviceChunkWTIndex[argStreamId], (WTLengthVec[argChunkId]) * sizeof(unsigned short int), hipMemcpyDeviceToHost, stream);
	hipMemcpyAsync(WTChunkVec[argChunkId].WTValue, deviceChunkWTValue[argStreamId], (WTLengthVec[argChunkId]) * sizeof(unsigned short int), hipMemcpyDeviceToHost, stream);

}


void WTAll::CPU2Disk(string argFilePrefix) {

	ofstream OutputNZWTCount((argFilePrefix + string("/NZWTCount.txt")).c_str(), ios::binary);
	for (int i = 0; i < numOfWordS; i++) {
		OutputNZWTCount << NZWTCount[i] << "\n";
	}
	OutputNZWTCount.close();
	ofstream OutputWTIndexValue((argFilePrefix + string("/WTIndexValue.txt")).c_str(), ios::binary);
	for (int i = 0; i < maxWTLength; i++) {
		OutputWTIndexValue << WTIndex[i] << " " << WTValue[i] << "\n";
	}
	OutputWTIndexValue.close();

	ofstream OutputWTRowSum((argFilePrefix + string("/WTRowSum.txt")).c_str(), ios::binary);

	for (int i = 0; i < K; i++) {
		OutputWTRowSum << WTRowSum[i]<< "\n";
	}
	OutputWTRowSum.close();

}


void WTAll::CPU2DiskChunk(string argFilePrefix, int argChunkId) {

	int chunkId = argChunkId;
	string chunkFolderName = argFilePrefix + "/chunk" + to_string(chunkId);

	ofstream OutputNZWTCount((chunkFolderName + string("/NZWTCount.txt")).c_str(), ios::binary);
	for (int i = 0; i < numOfWordS; i++) {
		OutputNZWTCount << WTChunkVec[chunkId].NZWTCount[i] << "\n";
	}
	OutputNZWTCount.close();
	ofstream OutputWTIndexValue((chunkFolderName + string("/WTIndexValue.txt")).c_str(), ios::binary);
	for (int i = 0; i < WTLengthVec[chunkId]; i++) {
		OutputWTIndexValue << WTChunkVec[chunkId].WTIndex[i] << " " << WTChunkVec[chunkId].WTValue[i] << "\n";
	}
	OutputWTIndexValue.close();
}







