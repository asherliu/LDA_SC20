#include "hip/hip_runtime.h"

#include "WT.cuh"
WTAll::WTAll(int argmaxWTLength, int argWordLength, int argNumChunks, int argMaxChunkWTLength, int argNumOfWordS, int argNumGPUS) {
	maxWTLength = argmaxWTLength;
	wordLength = argWordLength;
	numChunks = argNumChunks;
	maxChunkWTLength = argMaxChunkWTLength;
	numOfWordS = argNumOfWordS;
	numGPUs = argNumGPUS;
	WTLengthVec = new int[numChunks];
	WTRowSum = new int[K];

	NZWTCount = new int[numOfWordS];
	WTIndex = new int[maxWTLength];
	WTValue = new int[maxWTLength];
	WTCount = new int[wordLength];
	WTOffset = new int[wordLength];
	
	


	tmpChunkNZWTCount = new int[numOfWordS];
	tmpChunkWTIndex = new int[maxChunkWTLength];
	tmpChunkWTValue = new int[maxChunkWTLength];
	tmpChunkWTCount = new int[numOfWordS];
	tmpChunkWTOffset = new int[numOfWordS];










	////-----chunkWT-----for test--------
	//chunkNZWTCount = new int[wordLength];
	//chunkWTIndex = new int[maxChunkWTLength];
	//chunkWTValue = new int[maxChunkWTLength];
	////-----chunkWT-----for test--------


}




void WTAll::CPUMemSet() {

	memset(NZWTCount, 0, numOfWordS * sizeof(int));
	memset(WTIndex, 0, maxWTLength * sizeof(int));
	memset(WTValue, 0, maxWTLength * sizeof(int));
	memset(WTCount, 0, wordLength * sizeof(int));
	memset(WTOffset, 0, wordLength * sizeof(int));
	memset(WTRowSum, 0, K * sizeof(int));

}
//void WTAll::GPUMemAllocate(int argGPUId) {
//	
//	GPUId = argGPUId;
//	hipSetDevice(GPUId);
//	hipMalloc((void**)&deviceNZWTCount, (numOfWordS) * sizeof(int));
//	hipMalloc((void**)&deviceWTIndex, (maxWTLength) * sizeof(int));
//	hipMalloc((void**)&deviceWTValue, (maxWTLength) * sizeof(int));
//	hipMalloc((void**)&deviceWTCount, (wordLength) * sizeof(int));
//	hipMalloc((void**)&deviceWTOffset, (wordLength) * sizeof(int));
//
//	hipMalloc((void**)&deviceWTRowSum, (K) * sizeof(int));
//	hipMalloc((void**)&deviceBlockCount, (1) * sizeof(int));
//	hipMalloc((void**)&deviceWarpCount, (1) * sizeof(int));
//
//	hipMalloc((void**)&deviceChunkWTCount, (numOfWordS) * sizeof(int));
//	hipMalloc((void**)&deviceChunkWTOffset, (numOfWordS) * sizeof(int));
//	hipMalloc((void**)&deviceChunkNZWTCount, (numOfWordS) * sizeof(int));
//	hipMalloc((void**)&deviceChunkWTIndex, (maxChunkWTLength) * sizeof(int));
//	hipMalloc((void**)&deviceChunkWTValue, (maxChunkWTLength) * sizeof(int));
//
//
//	WTMemory = (6 * wordLength + 2 * maxWTLength + K + 2 * maxChunkWTLength) /1000000000.0 * sizeof(int);
//	printf("WT memory usage(Sparse):%f GB\n", WTMemory);
//	WTMemory = K /1000000000.0 * wordLength * sizeof(int);
//	printf("WT memory usage(Dense):%f GB\n", WTMemory);
//
//}
//void WTAll::GPUMemset()
//{
//	hipSetDevice(GPUId);
//	hipMemset(deviceNZWTCount, 0, (numOfWordS) * sizeof(int));
//	hipMemset(deviceWTIndex, 0, (maxWTLength) * sizeof(int));
//	hipMemset(deviceWTValue, 0, (maxWTLength) * sizeof(int));
//	hipMemset(deviceWTRowSum, 0, (K) * sizeof(int));
//
//}
//
//void WTAll::chunkGPUMemset()
//{
//	/*hipSetDevice(GPUId);*/
//	hipMemset(deviceChunkNZWTCount, 0, (numOfWordS) * sizeof(int));
//	hipMemset(deviceChunkWTIndex, 0, (maxChunkWTLength) * sizeof(int));
//	hipMemset(deviceChunkWTValue, 0, (maxChunkWTLength) * sizeof(int));
//	//hipMemset(deviceWTRowSum, 0, (K) * sizeof(int));
//
//}


void WTAll::loadWTLength(string argFilePrefix) {

	ifstream WTLength((argFilePrefix + string("/WTLength.txt")).c_str(), ios::binary);//store max Doc and DT length	
	for (int chunkId = 0; chunkId < numChunks; chunkId++) {
		WTLength >> WTLengthVec[chunkId];

	}
	WTLength.close();
}

void WTAll::loadWTCountOffset(string argFilePrefix) {

	//--------load chunkWTCountOffset--------------
	for (int chunkId = 0; chunkId < numChunks; chunkId++) {

		WTChunkData chunkWTData(chunkId, wordLength, maxChunkWTLength, WTLengthVec[chunkId], numOfWordS);
		chunkWTData.CPUMemSet();
		chunkWTData.loadWTCountOffset(argFilePrefix);
		WTChunkVec.push_back(chunkWTData);
	}
	//--------load chunkWTCountOffset--------------



	//--------load WTCountOffset--------------

	ifstream WTCountOffset((argFilePrefix + string("/WTCountOffset.txt")).c_str(), ios::binary);//store Word offset of TL
	blockCount = 0;
	for (int i = 0; i < wordLength; i++)
	{
		WTCountOffset >> WTCount[i] >> WTOffset[i];

		if (i >= wordLength - numOfWordS) {
			if (WTCount[i] > 32) {
				blockCount++;
			}
		}
		
	}
	WTCountOffset.close();
	warpCount = numOfWordS - blockCount;
	printf("WT Count and Offset loaded!...\n");

	//--------load WTCountOffset--------------

}


void WTAll::InitWTGPU()
{
	for (int GPUId = 0; GPUId < numGPUs; GPUId++) {

		WTGPUChunkData GPUChunkWTData(GPUId, wordLength, maxChunkWTLength, WTLengthVec[GPUId], numOfWordS);
		GPUChunkWTData.GPUMemAllocate(GPUId);
		GPUChunkWTData.GPUMemset(GPUId);
		GPUChunkWTData.chunkGPUMemset(GPUId);
		WTGPUChunkVec.push_back(GPUChunkWTData);
	}

}

void WTAll::GPUMemAllocate() {

	hipSetDevice(0);
	hipMalloc((void**)&deviceZeroWTRowSum, (K) * sizeof(int));
	hipMalloc((void**)&deviceZeroChunkWTCount, (numOfWordS) * sizeof(int));
	hipMalloc((void**)&deviceZeroChunkWTOffset, (numOfWordS) * sizeof(int));
	hipMalloc((void**)&deviceZeroChunkNZWTCount, (numOfWordS) * sizeof(int));
	hipMalloc((void**)&deviceZeroChunkWTIndex, (maxChunkWTLength) * sizeof(int));
	hipMalloc((void**)&deviceZeroChunkWTValue, (maxChunkWTLength) * sizeof(int));

}





void WTAll::GPUDataTransfer(int argGPUId, hipStream_t &stream) {
	hipSetDevice(0);

	hipMemcpy(deviceZeroChunkWTCount, WTGPUChunkVec[argGPUId].deviceChunkWTCount, (numOfWordS) * sizeof(int), hipMemcpyDeviceToDevice);
	hipMemcpy(deviceZeroChunkWTOffset, WTGPUChunkVec[argGPUId].deviceChunkWTOffset, (numOfWordS) * sizeof(int), hipMemcpyDeviceToDevice);
	hipMemcpy(deviceZeroChunkNZWTCount, WTGPUChunkVec[argGPUId].deviceChunkNZWTCount, (numOfWordS) * sizeof(int), hipMemcpyDeviceToDevice);
	hipMemcpy(deviceZeroChunkWTIndex, WTGPUChunkVec[argGPUId].deviceChunkWTIndex, (maxChunkWTLength) * sizeof(int), hipMemcpyDeviceToDevice);
	hipMemcpy(deviceZeroChunkWTValue, WTGPUChunkVec[argGPUId].deviceChunkWTValue, (maxChunkWTLength) * sizeof(int), hipMemcpyDeviceToDevice);

	/*hipMemcpyAsync(deviceZeroChunkWTCount, WTGPUChunkVec[argGPUId].deviceChunkWTCount, (numOfWordS) * sizeof(int), hipMemcpyDeviceToDevice, stream);
	hipMemcpyAsync(deviceZeroChunkWTOffset, WTGPUChunkVec[argGPUId].deviceChunkWTOffset, (numOfWordS) * sizeof(int), hipMemcpyDeviceToDevice, stream);
	hipMemcpyAsync(deviceZeroChunkNZWTCount, WTGPUChunkVec[argGPUId].deviceChunkNZWTCount, (numOfWordS) * sizeof(int), hipMemcpyDeviceToDevice, stream);
	hipMemcpyAsync(deviceZeroChunkWTIndex, WTGPUChunkVec[argGPUId].deviceChunkWTIndex, (maxChunkWTLength) * sizeof(int), hipMemcpyDeviceToDevice, stream);
	hipMemcpyAsync(deviceZeroChunkWTValue, WTGPUChunkVec[argGPUId].deviceChunkWTValue, (maxChunkWTLength) * sizeof(int), hipMemcpyDeviceToDevice, stream);*/


}

void WTAll::GPUDataDistribute(int argGPUId, hipStream_t &stream)
{
	hipSetDevice(0);
	/*hipMemcpyAsync(WTGPUChunkVec[argGPUId].deviceWTRowSum, WTGPUChunkVec[0].deviceWTRowSum,(K) * sizeof(int), hipMemcpyDeviceToDevice, stream);
	hipMemcpyAsync(WTGPUChunkVec[argGPUId].deviceNZWTCount, WTGPUChunkVec[0].deviceNZWTCount, (numOfWordS) * sizeof(int), hipMemcpyDeviceToDevice, stream);
	hipMemcpyAsync(WTGPUChunkVec[argGPUId].deviceWTIndex, WTGPUChunkVec[0].deviceWTIndex, (maxWTLength) * sizeof(int), hipMemcpyDeviceToDevice, stream);
	hipMemcpyAsync(WTGPUChunkVec[argGPUId].deviceWTValue, WTGPUChunkVec[0].deviceWTValue, (maxWTLength) * sizeof(int), hipMemcpyDeviceToDevice, stream);*/

	hipMemcpy(WTGPUChunkVec[argGPUId].deviceWTRowSum, WTGPUChunkVec[0].deviceWTRowSum, (K) * sizeof(int), hipMemcpyDeviceToDevice);
	hipMemcpy(WTGPUChunkVec[argGPUId].deviceNZWTCount, WTGPUChunkVec[0].deviceNZWTCount, (numOfWordS) * sizeof(int), hipMemcpyDeviceToDevice);
	hipMemcpy(WTGPUChunkVec[argGPUId].deviceWTIndex, WTGPUChunkVec[0].deviceWTIndex, (maxWTLength) * sizeof(int), hipMemcpyDeviceToDevice);
	hipMemcpy(WTGPUChunkVec[argGPUId].deviceWTValue, WTGPUChunkVec[0].deviceWTValue, (maxWTLength) * sizeof(int), hipMemcpyDeviceToDevice);

}






//void WTAll::blockWarpCountCPU2GPU() {
//	hipSetDevice(GPUId);
//	hipMemcpy(deviceBlockCount, &blockCount, (1) * sizeof(int), hipMemcpyHostToDevice);
//	hipMemcpy(deviceWarpCount, &warpCount, (1) * sizeof(int), hipMemcpyHostToDevice);
//
//}

void WTAll::CPU2GPUCountOffset(int argGPUId) {

	hipSetDevice(argGPUId);
	hipMemcpy(WTGPUChunkVec[argGPUId].deviceWTCount, WTCount, (wordLength) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(WTGPUChunkVec[argGPUId].deviceWTOffset, WTOffset, (wordLength) * sizeof(int), hipMemcpyHostToDevice);

}

void WTAll::WTCPU2GPU(int argGPUId) {
	hipSetDevice(argGPUId);
	hipMemcpy(WTGPUChunkVec[argGPUId].deviceNZWTCount, NZWTCount, (numOfWordS) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(WTGPUChunkVec[argGPUId].deviceWTIndex, WTIndex, (maxWTLength) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(WTGPUChunkVec[argGPUId].deviceWTValue, WTValue, (maxWTLength) * sizeof(int), hipMemcpyHostToDevice);
	
}

void WTAll::WTGPU2CPU(int argGPUId) {
	hipSetDevice(argGPUId);
	hipMemcpy(NZWTCount, WTGPUChunkVec[argGPUId].deviceNZWTCount, (numOfWordS) * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(WTIndex, WTGPUChunkVec[argGPUId].deviceWTIndex, (maxWTLength) * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(WTValue, WTGPUChunkVec[argGPUId].deviceWTValue, (maxWTLength) * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(WTRowSum, WTGPUChunkVec[argGPUId].deviceWTRowSum, (K) * sizeof(int), hipMemcpyDeviceToHost);
}


void WTAll::chunkCPU2GPUCountOffset(int argGPUId) {

	/*int chunkId = argChunkId;*/
	hipSetDevice(argGPUId);
	hipMemcpy(WTGPUChunkVec[argGPUId].deviceChunkWTCount, WTChunkVec[argGPUId].WTCount, (numOfWordS) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(WTGPUChunkVec[argGPUId].deviceChunkWTOffset, WTChunkVec[argGPUId].WTOffset, (numOfWordS) * sizeof(int), hipMemcpyHostToDevice);

}

void WTAll::GPUDataTransferBackCPU(int argGPUId) {

	/*int chunkId = argChunkId;*/
	hipSetDevice(argGPUId);
	
	hipMemcpy(tmpChunkWTCount, WTGPUChunkVec[argGPUId].deviceChunkWTCount, (numOfWordS) * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(tmpChunkWTOffset, WTGPUChunkVec[argGPUId].deviceChunkWTOffset, (numOfWordS) * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(tmpChunkNZWTCount, WTGPUChunkVec[argGPUId].deviceChunkNZWTCount, (numOfWordS) * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(tmpChunkWTIndex, WTGPUChunkVec[argGPUId].deviceChunkWTIndex, (maxChunkWTLength) * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(tmpChunkWTValue, WTGPUChunkVec[argGPUId].deviceChunkWTValue, (maxChunkWTLength) * sizeof(int), hipMemcpyDeviceToHost);

}


void WTAll::GPUDataTransferToGPU(int argGPUId) {

	hipSetDevice(0);

	hipMemcpy(deviceZeroChunkWTCount, tmpChunkWTCount,(numOfWordS) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(deviceZeroChunkWTOffset, tmpChunkWTOffset, (numOfWordS) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(deviceZeroChunkNZWTCount, tmpChunkNZWTCount,  (numOfWordS) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(deviceZeroChunkWTIndex, tmpChunkWTIndex, (maxChunkWTLength) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(deviceZeroChunkWTValue, tmpChunkWTValue, (maxChunkWTLength) * sizeof(int), hipMemcpyHostToDevice);

}


void WTAll::GPUDataDistributeBackCPU(int argGPUId, hipStream_t &stream)
{
	hipSetDevice(0);
	/*hipMemcpyAsync(WTGPUChunkVec[argGPUId].deviceWTRowSum, WTGPUChunkVec[0].deviceWTRowSum,(K) * sizeof(int), hipMemcpyDeviceToDevice, stream);
	hipMemcpyAsync(WTGPUChunkVec[argGPUId].deviceNZWTCount, WTGPUChunkVec[0].deviceNZWTCount, (numOfWordS) * sizeof(int), hipMemcpyDeviceToDevice, stream);
	hipMemcpyAsync(WTGPUChunkVec[argGPUId].deviceWTIndex, WTGPUChunkVec[0].deviceWTIndex, (maxWTLength) * sizeof(int), hipMemcpyDeviceToDevice, stream);
	hipMemcpyAsync(WTGPUChunkVec[argGPUId].deviceWTValue, WTGPUChunkVec[0].deviceWTValue, (maxWTLength) * sizeof(int), hipMemcpyDeviceToDevice, stream);*/

	hipMemcpy(WTRowSum, WTGPUChunkVec[0].deviceWTRowSum, (K) * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(NZWTCount, WTGPUChunkVec[0].deviceNZWTCount, (numOfWordS) * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(WTIndex, WTGPUChunkVec[0].deviceWTIndex, (maxWTLength) * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(WTValue, WTGPUChunkVec[0].deviceWTValue, (maxWTLength) * sizeof(int), hipMemcpyDeviceToHost);

}


void WTAll::GPUDataDistributeToGPU(int argGPUId, hipStream_t &stream) {


	hipSetDevice(argGPUId);


	hipMemcpy(WTGPUChunkVec[argGPUId].deviceWTRowSum, WTRowSum,(K) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(WTGPUChunkVec[argGPUId].deviceNZWTCount, NZWTCount, (numOfWordS) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(WTGPUChunkVec[argGPUId].deviceWTIndex, WTIndex, (maxWTLength) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(WTGPUChunkVec[argGPUId].deviceWTValue, WTValue,(maxWTLength) * sizeof(int), hipMemcpyHostToDevice);



}



void WTAll::chunkWTCPU2GPU(int argGPUId) {

	/*int chunkId = argChunkId;*/
	hipSetDevice(argGPUId);
	hipMemcpy(WTGPUChunkVec[argGPUId].deviceChunkNZWTCount, WTChunkVec[argGPUId].NZWTCount, (numOfWordS) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(WTGPUChunkVec[argGPUId].deviceChunkWTIndex, WTChunkVec[argGPUId].WTIndex, (WTLengthVec[argGPUId]) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(WTGPUChunkVec[argGPUId].deviceChunkWTValue, WTChunkVec[argGPUId].WTValue, (WTLengthVec[argGPUId]) * sizeof(int), hipMemcpyHostToDevice);

}

void WTAll::verifyWTSum() {

	hipSetDevice(0);

	hipMemcpy(WTRowSum, WTGPUChunkVec[0].deviceWTRowSum, (K) * sizeof(int), hipMemcpyDeviceToHost);
	int sum = 0;
	for (int i = 0; i < K; i++) {
		sum += WTRowSum[i];

	}
	printf("\nRow sum:%d\n", sum);



}







void WTAll::chunkWTGPU2CPU(int argGPUId) {

	/*int chunkId = argChunkId;*/
	hipSetDevice(argGPUId);
	hipMemcpy(WTChunkVec[argGPUId].NZWTCount, WTGPUChunkVec[argGPUId].deviceChunkNZWTCount, (numOfWordS) * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(WTChunkVec[argGPUId].WTIndex, WTGPUChunkVec[argGPUId].deviceChunkWTIndex, (WTLengthVec[argGPUId]) * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(WTChunkVec[argGPUId].WTValue, WTGPUChunkVec[argGPUId].deviceChunkWTValue, (WTLengthVec[argGPUId]) * sizeof(int), hipMemcpyDeviceToHost);

}


void WTAll::CPU2Disk(string argFilePrefix) {

	ofstream OutputNZWTCount((argFilePrefix + string("/NZWTCount.txt")).c_str(), ios::binary);
	for (int i = 0; i < numOfWordS; i++) {
		OutputNZWTCount << NZWTCount[i] << "\n";
	}
	OutputNZWTCount.close();
	ofstream OutputWTIndexValue((argFilePrefix + string("/WTIndexValue.txt")).c_str(), ios::binary);
	for (int i = 0; i < maxWTLength; i++) {
		OutputWTIndexValue << WTIndex[i] << " " << WTValue[i] << "\n";
	}
	OutputWTIndexValue.close();

	ofstream OutputWTRowSum((argFilePrefix + string("/WTRowSum.txt")).c_str(), ios::binary);

	for (int i = 0; i < K; i++) {
		OutputWTRowSum << WTRowSum[i]<< "\n";
	}
	OutputWTRowSum.close();

}


void WTAll::CPU2DiskChunk(string argFilePrefix, int argChunkId) {

	int chunkId = argChunkId;
	string chunkFolderName = argFilePrefix + "/chunk" + to_string(chunkId);

	ofstream OutputNZWTCount((chunkFolderName + string("/NZWTCount.txt")).c_str(), ios::binary);
	for (int i = 0; i < numOfWordS; i++) {
		OutputNZWTCount << WTChunkVec[chunkId].NZWTCount[i] << "\n";
	}
	OutputNZWTCount.close();
	ofstream OutputWTIndexValue((chunkFolderName + string("/WTIndexValue.txt")).c_str(), ios::binary);
	for (int i = 0; i < WTLengthVec[chunkId]; i++) {
		OutputWTIndexValue << WTChunkVec[chunkId].WTIndex[i] << " " << WTChunkVec[chunkId].WTValue[i] << "\n";
	}
	OutputWTIndexValue.close();
}







