
#include "DTUpdateKernel.cuh"
void UpdateDTKernel(DTChunk &argDT,Document &argDoc, int argGPUId, hipStream_t &stream) {

	int blockCounter = 0;
	int GPUId = argGPUId;
	int chunkId = argGPUId;
	int iterDT = (argDT.docLengthVec[chunkId] - 1) / GridDim + 1;
	hipSetDevice(GPUId);
	for (int i = 0; i < iterDT; i++) {
		/*hipMemcpy(argDoc.d_blockCounter, &blockCounter, sizeof(int), hipMemcpyHostToDevice);*/
		DT_Update_Kernel << <GridDim, BlockDim>> > (argDoc.GPUChunkVec[GPUId].deviceMapWord2Doc, argDoc.GPUChunkVec[GPUId].deviceTLTopic,  argDT.DTGPUChunkVec[GPUId].deviceNZDTCount, argDT.DTGPUChunkVec[GPUId].deviceDTIndex, argDT.DTGPUChunkVec[GPUId].deviceDTValue, argDoc.GPUChunkVec[GPUId].deviceTLDocCount, argDoc.GPUChunkVec[GPUId].deviceTLDocOffset, argDT.DTGPUChunkVec[GPUId].deviceDTCount, argDT.DTGPUChunkVec[GPUId].deviceDTOffset, blockCounter, argDT.docLengthVec[chunkId], argDoc.GPUChunkVec[GPUId].d_dense);
		
		hipDeviceSynchronize();
		blockCounter++;
	}
	
}

