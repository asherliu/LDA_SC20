#include "DataGPUChunk.cuh"

GPUChunk::GPUChunk(int argMaxTLLength, int argmaxDocLength, int argWordLength) {


	maxTLLength = argMaxTLLength;
	maxDocLength = argmaxDocLength;
	wordLength = argWordLength;


}






void GPUChunk::GPUMemAllocate(int argGPUId) {
	GPUId = argGPUId;
	hipSetDevice(GPUId);
	hipMalloc((void**)&deviceTLTopic, (maxTLLength) * sizeof(int));
	hipMalloc((void**)&deviceTLDocCount, (maxDocLength) * sizeof(int));
	hipMalloc((void**)&deviceTLDocOffset, (maxDocLength) * sizeof(int));
	hipMalloc((void**)&deviceTLWordCount, (wordLength) * sizeof(int));
	hipMalloc((void**)&deviceTLWordOffset, (wordLength) * sizeof(int));
	hipMalloc((void**)&deviceMapWord2Doc, (maxTLLength) * sizeof(int));
	hipMalloc((void**)&deviceMapDoc2Word, (maxTLLength) * sizeof(int));
	hipMalloc((void**)&devicePerplexity, (maxTLLength) * sizeof(float));
	hipMalloc((void**)&devicePerplexityMid, sizeof(float)*(GridDim*BlockDim / 32));

	hipMalloc((void **)&d_dense, sizeof(int)*(GridDim*K));
	hipMalloc((void **)&deviceWTHeadDense, sizeof(float)*(GridDim*K));

}
