
#include "WTAddKernel.cuh"
void WTAdditionKernel(WTAll &argWT, Document &argDoc, hipStream_t &stream) {


	int blockCounter = 0;
	int iterBlock = (argWT.numOfWordS - 1) / GridDim + 1;
	int* deviceWordLength;
	int numOfWordD = argWT.wordLength-argWT.numOfWordS;

	hipSetDevice(0);
	/*hipMalloc((void**)&deviceWordLength, (1) * sizeof(int));
	
	hipMemcpy(deviceWordLength, &argWT.numOfWordS, sizeof(int),hipMemcpyHostToDevice);*/
	for (int i = 0; i < iterBlock; i++) {
		/*hipMemcpy(argDoc.d_blockCounter, &blockCounter, (1) * sizeof(int), hipMemcpyHostToDevice);*/
		sparseMatrixAdd << <GridDim, BlockDim>> >(argWT.WTGPUChunkVec[0].deviceWTCount, argWT.WTGPUChunkVec[0].deviceWTOffset, argWT.WTGPUChunkVec[0].deviceNZWTCount, argWT.WTGPUChunkVec[0].deviceWTIndex, argWT.WTGPUChunkVec[0].deviceWTValue, argWT.deviceZeroChunkWTCount, argWT.deviceZeroChunkWTOffset, argWT.deviceZeroChunkNZWTCount, argWT.deviceZeroChunkWTIndex, argWT.deviceZeroChunkWTValue, argDoc.GPUChunkVec[0].d_dense, argWT.numOfWordS, blockCounter, argWT.WTGPUChunkVec[0].deviceWTRowSum, numOfWordD);
		H_ERR(hipDeviceSynchronize());
		blockCounter++;
	}


}


void WTDenAdditionKernel(WTD &argWTDen, WTAll &argWT, Document &argDoc, hipStream_t &stream) {

	hipSetDevice(0);
	denseMatrixAddKernel << <GridDim, BlockDim >> > (argWTDen.WTDenseGPUChunkVec[0].deviceWTDense, argWTDen.deviceZeroWTDense, argWT.WTGPUChunkVec[0].deviceWTOffset, argWTDen.numOfWordD);
	H_ERR(hipDeviceSynchronize());
}