#include "hip/hip_runtime.h"
#include "DT.cuh"


DTChunk::DTChunk(int argmaxDTLength, int argMaxDocLength, int argNumChunks, int argNumGPUs) {

	maxDTLength = argmaxDTLength;
	maxDocLength = argMaxDocLength;
	numChunks = argNumChunks;
	numGPUs = argNumGPUs;
	NZDTCount = new int[maxDocLength];
	DTIndex = new int[maxDTLength];
	DTValue = new int[maxDTLength];
	//DTCount = new int[maxDocLength];
	//DTOffset = new int[maxDocLength];
	DTLengthVec = new int[numChunks];
	docLengthVec = new int[numChunks];
}

void DTChunk::loadDocDTLength(string argFilePrefix) {
	ifstream DTLength((argFilePrefix + string("/DTLength.txt")).c_str(), ios::binary);//store max Doc and DT length
	ifstream docLength((argFilePrefix + string("/docLength.txt")).c_str(), ios::binary);//store max Doc and DT length
	for (int chunkId = 0; chunkId < numChunks; chunkId++) {
		DTLength >> DTLengthVec[chunkId];
		docLength >> docLengthVec[chunkId];
	}
	DTLength.close();
	docLength.close();
}



void DTChunk::CPUMemSet() {

	memset(NZDTCount, 0, maxDocLength * sizeof(int));
	memset(DTIndex, 0, maxDTLength * sizeof(int));
	memset(DTValue, 0, maxDTLength * sizeof(int));
	//memset(DTCount, 0, maxDocLength * sizeof(int));
	//memset(DTOffset, 0, maxDocLength * sizeof(int));

}

void DTChunk::InitDTGPU()
{
	for (int GPUId = 0; GPUId < numGPUs; GPUId++) {

		DTGPUChunk GPUChunkDT(maxDTLength, maxDocLength, GPUId);
		GPUChunkDT.GPUMemAllocate(GPUId);
		GPUChunkDT.GPUMemSet(GPUId);

		DTGPUChunkVec.push_back(GPUChunkDT);
	}

}




//void DTChunk::GPUMemAllocate(int argGPUId) {
//
//	GPUId = argGPUId;
//	hipSetDevice(GPUId);
//	hipMalloc((void**)&deviceNZDTCount, (maxDocLength) * sizeof(int));
//	hipMalloc((void**)&deviceDTIndex, (maxDTLength) * sizeof(int));
//	hipMalloc((void**)&deviceDTValue, (maxDTLength) * sizeof(int));
//	hipMalloc((void**)&deviceDTCount, (maxDocLength) * sizeof(int));
//	hipMalloc((void**)&deviceDTOffset, (maxDocLength) * sizeof(int));
//
//	DTMemory = (3 * maxDocLength + 2 * maxDTLength) * sizeof(int) / 1000000000.0;
//	printf("DT memory usage:%f GB\n", DTMemory);
//
//}

void DTChunk::loadDTCountOffset(string argFilePrefix) {

	/*chunkId = argChunkId;*/
	
	for (int chunkId = 0; chunkId < numChunks; chunkId++) {
		string chunkFolderName = argFilePrefix + "/chunk" + to_string(chunkId);
		ifstream DTCountOffset((chunkFolderName + string("/DTCountOffset.txt")).c_str(), ios::binary);//store Word offset of TL
		int* DTCount = new int[docLengthVec[chunkId]];
		int* DTOffset = new int[docLengthVec[chunkId]];
		memset(DTCount, 0, docLengthVec[chunkId] * sizeof(int));
		memset(DTOffset, 0, docLengthVec[chunkId] * sizeof(int));

		for (int i = 0; i < docLengthVec[chunkId]; i++)
		{
			DTCountOffset >> DTCount[i] >> DTOffset[i];
		}
		DTCountOffset.close();
		DTCountVec.push_back(DTCount);
		DTOffsetVec.push_back(DTOffset);


	}
	


}




void DTChunk::CPU2GPU(int argGPUId) {

	hipSetDevice(argGPUId);
	GPUId = argGPUId;
	//docLength = argDocLength;
	hipMemcpy(DTGPUChunkVec[argGPUId].deviceNZDTCount, NZDTCount, (docLengthVec[chunkId]) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(DTGPUChunkVec[argGPUId].deviceDTIndex, DTIndex, (DTLengthVec[chunkId]) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(DTGPUChunkVec[argGPUId].deviceDTValue, DTValue, (DTLengthVec[chunkId]) * sizeof(int), hipMemcpyHostToDevice);
	

}


//void DTChunk::GPUMemSet(int argChunkId)
//{
//	chunkId = argChunkId;
//	hipMemset(deviceNZDTCount, 0, (maxDocLength) * sizeof(int));
//	hipMemset(deviceDTIndex, 0, (maxDTLength) * sizeof(int));
//	hipMemset(deviceDTValue, 0, (maxDTLength) * sizeof(int));
//
//}






void DTChunk::CPU2GPUDTCountOffset(int argGPUId) {
	GPUId = argGPUId;
	hipSetDevice(argGPUId);
	//docLength = argDocLength;

	hipMemcpy(DTGPUChunkVec[argGPUId].deviceDTCount, DTCountVec[argGPUId], (docLengthVec[argGPUId]) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(DTGPUChunkVec[argGPUId].deviceDTOffset, DTOffsetVec[argGPUId], (docLengthVec[argGPUId]) * sizeof(int), hipMemcpyHostToDevice);

}


void DTChunk::GPU2CPU(int argGPUId) {
	GPUId = argGPUId;
	hipSetDevice(argGPUId);
	//docLength = argDocLength;
	hipMemcpy(NZDTCount, DTGPUChunkVec[argGPUId].deviceNZDTCount, (docLengthVec[argGPUId]) * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(DTIndex, DTGPUChunkVec[argGPUId].deviceDTIndex, (DTLengthVec[argGPUId]) * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(DTValue, DTGPUChunkVec[argGPUId].deviceDTValue, (DTLengthVec[argGPUId]) * sizeof(int), hipMemcpyDeviceToHost);

}

void DTChunk::CPU2Disk(string argFilePrefix,int argChunkId) {
	chunkId = argChunkId;
	//docLength = argDocLength;
	string chunkFolderName = argFilePrefix + "/chunk" + to_string(chunkId);
	ofstream OutputNZDTCount((chunkFolderName + string("/NZDTCount.txt")).c_str(), ios::binary);
	for (int i = 0; i < docLengthVec[chunkId]; i++) {
		OutputNZDTCount << NZDTCount[i] << "\n";
	}
	OutputNZDTCount.close();
	ofstream OutputDTIndexValue((chunkFolderName + string("/DTIndexValue.txt")).c_str(), ios::binary);
	for (int i = 0; i < DTLengthVec[chunkId]; i++) {
		OutputDTIndexValue << DTIndex[i] <<" "<<DTValue[i]<< "\n";
	}
	OutputDTIndexValue.close();
}