
#include "DTChunk.cuh"
DTGPUChunk::DTGPUChunk(int argmaxDTLength, int argMaxDocLength, int argGPUId) {

	maxDTLength = argmaxDTLength;
	maxDocLength = argMaxDocLength;
	GPUId = argGPUId;
	//NZDTCount = new int[maxDocLength];
	//DTIndex = new int[maxDTLength];
	//DTValue = new int[maxDTLength];
	////DTCount = new int[maxDocLength];
	////DTOffset = new int[maxDocLength];
	//DTLengthVec = new int[numChunks];
	//docLengthVec = new int[numChunks];
}




void DTGPUChunk::GPUMemAllocate(int argGPUId) {

	GPUId = argGPUId;
	hipSetDevice(argGPUId);
	hipMalloc((void**)&deviceNZDTCount, (maxDocLength) * sizeof(int));
	hipMalloc((void**)&deviceDTIndex, (maxDTLength) * sizeof(int));
	hipMalloc((void**)&deviceDTValue, (maxDTLength) * sizeof(int));
	hipMalloc((void**)&deviceDTCount, (maxDocLength) * sizeof(int));
	hipMalloc((void**)&deviceDTOffset, (maxDocLength) * sizeof(int));

	//DTMemory = (3 * maxDocLength + 2 * maxDTLength) * sizeof(int) / 1000000000.0;
	//printf("DT memory usage:%f GB\n", DTMemory);

}

void DTGPUChunk::GPUMemSet(int argGPUId)
{
	GPUId = argGPUId;
	hipSetDevice(argGPUId);
	hipMemset(deviceNZDTCount, 0, (maxDocLength) * sizeof(int));
	hipMemset(deviceDTIndex, 0, (maxDTLength) * sizeof(int));
	hipMemset(deviceDTValue, 0, (maxDTLength) * sizeof(int));

}
