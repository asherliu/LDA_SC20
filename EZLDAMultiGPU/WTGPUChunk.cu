
#include "WTGPUChunk.cuh"


WTGPUChunkData::WTGPUChunkData(int argGPUId, int argWordLength, int argMaxChunkWTLength, int argWTLength, int argNumOfWordS) {

	GPUId = argGPUId;
	wordLength = argWordLength;
	maxChunkWTLength = argMaxChunkWTLength;
	WTLength = argWTLength;
	numOfWordS = argNumOfWordS;


}


void WTGPUChunkData::GPUMemAllocate(int argGPUId) {

	GPUId = argGPUId;
	hipSetDevice(argGPUId);
	hipMalloc((void**)&deviceNZWTCount, (numOfWordS) * sizeof(int));
	hipMalloc((void**)&deviceWTIndex, (maxWTLength) * sizeof(int));
	hipMalloc((void**)&deviceWTValue, (maxWTLength) * sizeof(int));
	hipMalloc((void**)&deviceWTCount, (wordLength) * sizeof(int));
	hipMalloc((void**)&deviceWTOffset, (wordLength) * sizeof(int));

	hipMalloc((void**)&deviceWTRowSum, (K) * sizeof(int));


	hipMalloc((void**)&deviceChunkWTCount, (numOfWordS) * sizeof(int));
	hipMalloc((void**)&deviceChunkWTOffset, (numOfWordS) * sizeof(int));
	hipMalloc((void**)&deviceChunkNZWTCount, (numOfWordS) * sizeof(int));
	hipMalloc((void**)&deviceChunkWTIndex, (maxChunkWTLength) * sizeof(int));
	hipMalloc((void**)&deviceChunkWTValue, (maxChunkWTLength) * sizeof(int));



}
void WTGPUChunkData::GPUMemset(int argGPUId)
{
	hipSetDevice(argGPUId);
	hipMemset(deviceNZWTCount, 0, (numOfWordS) * sizeof(int));
	hipMemset(deviceWTIndex, 0, (maxWTLength) * sizeof(int));
	hipMemset(deviceWTValue, 0, (maxWTLength) * sizeof(int));
	hipMemset(deviceWTRowSum, 0, (K) * sizeof(int));

}

void WTGPUChunkData::chunkGPUMemset(int argGPUId)
{

	hipSetDevice(argGPUId);
	hipMemset(deviceChunkNZWTCount, 0, (numOfWordS) * sizeof(int));
	hipMemset(deviceChunkWTIndex, 0, (maxChunkWTLength) * sizeof(int));
	hipMemset(deviceChunkWTValue, 0, (maxChunkWTLength) * sizeof(int));
	//hipMemset(deviceWTRowSum, 0, (K) * sizeof(int));

}
