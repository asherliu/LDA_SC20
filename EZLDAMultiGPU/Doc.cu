#include "hip/hip_runtime.h"
#include "Doc.cuh"

Document::Document(string argFilePrefix, int argNumChunks, int argMaxTLLength, int argmaxDocLength, int argWordLength, int argNumGPUs) {

	filePrefix = argFilePrefix;
	numChunks = argNumChunks;
	maxTLLength = argMaxTLLength;
	maxDocLength = argmaxDocLength;
	wordLength = argWordLength;
	numGPUs = argNumGPUs;
	perplexityMid = new float[GridDim*BlockDim / 32];
	//perplexityMid2 = new float[GridDim*BlockDim / 32];
	perplexity = new float[maxTLLength];

	
}

void Document::loadDocument() {


	TLLengthVec = new int[numChunks];
	docLengthVec = new int[numChunks];
	numOfTokenVecD = new int[numChunks];
	numOfTokenVecS = new int[numChunks];

	ifstream docLength((filePrefix + string("/docLength.txt")).c_str(), ios::binary);//store max Doc and DT length
	ifstream TLLength((filePrefix + string("/TLLength.txt")).c_str(), ios::binary);
	ifstream TLSplit((filePrefix + string("/TLSplit.txt")).c_str(), ios::binary);

	for (int chunkId = 0; chunkId < numChunks; chunkId++) {

		TLLength >> TLLengthVec[chunkId];
		docLength >> docLengthVec[chunkId];
		TLSplit >> numOfTokenVecD[chunkId] >> numOfTokenVecS[chunkId];
		DocChunk tmpDocChunk(TLLengthVec[chunkId], docLengthVec[chunkId], wordLength);
		tmpDocChunk.CPUMemSet();
		tmpDocChunk.loadChunk(filePrefix, chunkId);
		docChunkVec.push_back(tmpDocChunk);

	}
	printf("All chunks loaded!");
	docLength.close();
	TLLength.close();

}


void Document::InitGPU()
{
	for (int GPUId = 0; GPUId < numGPUs; GPUId++) {

		GPUChunk ChunkGPU(maxTLLength, maxDocLength, wordLength);
		ChunkGPU.GPUMemAllocate(GPUId);
		GPUChunkVec.push_back(ChunkGPU);
	}

}








void Document::CPU2GPUPerplexity(int argGPUId) {

	hipSetDevice(argGPUId);
	memset(perplexityMid, 0, GridDim*BlockDim / 32 * sizeof(float));

	//memset(perplexityMid2, 0, GridDim*BlockDim / 32 * sizeof(float));

	hipMemcpy(GPUChunkVec[argGPUId].devicePerplexityMid, perplexityMid, (GridDim*BlockDim / 32) * sizeof(float), hipMemcpyHostToDevice);
    hipMemset(GPUChunkVec[argGPUId].devicePerplexity,0,maxTLLength*sizeof(float));

}


void Document::GPU2CPUPerplexity(int argGPUId) {
	//hipSetDevice(argGPUId);
	//if (argGPUId == 0) {
	//	hipMemcpy(perplexityMid, GPUChunkVec[argGPUId].devicePerplexityMid, (GridDim*BlockDim / 32) * sizeof(float), hipMemcpyDeviceToHost);
	//}
	//
	//else {
	//	hipMemcpy(perplexityMid2, GPUChunkVec[argGPUId].devicePerplexityMid, (GridDim*BlockDim / 32) * sizeof(float), hipMemcpyDeviceToHost);
	//}
	//

	///*hipMemcpy(perplexity, GPUChunkVec[argGPUId].devicePerplexity, maxTLLength*sizeof(float), hipMemcpyDeviceToHost);*/
	//sumPerplexity = 0.0;
	//if (argGPUId == 0) {
	//	for (int i = 0; i < GridDim*BlockDim / 32; i++) {
	//		// printf("Perplexity:%f \n", h_PerplexityMid[i]);
	//		sumPerplexity += perplexityMid[i] / 467723.0;
	//	}
	//}

	//else {
	//	for (int i = 0; i < GridDim*BlockDim / 32; i++) {
	//		// printf("Perplexity:%f \n", h_PerplexityMid[i]);
	//		sumPerplexity += perplexityMid2[i] / 467723.0;
	//	}
	//}
	//

	//printf("Parallel Perplexity:%f \n", sumPerplexity);




	hipSetDevice(argGPUId);
	
	hipMemcpy(perplexityMid, GPUChunkVec[argGPUId].devicePerplexityMid, (GridDim*BlockDim / 32) * sizeof(float), hipMemcpyDeviceToHost);
	

	//else {
	//	hipMemcpy(perplexityMid2, GPUChunkVec[argGPUId].devicePerplexityMid, (GridDim*BlockDim / 32) * sizeof(float), hipMemcpyDeviceToHost);
	//}


	/*hipMemcpy(perplexity, GPUChunkVec[argGPUId].devicePerplexity, maxTLLength*sizeof(float), hipMemcpyDeviceToHost);*/
	sumPerplexity = 0.0;
	for (int i = 0; i < GridDim*BlockDim / 32; i++) {
			// printf("Perplexity:%f \n", h_PerplexityMid[i]);
			sumPerplexity += perplexityMid[i] / 467723.0;
	}





	printf("Parallel Perplexity:%f \n", sumPerplexity);




}

void Document::CPU2DiskPerplexity(string argFilePrefix) {

	ofstream OutPutPerplexity((argFilePrefix + string("/Perplexity.txt")).c_str(), ios::binary);
	for (int i = 0; i < maxTLLength; i++) {
		OutPutPerplexity << perplexity[i] << "\n";
	}
	OutPutPerplexity.close();
}

//void Document::GPUMemAllocate(int argGPUId) {
//	GPUId = argGPUId;
//	hipSetDevice(GPUId);
//	hipMalloc((void**)&deviceTLTopic, (maxTLLength) * sizeof(int));
//	hipMalloc((void**)&deviceTLDocCount, (maxDocLength) * sizeof(int));
//	hipMalloc((void**)&deviceTLDocOffset, (maxDocLength) * sizeof(int));
//	hipMalloc((void**)&deviceTLWordCount, (wordLength) * sizeof(int));
//	hipMalloc((void**)&deviceTLWordOffset, (wordLength) * sizeof(int));
//	hipMalloc((void**)&deviceMapWord2Doc, (maxTLLength) * sizeof(int));
//	hipMalloc((void**)&deviceMapDoc2Word, (maxTLLength) * sizeof(int));
//	hipMalloc((void**)&devicePerplexity, (maxTLLength) * sizeof(float));
//	hipMalloc((void**)&devicePerplexityMid, sizeof(float)*(GridDim*BlockDim / 32));
//	
//	hipMalloc((void **)&d_blockCounter, sizeof(int)*(1));
//	hipMalloc((void **)&d_warpCounter, sizeof(int)*(1));
//	hipMalloc((void **)&d_dense, sizeof(int)*(GridDim*K));
//	hipMalloc((void **)&deviceWTHeadDense, sizeof(float)*(GridDim*K));
//
//
//	TLMemory = ((3 * maxTLLength + 2 * maxDocLength + 2 * wordLength + GridDim*K) * sizeof(int) + (maxTLLength + GridDim*BlockDim / 32 + GridDim*K) * sizeof(float))/ 1000000000.0;
//
//	printf("Token list memory usage:%f GB\n", TLMemory);
//
//
//}


void Document::CPU2GPU(int argGPUId, int argChunkId) {

	hipSetDevice(argGPUId);
	

	hipMemcpy(GPUChunkVec[argGPUId].deviceTLTopic, docChunkVec[argChunkId].TLTopic, (TLLengthVec[argChunkId]) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(GPUChunkVec[argGPUId].deviceTLDocCount, docChunkVec[argChunkId].TLDocCount, (docLengthVec[argChunkId]) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(GPUChunkVec[argGPUId].deviceTLDocOffset, docChunkVec[argChunkId].TLDocOffset, (docLengthVec[argChunkId]) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(GPUChunkVec[argGPUId].deviceTLWordCount, docChunkVec[argChunkId].TLWordCount, (wordLength) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(GPUChunkVec[argGPUId].deviceTLWordOffset, docChunkVec[argChunkId].TLWordOffset, (wordLength) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(GPUChunkVec[argGPUId].deviceMapWord2Doc, docChunkVec[argChunkId].mapWord2Doc, (TLLengthVec[argChunkId]) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(GPUChunkVec[argGPUId].deviceMapDoc2Word, docChunkVec[argChunkId].mapDoc2Word, (TLLengthVec[argChunkId]) * sizeof(int), hipMemcpyHostToDevice);


}

void Document::GPU2CPU(int argGPUId, int argChunkId) {
	hipSetDevice(argGPUId);
	hipMemcpy(docChunkVec[argChunkId].TLTopic, GPUChunkVec[argGPUId].deviceTLTopic, (TLLengthVec[argChunkId]) * sizeof(int), hipMemcpyDeviceToHost);


}
