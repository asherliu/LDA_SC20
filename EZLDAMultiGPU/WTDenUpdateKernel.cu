#include "WTDenUpdateKernel.cuh"
void UpdateWTDenKernel(WTD &argWTDen, WTAll &argWT, Document &argDoc, int argChunkId, int argGPUId, hipStream_t &stream) {

	int iter= (argWTDen.numOfWordD - 1) / GridDim + 1;// number of iterations for block.
	
	int counter = 0;
	int GPUId = argGPUId;
	hipSetDevice(GPUId);
	/*int numOfTokenD = argWTDen.numOfWordD;*/
	for (int i = 0; i < iter; i++)
	{

		
		WTDen_Update_Kernel << <GridDim, BlockDim >> >(argDoc.GPUChunkVec[GPUId].deviceTLTopic, argWTDen.WTDenseGPUChunkVec[GPUId].deviceWTDenseCopy, argDoc.GPUChunkVec[GPUId].deviceTLWordCount, argDoc.GPUChunkVec[GPUId].deviceTLWordOffset, argWT.WTGPUChunkVec[GPUId].deviceWTOffset, argWTDen.numOfWordD,  counter);

		H_ERR(hipDeviceSynchronize());
		counter++;

	}

}

void UpdateWTDenRowSumKernel(WTD &argWTDen, WTAll &argWT, int argGPUId, hipStream_t &stream)
{
	int GPUId = argGPUId;
	hipSetDevice(GPUId);

	WTDen_Sum_Update_Kernel << <GridDim, BlockDim>> >(argWTDen.WTDenseGPUChunkVec[0].deviceWTDense, argWT.WTGPUChunkVec[GPUId].deviceWTRowSum, argWT.WTGPUChunkVec[GPUId].deviceWTOffset, argWTDen.numOfWordD);
	H_ERR(hipDeviceSynchronize());
}



