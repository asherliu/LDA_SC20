#include "WTUpdateKernel.cuh"
void UpdateWTKernel(WTAll &argWT, Document &argDoc, int argChunkId, int argGPUId, hipStream_t &stream) {

	int iterBlock = (argWT.blockCount - 1) / GridDim + 1;// number of iterations for block.
	//int iterBlock = 9;// number of iterations for block.
	int GridWarpDim = GridDim*BlockDim / 32;
	int iterAll = (argWT.blockCount - 1) / GridDim + 1 + (argWT.warpCount - 1) / GridWarpDim + 1; // number of total iterations.

	int blockCounter = 0;
	int warpCounter = 0;
	int GPUId = argGPUId;
	int numOfTokenD = argDoc.numOfTokenVecD[argChunkId];

	hipSetDevice(GPUId);

	for (int i = 0; i < iterAll; i++)
	{
		if (i < iterBlock)
		{
			/*H_ERR(hipMemcpy(argDoc.d_blockCounter, &blockCounter, sizeof(int), hipMemcpyHostToDevice));*/
			tokenlist_to_matrix << <GridDim, BlockDim >> > (argDoc.GPUChunkVec[GPUId].deviceTLTopic, argWT.WTGPUChunkVec[GPUId].deviceChunkNZWTCount, argWT.WTGPUChunkVec[GPUId].deviceChunkWTIndex, argWT.WTGPUChunkVec[GPUId].deviceChunkWTValue,  argWT.WTGPUChunkVec[GPUId].deviceChunkWTCount, argWT.WTGPUChunkVec[GPUId].deviceChunkWTOffset, argWT.WTGPUChunkVec[GPUId].deviceWTRowSum, blockCounter, argWT.blockCount, argDoc.GPUChunkVec[GPUId].d_dense, numOfTokenD);
			 H_ERR(hipDeviceSynchronize());
			 blockCounter++;

	
		}
		else
		{
			/*hipMemcpy(argDoc.d_warpCounter, &warpCounter, sizeof(int), hipMemcpyHostToDevice);*/
			tokenlist_to_matrix_warp << <GridDim, BlockDim >> > (argDoc.GPUChunkVec[GPUId].deviceTLTopic, argWT.WTGPUChunkVec[GPUId].deviceChunkNZWTCount, argWT.WTGPUChunkVec[GPUId].deviceChunkWTIndex, argWT.WTGPUChunkVec[GPUId].deviceChunkWTValue,  argWT.WTGPUChunkVec[GPUId].deviceChunkWTCount, argWT.WTGPUChunkVec[GPUId].deviceChunkWTOffset, argWT.WTGPUChunkVec[GPUId].deviceWTRowSum, warpCounter, argWT.blockCount, argWT.warpCount, numOfTokenD);
			/*printf("abc %d", warpCounter);*/
			H_ERR(hipDeviceSynchronize());
			warpCounter++;
		}
		H_ERR(hipDeviceSynchronize());
		
		
	}

}





