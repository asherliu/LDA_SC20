#include "WTDense.cuh"

WTD::WTD(int argNumOfWordD, int argWordLength, int argNumGPUs) {
	numOfWordD = argNumOfWordD;
	wordLength = argWordLength;
	numGPUs = argNumGPUs;
	WTDenseLength = argNumOfWordD*K;
	WTDense = new int[WTDenseLength];
	WTDenseCopy = new int[WTDenseLength];
	/*WTRowSumDense = new int[K];*/
}

void WTD::CPUMemSet() {

	memset(WTDense, 0, WTDenseLength * sizeof(int));
	memset(WTDenseCopy, 0, WTDenseLength * sizeof(int));
	//memset(WTRowSumDense, 0, K * sizeof(int));

}


//void WTD::GPUMemAllocate(int argGPUId) {
//	GPUId = argGPUId;
//	hipSetDevice(GPUId);
//	hipMalloc((void**)&deviceWTDense, (WTDenseLength) * sizeof(int));
//	hipMalloc((void**)&deviceWTDenseCopy, (WTDenseLength) * sizeof(int));
//	//hipMalloc((void**)&deviceWTRowSumDense, (K) * sizeof(int));
//
//	WTMemory = (2*WTDenseLength + K ) / 1000000000.0 * sizeof(int);
//	printf("WT memory usage(Sparse Part):%f GB\n", WTMemory);
//	
//}


void WTD::InitWTGPU()
{
	for (int GPUId = 0; GPUId < numGPUs; GPUId++) {

		WTDChunk WTDenseGPUChunk(numOfWordD, wordLength, GPUId);
		WTDenseGPUChunk.GPUMemAllocate(GPUId);
		WTDenseGPUChunk.GPUMemInit(GPUId);
		WTDenseGPUChunkVec.push_back(WTDenseGPUChunk);
	}

}

void WTD::GPUMemAllocate() {

	hipSetDevice(0);
	hipMalloc((void**)&deviceZeroWTDense, (WTDenseLength) * sizeof(int));

}

void WTD::GPUDataTransfer(int argGPUId, hipStream_t &stream) {
	hipSetDevice(0);
	hipMemcpyAsync(deviceZeroWTDense, WTDenseGPUChunkVec[argGPUId].deviceWTDenseCopy, (WTDenseLength) * sizeof(int), hipMemcpyDeviceToDevice, stream);

}

void WTD::GPUDataDistribute(int argGPUId, hipStream_t &stream) {

	hipSetDevice(0);
	hipMemcpyAsync(WTDenseGPUChunkVec[argGPUId].deviceWTDense, WTDenseGPUChunkVec[0].deviceWTDense, (WTDenseLength) * sizeof(int), hipMemcpyDeviceToDevice, stream);

}



void WTD::GPUDataTransferBackCPU(int argGPUId) {
	hipSetDevice(argGPUId);
	hipMemcpy(WTDenseCopy, WTDenseGPUChunkVec[argGPUId].deviceWTDenseCopy, (WTDenseLength) * sizeof(int), hipMemcpyDeviceToHost);

}


void WTD::GPUDataTransferToGPU(int argGPUId) {
	hipSetDevice(0);
	hipMemcpy(deviceZeroWTDense, WTDenseCopy, (WTDenseLength) * sizeof(int), hipMemcpyHostToDevice);

}



void WTD::GPUDataDistributeBackCPU(int argGPUId) {

	hipSetDevice(0);
	hipMemcpy(WTDense, WTDenseGPUChunkVec[0].deviceWTDense, (WTDenseLength) * sizeof(int), hipMemcpyDeviceToHost);

}

void WTD::GPUDataDistributeToGPU(int argGPUId) {

	hipSetDevice(argGPUId);
	hipMemcpy(WTDenseGPUChunkVec[argGPUId].deviceWTDense, WTDense, (WTDenseLength) * sizeof(int), hipMemcpyHostToDevice);

}


















//void WTD::GPUMemInit()
//{
//
//
//
//
//	hipSetDevice(GPUId);
//	hipMemset(deviceWTDense, 0, (WTDenseLength) * sizeof(int));
//	hipMemset(deviceWTDenseCopy, 0, (WTDenseLength) * sizeof(int));
//	//hipMemset(deviceWTRowSumDense, 0, (K) * sizeof(int));
//}

void WTD::GPUMemCopy(int argGPUId)
{
	hipSetDevice(argGPUId);
	hipMemcpy(WTDenseGPUChunkVec[argGPUId].deviceWTDense, WTDenseGPUChunkVec[argGPUId].deviceWTDenseCopy, (WTDenseLength) * sizeof(int), hipMemcpyDeviceToDevice);
	
}

void WTD::GPUMemset(int argGPUId)
{
	hipSetDevice(argGPUId);
	hipMemset(WTDenseGPUChunkVec[argGPUId].deviceWTDenseCopy, 0, (WTDenseLength) * sizeof(int));
	//hipMemset(deviceWTRowSumDense, 0, (K) * sizeof(int));
}


void WTD::WTDenGPU2CPU(int argGPUId)
{
	hipSetDevice(argGPUId);
	hipMemcpy(WTDense, WTDenseGPUChunkVec[argGPUId].deviceWTDense, (WTDenseLength) * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(WTDenseCopy, WTDenseGPUChunkVec[argGPUId].deviceWTDenseCopy, (WTDenseLength) * sizeof(int), hipMemcpyDeviceToHost);
	

}
void WTD::WTDenCPU2Disk(string argFilePrefix) {

	ofstream WTDen((argFilePrefix + string("/WTDen.txt")).c_str(), ios::binary);
	for (int i = 0; i < WTDenseLength; i++) {
		WTDen << WTDense[i] << "\n";
	}
	WTDen.close();
	ofstream WTDenCopy((argFilePrefix + string("/WTDenCopy.txt")).c_str(), ios::binary);
	for (int i = 0; i < WTDenseLength; i++) {
		WTDenCopy << WTDenseCopy[i] << "\n";
	}
	WTDen.close();

}