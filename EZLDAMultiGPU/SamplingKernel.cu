


#include "SamplingKernel.cuh"


#define gpuErr(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


void SampleKernelD(WTD &argWTDen, WTAll &argWT, DTChunk &argDT, Document &argDoc, hiprandState* randState, int argChunkId, int argGPUId, hipStream_t &stream)
{

	unsigned int blockCounter = 0;
	int iterWT = (argWTDen.numOfWordD - 1) / GridDim + 1;
	float Perplexity = 0.0;
	unsigned int* deviceCounter;
	hipSetDevice(argGPUId);
	hipMalloc(&deviceCounter, sizeof(unsigned int));
	// srand(time(NULL));

	// hiprandState* randState;
	// hipMalloc(&randState, sizeof(hiprandState)*GridDim*BlockDim);
	// H_ERR(hipDeviceSynchronize());
   
 //    gpuErr(hipPeekAtLastError());
	
	initRandState << <GridDim, BlockDim >> >(randState);
	H_ERR(hipDeviceSynchronize());

	// for (int i = 0; i < iterWT; i++) {

	hipMemcpy(deviceCounter, &blockCounter, sizeof(unsigned int), hipMemcpyHostToDevice);

	LDAKernelTrainD << <GridDim, BlockDim>> > (alpha, beta, argDoc.GPUChunkVec[argGPUId].deviceMapWord2Doc, argDoc.GPUChunkVec[argGPUId].deviceTLTopic, argDT.DTGPUChunkVec[argGPUId].deviceNZDTCount, argDT.DTGPUChunkVec[argGPUId].deviceDTIndex, argDT.DTGPUChunkVec[argGPUId].deviceDTValue, argDoc.GPUChunkVec[argGPUId].deviceTLDocCount, argDoc.GPUChunkVec[argGPUId].deviceTLDocOffset, argDT.DTGPUChunkVec[argGPUId].deviceDTCount, argDT.DTGPUChunkVec[argGPUId].deviceDTOffset, argWTDen.WTDenseGPUChunkVec[argGPUId].deviceWTDense, argWTDen.WTDenseGPUChunkVec[argGPUId].deviceWTDenseCopy, argDoc.GPUChunkVec[argGPUId].deviceTLWordCount, argDoc.GPUChunkVec[argGPUId].deviceTLWordOffset, argWT.WTGPUChunkVec[argGPUId].deviceWTCount, argWT.WTGPUChunkVec[argGPUId].deviceWTOffset, argWT.WTGPUChunkVec[argGPUId].deviceWTRowSum, deviceCounter, argDoc.GPUChunkVec[argGPUId].deviceMapDoc2Word, argDoc.docLengthVec[argGPUId], argWT.wordLength, argDoc.GPUChunkVec[argGPUId].devicePerplexity, randState, argDoc.GPUChunkVec[argGPUId].deviceWTHeadDense, argWTDen.numOfWordD);
	// H_ERR(hipDeviceSynchronize());
	// 	blockCounter++;

	// }
	
	H_ERR(hipDeviceSynchronize());

}
//(double alpha, double beta, int* d_Index, int* d_TopicIndex, int* d_SparseDTCount, int* d_SparseDTIndex, int* d_SparseDTValue, int* d_TokenCountDT, int* d_TokenOffsetDT, int* d_DocListCount, int* d_DocListOffset, int* d_WTDense, int* d_WTDenseCopy, int* d_TokenCount, int* d_TokenOffset, int* d_WordListCount, int* d_WordListOffset, int* d_WTRowSum, int* d_blockCounter, int*d_DocIndex, int D, int W, double* d_Perplexity, hiprandState *randState, double *WTHeadDense, int numOfWordD);


void SampleKernel(WTAll &argWT, DTChunk &argDT, Document &argDoc, hiprandState* randState, int argChunkId, int argGPUId, hipStream_t &stream) {

	int blockCounter = 0;
	int iterWT = (argWT.numOfWordS - 1) / GridDim + 1;
	float Perplexity = 0.0;
	int numOfWordD = argWT.wordLength- argWT.numOfWordS;
	// srand(time(NULL));

	// hiprandState* randState;
	// hipMalloc(&randState, sizeof(hiprandState)*GridDim*BlockDim);
	// H_ERR(hipDeviceSynchronize());
 //    gpuErr(hipPeekAtLastError());
	hipSetDevice(argGPUId);
	initRandState << <GridDim, BlockDim >> >(randState);
	H_ERR(hipDeviceSynchronize());

	for (int i = 0; i < iterWT; i++) {

		/*hipMemcpy(argDoc.d_blockCounter, &blockCounter, sizeof(int), hipMemcpyHostToDevice);*/

		LDAKernelTrain << <GridDim, BlockDim >> > (alpha, beta, argDoc.GPUChunkVec[argGPUId].deviceMapWord2Doc, argDoc.GPUChunkVec[argGPUId].deviceTLTopic, argDT.DTGPUChunkVec[argGPUId].deviceNZDTCount, argDT.DTGPUChunkVec[argGPUId].deviceDTIndex, argDT.DTGPUChunkVec[argGPUId].deviceDTValue, argDoc.GPUChunkVec[argGPUId].deviceTLDocCount, argDoc.GPUChunkVec[argGPUId].deviceTLDocOffset, argDT.DTGPUChunkVec[argGPUId].deviceDTCount, argDT.DTGPUChunkVec[argGPUId].deviceDTOffset, argWT.WTGPUChunkVec[argGPUId].deviceNZWTCount, argWT.WTGPUChunkVec[argGPUId].deviceWTIndex, argWT.WTGPUChunkVec[argGPUId].deviceWTValue, argDoc.GPUChunkVec[argGPUId].deviceTLWordCount, argDoc.GPUChunkVec[argGPUId].deviceTLWordOffset, argWT.WTGPUChunkVec[argGPUId].deviceWTCount, argWT.WTGPUChunkVec[argGPUId].deviceWTOffset, argWT.WTGPUChunkVec[argGPUId].deviceWTRowSum, blockCounter, argDoc.GPUChunkVec[argGPUId].deviceMapDoc2Word, argDoc.docLengthVec[argGPUId], argWT.wordLength, argDoc.GPUChunkVec[argGPUId].devicePerplexity, randState, argDoc.GPUChunkVec[argGPUId].deviceWTHeadDense, numOfWordD, argWT.numOfWordS);
		H_ERR(hipDeviceSynchronize());
		blockCounter++;

	}
	hipSetDevice(argGPUId);
	LDATrainPerplexityReduce1 << <GridDim, BlockDim>> > (argDoc.GPUChunkVec[argGPUId].devicePerplexity, argDoc.GPUChunkVec[argGPUId].devicePerplexityMid, argDoc.TLLengthVec[argGPUId]);

	H_ERR(hipDeviceSynchronize());


}








