#include "hip/hip_runtime.h"

#ifndef _MAINKERNEL_H_
#define _MAINKERNEL_H_

#include "DTUpdateKernel.cuh"
#include "WTUpdateKernel.cuh"
#include "WTDenUpdateKernel.cuh"
#include "WTAddKernel.cuh"
#include "SamplingKernel.cuh"
#include "WTDense.cuh"
using namespace std;

void fileErrorCheck(ifstream& argFileStream, string folderName)
{
	if (!argFileStream.is_open())
	{
		cout << "File " << folderName << " open failed" << endl;
		exit(0);
	}
}

int main(int argc, char *argv[]) {

	clock_t startTime, startTime1,endTime;
	double WTTime=0.0;
	double samplingTime=0.0;
	double DTTime=0.0;
	double totalTime=0.0;

	int maxTLLength;
	int maxDTLength;
	int maxWTLength;
	int maxDocLength;
	int wordLength;
	int maxChunkWTLength;
	int numOfWordD;
	int numOfWordS;
	int numChunks = 3;
	int numIters = 200;
	const int numGPUs = 3;
	//int numChunksPerGPU = numChunks / numGPUs;

	string chunkFilePrefix = "C:/shilong/LDA/DSLDACode/datasetDS/docword_kos";// folder that store preprocessed chunks

	ifstream lengthVec((chunkFilePrefix + string("/lengthVec.txt")).c_str(), ios::binary);//store max Doc and DT length

	ofstream timeRecord((chunkFilePrefix + string("/timeRecord.txt")).c_str(), ios::binary);

	fileErrorCheck(lengthVec, "/lengthVec.txt");

	lengthVec >> maxTLLength >> maxDTLength >> maxWTLength >> maxDocLength >> wordLength>>maxChunkWTLength>> numOfWordD>> numOfWordS;
	lengthVec.close();

	Document document(chunkFilePrefix,numChunks,maxTLLength,maxDocLength,wordLength,numGPUs);

	document.loadDocument();
	document.InitGPU();

	hipStream_t stream[numGPUs];
	for (int i = 0; i < numGPUs; i++) {
		hipSetDevice(i);
		hipStreamCreate(&stream[i]);
	}



	DTChunk chunkDT(maxDTLength, maxDocLength, numChunks,numGPUs);
	chunkDT.loadDocDTLength(chunkFilePrefix);// load DT and Doc length from disk to cpu
	chunkDT.loadDTCountOffset(chunkFilePrefix);// load DT count and offset from disk to cpu
	chunkDT.InitDTGPU();// allocate DT and Count and offset ; Init DT 




	//chunkDT.CPUMemSet();
	/*for (int i = 0; i < numGPUs; i++) {
		chunkDT.GPUMemAllocate(i);
	}*/
	
	
	WTD WTDen(numOfWordD, wordLength,numGPUs);
	WTDen.InitWTGPU();
	WTDen.GPUMemAllocate();
	//for (int i = 0; i < numGPUs; i++) {
	//	WTDen.GPUMemAllocate(i);
	//	WTDen.GPUMemInit();//may occur bug
	//}
	


	
	WTAll WT(maxWTLength, wordLength, numChunks, maxChunkWTLength,numOfWordS,numGPUs);
	WT.loadWTLength(chunkFilePrefix);
	WT.loadWTCountOffset(chunkFilePrefix);
	WT.InitWTGPU();
	WT.GPUMemAllocate();
	/*for (int i = 0; i < numGPUs; i++) {
		WT.GPUMemAllocate(i);
		WT.GPUMemset();
		WT.blockWarpCountCPU2GPU();
		WT.CPU2GPUCountOffset();
	}	*/

	/*WT.CPUMemSet();*/


	hiprandState* randState[numGPUs];
	srand(time(NULL));
	for (int i = 0; i < numGPUs; i++) {
		hipSetDevice(i);
		hipMalloc(&randState[i], sizeof(hiprandState)*GridDim*BlockDim);//may have bugs
	}



	H_ERR(hipDeviceSynchronize());

	printf("Total memory usage : %f GB\n", document.TLMemory + WT.WTMemory + chunkDT.DTMemory);


	
	for (int chunkId = 0; chunkId < numChunks; chunkId++) {
		document.CPU2GPU(chunkId,chunkId);
		WT.chunkCPU2GPUCountOffset(chunkId);
		WT.CPU2GPUCountOffset(chunkId);
		H_ERR(hipDeviceSynchronize());
		//WT.WTGPUChunkVec[chunkId].chunkGPUMemset(chunkId);
		//--------------update WTDen matrix ---------
		printf("1");
		UpdateWTDenKernel(WTDen, WT, document, chunkId, chunkId, stream[chunkId]);
		H_ERR(hipDeviceSynchronize());
		printf("2");
		//--------------update WTDen matrix-----------

		//--------------update WT matrix--------
		
		//WT.chunkCPU2GPUCountOffset(chunkId);
		//WT.chunkGPUMemset();
		UpdateWTKernel(WT, document,chunkId, chunkId, stream[chunkId]);
		H_ERR(hipDeviceSynchronize());
		//WT.chunkWTGPU2CPU(chunkId);// marker
		//
		//WT.CPU2DiskChunk(chunkFilePrefix, chunkId);// marker
		/*printf("\n what's this %d\n", chunkId);*/
		//--------------update WT matrix-----------

	}
	printf("3");
	H_ERR(hipDeviceSynchronize());



	for (int chunkId = 0; chunkId < numChunks; chunkId++) {
		printf("4");
		chunkDT.CPU2GPUDTCountOffset(chunkId);
	}

	H_ERR(hipDeviceSynchronize());

	/*printf("3");*/


	///*WTDen.WTDenGPU2CPU();
	//WTDen.WTDenCPU2Disk(chunkFilePrefix);*/
	//printf("WT ended!\n");

	////WT.CPU2GPUCountOffset();
	//startTime = clock();
	for (int iter = 0; iter < numIters; iter++) {


			
		/*startTime1=clock();*/
		
		// MemorySet WTDenCopy
	/*	for (int chunkId = 0; chunkId < numChunks; chunkId++) {
			WTDen.WTDenseGPUChunkVec[chunkId].GPUMemset(chunkId);
			
		}*/

		//--------------MemSet WT and WTRowSum -----------
		for (int chunkId = 0; chunkId < numChunks; chunkId++) {
			WT.WTGPUChunkVec[chunkId].GPUMemset(chunkId);
			H_ERR(hipDeviceSynchronize());
			WTDen.WTDenseGPUChunkVec[chunkId].GPUMemsetWTDense(chunkId);
			H_ERR(hipDeviceSynchronize());
		}
		//--------------MemSet WT and WTRowSum -----------

		H_ERR(hipDeviceSynchronize());



		for (int chunkId = 0; chunkId < numChunks; chunkId++) {
			/*WT.GPUDataTransfer(chunkId, stream[0]);*/
			WT.GPUDataTransferBackCPU(chunkId);
			H_ERR(hipDeviceSynchronize());
			WT.GPUDataTransferToGPU(chunkId);
			H_ERR(hipDeviceSynchronize());
			/*WTDen.GPUDataTransfer(chunkId, stream[0]);*/
			WTDen.GPUDataTransferBackCPU(chunkId);
			H_ERR(hipDeviceSynchronize());
			WTDen.GPUDataTransferToGPU(chunkId);
			H_ERR(hipDeviceSynchronize());
			printf("\nchunkID:%d\n", chunkId);
			WTAdditionKernel(WT, document, stream[0]);
			H_ERR(hipDeviceSynchronize());
			printf("\nchunkID:%d\n", chunkId);
			WTDenAdditionKernel(WTDen, WT, document, stream[0]);
			H_ERR(hipDeviceSynchronize());
		}

		
		



		//for (int chunkId = 0; chunkId < 3; chunkId++) {
		//	WT.GPUDataTransfer(chunkId, stream[0]);
		//	WTDen.GPUDataTransfer(chunkId, stream[0]);
		//	
		//	H_ERR(hipDeviceSynchronize());
		//	printf("\nchunkID:%d\n", chunkId);
		//	WTAdditionKernel(WT, document, stream[0]);
		//	H_ERR(hipDeviceSynchronize());
		//	printf("\nchunkID:%d\n", chunkId);
		//	WTDenAdditionKernel(WTDen, WT, document, stream[0]);

		//}
		H_ERR(hipDeviceSynchronize());
		printf("5");


		UpdateWTDenRowSumKernel(WTDen, WT, 0, stream[0]);
		H_ERR(hipDeviceSynchronize());
		WT.verifyWTSum();
		H_ERR(hipDeviceSynchronize());

		for (int chunkId = 0; chunkId < numChunks; chunkId++) {
			WT.GPUDataDistributeBackCPU(chunkId, stream[0]);
			H_ERR(hipDeviceSynchronize());
			WT.GPUDataDistributeToGPU(chunkId, stream[0]);
			H_ERR(hipDeviceSynchronize());
			WTDen.GPUDataDistributeBackCPU(chunkId);
			H_ERR(hipDeviceSynchronize());
			WTDen.GPUDataDistributeToGPU(chunkId);
			H_ERR(hipDeviceSynchronize());


			/*WTDen.GPUDataDistribute(chunkId, stream[0]);*/


			//WT.GPUDataDistribute(chunkId, stream[0]);
		}
		printf("6");
		H_ERR(hipDeviceSynchronize());


		//--------------MemSet WTDenseCopy and chunkWT -----------
		for (int chunkId = 0; chunkId < numChunks; chunkId++) {
			WTDen.WTDenseGPUChunkVec[chunkId].GPUMemsetWTDenseCopy(chunkId);
			H_ERR(hipDeviceSynchronize());
			WT.WTGPUChunkVec[chunkId].chunkGPUMemset(chunkId);
			H_ERR(hipDeviceSynchronize());
		}
		//--------------MemSet WTDenseCopy and chunkWT---------- -
		

		printf("7");
		H_ERR(hipDeviceSynchronize());


		for (int chunkId = 0; chunkId < numChunks; chunkId++)
		{
			document.CPU2GPUPerplexity(chunkId);
			H_ERR(hipDeviceSynchronize());
			chunkDT.DTGPUChunkVec[chunkId].GPUMemSet(chunkId);
			H_ERR(hipDeviceSynchronize());
		}
		printf("8");
		H_ERR(hipDeviceSynchronize());





		/*for (int chunkId = 0; chunkId < numChunks; chunkId++) {

			chunkDT.DTGPUChunkVec[chunkId].GPUMemSet(chunkId);
		}
		printf("9");
		H_ERR(hipDeviceSynchronize());*/


		

		for (int chunkId = 0; chunkId < numChunks; chunkId++) {


			UpdateDTKernel(chunkDT, document, chunkId, stream[chunkId]);
			H_ERR(hipDeviceSynchronize());
		
			SampleKernelD(WTDen, WT, chunkDT, document, randState[chunkId], chunkId, chunkId, stream[chunkId]);
			H_ERR(hipDeviceSynchronize());
			SampleKernel(WT, chunkDT, document, randState[chunkId], chunkId, chunkId, stream[chunkId]);
			H_ERR(hipDeviceSynchronize());
			UpdateWTKernel(WT, document, chunkId, chunkId, stream[chunkId]);
			H_ERR(hipDeviceSynchronize());

		}
		H_ERR(hipDeviceSynchronize());

		for (int chunkId = 0; chunkId < numChunks; chunkId++)
		{
			document.GPU2CPUPerplexity(chunkId);
			
		}
		H_ERR(hipDeviceSynchronize());


	


		}
		
		
	
	
}
#endif
