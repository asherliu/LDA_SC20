#include "hip/hip_runtime.h"

#ifndef _MAINKERNEL_H_
#define _MAINKERNEL_H_
#define HAVE_STRUCT_TIMESPEC

#include "DTUpdateKernel.cuh"
#include "WTUpdateKernel.cuh"
#include "WTDenUpdateKernel.cuh"
#include "WTAddKernel.cuh"
#include "SamplingKernel.cuh"
#include "WTDense.cuh"
#include "Argument.cuh"
#include <stdio.h>
#include <stdlib.h>
//#include <pthread.h>
#include <thread>

using namespace std;

void fileErrorCheck(ifstream& argFileStream, string folderName)
{
	if (!argFileStream.is_open())
	{
		cout << "File " << folderName << " open failed" << endl;
		exit(0);
	}
}




int main(int argc, char *argv[]) {

	clock_t startTime, startTime1,endTime;
	double transferTimeCPU2GPU=0.0;
	double transferTimeGPU2CPU=0.0;
	double WTTime=0.0;
	double samplingTimeD=0.0;
	double samplingTimeS=0.0;
	double DTTime=0.0;
	double totalTime=0.0;
	double UpdateMTime=0.0;

	int maxTLLength;
	int maxDTLength;
	int maxWTLength;
	int maxDocLength;
	int wordLength;
	int maxChunkWTLength;
	int numOfWordD;
	int numOfWordS;
	int numChunks = 16;
	int numIters = 300;

	int chunksPerStream = numChunks / numStreams;

	string chunkFilePrefix = "/gpfs/alpine/proj-shared/csc289/lda/datasets/data200k";

	
	ofstream SamplingDRecord((chunkFilePrefix + string("/SamplingDRecord.txt")).c_str(), ios::binary);

	ifstream lengthVec((chunkFilePrefix + string("/lengthVec.txt")).c_str(), ios::binary);//store max Doc and DT length

	ofstream timeRecord((chunkFilePrefix + string("/timeRecord.txt")).c_str(), ios::binary);
	ofstream maxPercentRecord((chunkFilePrefix + string("/maxPercentRecord.txt")).c_str(), ios::binary);

	ofstream warpTimeRecord((chunkFilePrefix + string("/warpTimeRecord.txt")).c_str(), ios::binary);

	fileErrorCheck(lengthVec, "/lengthVec.txt");

	lengthVec >> maxTLLength >> maxDTLength >> maxWTLength >> maxDocLength >> wordLength>>maxChunkWTLength>> numOfWordD>> numOfWordS;
	lengthVec.close();

	Document document(chunkFilePrefix,numChunks,maxTLLength,maxDocLength,wordLength);


	//Document doc[2] = {Document(chunkFilePrefix,numChunks,maxTLLength,maxDocLength,wordLength), Document(chunkFilePrefix,numChunks,maxTLLength,maxDocLength,wordLength)};
	//doc[1].loadDocument();
	//doc[1].GPUMemAllocate();
	//doc[1].deviceCounterMemAllocate();
	//doc[2].loadDocument();
	//doc[2].GPUMemAllocate();
	//doc[2].deviceCounterMemAllocate();

	document.loadDocument();
	document.GPUMemAllocate();
	document.deviceCounterMemAllocate();
	H_ERR(hipDeviceSynchronize());

	DTChunk chunkDT(maxDTLength,maxDocLength,numChunks);
	chunkDT.loadDocDTLength(chunkFilePrefix);
	chunkDT.CPUMemSet();
	chunkDT.GPUMemAllocate();	
	chunkDT.loadDTCountOffset(chunkFilePrefix);
//	WTD WTDen(numOfWordD, wordLength);
//	WTDen.GPUMemAllocate();
//	WTDen.GPUMemInit();

	hiprandState* randState[numStreams];
	hipStream_t streams[numStreams];
	hipStream_t syncStream;
	hipStreamCreateWithPriority(&syncStream, hipStreamDefault,0);
	hipEvent_t stopEvents[numStreams];
	hipEvent_t startEvent;
	hipStreamCreate(&syncStream);
	hipEventCreate(&startEvent);
	for (int i = 0; i < numStreams; i++)
	{
		hipStreamCreateWithPriority(&streams[i], hipStreamDefault, i);
		hipEventCreate(&stopEvents[i]);
		hipMalloc(&randState[i], sizeof(hiprandState)*GridDim*BlockDim);
	}

	int clockRate;
	hipDeviceGetAttribute(&clockRate, hipDeviceAttributeClockRate, 0);

	printf("clockRate:%d\n", clockRate);
	WTAll WT(maxWTLength+ numOfWordD*K, wordLength, numChunks, maxChunkWTLength,wordLength);

	WT.CPUMemSet();
	WT.GPUMemAllocate();
	WT.GPUMemset(streams[0]);
	//WT.loadWTLength(chunkFilePrefix);
	WT.loadWTCountOffset(chunkFilePrefix);
	/*WT.blockWarpCountCPU2GPU();*/
	WT.CPU2GPUCountOffset(streams[0]);
	srand(time(NULL));
	float iterTime=0.0;
	hipEvent_t iterStart, iterStop;
	hipEventCreate(&iterStart);
	hipEventCreate(&iterStop);

	H_ERR(hipDeviceSynchronize());

	printf("Total memory usage : %f GB\n", document.TLMemory + WT.WTMemory + chunkDT.DTMemory);

	/*for (int chunkId = 0; chunkId < numChunks; chunkId++)*/
//	for (int batchId = 0; batchId < chunksPerStream; batchId++)
//	{
//		for (int streamId = 0; streamId < numStreams; streamId++)
//		{
//			int chunkId = batchId*numStreams + streamId;
//			document.CPU2GPU(chunkId, streamId, streams[0]);
////			WT.chunkCPU2GPUCountOffset(chunkId, streamId, streams[0]);
////			WT.chunkGPUMemset(streamId, streams[0]);
////			//--------------update WTDen matrix ---------
////			UpdateWTDenKernel(WTDen, WT, document, chunkId, streamId, streams[0]);
//			//--------------update WTDen matrix-----------
//
//			//--------------update WT matrix--------
//			UpdateWTKernel(WT, document, chunkId, streamId, streams[0]);
////			WT.chunkWTGPU2CPU(chunkId, streamId, streams[0]);// marker
//			//--------------update WT matrix-----------
//		}
//	}
//	H_ERR(hipDeviceSynchronize());
	printf("WT ended!\n");

	//WT.CPU2GPUCountOffset();
	startTime = clock();
	/*pthread_t thread[numStreams];*/
	thread threadBlock[numStreams];
	for (int iter = 0; iter < numIters; iter++) {

		hipEventRecord(iterStart,streams[1]);
		startTime1=clock();
		//printf("chunk WT updated!\n");
		WT.GPUMemset(streams[1]);
		for (int batchId = 0; batchId < chunksPerStream; batchId++) {
			for (int streamId = 0; streamId < numStreams; streamId++)
			{
				int chunkId = batchId*numStreams + streamId;
				document.CPU2GPU(chunkId, streamId, streams[1]);
	//			WT.chunkCPU2GPUCountOffset(chunkId, streamId, streams[0]);
	//			WT.chunkGPUMemset(streamId, streams[0]);
	//			//--------------update WTDen matrix ---------
	//			UpdateWTDenKernel(WTDen, WT, document, chunkId, streamId, streams[0]);
				//--------------update WTDen matrix-----------

				//--------------update WT matrix--------
				UpdateWTKernel(WT, document, chunkId, streamId, streams[1]);
	//			WT.chunkWTGPU2CPU(chunkId, streamId, streams[0]);// marker
				//--------------update WT matrix-----------
			}
		}






		//--------------update WTDenSum -----------
		//UpdateWTDenRowSumKernel(WTDen,WT, streams[1]);
		UpdateWTRowSumKernel(WT, streams[1]);
		//--------------update WTDenSum -----------
		/*H_ERR(hipDeviceSynchronize());*/
		//--------------update WTSum -----------

		//for (int chunkId = 0; chunkId < numChunks; chunkId++) 
//		for (int batchId = 0; batchId < chunksPerStream; batchId++) {
//			for (int streamId = 0; streamId < numStreams; streamId++)
//			{
//				int chunkId = batchId*numStreams + streamId;
//				WT.chunkCPU2GPUCountOffset(chunkId, streamId, streams[1]);
//				WT.chunkGPUMemset(streamId, streams[1]);
//				WT.chunkWTCPU2GPU(chunkId, streamId, streams[1]);
//				WTAdditionKernel(WT, document, streamId, streams[1]);
//			}
//		}
		//--------------update WTSum -----------
		//WT.WTGPU2CPU();// marker
		//WT.CPU2Disk(chunkFilePrefix);// marker

		printf("WT updated!\n");
		endTime = clock();
		WTTime+=(double)(endTime-startTime1)/CLOCKS_PER_SEC;

		document.CPU2GPUPerplexity(streams[1]);
		hipEventRecord(startEvent, streams[1]);

		/*samplingTimeD=0;*/
		/*hipDeviceSynchronize();*/
		
		

		
		for (int batchId = 0; batchId < chunksPerStream; batchId++) {

			for (int streamId = 0; streamId < numStreams; streamId++) {
				int chunkId = batchId*numStreams + streamId;
				printf("step: %d\n", chunkId);
				//--------------update DT matrix-----------

				/*int startTime1 = clock();*/

				//int endTime = clock();
				//int transferTimeCPU2GPU += (double)(endTime - startTime1) / CLOCKS_PER_SEC;
				

				printf("%d\n", 1);
				chunkDT.GPUMemSet(chunkId, streamId, streams[streamId]);
				printf("%d\n", 2);
				chunkDT.CPU2GPUDTCountOffset(chunkId, streamId, streams[streamId]);
				printf("%d\n", 3);
				
				if ((chunkId<numStreams)&&(streamId!=1))hipStreamWaitEvent(streams[streamId], startEvent, 0);

				document.CPU2GPU(chunkId, streamId, streams[streamId]);

				MaxTopicKernel(WT, document, streamId, streams[streamId]);

				UpdateDTKernel(chunkDT, document, streamId, streams[streamId]);
				//chunkDT.CPU2GPU(chunkId, document.docLengthVec[chunkId]);
				/*startTime1 = clock();*/
				
				/*endTime = clock();*/
				printf("%d\n", 4);
				//chunkDT.GPU2CPU(chunkId);
				//chunkDT.CPU2Disk(chunkFilePrefix, chunkId);// marker

				//--------------update DT matrix-----------
				// endTime = clock();
				printf("%d\n", 5);
				//DTTime += (double)(endTime - startTime1) / CLOCKS_PER_SEC;

				/*startTime1 = clock();*/
				

				UpdateProbKernelD(WT, chunkDT, document, randState[streamId], streamId, streams[streamId]);
				/*endTime = clock();
				UpdateMTime += (double)(endTime - startTime1) / CLOCKS_PER_SEC;*/
				printf("%d\n", 6);
				/*startTime1 = clock();*/

				//--------------sampling-----------

				/*if (chunkId == 0) {
				document.GPU2CPUEffectiveTokenIndex();
				document.CPU2DiskEffectiveTokenIndex(chunkFilePrefix);
				}
				*/

				//--------------sampling-----------



				//SampleKernelD(WTDen, WT, chunkDT, document, randState[streamId], streamId, streams[streamId]);

				/*endTime = clock();
				samplingTimeD += (double)(endTime - startTime1) / CLOCKS_PER_SEC;*/
				// if (chunkId == 0) {
				// 	document.GPU2CPUTime();
				// 	for (int i = 0; i < GridDim*BlockDim / 32; i++) {
				// 		warpTimeRecord << document.timeRecord[i] << " ";
				// 	}
				// 	warpTimeRecord << "\n";
				// }

				printf("%d\n", 7);
				//WTDen.WTDenGPU2CPU();// marker
				//WTDen.WTDenCPU2Disk(chunkFilePrefix);// marker
				/*startTime1 = clock();*/

				SampleKernelD(WT, chunkDT, document, randState[streamId], streamId, streams[streamId]);
				printf("%d\n", 8);
				/*endTime = clock();*/
				//WT.WTGPU2CPU();// marker
				//WT.CPU2Disk(chunkFilePrefix);// marker

				//--------------sampling-----------

				//	endTime = clock();
				/*samplingTimeS += (double)(endTime - startTime1) / CLOCKS_PER_SEC;

				startTime1 = clock();

				endTime = clock();
				transferTimeGPU2CPU += (double)(endTime - startTime1) / CLOCKS_PER_SEC;*/

				/*startTime1 = clock();*/
				//--------------update chunkWT matrix-----------
//				WT.chunkCPU2GPUCountOffset(chunkId, streamId, streams[streamId]);
//				WT.chunkGPUMemset(streamId, streams[streamId]);
//				UpdateWTDenKernel1(WTDen, WT, document, chunkId, streamId, streams[streamId]);
//				UpdateWTKernel(WT, document, chunkId, streamId, streams[streamId]);
//
//				WT.chunkWTGPU2CPU(chunkId, streamId, streams[streamId]);

				document.GPU2CPU(chunkId, streamId, streams[streamId]);
				if (streamId != 1) hipEventRecord(stopEvents[streamId], streams[streamId]);
				//WT.CPU2DiskChunk(chunkFilePrefix, chunkId);
				//--------------update chunkWT matrix-----------
				//endTime = clock();
				//WTTime += (double)(endTime - startTime1) / CLOCKS_PER_SEC;
				/*return NULL;*/
			}



		}




		//for (int batchId = 0; batchId < chunksPerStream; batchId++) {

		//	for (int streamId = 0; streamId < numStreams; streamId++) {
		//		int chunkId = batchId*numStreams + streamId;
		//		threadBlock[streamId] = thread(thrd_func, ref(document), ref(chunkDT), ref(WT), ref(WTDen), randState, streams, chunkId, streamId);
		//		/*if (pthread_create(&thread[streamId], NULL, thrd_func, ())) {
		//		fprintf(stderr, "Error creating threadn");
		//		return 1;
		//		}*/
		//	}

		//	for (int streamId = 0; streamId < numStreams; streamId++) {
		//		threadBlock[streamId].join();
		//	}

		//}


		for (int streamId = 0; streamId < numStreams; streamId++) {
			if (streamId != 1) {
				//hipEventRecord(stopEvents[streamId], streams[streamId]);
				hipStreamWaitEvent(streams[1], stopEvents[streamId], 0);
			}
			
		}
		hipEventRecord(iterStop,streams[1]);
		hipEventSynchronize(iterStop);
		hipEventElapsedTime(&iterTime,iterStart,iterStop);


		




		/*hipDeviceSynchronize();*/
//		WT.GPUMemCopy(streams[1]);
//		WT.GPUMemset(streams[1]);
		for (int streamId = 0; streamId < numStreams; streamId++) {
			PerplexityKernel(document, streamId, streams[1]);
		}
		

	/*	document.PercentageCalculate();*/

		printf("done!!!!!");
		/*document.GPU2CPUPerplexity();*/

		// document.CPU2DiskPerplexity(chunkFilePrefix);

		//endTime = clock();
		//totalTime=(double)(endTime-startTime)/CLOCKS_PER_SEC;

		//maxPercentRecord << document.increasePercent << " " << document.topicUnchangedPercent << " " << document.perplexityAve[0] << "\n";
		//timeRecord << WTTime << " " << DTTime << " " <<UpdateMTime<< " " <<samplingTimeD << " " << samplingTimeS << " " << transferTimeCPU2GPU << " " << transferTimeGPU2CPU << " " <<totalTime << " " << document.sumPerplexity<< "\n";
		//
		SamplingDRecord << iterTime << "\n";

		//printf("WTTime: %f, DTTime: %f, samplingTimeD:%f, samplingTimeS:%f,transferTimeCPU2GPU:%f,transferTimeGPU2CPU:%f,totalTime:%f\n",WTTime,DTTime,samplingTimeD,samplingTimeS,transferTimeCPU2GPU,transferTimeGPU2CPU,totalTime);

	}

	timeRecord.close();
	warpTimeRecord.close();
	hipDeviceReset();
}
#endif




//
//
//volatile __shared__ int p_input[ShaMemSize];
//volatile __shared__ int p_index[ShaMemSize];
//volatile __shared__ int p_value[ShaMemSize];
//volatile __shared__ int p_index_tmp[ShaMemSize];
//volatile __shared__ int p_value_tmp[ShaMemSize];
////volatile __shared__ int p_dense[K];
//int tid = threadIdx.x;
//int globalId = threadIdx.x + blockIdx.x * blockDim.x;
//int blockId = blockIdx.x;
//int indicator = 0;
//int GridDim = gridDim.x;
//
///*int wordIdWT = blockId + (*d_counter_0)*GridDim ;*/
///*long long tokenStart = d_TokenOffset[wordId];
//long long tokenEnd = d_TokenOffset[wordId] + d_TokenCount[wordId];*/
//
//
//
//if ((blockId > (*d_token_amount_0 - 1 - *d_counter_0*gridDim.x)) || (d_slotcount[blockId + (*d_counter_0)*GridDim] == 0))
//{
//	return;
//}
//int wordId = blockId + (*d_counter_0)*GridDim;
//p_input[tid] = 0;
//p_index[tid] = 0;
//p_value[tid] = 0;
//p_index_tmp[tid] = 0;
//p_value_tmp[tid] = 0;
//for (int k = tid; k < K; k += blockDim.x)
//{
//	d_dense[k + K*blockId] = 0;
//}
//
//__syncthreads();
//
//for (int i = tid; i < ((d_slotcount[wordId] - 1) / blockDim.x + 1)*blockDim.x; i += blockDim.x) {
//	if (i < d_slotcount[wordId]) {
//		int tmpIndex = d_slotoffset[wordId] + i + numOfTokenD;
//		p_input[tid] = d_a[tmpIndex];
//		//atomicAdd(&d_row_sum[p_input[tid] - 1], 1);
//	}
//
//	__syncthreads();
//	radix_sort(p_input);
//	__syncthreads();
//	index_value_count(p_input, p_index, p_value);
//	__syncthreads();
//	if (((d_slotcount[wordId] - indicator*blockDim.x) < blockDim.x) && (tid<(blockDim.x - 1)))
//	{
//		p_index_tmp[tid] = p_index[tid + 1];
//		p_value_tmp[tid] = p_value[tid + 1];
//	}
//	__syncthreads();
//
//	if (((d_slotcount[wordId] - indicator*blockDim.x) < blockDim.x) && (tid<(blockDim.x - 1)))
//	{
//		p_index[tid] = p_index_tmp[tid];
//		p_value[tid] = p_value_tmp[tid];
//	}
//	__syncthreads();
//
//	if (((d_slotcount[wordId] - indicator*blockDim.x) < blockDim.x) && (tid == (blockDim.x - 1)))
//	{
//		p_index[tid] = 0;
//		p_value[tid] = 0;
//	}
//	__syncthreads();
//	if (p_index[tid])
//	{
//		//atomicAdd(&p_dense[p_index[tid] - 1], 1);
//		d_dense[p_index[tid] - 1 + K*blockId] += p_value[tid];
//	}
//	__syncthreads();
//	p_index[tid] = 0;
//	p_value[tid] = 0;
//	p_input[tid] = 0;
//	p_index_tmp[tid] = 0;
//	p_index_tmp[tid] = 0;
//	indicator++;
//	__syncthreads();
//}
//__syncthreads();
///*if (globalId == 0) printf("%d mark\n", *d_counter_0);
//__syncthreads();*/
//dense_sparse_kernel(d_dense, d_index, d_value, d_count, d_slotcount, d_slotoffset, d_counter_0);
//__syncthreads();
//
//
//








