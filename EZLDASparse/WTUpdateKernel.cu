#include "WTUpdateKernel.cuh"
void UpdateWTKernel(WTAll &argWT, Document &argDoc, int argChunkId, int argStreamId, hipStream_t& stream) {

	
	//unsigned int* deviceCounter;
	//hipMalloc(&deviceCounter, sizeof(unsigned int));
	hipMemsetAsync(argDoc.deviceCounterWTUpdateKernel[argStreamId], 0, sizeof(unsigned int),stream);
	/*hipMemcpyAsync(argDoc.deviceCounterWTUpdateKernel[argStreamId], &argDoc.counterWTUpdateKernel, sizeof(unsigned int), hipMemcpyHostToDevice, stream);*/

	/*int numOfTokenD = argDoc.numOfTokenVecD[argChunkId];
	int numOfWordS = argWT.blockCount + argWT.warpCount;*/

	WT_Update_Kernel << <GridDim, BlockDim, 0, stream >> > (argDoc.deviceTLTopic[argStreamId], argDoc.deviceTLWordCount[argStreamId], argDoc.deviceTLWordOffset[argStreamId], argWT.deviceNZWTCount, argWT.deviceWTIndex, argWT.deviceWTValue, argWT.deviceWTCount, argWT.deviceWTOffset, argWT.deviceWTRowSum, argDoc.deviceCounterWTUpdateKernel[argStreamId], argWT.numOfWordS, argDoc.d_dense[argStreamId], argDoc.numOfTokenVecD[argChunkId]);

	//H_ERR(hipDeviceSynchronize());
	
}

void UpdateWTRowSumKernel(WTAll &argWT, hipStream_t& stream)

{
	WTRow_Sum_Update_Kernel<< <GridDim, BlockDim, 0, stream >> > (argWT.deviceNZWTCount, argWT.deviceWTOffset, argWT.deviceWTIndex, argWT.deviceWTValue, argWT.deviceWTRowSum, argWT.wordLength);

}

//void UpdateWTKernel(WTAll &argWT, Document &argDoc, int argChunkId) {
//
//	int iterBlock = (argWT.blockCount - 1) / GridDim + 1;// number of iterations for block.
//														 //int iterBlock = 9;// number of iterations for block.
//	int GridWarpDim = GridDim*BlockDim / 32;
//	int iterAll = (argWT.blockCount - 1) / GridDim + 1 + (argWT.warpCount - 1) / GridWarpDim + 1; // number of total iterations.
//
//	int blockCounter = 0;
//	int warpCounter = 0;
//	int numOfTokenD = argDoc.numOfTokenVecD[argChunkId];
//	for (int i = 0; i < iterAll; i++)
//	{
//		if (i < iterBlock)
//		{
//			H_ERR(hipMemcpy(argDoc.d_blockCounter, &blockCounter, sizeof(int), hipMemcpyHostToDevice));
//			tokenlist_to_matrix << <GridDim, BlockDim >> > (argDoc.deviceTLTopic, argWT.deviceChunkNZWTCount, argWT.deviceChunkWTIndex, argWT.deviceChunkWTValue, argWT.deviceChunkWTCount, argWT.deviceChunkWTOffset, argWT.deviceWTRowSum, argDoc.d_blockCounter, argWT.deviceBlockCount, argDoc.d_dense, numOfTokenD);
//			H_ERR(hipDeviceSynchronize());
//			blockCounter++;
//
//
//		}
//		else
//		{
//			hipMemcpy(argDoc.d_warpCounter, &warpCounter, sizeof(int), hipMemcpyHostToDevice);
//			tokenlist_to_matrix_warp << <GridDim, BlockDim >> > (argDoc.deviceTLTopic, argWT.deviceChunkNZWTCount, argWT.deviceChunkWTIndex, argWT.deviceChunkWTValue, argWT.deviceChunkWTCount, argWT.deviceChunkWTOffset, argWT.deviceWTRowSum, argDoc.d_warpCounter, argWT.deviceBlockCount, argWT.deviceWarpCount, numOfTokenD);
//			/*printf("abc %d", warpCounter);*/
//			H_ERR(hipDeviceSynchronize());
//			warpCounter++;
//		}
//		H_ERR(hipDeviceSynchronize());
//
//
//	}
//
//}
//




