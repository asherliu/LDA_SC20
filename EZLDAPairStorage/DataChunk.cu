#include"DataChunk.cuh"

DocChunk::DocChunk(int argTLLength, int argDocLength, int argWordLength) {

	TLLength = argTLLength;
	docLength = argDocLength;
	wordLength = argWordLength;


}

void DocChunk::CPUMemSet() {

	hipHostMalloc((void**)&TLTopic, TLLength* sizeof(unsigned short int));
	hipHostMalloc((void**)&TLMaxTopic, TLLength * sizeof(unsigned short int));

	hipHostMalloc((void**)&TLDocCount, docLength * sizeof(int));
	hipHostMalloc((void**)&TLDocOffset, docLength * sizeof(int));

	hipHostMalloc((void**)&TLWordCount, wordLength * sizeof(int));
	hipHostMalloc((void**)&TLWordOffset, wordLength * sizeof(int));

	hipHostMalloc((void**)&mapWord2Doc, TLLength * sizeof(int));
	hipHostMalloc((void**)&mapDoc2Word, TLLength * sizeof(int));

	/*TLTopic = new unsigned short int[TLLength];
	TLMaxTopic = new unsigned short int[TLLength];

	TLDocCount = new int[docLength];
	TLDocOffset = new int[docLength];
	TLWordCount = new int[wordLength];
	TLWordOffset = new int[wordLength];
	mapWord2Doc = new int[TLLength];
	mapDoc2Word = new int[TLLength];*/

	memset(TLTopic, 0, TLLength * sizeof(unsigned short int));
	memset(TLMaxTopic, 0, TLLength * sizeof(unsigned short int));

	memset(TLDocCount, 0, docLength * sizeof(int));
	memset(TLDocOffset, 0, docLength * sizeof(int));
	memset(TLWordCount, 0, wordLength * sizeof(int));
	memset(TLWordOffset, 0, wordLength * sizeof(int));
	memset(mapWord2Doc, 0, TLLength * sizeof(int));
	memset(mapDoc2Word, 0, TLLength * sizeof(int));
}


void DocChunk::loadChunk(string argFilePrefix, int argChunkId) {


	chunkId = argChunkId;
	string chunkFolderName = argFilePrefix + "/chunk" + to_string(chunkId);
	CPUMemSet();
	printf("loading chunk %d ...\n", chunkId);

	ifstream TL((chunkFolderName + string("/TL.txt")).c_str(), ios::binary);//Store TL and word2doc map

	ifstream word2DocMap((chunkFolderName + string("/word2DocMap.txt")).c_str(), ios::binary);//Store TL and word2doc map

	ifstream doc2WordMap((chunkFolderName + string("/doc2WordMap.txt")).c_str(), ios::binary);//Store TL and word2doc map



	for (int i = 0; i < TLLength; i++) {

		TL >> TLTopic[i];
		word2DocMap >> mapWord2Doc[i];
		doc2WordMap >> mapDoc2Word[i];
		TLMaxTopic[i] = TLTopic[i];
	}
	TL.close();
	word2DocMap.close();
	doc2WordMap.close();

	ifstream wordCountOffset((chunkFolderName + string("/wordCountOffset.txt")).c_str(), ios::binary);//store Word offset of TL

	for (int i = 0; i < wordLength; i++)
	{
		wordCountOffset >> TLWordCount[i] >> TLWordOffset[i];
	}
	wordCountOffset.close();

	ifstream docCountOffset((chunkFolderName + string("/docCountOffset.txt")).c_str(), ios::binary);//store Doc offset of TL and DT offset

	for (int i = 0; i < docLength; i++)
	{
		docCountOffset >> TLDocCount[i] >> TLDocOffset[i];
	}
	docCountOffset.close();
	printf("chunk %d loaded!...\n", chunkId);

}
