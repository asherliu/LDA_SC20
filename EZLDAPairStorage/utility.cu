#include "hip/hip_runtime.h"
#include "utility.cuh"

#define	BUFF_SIZE_LONG	100000

__device__ struct maxStruct {
	float maxProb=0.0;
	unsigned short int maxK=0;

};
__global__ void WT_Update_Kernel(unsigned short int *d_a, int *d_count, unsigned short int *d_index, unsigned short int *d_value, int *d_slotcount, int *d_slotoffset, int *d_row_sum, unsigned int *d_counter_0, int d_token_amount_0, int *d_dense, int numOfTokenD) {

	int globalId = threadIdx.x + blockIdx.x * blockDim.x;
	int laneId = threadIdx.x % 32;
	int warpId = globalId / 32;
	int iterCounter = 0;
	unsigned int Counter;


	if (laneId == 0) {

		Counter = atomicAdd(&d_counter_0[0], 1);
	}
	Counter = __shfl(Counter, 0);

	while (Counter < d_token_amount_0)
		//while (warpId + iterCounter * gridDim.x*blockDim.x / 32< argD)
	{
		int wordId = Counter;
	
		for (int k = laneId; k < K; k += 32)
		{
			d_dense[k + K*warpId] = 0;
		}

		for (int i = d_slotoffset[wordId] + laneId; i < d_slotoffset[wordId] + d_slotcount[wordId]; i += 32)
		{

			unsigned short int topic = d_a[i+numOfTokenD];
			if ((topic < 1) || (topic > K)) printf("wrong Index:%d", topic);
			atomicAdd(&d_dense[K*warpId + topic - 1], 1);
		}

		int noneZeroCount = 0;
		for (int k = laneId; k < K; k += 32) {
			int value = d_dense[K*warpId + k];
			int flag = value > 0;
			int tmpNoneZeroCount = __popc(__ballot(value));

			if (tmpNoneZeroCount == 0) continue;

			flag += __shfl_up(flag, 1, 32)*(laneId >= 1);
			flag += __shfl_up(flag, 2, 32)*(laneId >= 2);
			flag += __shfl_up(flag, 4, 32)*(laneId >= 4);
			flag += __shfl_up(flag, 8, 32)*(laneId >= 8);
			flag += __shfl_up(flag, 16, 32)*(laneId >= 16);

			if (value) {
				int idx = d_slotoffset[wordId] + noneZeroCount + flag - 1;
				d_index[idx] = k + 1;
				d_value[idx] = value;
			}
			noneZeroCount += tmpNoneZeroCount;

		}
		/*if(laneId==0) d_count[docId] = noneZeroCount;*/
		if (laneId == 0) {
			d_count[wordId] = noneZeroCount;
			Counter = atomicAdd(&d_counter_0[0], 1);
		}
		Counter = __shfl(Counter, 0);

		/*iterCounter ++;*/

	}
	
}
__device__ short atomicAddShort(short* address, short val)

{

    unsigned int *base_address = (unsigned int *)((size_t)address & ~2);

    unsigned int old, assumed, new_;



    old = *base_address;

    do {

        assumed = old;

        new_ = ((size_t)address & 2) ? old + ((unsigned int)val << 16) : (old & 0xffff0000) | ((old & 0xffff) + val);

       old = atomicCAS(base_address, assumed, new_);

    } while (assumed != old);

    return old;

}

__global__ void DT_Update_Kernel(int *d_Index, unsigned short int *d_a, int *d_count, unsigned short int *d_index, int *d_value, int *d_slotcount, int *d_slotoffset, int *d_sparse_slotcount, int *d_sparse_slotoffset, unsigned int *d_counter_0, int argD, int *d_dense, unsigned short int* deviceMaxTokenCount, unsigned short int* deviceMaxTopic, unsigned short int* deviceSecondMaxTopic, unsigned short int* deviceSecondMaxTokenCount)
{

	int globalId = threadIdx.x + blockIdx.x * blockDim.x;
	int laneId = threadIdx.x % 32;
	int warpId = threadIdx.x / 32;
	int iterCounter = 0;
	unsigned int Counter;
	//__shared__ unsigned int DT[K*BlockDim/32];
	__shared__ int DT[K*BlockDim/32];
	__shared__ short int Index[K*BlockDim/32];
	//unsigned short int one=1;

	if (laneId == 0) {

		Counter = atomicAdd(&d_counter_0[0], 1);
	}
	Counter = __shfl(Counter, 0);

	while (Counter < argD)
	//while (warpId + iterCounter * gridDim.x*blockDim.x / 32< argD)
	{
		/*warpId = Counter;*/

		int docId = Counter;

		for (int k = laneId; k < K; k += 32)
		{
			DT[k + K*warpId] = 0;
		}
		// finish0 = clock64();
		// costtime0 += (double)(finish0 - start0);
		

		int numIter=d_slotcount[docId]/64*64;
		//int countLeft=d_slotcount[docId]-numIter;
		
		for (int i = d_slotoffset[docId] + laneId; (i+32)< d_slotoffset[docId] + numIter; i += 64)
		{	
			//start0 = clock64();
			int idx  = __ldg(&d_Index[i]);
			int idx1 = __ldg(&d_Index[i+32]);
			//finish0 = clock64();
			//costtime0 += (double)(finish0 - start0);
			//finish0 = clock64();
			unsigned short int topic = __ldg(&d_a[idx])-1;
			unsigned short int topic1 = __ldg(&d_a[idx1])-1;
			//finish1 = clock64();
			//costtime1 += (double)(finish1 - finish0);
			//if ((topic < 1) || (topic > K)) printf("wrong Index:%d", topic);
			//atomicAddShort(&DT[topic+ K*warpId], 1);
			atomicAdd(&DT[topic+ K*warpId], 1);
			atomicAdd(&DT[topic1+ K*warpId], 1);
			//atomicCAS(&DT[topic+ K*warpId], DT[topic+ K*warpId],  one);
		}
		for (int i = d_slotoffset[docId] + numIter + laneId; i< d_slotoffset[docId] + d_slotcount[docId]; i += 32)
		{	
			//start0 = clock64();
			int idx  = __ldg(&d_Index[i]);
			// int idx1 = __ldg(&d_Index[i+32]);
			//finish0 = clock64();
			//costtime0 += (double)(finish0 - start0);
			//finish0 = clock64();
			unsigned short int topic = __ldg(&d_a[idx])-1;
			// unsigned short int topic1 = __ldg(&d_a[idx1])-1;
			//finish1 = clock64();
			//costtime1 += (double)(finish1 - finish0);
			//if ((topic < 1) || (topic > K)) printf("wrong Index:%d", topic);
			//atomicAddShort(&DT[topic+ K*warpId], 1);
			atomicAdd(&DT[topic+ K*warpId], 1);
			// atomicAdd(&DT[topic1+ K*warpId], 1);
			//atomicCAS(&DT[topic+ K*warpId], DT[topic+ K*warpId],  one);
		}




		//finish1 = clock64();
		// finish1 = clock64();
		// costtime1 += (double)(finish1 - finish0);

		for (int i = d_slotoffset[docId] + laneId; i < d_slotoffset[docId] + d_slotcount[docId]; i += 32)
		{	
			// start0 = clock64();
			unsigned short int topic = __ldg(&deviceMaxTopic[d_Index[i]])-1;
			unsigned short int secondTopic = __ldg(&deviceSecondMaxTopic[d_Index[i]])-1;
			// finish0 = clock64();
			// costtime0 += (double)(finish0 - start0);
			// finish0 = clock64();
			deviceMaxTokenCount[d_Index[i]]= DT[topic + K*warpId];
			deviceSecondMaxTokenCount[d_Index[i]] = DT[secondTopic+ K*warpId];
			// finish1 = clock64();
			// costtime1 += (double)(finish1 - finish0);
		}
		// finish2 = clock64();
		// costtime2 += (double)(finish2 - finish1);

		int noneZeroCount = 0;
		for (int k = laneId; k < K; k += 32) {
			
			int value = DT[k + K*warpId];
			int flag = value > 0;
			
			int tmpNoneZeroCount = __popc(__ballot(value));

			

			if (tmpNoneZeroCount == 0) continue;
			
	
			
			flag += __shfl_up(flag, 1, 32)*(laneId >= 1);
			flag += __shfl_up(flag, 2, 32)*(laneId >= 2);
			flag += __shfl_up(flag, 4, 32)*(laneId >= 4);
			flag += __shfl_up(flag, 8, 32)*(laneId >= 8);
			flag += __shfl_up(flag, 16, 32)*(laneId >= 16);

			//finish2 = clock64();
			// if (value > 0) {

			// 	int idx = d_sparse_slotoffset[docId] + noneZeroCount+ flag-1;
				
			// 	d_index[idx] = k+1;
			// 	d_value[idx] = value;
				

			// }

			if (value > 0) {

				int idx = noneZeroCount+ flag-1+K*warpId;
				
				Index[idx] = k+1;
				DT[idx] = value;
				

			}

			// int idx = d_sparse_slotoffset[docId] + noneZeroCount+ flag-1;
				
			// d_index[idx] = k+1;
			// d_value[idx] = value;

			
			
			noneZeroCount += tmpNoneZeroCount;
			
		}
		/*if(laneId==0) d_count[docId] = noneZeroCount;*/
		if (laneId == 0) {
			d_count[docId] = noneZeroCount;
			Counter = atomicAdd(&d_counter_0[0], 1);
		}
		Counter = __shfl(Counter, 0);


		for (int i = d_sparse_slotoffset[docId] + laneId; i < d_sparse_slotoffset[docId] + noneZeroCount; i += 32){
			int idx=K*warpId + i - d_sparse_slotoffset[docId];
			d_index[i]=Index[idx];
			d_value[i]=DT[idx];
		}

//		finish3 = clock64();
//		costtime3 += (double)(finish3 - finish2);
		
		/*iterCounter ++;*/
		/*iterCounter ++;*/

	}
	//if (threadIdx.x + blockDim.x*blockIdx.x == 0) printf("costtime0,costtime1,costtime2,costtime3:%f,%f,%f,%f\n", costtime0 / (158200000 * 1.0), costtime1 / (158200000 * 1.0),costtime2 / (158200000 * 1.0), costtime3 / (158200000 * 1.0));

}




















__global__ void MaxTopicDense_Update_Kernel(unsigned short int* deviceWordMaxTopic, unsigned short int* deviceWordSecondMaxTopic, unsigned short int* deviceMaxTopic, int *deviceWTDense, int *deviceTLCount, int *deviceTLOffset, int *deviceWTOffset, int numOfWordD, unsigned int* deviceCounter, int *deviceWTRowSum,int wordLength, float beta, unsigned short int* deviceWordThirdMaxTopic, unsigned short int* deviceSecondMaxTopic) {

	int laneId = threadIdx.x % 32;
	int localId = threadIdx.x / 32;
	volatile __shared__ float WTHead[K];
	volatile __shared__ float MaxTree[32];
	volatile __shared__ float MaxWT[3];
	volatile __shared__ unsigned short int MaxKTree[32];
	volatile __shared__ unsigned short int MaxK[3];
	volatile __shared__ unsigned int Counter[1];
	if (threadIdx.x == 0) {
		Counter[0] = atomicAdd(&deviceCounter[0], 1);
	}
	__syncthreads();

	while (Counter[0] < numOfWordD)
	{
		int wordId = Counter[0];
		if (localId == 0) {
			MaxTree[laneId] = 0;
			MaxKTree[laneId] = 0;
		}

		int tokenStart = deviceTLOffset[wordId];
		int tokenEnd = deviceTLOffset[wordId] + deviceTLCount[wordId];
		int WTStart = deviceWTOffset[wordId];
		// Reconstruct dense WT vector from sparse WT matrix
		for (int i = threadIdx.x; i < K; i += blockDim.x)
		{
			WTHead[i] = (deviceWTDense[WTStart + i] + beta) / (deviceWTRowSum[i] + wordLength*beta);
		}
		__syncthreads();
		// Find maxK
		for (int i = localId; i < K / 32; i += blockDim.x / 32) {
			unsigned short int   tmpK = i * 32 + laneId;
			float tmpMax = 0.0;
			float tmpMax1 = 0.0;
			unsigned short int tmpK1 = 0;
			tmpMax = WTHead[tmpK];
			tmpMax1 = __shfl_down(tmpMax, 16);
			tmpK1 = __shfl_down(tmpK, 16);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 8);
			tmpK1 = __shfl_down(tmpK, 8);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 4);
			tmpK1 = __shfl_down(tmpK, 4);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 2);
			tmpK1 = __shfl_down(tmpK, 2);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 1);
			tmpK1 = __shfl_down(tmpK, 1);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax = __shfl(tmpMax, 0);
			tmpK = __shfl(tmpK, 0);
			MaxTree[i] = tmpMax;
			MaxKTree[i] = tmpK;
		}
		__syncthreads();

		if (localId == 0) {

			float tmpMax = 0.0;
			float tmpMax1 = 0.0;
			unsigned short int tmpMaxK = 0;
			unsigned short int tmpMaxK1 = 0;
			tmpMax = MaxTree[laneId];
			tmpMaxK = MaxKTree[laneId];
			tmpMax1 = __shfl_down(tmpMax, 16);
			tmpMaxK1 = __shfl_down(tmpMaxK, 16);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpMaxK = tmpMaxK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 8);
			tmpMaxK1 = __shfl_down(tmpMaxK, 8);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpMaxK = tmpMaxK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 4);
			tmpMaxK1 = __shfl_down(tmpMaxK, 4);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpMaxK = tmpMaxK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 2);
			tmpMaxK1 = __shfl_down(tmpMaxK, 2);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpMaxK = tmpMaxK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 1);
			tmpMaxK1 = __shfl_down(tmpMaxK, 1);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpMaxK = tmpMaxK1;
			}
			tmpMaxK = __shfl(tmpMaxK, 0);
			if (laneId == 0) {
				MaxWT[laneId] = tmpMax;
				MaxK[laneId] = tmpMaxK;
			}

		}
		__syncthreads();
		if (localId == 0) {

			float tmpMax = 0.0;
			float tmpMax1 = 0.0;
			unsigned short int tmpPosition = MaxK[0] / 32;
			unsigned short int tmpK = (tmpPosition) * 32 + laneId;
			unsigned short int tmpK1 = 0;
			

			tmpMax = WTHead[tmpK];
			if (tmpK == MaxK[0]) tmpMax = 0.0;
			tmpMax1 = __shfl_down(tmpMax, 16);
			tmpK1 = __shfl_down(tmpK, 16);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 8);
			tmpK1 = __shfl_down(tmpK, 8);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 4);
			tmpK1 = __shfl_down(tmpK, 4);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 2);
			tmpK1 = __shfl_down(tmpK, 2);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 1);
			tmpK1 = __shfl_down(tmpK, 1);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax = __shfl(tmpMax, 0);
			tmpK = __shfl(tmpK, 0);
			MaxTree[tmpPosition] = tmpMax;
			MaxKTree[tmpPosition] = tmpK;

			tmpMax = 0.0;
			tmpMax1 = 0.0;
			tmpMax = MaxTree[laneId];
			tmpMax1 = __shfl_down(tmpMax, 16);
			tmpK = MaxKTree[laneId];
			tmpK1 = __shfl_down(tmpK, 16);

			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 8);
			tmpK1 = __shfl_down(tmpK, 8);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 4);
			tmpK1 = __shfl_down(tmpK, 4);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 2);
			tmpK1 = __shfl_down(tmpK, 2);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 1);
			tmpK1 = __shfl_down(tmpK, 1);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			if (laneId == 0) {
				MaxWT[1] = tmpMax;
				MaxK[1] = tmpK;
			}
		}
		__syncthreads();

		if (localId == 0) {

			float tmpMax = 0.0;
			float tmpMax1 = 0.0;
			unsigned short int tmpPosition = MaxK[1] / 32;
			unsigned short int tmpK = (tmpPosition) * 32 + laneId;
			unsigned short int tmpK1 = 0;


			tmpMax = WTHead[tmpK];
			if (tmpK == MaxK[0]) tmpMax = 0.0;
			if (tmpK == MaxK[1]) tmpMax = 0.0;
			tmpMax1 = __shfl_down(tmpMax, 16);
			tmpK1 = __shfl_down(tmpK, 16);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 8);
			tmpK1 = __shfl_down(tmpK, 8);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 4);
			tmpK1 = __shfl_down(tmpK, 4);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 2);
			tmpK1 = __shfl_down(tmpK, 2);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 1);
			tmpK1 = __shfl_down(tmpK, 1);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax = __shfl(tmpMax, 0);
			tmpK = __shfl(tmpK, 0);
			MaxTree[tmpPosition] = tmpMax;
			MaxKTree[tmpPosition] = tmpK;

			tmpMax = 0.0;
			tmpMax1 = 0.0;
			tmpMax = MaxTree[laneId];
			tmpMax1 = __shfl_down(tmpMax, 16);
			tmpK = MaxKTree[laneId];
			tmpK1 = __shfl_down(tmpK, 16);

			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 8);
			tmpK1 = __shfl_down(tmpK, 8);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 4);
			tmpK1 = __shfl_down(tmpK, 4);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 2);
			tmpK1 = __shfl_down(tmpK, 2);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 1);
			tmpK1 = __shfl_down(tmpK, 1);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			if (laneId == 0) {
				MaxWT[2] = tmpMax;
				MaxK[2] = tmpK;
			}
		}

		__syncthreads();


		//float WTMax = MaxWT[0];
		//float WTMax2 = MaxWT[1];
		//float WTMax3 = MaxWT[2];
		unsigned short int WTMaxK = MaxK[0];
		unsigned short int WTSecondMaxK = MaxK[1];
		unsigned short int WTThirdMaxK = MaxK[2];
		if (threadIdx.x == 0) {
			deviceWordMaxTopic[wordId] = WTMaxK+1;
			deviceWordSecondMaxTopic[wordId] = WTSecondMaxK+1;
			deviceWordThirdMaxTopic[wordId] = WTThirdMaxK + 1;
		}
		for (int i = tokenStart+ threadIdx.x; i < tokenEnd; i += blockDim.x) {
			deviceMaxTopic[i] = WTMaxK+1;
			deviceSecondMaxTopic[i] = WTSecondMaxK + 1;

		}

		if (threadIdx.x == 0) Counter[0] = atomicAdd(&deviceCounter[0], 1);
		__syncthreads();
	}

}
__global__ void MaxTopicSparse_Update_Kernel(unsigned short int* deviceWordMaxTopic, unsigned short int* deviceWordSecondMaxTopic, unsigned short int* deviceMaxTopic, int *deviceTLCount, int *deviceTLOffset, int *deviceWTOffset, int numOfWordD, unsigned int* deviceCounter, int *deviceWTRowSum, int wordLength, int numOfWordS, int* d_WordListOffset, int* d_SparseWTCount, unsigned short int* d_SparseWTIndex, unsigned short int* d_SparseWTValue, float beta, unsigned short int* deviceWordThirdMaxTopic, unsigned short int* deviceSecondMaxTopic) {


	int laneId = threadIdx.x % 32;
	int localId = threadIdx.x / 32;
	volatile __shared__ float WTHead[K];
	volatile __shared__ float MaxTree[32];
	volatile __shared__ float MaxWT[3];
	volatile __shared__ unsigned short int MaxKTree[32];
	volatile __shared__ unsigned short int MaxK[3];
	volatile __shared__ unsigned int Counter[1];
	if (threadIdx.x == 0) {
		Counter[0] = atomicAdd(&deviceCounter[0], 1);
	}
	__syncthreads();

	while (Counter[0] < numOfWordS)
	{
		int wordId = Counter[0] + numOfWordD;
		if (localId == 0) {
			MaxTree[laneId] = 0;
			MaxKTree[laneId] = 0;
		}

		long long tokenStart = deviceTLOffset[wordId];
		long long tokenEnd = deviceTLOffset[wordId] + deviceTLCount[wordId];
		long long WTStart = d_WordListOffset[wordId] - d_WordListOffset[numOfWordD];
		long long WTEnd = d_WordListOffset[wordId] - d_WordListOffset[numOfWordD] + d_SparseWTCount[wordId - numOfWordD];
		// Reconstruct dense WT vector from sparse WT matrix
		for (int i = threadIdx.x; i < K; i += blockDim.x)
		{
			WTHead[i] = beta / (deviceWTRowSum[i] + wordLength*beta);

		}
		__syncthreads();

		for (int i = threadIdx.x + WTStart; i < WTEnd; i += blockDim.x)
		{
			WTHead[d_SparseWTIndex[i] - 1] = (d_SparseWTValue[i] + beta) / (deviceWTRowSum[d_SparseWTIndex[i] - 1] + wordLength*beta);

		}
		__syncthreads();

		// Find maxK
		for (int i = localId; i < K / 32; i += blockDim.x / 32) {
			unsigned short int   tmpK = i * 32 + laneId;
			float tmpMax = 0.0;
			float tmpMax1 = 0.0;
			unsigned short int tmpK1 = 0;
			tmpMax = WTHead[tmpK];
			tmpMax1 = __shfl_down(tmpMax, 16);
			tmpK1 = __shfl_down(tmpK, 16);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 8);
			tmpK1 = __shfl_down(tmpK, 8);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 4);
			tmpK1 = __shfl_down(tmpK, 4);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 2);
			tmpK1 = __shfl_down(tmpK, 2);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 1);
			tmpK1 = __shfl_down(tmpK, 1);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax = __shfl(tmpMax, 0);
			tmpK = __shfl(tmpK, 0);
			MaxTree[i] = tmpMax;
			MaxKTree[i] = tmpK;
		}
		__syncthreads();

		if (localId == 0) {

			float tmpMax = 0.0;
			float tmpMax1 = 0.0;
			unsigned short int tmpMaxK = 0;
			unsigned short int tmpMaxK1 = 0;
			tmpMax = MaxTree[laneId];
			tmpMaxK = MaxKTree[laneId];
			tmpMax1 = __shfl_down(tmpMax, 16);
			tmpMaxK1 = __shfl_down(tmpMaxK, 16);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpMaxK = tmpMaxK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 8);
			tmpMaxK1 = __shfl_down(tmpMaxK, 8);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpMaxK = tmpMaxK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 4);
			tmpMaxK1 = __shfl_down(tmpMaxK, 4);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpMaxK = tmpMaxK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 2);
			tmpMaxK1 = __shfl_down(tmpMaxK, 2);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpMaxK = tmpMaxK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 1);
			tmpMaxK1 = __shfl_down(tmpMaxK, 1);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpMaxK = tmpMaxK1;
			}
			tmpMaxK = __shfl(tmpMaxK, 0);
			if (laneId == 0) {
				MaxWT[laneId] = tmpMax;
				MaxK[laneId] = tmpMaxK;
			}

		}
		__syncthreads();
		if (localId == 0) {

			float tmpMax = 0.0;
			float tmpMax1 = 0.0;
			unsigned short int tmpPosition = MaxK[0] / 32;
			unsigned short int tmpK = (tmpPosition) * 32 + laneId;
			unsigned short int tmpK1 = 0;


			tmpMax = WTHead[tmpK];
			if (tmpK == MaxK[0]) tmpMax = 0.0;
			tmpMax1 = __shfl_down(tmpMax, 16);
			tmpK1 = __shfl_down(tmpK, 16);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 8);
			tmpK1 = __shfl_down(tmpK, 8);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 4);
			tmpK1 = __shfl_down(tmpK, 4);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 2);
			tmpK1 = __shfl_down(tmpK, 2);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 1);
			tmpK1 = __shfl_down(tmpK, 1);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax = __shfl(tmpMax, 0);
			tmpK = __shfl(tmpK, 0);
			MaxTree[tmpPosition] = tmpMax;
			MaxKTree[tmpPosition] = tmpK;

			tmpMax = 0.0;
			tmpMax1 = 0.0;
			tmpMax = MaxTree[laneId];
			tmpMax1 = __shfl_down(tmpMax, 16);
			tmpK = MaxKTree[laneId];
			tmpK1 = __shfl_down(tmpK, 16);

			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 8);
			tmpK1 = __shfl_down(tmpK, 8);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 4);
			tmpK1 = __shfl_down(tmpK, 4);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 2);
			tmpK1 = __shfl_down(tmpK, 2);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 1);
			tmpK1 = __shfl_down(tmpK, 1);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			if (laneId == 0) {
				MaxWT[1] = tmpMax;
				MaxK[1] = tmpK;
			}
		}
		__syncthreads();


		if (localId == 0) {

			float tmpMax = 0.0;
			float tmpMax1 = 0.0;
			unsigned short int tmpPosition = MaxK[1] / 32;
			unsigned short int tmpK = (tmpPosition) * 32 + laneId;
			unsigned short int tmpK1 = 0;


			tmpMax = WTHead[tmpK];
			if (tmpK == MaxK[0]) tmpMax = 0.0;
			if (tmpK == MaxK[1]) tmpMax = 0.0;
			tmpMax1 = __shfl_down(tmpMax, 16);
			tmpK1 = __shfl_down(tmpK, 16);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 8);
			tmpK1 = __shfl_down(tmpK, 8);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 4);
			tmpK1 = __shfl_down(tmpK, 4);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 2);
			tmpK1 = __shfl_down(tmpK, 2);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 1);
			tmpK1 = __shfl_down(tmpK, 1);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax = __shfl(tmpMax, 0);
			tmpK = __shfl(tmpK, 0);
			MaxTree[tmpPosition] = tmpMax;
			MaxKTree[tmpPosition] = tmpK;

			tmpMax = 0.0;
			tmpMax1 = 0.0;
			tmpMax = MaxTree[laneId];
			tmpMax1 = __shfl_down(tmpMax, 16);
			tmpK = MaxKTree[laneId];
			tmpK1 = __shfl_down(tmpK, 16);

			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 8);
			tmpK1 = __shfl_down(tmpK, 8);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 4);
			tmpK1 = __shfl_down(tmpK, 4);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 2);
			tmpK1 = __shfl_down(tmpK, 2);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 1);
			tmpK1 = __shfl_down(tmpK, 1);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			if (laneId == 0) {
				MaxWT[2] = tmpMax;
				MaxK[2] = tmpK;
			}
		}

		__syncthreads();

	/*	float WTMax = MaxWT[0];
		float WTMax2 = MaxWT[1];
		float WTMax3 = MaxWT[2];*/
		unsigned short int WTMaxK = MaxK[0];
		unsigned short int WTSecondMaxK = MaxK[1];
		unsigned short int WTThirdMaxK = MaxK[2];
		if (threadIdx.x == 0) {
			deviceWordMaxTopic[wordId] = WTMaxK + 1;
			deviceWordSecondMaxTopic[wordId] = WTSecondMaxK + 1;
			deviceWordThirdMaxTopic[wordId] = WTThirdMaxK + 1;
		}
		for (int i = tokenStart + threadIdx.x; i < tokenEnd; i += blockDim.x) {
			deviceMaxTopic[i] = WTMaxK + 1;
			deviceSecondMaxTopic[i] = WTSecondMaxK + 1;
		}
		if (threadIdx.x == 0) Counter[0] = atomicAdd(&deviceCounter[0], 1);
		__syncthreads();

	}

}




__global__ void WTDen_Update_Kernel(unsigned short int *deviceTopic, int *deviceWTDense, int *deviceTLCount, int *deviceTLOffset, int *deviceWTOffset, int numOfWordD, unsigned int* deviceCounter)
{
	int globalId = threadIdx.x + blockIdx.x * blockDim.x;
	int laneId = threadIdx.x % 32;
	int warpId = globalId / 32;
	unsigned int Counter;


	if (laneId == 0) {

		Counter = atomicAdd(&deviceCounter[0], 1);
	}
	Counter = __shfl(Counter, 0);

	while (Counter < numOfWordD)
		
	{
		int wordId = Counter;
		unsigned short int tmpTopic;
		int tmpWTOffset = deviceWTOffset[wordId];
		int tmpTLOffset = deviceTLOffset[wordId];

		for (int k = laneId; k < deviceTLCount[wordId]; k += 32)
		{
			tmpTopic = deviceTopic[tmpTLOffset + k];
			atomicAdd(&deviceWTDense[tmpWTOffset + tmpTopic - 1], 1);
		}

		if (laneId == 0)  Counter = atomicAdd(&deviceCounter[0], 1);
		Counter = __shfl(Counter, 0);

	}



}

__global__ void WTDen_Sum_Update_Kernel(int *deviceWTDense, int *deviceWTRowSum, int *deviceWTOffset, int numOfWordD)
{

	int input;
	int tid = threadIdx.x;
	int globalId = threadIdx.x + blockIdx.x * blockDim.x;
	int blockId = blockIdx.x;
	int tmpIndex;

	for (int k = globalId; k < K; k += GridDim*BlockDim)
	{
		for (int i = 0; i < numOfWordD; i ++)
		{
			tmpIndex = deviceWTOffset[i]  + k;
			deviceWTRowSum[k] += deviceWTDense[tmpIndex];

		}
	}
	__syncthreads();



}





__global__ void sparseMatrixAdd(int* argCount0, int* argOffset0, int* argNZCount0, unsigned short int* argIndex0, unsigned short int* argValue0, int* argCount1, int* argOffset1, int* argNZCount1, unsigned short int* argIndex1, unsigned short int* argValue1, int* argDense, int argNumRows, unsigned int* deviceCounter, int* argWTRowSum, int numOfWordD)
{

	int globalId = threadIdx.x + blockIdx.x * blockDim.x;
	int laneId = threadIdx.x % 32;
	int warpId = globalId / 32;
	int iterCounter = 0;
	unsigned int Counter;

	if (laneId == 0) {

		Counter = atomicAdd(&deviceCounter[0], 1);
	}
	Counter = __shfl(Counter, 0);

	while (Counter < argNumRows)
		//while (warpId + iterCounter * gridDim.x*blockDim.x / 32< argD)
	{
		int wordId = Counter;

		for (int k = laneId; k < K; k += 32)
		{
			argDense[k + K*warpId] = 0;
		}

		for (int k = laneId; k < argNZCount0[wordId]; k += 32)
		{
			int tmpIdx = argOffset0[wordId + numOfWordD] - K*numOfWordD + k;
			argDense[K*warpId + argIndex0[tmpIdx] - 1] += argValue0[tmpIdx];
		}

		for (int k = laneId; k < argNZCount1[wordId]; k += 32)
		{

			int tmpIdx = argOffset1[wordId] + k;
			atomicAdd(&argWTRowSum[argIndex1[tmpIdx] - 1], argValue1[tmpIdx]);
			argDense[K*warpId + argIndex1[tmpIdx] - 1] += argValue1[tmpIdx];
		}
		int noneZeroCount = 0;
		for (int k = laneId; k < K; k += 32) {
			int value = argDense[K*warpId + k];
			int flag = value > 0;
			int tmpNoneZeroCount = __popc(__ballot(value));

			if (tmpNoneZeroCount == 0) continue;

			flag += __shfl_up(flag, 1, 32)*(laneId >= 1);
			flag += __shfl_up(flag, 2, 32)*(laneId >= 2);
			flag += __shfl_up(flag, 4, 32)*(laneId >= 4);
			flag += __shfl_up(flag, 8, 32)*(laneId >= 8);
			flag += __shfl_up(flag, 16, 32)*(laneId >= 16);

			if (value) {
				int idx = argOffset0[wordId + numOfWordD] - K*numOfWordD + noneZeroCount + flag - 1;
				argIndex0[idx] = k + 1;
				argValue0[idx] = value;
			}
			noneZeroCount += tmpNoneZeroCount;

		}

		if (laneId == 0) {
			argNZCount0[wordId] = noneZeroCount;
			Counter = atomicAdd(&deviceCounter[0], 1);
		}
		Counter = __shfl(Counter, 0);


	}



}




__global__ void initRandState(hiprandState *state)
{
	int tid = blockIdx.x*blockDim.x + threadIdx.x;
	hiprand_init(clock() + tid, tid, 0, &state[tid]);
}



__global__ void LDAKernelTrain(float alpha, float beta, int* d_Index, unsigned short int* d_TopicIndex, int* d_SparseDTCount, unsigned short int* d_SparseDTIndex, int* d_SparseDTValue, int* d_TokenCountDT, int* d_TokenOffsetDT, int* d_DocListCount, int* d_DocListOffset, int* d_SparseWTCount, unsigned short int* d_SparseWTIndex, unsigned short int* d_SparseWTValue, int* d_TokenCount, int* d_TokenOffset, int* d_WordListCount, int* d_WordListOffset, int* d_WTRowSum, unsigned int* d_blockCounter, int*d_DocIndex, int D, int W, float* d_Perplexity, hiprandState *randState, float *WTHeadDense, int numOfWordD, int numOfWordS,  unsigned short int* deviceMaxTokenCount, unsigned short int* deviceMaxTopic, unsigned short int* deviceWordMaxTopic, unsigned short int* deviceWordSecondMaxTopic)

{
	int tid = threadIdx.x;
	int laneId = threadIdx.x % 32;
	int localId = threadIdx.x / 32;
	int blockId = blockIdx.x;
	volatile __shared__ float WTHead[K];
	volatile __shared__ float QTree[32];
	volatile __shared__ float WTMax[2];
	volatile __shared__ float STree[ShaMemSize / 32][K / 32];
	volatile __shared__ float prefixSumSample[ShaMemSize / 32][32];
	volatile __shared__ unsigned int Counter[1];
	__shared__ unsigned int WarpCounter[1];

	if (tid == 0) {
		Counter[0] = atomicAdd(&d_blockCounter[0], 1);	
	}
	__syncthreads();

	float sumPerplexity = 0.0;

	while (Counter[0]<numOfWordS)
	{
		int wordId = Counter[0]+ numOfWordD;
		if (localId == 0) {
			QTree[laneId] = 0;

		}
		float p_temp1 = 0.0;
		prefixSumSample[localId][laneId] = 0.0;
		long long tokenStart = d_TokenOffset[wordId];
		long long tokenEnd = d_TokenOffset[wordId] + d_TokenCount[wordId];
		long long WTStart = d_WordListOffset[wordId] - d_WordListOffset[numOfWordD];
		long long WTEnd = d_WordListOffset[wordId] - d_WordListOffset[numOfWordD] + d_SparseWTCount[wordId - numOfWordD];

		unsigned short int maxK = deviceWordMaxTopic[wordId];
		unsigned short int secondMaxK = deviceWordSecondMaxTopic[wordId];

		
		for (int i = tid; i < K; i += blockDim.x)
		{
			WTHead[i] = beta / (d_WTRowSum[i] + W*beta);
			
		}

		__syncthreads();

		for (int i = tid + WTStart; i < WTEnd; i += blockDim.x)
		{
			WTHead[d_SparseWTIndex[i] - 1] = (d_SparseWTValue[i] + beta) / (d_WTRowSum[d_SparseWTIndex[i] - 1] + W*beta);
			
		}
		
		__syncthreads();
		if (threadIdx.x == 0) {
			WTMax[0] = WTHead[maxK - 1];
			WTMax[1] = WTHead[secondMaxK - 1];
			WTHead[maxK - 1] = 0.0;
		}
		__syncthreads();
		

		for (int i = localId; i < K / 32; i += blockDim.x / 32) {
			unsigned short int   tmpK = i * 32 + laneId;
			//__syncthreads();
			float tmpVal = 0.0;
			tmpVal = alpha*WTHead[tmpK];
			tmpVal += __shfl_down(tmpVal, 16);
			tmpVal += __shfl_down(tmpVal, 8);
			tmpVal += __shfl_down(tmpVal, 4);
			tmpVal += __shfl_down(tmpVal, 2);
			tmpVal += __shfl_down(tmpVal, 1);
			tmpVal = __shfl(tmpVal, 0);
			QTree[i] = tmpVal;

			
		}
		__syncthreads();


		if (localId == 0) {

			float value = QTree[laneId];
			value += __shfl_up(value, 1, 32)*(laneId >= 1);
			value += __shfl_up(value, 2, 32)*(laneId >= 2);
			value += __shfl_up(value, 4, 32)*(laneId >= 4);
			value += __shfl_up(value, 8, 32)*(laneId >= 8);
			value += __shfl_up(value, 16, 32)*(laneId >= 16);

			QTree[laneId] = value;



		}



		if (tid == 0) WarpCounter[0] = tokenStart;
		
		__syncthreads();


		float Q = QTree[31];
		int tokenIdx;

		if (laneId == 0)
		{
			tokenIdx = atomicAdd(&WarpCounter[0], 1);

		}
		tokenIdx = __shfl(tokenIdx, 0);

		//unsigned short int maxTopic = deviceMaxTopic[tokenIdx];
		float WTMaxProb = WTMax[0];
		float WTSecondMaxProb = WTMax[1];

		while (tokenIdx<tokenEnd)
		{

			//int docId = __ldg(&d_Index[d_TopicIndex[tokenIdx]]);
			int oldZ=d_TopicIndex[tokenIdx];
			int docId = d_DocIndex[tokenIdx];

			
			unsigned short int maxTokenCount = deviceMaxTokenCount[tokenIdx];
			int totalTokenCount = d_TokenCountDT[docId - 1];
			float maxProbability = (maxTokenCount+alpha)*WTMaxProb;
			float maxS = (totalTokenCount - maxTokenCount)*WTSecondMaxProb;
			float thresProb = maxProbability / (maxProbability + maxS + Q);

			if (maxS < 0) printf("wrong maxS\n");
			float u;
			if (laneId == 0)u = hiprand_uniform(&(randState[threadIdx.x + blockDim.x*blockIdx.x])) / 1.00001;
			u = __shfl(u, 0);

			int newZ = 1;
			unsigned short int sFlag = 1;

			if (u < thresProb) {
		
				newZ = maxK;
				//if (laneId == 0) {
				//	//newZ = maxTopic;
				//	/*atomicAdd(&d_WTDenseCopy[WTStart + newZ - 1], 1);*/
				//	//d_TopicIndex[tokenIdx] = newZ;
				//	tokenIdx = atomicAdd(&WarpCounter[0], 1);
				//}
				//tokenIdx = __shfl(tokenIdx, 0);
				/*continue;*/
				sFlag = 0;
				goto endloop;

			}
			else{
			//computing S.
				float S = 0;
				long long DTStart = d_DocListOffset[docId - 1];
				long long DTEnd = d_DocListOffset[docId - 1] + ((d_SparseDTCount[docId - 1] - 1) / 32 + 1) * 32;

				//long long DTEnd = d_DocListOffset[docId - 1] + d_SparseDTCount[docId - 1];


				STree[localId][laneId] = 0;
				// //__syncthreads();
				int SIdx = 0;
				float  tmpP1k = 0.0;
				int   colVal;
				unsigned short int  colK;
				
				//maxStruct maxValue;
				for (int tmpIdx = DTStart + laneId; tmpIdx < DTEnd - 32; tmpIdx += 32) {

					colVal = d_SparseDTValue[tmpIdx];
					colK = d_SparseDTIndex[tmpIdx];
					tmpP1k = colVal*WTHead[colK - 1];	
					//if (colK == maxTopic) tmpP1k = 0.0;


					tmpP1k += __shfl_down(tmpP1k, 16);
					tmpP1k += __shfl_down(tmpP1k, 8);
					tmpP1k += __shfl_down(tmpP1k, 4);
					tmpP1k += __shfl_down(tmpP1k, 2);
					tmpP1k += __shfl_down(tmpP1k, 1);
					tmpP1k = __shfl(tmpP1k, 0);

					S += tmpP1k;
					STree[localId][SIdx] = S;

					SIdx++;
				}

				tmpP1k = 0.0;
				colVal = d_SparseDTValue[DTEnd - 32 + laneId];
				colK = d_SparseDTIndex[DTEnd - 32 + laneId];
				if (colK != 0) tmpP1k = colVal*WTHead[colK - 1];
				//if (colK == maxTopic) tmpP1k = 0.0;

				tmpP1k += __shfl_down(tmpP1k, 16);
				tmpP1k += __shfl_down(tmpP1k, 8);
				tmpP1k += __shfl_down(tmpP1k, 4);
				tmpP1k += __shfl_down(tmpP1k, 2);
				tmpP1k += __shfl_down(tmpP1k, 1);
				tmpP1k = __shfl(tmpP1k, 0);
				S += tmpP1k;
				STree[localId][SIdx] = S;


				//__syncthreads();
				/*STmp = S;

				S = __shfl(STmp, 0);*/
				S = __shfl(S, 0);
				//__syncthreads();
				//randomly generate u.


				/*if (maxProbability / (maxProbability + S + Q) < thresProb) printf("Wrong Prob!!!!%f,%f,%d\n", maxProbability / (maxProbability + S + Q), thresProb, maxK);*/

				if (u < maxProbability / (maxProbability + S + Q)) {

					newZ = maxK;

				}

				else if ((u >= maxProbability / (maxProbability + S + Q)) && (u< (maxProbability + S) / (maxProbability + S + Q)))
				{
					//float transU = u*(S + Q);
					float transU = u*(maxProbability + S + Q) - maxProbability;

					float tmpSumHigh, tmpSumLow = 0.0;
					tmpSumHigh = STree[localId][laneId];
					tmpSumLow = __shfl_up(tmpSumHigh, 1, 32);
					if (laneId == 0)tmpSumLow = 0;
					int voteFlag = 0;
					if ((transU < tmpSumHigh)) voteFlag = 1;
					int lvl1Idx = __ffs(__ballot(voteFlag)) - 1;
					//int overflowFlag = 0;

					if (lvl1Idx < 0) lvl1Idx = (DTEnd - DTStart) / 32 - 1;
					
					transU = transU - tmpSumLow;

					transU = __shfl(transU, lvl1Idx);
					int tmpIdx = DTStart + lvl1Idx * 32 + laneId;
					int tmpNewZ = d_SparseDTIndex[tmpIdx];
					int colVal = d_SparseDTValue[tmpIdx];
					float p1k = 0.0;
					if (tmpNewZ != 0)
					{
						p1k = colVal*WTHead[tmpNewZ - 1];
					}
					prefixSumSample[localId][laneId] = p1k;
					float value = prefixSumSample[localId][laneId];
					value += __shfl_up(value, 1, 32)*(laneId >= 1);
					value += __shfl_up(value, 2, 32)*(laneId >= 2);
					value += __shfl_up(value, 4, 32)*(laneId >= 4);
					value += __shfl_up(value, 8, 32)*(laneId >= 8);
					value += __shfl_up(value, 16, 32)*(laneId >= 16);
					prefixSumSample[localId][laneId] = value;
					float tmpSum = prefixSumSample[localId][laneId];
					voteFlag = 0;
					if (transU < tmpSum) voteFlag = 1;
					int offset = __ffs(__ballot(voteFlag)) - 1;
					// int tmpoffset=0;
					if (offset<0) offset = 0;

					// tmpoffset=__ldg(&d_SparseDTCount[docId - 1])-lvl1Idx*32-1;
					newZ = __shfl(tmpNewZ, offset);
					// if ((newZ < 1) || (newZ > K)) {
					// 	printf("wrong Index from sampling Dense:%d,%f,%f,%f,%f\n", newZ, u - S / (S + Q),u,S,Q);
					// 	printf("TmpNewZ and offset: %d,%d\n",tmpNewZ,offset);
					// 	printf("transU and tmpSum and voteFlag: %.10f,%.10f,%d\n",transU,tmpSum,voteFlag);
					// }
					if ((newZ == 0) || (newZ > K)) {
						int tmpoffset = d_SparseDTCount[docId - 1] - lvl1Idx * 32 - 1;
						newZ = __shfl(tmpNewZ, tmpoffset);
						// printf("Dense part:NewZ , tmpNewZ and tmpoffset: %d,%d,%d\n",newZ,tmpNewZ,tmpoffset);
					}

				}

				else //bucket Q
				{

					//float transU = (u - S / (S + Q))*(S + Q);

					float transU = (u - (maxProbability + S) / (maxProbability + S + Q))*(maxProbability + S + Q);
					//level 1: decide position
					float tmpSumHigh, tmpSumLow = 0.0;
					tmpSumHigh = QTree[laneId];
					tmpSumLow = __shfl_up(tmpSumHigh, 1, 32);
					if (laneId == 0)tmpSumLow = 0;
					//voting for lvl1Idx
					int voteFlag = 0;
					if (transU < tmpSumHigh) voteFlag = 1;
					int lvl1Idx = __ffs(__ballot(voteFlag)) - 1;
					if (lvl1Idx < 0) lvl1Idx = 31;
					transU = transU - tmpSumLow;
					transU = __shfl(transU, lvl1Idx);
					prefixSumSample[localId][laneId] = alpha*WTHead[32 * lvl1Idx + laneId];
					//accumulation

					float value = prefixSumSample[localId][laneId];
					value += __shfl_up(value, 1, 32)*(laneId >= 1);
					value += __shfl_up(value, 2, 32)*(laneId >= 2);
					value += __shfl_up(value, 4, 32)*(laneId >= 4);
					value += __shfl_up(value, 8, 32)*(laneId >= 8);
					value += __shfl_up(value, 16, 32)*(laneId >= 16);

					prefixSumSample[localId][laneId] = value;

					voteFlag = 0;
					tmpSumLow = 0;
					tmpSumHigh = prefixSumSample[localId][laneId];
					tmpSumLow = __shfl_up(tmpSumHigh, 1, 32);
					if (laneId == 0)tmpSumLow = 0;

					if (transU < tmpSumHigh)voteFlag = 1;
					int lvl2Idx = __ffs(__ballot(voteFlag)) - 1;
					if (lvl2Idx < 0)lvl2Idx = 31;
					newZ = lvl1Idx * 32 + lvl2Idx + 1;

					if ((newZ < 1) || (newZ > K)) {
						printf("wrong Index from sampling Dense else :%d,%f,%f,%f,%f\n", newZ, u - S / (S + Q), u, S, Q);
					}


				}
			}

		endloop:

			if (laneId == 0) {
				d_TopicIndex[tokenIdx] = newZ;
				//deviceMaxTopic[tokenIdx] = newZ;

				/*sumPerplexity += log((S + maxProbability + Q) / (totalTokenCount + K*alpha));*/

				sumPerplexity += 1.0;
				/*if(oldZ==newZ) sumPerplexity += 1.0;*/
				//d_Perplexity[tokenIdx] = log((S + Q) / (d_TokenCountDT[docId - 1] + K*alpha));

				//d_Perplexity[tokenIdx] = 1.0;
				// printf("Perplexity:%f, %d, %d, %d, %d\n",d_Perplexity[tokenIdx],tokenStart,tokenIdx,newZ,wordId);
				// printf("Perplexity: %d\n",tokenStart);

				tokenIdx = atomicAdd(&WarpCounter[0], 1);

				// sumPerplexity += log((S + Q) / (d_TokenCountDT[docId - 1] + K*alpha));

			}


			tokenIdx = __shfl(tokenIdx, 0);

		}

		if (tid == 0) Counter[0] = atomicAdd(&d_blockCounter[0], 1);

		__syncthreads();

	}

	if (laneId == 0) QTree[localId] = sumPerplexity;

	__syncthreads();

	if (localId == 0) {
		float perplexity = 0.0;
		perplexity = QTree[laneId] * (laneId < BlockDim / 32);
		perplexity += __shfl_down(perplexity, 16);
		perplexity += __shfl_down(perplexity, 8);
		perplexity += __shfl_down(perplexity, 4);
		perplexity += __shfl_down(perplexity, 2);
		perplexity += __shfl_down(perplexity, 1);
		if (laneId == 0) d_Perplexity[blockId] += perplexity;
	}


}

__device__ volatile int sem = 0;
__device__ unsigned int subCount = 0;

__device__ void acquire_semaphore(volatile int *lock) {
	while (atomicCAS((int *)lock, 0, 1) != 0);
}

__device__ void release_semaphore(volatile int *lock) {
	*lock = 0;
	__threadfence();
}

__global__ void LDAKernelTrainD(float alpha, float beta, int* d_Index, unsigned short int* d_TopicIndex, int* d_SparseDTCount, unsigned short int* d_SparseDTIndex, int* d_SparseDTValue, int* d_TokenCountDT, int* d_TokenOffsetDT, int* d_DocListCount, int* d_DocListOffset, int* d_WTDense, int* d_WTDenseCopy, int* d_TokenCount, int* d_TokenOffset, int* d_WordListCount, int* d_WordListOffset, int* d_WTRowSum, unsigned int* d_blockCounter, int*d_DocIndex, int D, int W, float* d_Perplexity, hiprandState *randState, float *WTHeadDense, int numOfWordD, unsigned short int* deviceMaxTokenCount, unsigned short int* deviceMaxTopic, unsigned short int* deviceWordMaxTopic, unsigned short int* deviceWordSecondMaxTopic, float* deviceMaxProb, float* deviceThresProb,float* deviceTimeRecord, int tokenSegment, float* deviceRandomfloat, int* deviceEffectiveTokenIndex, int* deviceNewTokenCount)

{
	
	int laneId = threadIdx.x % 32;
	int localId = threadIdx.x / 32;

	volatile __shared__ float WTHead[K];
	volatile __shared__ float QTree[32];



	volatile __shared__ float WTMax[2];

	volatile __shared__ float STree[ShaMemSize / 32][K / 32];
	volatile __shared__ float prefixSumSample[ShaMemSize / 32][32];
	volatile __shared__ unsigned int Counter[1];
	__shared__ unsigned int tokenRegionStart[1];
	volatile __shared__ unsigned int tokenEndFlag[1];
	__shared__ unsigned int WarpCounter[1];

	/*
	clock_t start0, finish0, finish1, finish2, finish3, finish4;
	double costtime0 = 0.0, costtime1 = 0.0, costtime2 = 0.0, costtime3 = 0.0, costtime4 = 0.0, total=0.0;*/
	
	clock_t start0, finish0, finish1;
	double costtime0 = 0.0, costtime1 = 0.0;

	

	if (threadIdx.x == 0)
	{
		acquire_semaphore(&sem);
		tokenEndFlag[0] = 0;
		Counter[0] = d_blockCounter[0];
		unsigned int numRegions = (deviceNewTokenCount[Counter[0]] == 0) ? 0 : ((deviceNewTokenCount[Counter[0]] - 1) / tokenSegment);
		tokenRegionStart[0] = atomicInc(&subCount, numRegions);
		if (subCount == 0) {
			d_blockCounter[0] = d_blockCounter[0] + 1;
			tokenEndFlag[0] = 1;
		}
		release_semaphore(&sem);
	}
	__syncthreads();

	/*if(threadIdx.x==0){
		Counter[0]=atomicAdd(&d_blockCounter[0],1);
	}
	__syncthreads();*/

	float sumPerplexity = 0.0;

	//start0 = clock64();


	while (Counter[0]<numOfWordD)
	{
		/*start0 = clock64();*/

		int wordId = Counter[0] ;
		if (localId == 0) {
			QTree[laneId] = 0;
		}

		prefixSumSample[localId][laneId] = 0.0;
		int tokenStart = d_TokenOffset[wordId]+ tokenRegionStart[0]*tokenSegment;
		int tokenEnd = d_TokenOffset[wordId] + (tokenRegionStart[0]+1) * tokenSegment;
		if (tokenEndFlag[0]) tokenEnd = d_TokenOffset[wordId] + deviceNewTokenCount[wordId];
		int WTStart = d_WordListOffset[wordId];
		unsigned short int maxK = deviceWordMaxTopic[wordId];
		unsigned short int secondMaxK = deviceWordSecondMaxTopic[wordId];

		// Reconstruct dense WT vector from sparse WT matrix
		for (int i = threadIdx.x; i < K; i += blockDim.x)
		{
			WTHead[i] = (d_WTDense[WTStart + i] + beta) / (d_WTRowSum[i] + W*beta);
			//__syncthreads();
		}
		__syncthreads();

		if (threadIdx.x == 0) {
			WTMax[0] = WTHead[maxK - 1];
			WTMax[1] = WTHead[secondMaxK - 1];
			WTHead[maxK - 1] = 0.0;
		}
		__syncthreads();


		for (int i = localId; i < K / 32; i += blockDim.x / 32) {
			short int   tmpK = i * 32 + laneId;
			float tmpVal = 0.0;
			tmpVal = alpha*WTHead[tmpK];
			tmpVal += __shfl_down(tmpVal, 16);
			tmpVal += __shfl_down(tmpVal, 8);
			tmpVal += __shfl_down(tmpVal, 4);
			tmpVal += __shfl_down(tmpVal, 2);
			tmpVal += __shfl_down(tmpVal, 1);
			tmpVal = __shfl(tmpVal, 0);
			QTree[i] = tmpVal;

		}
		__syncthreads();

		if (localId == 0) {

			float value = QTree[laneId];
			value += __shfl_up(value, 1, 32)*(laneId >= 1);
			value += __shfl_up(value, 2, 32)*(laneId >= 2);
			value += __shfl_up(value, 4, 32)*(laneId >= 4);
			value += __shfl_up(value, 8, 32)*(laneId >= 8);
			value += __shfl_up(value, 16, 32)*(laneId >= 16);

			QTree[laneId] = value;

		}

		if (threadIdx.x == 0) WarpCounter[0] = tokenStart;
		__syncthreads();

		//float WTMax = MaxWT[0];
		//float WTMax2 = MaxWT[1];
		//unsigned short int WTMaxK = MaxK[0];

		float Q = QTree[31];
		int tokenIdx;

		if (laneId == 0)
		{
			tokenIdx = atomicAdd(&WarpCounter[0], 1);

		}
		tokenIdx = __shfl(tokenIdx, 0);

		
		// float WTMaxProb = WTMax[0];
		// float WTSecondMaxProb = WTMax[1];

		// for (int tokenIdx = tokenStart + localId; tokenIdx < tokenEnd; tokenIdx += blockDim.x / 32) //iterate over tokens
		// {

		/*finish0 = clock64();
		costtime0 += (double)(finish0 - start0);*/

		while (tokenIdx<tokenEnd)
		{
			////int oldZ = d_TopicIndex[tokenIdx];
		 //   
			////unsigned short int sFlag = 1;
			//start0 = clock64();
			//
			//if (deviceMflag[tokenIdx]) {
			//	if (laneId==0) tokenIdx = atomicAdd(&WarpCounter[0], 1);
			//	tokenIdx = __shfl(tokenIdx, 0);
			//	finish0 = clock64();
			//	costtime0 += (double)(finish0 - start0);
			//	continue;
			//	//newZ = maxK;
			//	//sFlag = 0;
			//	//goto endloop;

			//}


			int tokenNewIdx = deviceEffectiveTokenIndex[tokenIdx];

			unsigned short int newZ = 1;
			int docId = d_DocIndex[tokenNewIdx]-1;
			float maxProbability = deviceMaxProb[tokenNewIdx];
			float thresProb = deviceThresProb[tokenNewIdx];
			float u = deviceRandomfloat[tokenNewIdx];

		/*	if (laneId == 0)u = hiprand_uniform(&(randState[threadIdx.x + blockDim.x*blockIdx.x])) / 1.00001;
			u = __shfl(u, 0);*/

			//computing S.
			float S = 0;

			int DTStart = d_DocListOffset[docId];
			int DTEnd = d_DocListOffset[docId] + ((d_SparseDTCount[docId] - 1) / 32 + 1) * 32;

			STree[localId][laneId] = 0;
			short int SIdx = 0;
			float  tmpP1k = 0.0;
			short int  colVal;
			short int  colK;
			/*finish1 = clock64();
			costtime1 += (double)(finish1 - finish0);*/
			//maxStruct maxValue;
			for (int tmpIdx = DTStart + laneId; tmpIdx < DTEnd - 32; tmpIdx += 32) {

				colVal = d_SparseDTValue[tmpIdx];
				colK = d_SparseDTIndex[tmpIdx];
				tmpP1k = colVal*WTHead[colK - 1];
				//tmpP1k = tmpP1k*(colK != maxTopic);
				//if (colK == maxTopic) tmpP1k = 0.0;
				tmpP1k += __shfl_down(tmpP1k, 16);
				tmpP1k += __shfl_down(tmpP1k, 8);
				tmpP1k += __shfl_down(tmpP1k, 4);
				tmpP1k += __shfl_down(tmpP1k, 2);
				tmpP1k += __shfl_down(tmpP1k, 1);
				tmpP1k = __shfl(tmpP1k, 0);
				S += tmpP1k;
				STree[localId][SIdx] = S;
				SIdx++;
			}

			tmpP1k = 0.0;
			colVal = d_SparseDTValue[DTEnd - 32 + laneId];
			colK = d_SparseDTIndex[DTEnd - 32 + laneId];
			if (colK != 0) tmpP1k = colVal*WTHead[colK - 1];
			//if (colK == maxTopic) tmpP1k = 0.0;
			//tmpP1k = tmpP1k*(colK != maxTopic);

			tmpP1k += __shfl_down(tmpP1k, 16);
			tmpP1k += __shfl_down(tmpP1k, 8);
			tmpP1k += __shfl_down(tmpP1k, 4);
			tmpP1k += __shfl_down(tmpP1k, 2);
			tmpP1k += __shfl_down(tmpP1k, 1);
			tmpP1k = __shfl(tmpP1k, 0);
			S += tmpP1k;
			STree[localId][SIdx] = S;


			//__syncthreads();
			/*STmp = S;

			S = __shfl(STmp, 0);*/
			S = __shfl(S, 0);
			//__syncthreads();
			//randomly generate u.

			float totalProb = maxProbability+S+Q;

			if (maxProbability / totalProb <thresProb) printf("What!!!!%f,%f,%d,%d\n", maxProbability / totalProb, thresProb, maxK - 1,  wordId);

			/*finish2 = clock64();
			costtime2 += (double)(finish2 - finish1);*/

			/*if ((wordId == 40) && (laneId == 0) && (tokenIdx - tokenStart<50)) printf("thresProb: %f,%f,%d,%d,%f,%f\n", thresProb, (maxProbability + alpha*(WTHead[maxTopic - 1])) / (maxProbability + S + Q), maxTokenCount, totalTokenCount, WTHead[maxTopic - 1] / Q, WTMax / Q);*/

			//if (maxProbability / (maxProbability + S + Q) < thresProb) printf("Wrong Prob!!!!%f,%f\n", maxProbability / (maxProbability + S + Q), thresProb);

				

			//if (maxProbability / (maxProbability + S + Q) < thresProb) printf("Wrong Prob!!!!");

			if (u < maxProbability / totalProb) {

				newZ = maxK;

			}
				

			else if ((u>= maxProbability / totalProb) && (u< (maxProbability + S) / totalProb))
			{
				//float transU = u*(S + Q);
				float transU = u*totalProb- maxProbability;
				float tmpSumHigh, tmpSumLow = 0.0;
				tmpSumHigh = STree[localId][laneId];
				tmpSumLow = __shfl_up(tmpSumHigh, 1, 32);
				if (laneId == 0) tmpSumLow = 0;
				int voteFlag = 0;
				if ((transU < tmpSumHigh)) voteFlag = 1;
				int lvl1Idx = __ffs(__ballot(voteFlag)) - 1;

				if (lvl1Idx < 0) lvl1Idx = (DTEnd - DTStart) / 32 - 1;
				//tmpU1 = transU;
				transU = transU - tmpSumLow;
				/*tmpU = transU;*/
				transU = __shfl(transU, lvl1Idx);
				int tmpIdx = DTStart + lvl1Idx * 32 + laneId;
				int tmpNewZ = d_SparseDTIndex[tmpIdx];
				int colVal = d_SparseDTValue[tmpIdx];
				float p1k = 0.0;
				if (tmpNewZ != 0)
				{
					p1k = colVal*WTHead[tmpNewZ - 1];
				}

				//p1k = p1k*(colK != maxTopic);

				prefixSumSample[localId][laneId] = p1k;
				float value = prefixSumSample[localId][laneId];
				value += __shfl_up(value, 1, 32)*(laneId >= 1);
				value += __shfl_up(value, 2, 32)*(laneId >= 2);
				value += __shfl_up(value, 4, 32)*(laneId >= 4);
				value += __shfl_up(value, 8, 32)*(laneId >= 8);
				value += __shfl_up(value, 16, 32)*(laneId >= 16);
				prefixSumSample[localId][laneId] = value;
				float tmpSum = prefixSumSample[localId][laneId];
				voteFlag = 0;
				if (transU < tmpSum) voteFlag = 1;
				int offset = __ffs(__ballot(voteFlag)) - 1;
				// int tmpoffset=0;
				if(offset<0) offset=0;

				// tmpoffset=__ldg(&d_SparseDTCount[docId - 1])-lvl1Idx*32-1;
				newZ = __shfl(tmpNewZ, offset);
				// if ((newZ < 1) || (newZ > K)) {
				// 	printf("wrong Index from sampling Dense:%d,%f,%f,%f,%f\n", newZ, u - S / (S + Q),u,S,Q);
				// 	printf("TmpNewZ and offset: %d,%d\n",tmpNewZ,offset);
				// 	printf("transU and tmpSum and voteFlag: %.10f,%.10f,%d\n",transU,tmpSum,voteFlag);
				// }
				if ((newZ == 0) || (newZ > K)){
					int tmpoffset=d_SparseDTCount[docId]-lvl1Idx*32-1;
					newZ=__shfl(tmpNewZ, tmpoffset);
					// printf("Dense part:NewZ , tmpNewZ and tmpoffset: %d,%d,%d\n",newZ,tmpNewZ,tmpoffset);
				}

			}

			else //bucket Q
			{

				//float transU = (u - S / (S + Q))*(S + Q);

				float transU = (u - (maxProbability + S) / totalProb)*totalProb;
				//level 1: decide position
				float tmpSumHigh, tmpSumLow = 0.0;
				tmpSumHigh = QTree[laneId];
				tmpSumLow = __shfl_up(tmpSumHigh, 1, 32);
				if (laneId == 0)tmpSumLow = 0;
				//voting for lvl1Idx
				int voteFlag = 0;
				if (transU < tmpSumHigh) voteFlag = 1;
				int lvl1Idx = __ffs(__ballot(voteFlag)) - 1;
				if (lvl1Idx < 0) lvl1Idx = 31;
				/*tmpU1 = transU;*/
				transU = transU - tmpSumLow;
				/*tmpU = transU;*/
				transU = __shfl(transU, lvl1Idx);
				prefixSumSample[localId][laneId] = alpha*WTHead[32 * lvl1Idx + laneId];
				//accumulation

				float value = prefixSumSample[localId][laneId];
				value += __shfl_up(value, 1, 32)*(laneId >= 1);
				value += __shfl_up(value, 2, 32)*(laneId >= 2);
				value += __shfl_up(value, 4, 32)*(laneId >= 4);
				value += __shfl_up(value, 8, 32)*(laneId >= 8);
				value += __shfl_up(value, 16, 32)*(laneId >= 16);

				prefixSumSample[localId][laneId] = value;

				voteFlag = 0;
				tmpSumLow = 0;
				tmpSumHigh = prefixSumSample[localId][laneId];
				tmpSumLow = __shfl_up(tmpSumHigh, 1, 32);
				if (laneId == 0)tmpSumLow = 0;

				if (transU < tmpSumHigh)voteFlag = 1;
				int lvl2Idx = __ffs(__ballot(voteFlag)) - 1;
				if (lvl2Idx < 0)lvl2Idx = 31;
				newZ = lvl1Idx * 32 + lvl2Idx + 1;

				if ((newZ < 1) || (newZ > K)) {
					printf("wrong Index from sampling Dense else :%d,%f,%f,%f,%f\n", newZ, u - S / (S + Q),u,S,Q);
				}

			}

			if (laneId == 0) {
				d_TopicIndex[tokenNewIdx] = newZ;
				//deviceMaxTopic[tokenIdx] = newZ;

				atomicAdd(&d_WTDenseCopy[WTStart + newZ - 1], 1);

				//p_temp = S + Q;
				// d_S[tokenIdx] = Q;

				/*sumPerplexity+= log(totalProb / (d_TokenCountDT[docId] + K*alpha));*/
				sumPerplexity += 1.0;

				//if(oldZ==newZ) sumPerplexity += 1.0;

				//d_Perplexity[tokenIdx] = log((S + Q) / (d_TokenCountDT[docId - 1] + K*alpha));

				//d_Perplexity[tokenIdx] = 1.0;
				// printf("Perplexity:%f, %d, %d, %d, %d\n",d_Perplexity[tokenIdx],tokenStart,tokenIdx,newZ,wordId);
				// printf("Perplexity: %d\n",tokenStart);

				tokenIdx = atomicAdd(&WarpCounter[0], 1);

				// sumPerplexity += log((S + Q) / (d_TokenCountDT[docId - 1] + K*alpha));

			}

		//	// if(laneId==0) 
		//	// {

		//	// 	__threadfence_block();
		//	// }
		tokenIdx = __shfl(tokenIdx, 0);
		//	finish1 = clock64();
		//	costtime1 += (double)(finish1 - start0);


		//	/*finish4 = clock64();
		//	costtime4 += (double)(finish4 - finish3);*/
	
		//		/*finish3 = clock64();
		//		costtime3 += (double)(finish3 - finish2);*/

		///*endloop:*/
		//	/*finish3 = clock64();*/
		//	


	        
		}

		/*if(threadIdx.x==0) Counter[0]=atomicAdd(&d_blockCounter[0],1);

		__syncthreads();*/
		if (threadIdx.x == 0)
		{
			acquire_semaphore(&sem);
			tokenEndFlag[0] = 0;
			Counter[0] = d_blockCounter[0];
			unsigned int numRegions = (deviceNewTokenCount[Counter[0]] == 0) ? 0 : ((deviceNewTokenCount[Counter[0]] - 1) / tokenSegment);
			tokenRegionStart[0] = atomicInc(&subCount, numRegions);
			if (subCount == 0) {
				d_blockCounter[0] = d_blockCounter[0] + 1;
				tokenEndFlag[0] = 1;
			}
			release_semaphore(&sem);
		}
		__syncthreads();
		//if (localId == 0) {
		//	unsigned short int topic = 0;
		//	topic = maxTopicVec[laneId] * (laneId < BlockDim / 32);
		//	topic += __shfl_down(topic, 16);
		//	topic += __shfl_down(topic, 8);
		//	topic += __shfl_down(topic, 4);
		//	topic += __shfl_down(topic, 2);
		//	topic += __shfl_down(topic, 1);
		//	if (laneId == 0) atomicAdd(&d_WTDenseCopy[WTStart + MaxK[0]], topic);
		//}
		//__syncthreads();

	}

	if (laneId == 0) QTree[localId] = sumPerplexity;
	__syncthreads();
	
	if (localId == 0) {
		float perplexity = 0.0;
		perplexity = QTree[laneId] * (laneId < BlockDim / 32);
		perplexity += __shfl_down(perplexity, 16);
		perplexity += __shfl_down(perplexity, 8);
		perplexity += __shfl_down(perplexity, 4);
		perplexity += __shfl_down(perplexity, 2);
		perplexity += __shfl_down(perplexity, 1);
		if (laneId == 0) d_Perplexity[blockIdx.x] += perplexity;
	}

	/*finish0 = clock64();
	costtime0 = (double)(finish0 - start0);*/

	//if (threadIdx.x + blockDim.x*blockIdx.x == 0) printf("costtime0,costtime1:%f,%f", costtime0 / (158200000 * 1.0), costtime1 / (158200000 * 1.0)); 

	/*deviceTimeRecord[(threadIdx.x + blockDim.x*blockIdx.x)/32] = costtime0 / (158200000 * 1.0);

	if (threadIdx.x+blockDim.x*blockIdx.x == 0) printf("costtime0,costtime1,costtime2,costtime3,costtime4,total:%f,%f,%f,%f,%f,%f", costtime0/(158200000*1.0), costtime1 / (158200000 * 1.0), costtime2 / (158200000 * 1.0), costtime3 / (158200000 * 1.0), costtime4 / (158200000 * 1.0), (costtime0 + costtime1+ costtime2+ costtime3+ costtime4)/ (158200000 * 1.0));*/
	//if (threadIdx.x % 32 == 0)
	//	d_Perplexity[(threadIdx.x + blockDim.x*blockIdx.x) / 32] = sumPerplexity;
	////wordPerplexity[(threadIdx.x + blockDim.x*blockIdx.x) / 32] = sumPerplexity;
	//__syncthreads();

}


__global__ void LDATrainPerplexityReduce(float *perplexity,float numOfTokens,float* devicePerplexityAve) {

	int tid = threadIdx.x;
	int laneId = threadIdx.x % 32;
	int localId = threadIdx.x / 32;
	
	float S = 0.0;
	volatile __shared__ float perplexityMid[32];
	for (int i = tid; i < GridDim; i += BlockDim) {

		float tmpPerplexity = 0.0;
		tmpPerplexity = perplexity[i];
		tmpPerplexity += __shfl_down(tmpPerplexity, 16);
		tmpPerplexity += __shfl_down(tmpPerplexity, 8);
		tmpPerplexity += __shfl_down(tmpPerplexity, 4);
		tmpPerplexity += __shfl_down(tmpPerplexity, 2);
		tmpPerplexity += __shfl_down(tmpPerplexity, 1);
		S += tmpPerplexity;
	}
	if (laneId == 0) perplexityMid[localId] = S;
	__syncthreads();
	if (localId == 0) {
		float AveragePerplexity = 0.0;
		S = 0.0;
		S = perplexityMid[laneId] * (laneId < BlockDim / 32);
		//printf("\nS=:%f\n", S);
		S += __shfl_down(S, 16);
		S += __shfl_down(S, 8);
		S += __shfl_down(S, 4);
		S += __shfl_down(S, 2);
		S += __shfl_down(S, 1);

		
		if (laneId == 0)
		{
			AveragePerplexity = S / numOfTokens; 
			devicePerplexityAve[0]=AveragePerplexity;
			printf("\nAverage Perplexity:%f\n", AveragePerplexity);
		}
		
		
	}



}





__global__ void LDATrainPerplexityReduce1(float *perplexity, float *perplexityMid, int numVals) {


	int numWarps = gridDim.x*blockDim.x / 32;
	int tid = threadIdx.x + blockIdx.x*blockDim.x;
	int warpId = tid / 32;
	int laneId = tid % 32;

	int perWarpSize = ((numVals + numWarps - 1) / numWarps + 31) / 32 * 32;
	int perWarpSizeMax = (numVals + numWarps - 1) / numWarps;
	int startIdx = perWarpSizeMax*warpId;
	int endIdx = perWarpSizeMax*warpId + perWarpSize;
	int endMax = perWarpSizeMax*warpId + perWarpSizeMax;
	
	float totalProd = 0.0;
	for (long long i = startIdx + laneId; i < endIdx; i += 32) {

		float tmpProd = 0.0;
		if ((i < numVals) && (i < endMax))tmpProd = perplexity[i];

		tmpProd += __shfl_down(tmpProd, 16);
		tmpProd += __shfl_down(tmpProd, 8);
		tmpProd += __shfl_down(tmpProd, 4);
		tmpProd += __shfl_down(tmpProd, 2);
		tmpProd += __shfl_down(tmpProd, 1);
		tmpProd = __shfl(tmpProd, 0);
		totalProd += tmpProd;
		//__syncthreads();
	}
	__syncthreads();
	if (laneId == 0) perplexityMid[warpId] += totalProd;

}



__global__ void UpdateProbKernelTrainD(float alpha, float beta, int* d_Index, unsigned short int* d_TopicIndex, int* d_SparseDTCount, unsigned short int* d_SparseDTIndex, int* d_SparseDTValue, int* d_TokenCountDT, int* d_TokenOffsetDT, int* d_DocListCount, int* d_DocListOffset, int* d_WTDense, int* d_WTDenseCopy, int* d_TokenCount, int* d_TokenOffset, int* d_WordListCount, int* d_WordListOffset, int* d_WTRowSum, unsigned int* d_blockCounter, int*d_DocIndex, int D, int W, float* d_Perplexity, hiprandState *randState, float *WTHeadDense, int numOfWordD, unsigned short int* deviceMaxTokenCount, unsigned short int* deviceMaxTopic, unsigned short int* deviceWordMaxTopic, unsigned short int* deviceWordSecondMaxTopic,float* deviceMaxProb, float* deviceThresProb, unsigned short int* deviceSecondMaxTokenCount, unsigned short int* deviceWordThirdMaxTopic, float* deviceRandomfloat,  int* deviceEffectiveTokenIndex, int* deviceNewTokenCount)

{

	int laneId = threadIdx.x % 32;
	int localId = threadIdx.x / 32;
	volatile __shared__ float WTHead[K];
	volatile __shared__ float QTree[32];
	volatile __shared__ float WTMax[3];
	volatile __shared__ unsigned int Counter[1];
	__shared__ unsigned int WarpCounter[1];

	if (threadIdx.x == 0) {
		Counter[0] = atomicAdd(&d_blockCounter[0], 1);
	}
	__syncthreads();
	float sumPerplexity = 0.0;

	while (Counter[0]<numOfWordD)
	{
		int wordId = Counter[0];
		if (localId == 0) {
			QTree[laneId] = 0;
		}
		int tokenStart = d_TokenOffset[wordId];
		int tokenEnd = d_TokenOffset[wordId] + d_TokenCount[wordId];
		int WTStart = d_WordListOffset[wordId];
		unsigned short int maxK = deviceWordMaxTopic[wordId];
		unsigned short int secondMaxK = deviceWordSecondMaxTopic[wordId];
		unsigned short int thirdMaxK = deviceWordThirdMaxTopic[wordId];
		// Reconstruct dense WT vector from sparse WT matrix
		for (int i = threadIdx.x; i < K; i += blockDim.x)
		{
			WTHead[i] = (d_WTDense[WTStart + i] + beta) / (d_WTRowSum[i] + W*beta);
			//__syncthreads();
		}
		__syncthreads();

		if (threadIdx.x == 0) {
			WTMax[0] = WTHead[maxK - 1];
			WTMax[1] = WTHead[secondMaxK - 1];
			WTMax[2] = WTHead[thirdMaxK - 1];
			WTHead[maxK - 1] = 0.0;
			//WTHead[secondMaxK - 1] = 0.0;
		}
		__syncthreads();


		for (int i = localId; i < K / 32; i += blockDim.x / 32) {
			unsigned short int   tmpK = i * 32 + laneId;
			float tmpVal = 0.0;
			tmpVal = alpha*WTHead[tmpK];
			tmpVal += __shfl_down(tmpVal, 16);
			tmpVal += __shfl_down(tmpVal, 8);
			tmpVal += __shfl_down(tmpVal, 4);
			tmpVal += __shfl_down(tmpVal, 2);
			tmpVal += __shfl_down(tmpVal, 1);
			tmpVal = __shfl(tmpVal, 0);
			QTree[i] = tmpVal;

		}
		__syncthreads();

		if (localId == 0) {

			float value = QTree[laneId];
			value += __shfl_up(value, 1, 32)*(laneId >= 1);
			value += __shfl_up(value, 2, 32)*(laneId >= 2);
			value += __shfl_up(value, 4, 32)*(laneId >= 4);
			value += __shfl_up(value, 8, 32)*(laneId >= 8);
			value += __shfl_up(value, 16, 32)*(laneId >= 16);

			QTree[laneId] = value;

		}
		if (threadIdx.x == 0) WarpCounter[0] = 0;
		__syncthreads();
		float Q = QTree[31];
		int tokenIdx;
		float WTMaxProb = WTMax[0];
		float WTSecondMaxProb = WTMax[1];
		float WTThirdMaxProb = WTMax[2];
		
		
		for (int tokenIdx = tokenStart + threadIdx.x; tokenIdx < tokenEnd; tokenIdx += blockDim.x)  
		{
			
			int docId = d_DocIndex[tokenIdx]-1;
			int nonSkipTokenIdx;
			float u = hiprand_uniform(&(randState[threadIdx.x + blockDim.x*blockIdx.x])) / 1.00001;
			deviceRandomfloat[tokenIdx] = u;
			unsigned short int maxTokenCount = deviceMaxTokenCount[tokenIdx];
			unsigned short int maxSecondTokenCount = deviceSecondMaxTokenCount[tokenIdx];
			int totalTokenCount = d_TokenCountDT[docId];
			deviceMaxProb[tokenIdx] = (maxTokenCount + alpha)*WTMaxProb;
			//float maxS = (totalTokenCount - maxTokenCount)*WTSecondMaxProb;
			float maxS = (totalTokenCount - maxTokenCount - maxSecondTokenCount)*WTThirdMaxProb + maxSecondTokenCount*WTSecondMaxProb;
			deviceThresProb[tokenIdx] = deviceMaxProb[tokenIdx] / (deviceMaxProb[tokenIdx] + maxS + Q);
			if (u < deviceThresProb[tokenIdx]) {

				
				d_TopicIndex[tokenIdx] = maxK;
				atomicAdd(&d_WTDenseCopy[WTStart + maxK - 1], 1);
				//sumPerplexity += 1;
			}
			else {
				nonSkipTokenIdx = atomicAdd(&deviceNewTokenCount[wordId], 1);
				deviceEffectiveTokenIndex[nonSkipTokenIdx + tokenStart] = tokenIdx;
			}
		}
		if (threadIdx.x == 0) Counter[0] = atomicAdd(&d_blockCounter[0], 1);
		__syncthreads();

		/*if (threadIdx.x == 0) deviceNewTokenCount[wordId] = WarpCounter[0];
		__syncthreads();
*/
	}

	sumPerplexity += __shfl_down(sumPerplexity, 16);
	sumPerplexity += __shfl_down(sumPerplexity, 8);
	sumPerplexity += __shfl_down(sumPerplexity, 4);
	sumPerplexity += __shfl_down(sumPerplexity, 2);
	sumPerplexity += __shfl_down(sumPerplexity, 1);
	sumPerplexity = __shfl(sumPerplexity, 0);

	if (laneId == 0) QTree[localId] = sumPerplexity;
	__syncthreads();

	if (localId == 0) {
		float perplexity = 0.0;
		perplexity = QTree[laneId] * (laneId < BlockDim / 32);
		perplexity += __shfl_down(perplexity, 16);
		perplexity += __shfl_down(perplexity, 8);
		perplexity += __shfl_down(perplexity, 4);
		perplexity += __shfl_down(perplexity, 2);
		perplexity += __shfl_down(perplexity, 1);
		if (laneId == 0) d_Perplexity[blockIdx.x] += perplexity;
	}

}






