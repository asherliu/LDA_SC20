
#include "DTUpdateKernel.cuh"
void UpdateDTKernel(DTChunk &argDT,Document &argDoc, int argStreamId, hipStream_t& stream) {


	/*unsigned int* deviceCounter;
	hipMalloc(&deviceCounter, sizeof(unsigned int));*/
	hipMemsetAsync(argDoc.deviceCounterDTUpdateKernel[argStreamId], 0, sizeof(unsigned int), stream);

	/*hipMemcpyAsync(argDoc.deviceCounterDTUpdateKernel[argStreamId], &argDoc.counterDTUpdateKernel, sizeof(unsigned int),hipMemcpyHostToDevice, stream);*/

	DT_Update_Kernel << <GridDim, BlockDim, 0, stream >> > (argDoc.deviceMapWord2Doc[argStreamId], argDoc.deviceTLTopic[argStreamId],  argDT.deviceNZDTCount[argStreamId], argDT.deviceDTIndex[argStreamId], argDT.deviceDTValue[argStreamId], argDoc.deviceTLDocCount[argStreamId], argDoc.deviceTLDocOffset[argStreamId], argDT.deviceDTCount[argStreamId], argDT.deviceDTOffset[argStreamId], argDoc.deviceCounterDTUpdateKernel[argStreamId], argDT.docLengthVec[argDT.chunkId], argDoc.d_dense[argStreamId], argDoc.deviceMaxTokenCount[argStreamId], argDoc.deviceMaxTopic[argStreamId], argDoc.deviceSecondMaxTopic[argStreamId], argDoc.deviceSecondMaxTokenCount[argStreamId]);
	/*H_ERR(hipDeviceSynchronize());*/
	
}

