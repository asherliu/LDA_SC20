
#include "DTUpdateKernel.cuh"
void UpdateDTKernel(DTChunk &argDT,Document &argDoc) {

	int blockCounter = 0;

	int iterDT = (argDT.docLengthVec[argDT.chunkId] - 1) / GridDim + 1;
	for (int i = 0; i < iterDT; i++) {
		hipMemcpy(argDoc.d_blockCounter, &blockCounter, sizeof(int), hipMemcpyHostToDevice);
		DT_Update_Kernel << <GridDim, BlockDim >> > (argDoc.deviceMapWord2Doc, argDoc.deviceTLTopic,  argDT.deviceNZDTCount, argDT.deviceDTIndex, argDT.deviceDTValue, argDoc.deviceTLDocCount, argDoc.deviceTLDocOffset, argDT.deviceDTCount, argDT.deviceDTOffset, argDoc.d_blockCounter, argDT.docLengthVec[argDT.chunkId], argDoc.d_dense);
		
		hipDeviceSynchronize();
		blockCounter++;
	}
	
}

