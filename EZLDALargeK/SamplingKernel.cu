


#include "SamplingKernel.cuh"

void SampleKernelD(WTD &argWTDen, WTAll &argWT, DTChunk &argDT, Document &argDoc)
{

	int blockCounter = 0;
	int iterWT = (argWTDen.numOfWordD - 1) / GridDim + 1;
	float Perplexity = 0.0;
	srand(time(NULL));

	hiprandState* randState;
	hipMalloc(&randState, sizeof(hiprandState)*GridDim*BlockDim);
	initRandState << <GridDim, BlockDim >> >(randState);

	for (int i = 0; i < iterWT; i++) {

		hipMemcpy(argDoc.d_blockCounter, &blockCounter, sizeof(int), hipMemcpyHostToDevice);

		LDAKernelTrainD << <GridDim, BlockDim >> > (alpha, beta, argDoc.deviceMapWord2Doc, argDoc.deviceTLTopic, argDT.deviceNZDTCount, argDT.deviceDTIndex, argDT.deviceDTValue, argDoc.deviceTLDocCount, argDoc.deviceTLDocOffset, argDT.deviceDTCount, argDT.deviceDTOffset, argWTDen.deviceWTDense, argWTDen.deviceWTDenseCopy, argDoc.deviceTLWordCount, argDoc.deviceTLWordOffset, argWT.deviceWTCount, argWT.deviceWTOffset, argWT.deviceWTRowSum, argDoc.d_blockCounter, argDoc.deviceMapDoc2Word, argDoc.docLengthVec[argDT.chunkId], argWT.wordLength, argDoc.devicePerplexity, randState, argDoc.deviceWTHeadDense, argWTDen.numOfWordD);
		H_ERR(hipDeviceSynchronize());
		blockCounter++;

	}
	
	H_ERR(hipDeviceSynchronize());

}
//(double alpha, double beta, int* d_Index, int* d_TopicIndex, int* d_SparseDTCount, int* d_SparseDTIndex, int* d_SparseDTValue, int* d_TokenCountDT, int* d_TokenOffsetDT, int* d_DocListCount, int* d_DocListOffset, int* d_WTDense, int* d_WTDenseCopy, int* d_TokenCount, int* d_TokenOffset, int* d_WordListCount, int* d_WordListOffset, int* d_WTRowSum, int* d_blockCounter, int*d_DocIndex, int D, int W, double* d_Perplexity, hiprandState *randState, double *WTHeadDense, int numOfWordD);


void SampleKernel(WTAll &argWT, DTChunk &argDT, Document &argDoc) {

	int blockCounter = 0;
	int iterWT = (argWT.numOfWordS - 1) / GridDim + 1;
	float Perplexity = 0.0;
	int numOfWordD = argWT.wordLength- argWT.numOfWordS;
	srand(time(NULL));

	hiprandState* randState;
	hipMalloc(&randState, sizeof(hiprandState)*GridDim*BlockDim);
	initRandState << <GridDim, BlockDim >> >(randState);

	for (int i = 0; i < iterWT; i++) {

		hipMemcpy(argDoc.d_blockCounter, &blockCounter, sizeof(int), hipMemcpyHostToDevice);

		LDAKernelTrain << <GridDim, BlockDim >> > (alpha, beta, argDoc.deviceMapWord2Doc, argDoc.deviceTLTopic, argDT.deviceNZDTCount, argDT.deviceDTIndex, argDT.deviceDTValue, argDoc.deviceTLDocCount, argDoc.deviceTLDocOffset, argDT.deviceDTCount, argDT.deviceDTOffset, argWT.deviceNZWTCount, argWT.deviceWTIndex, argWT.deviceWTValue, argDoc.deviceTLWordCount, argDoc.deviceTLWordOffset, argWT.deviceWTCount, argWT.deviceWTOffset, argWT.deviceWTRowSum, argDoc.d_blockCounter, argDoc.deviceMapDoc2Word, argDoc.docLengthVec[argDT.chunkId], argWT.wordLength, argDoc.devicePerplexity, randState, argDoc.deviceWTHeadDense, numOfWordD, argWT.numOfWordS);
		H_ERR(hipDeviceSynchronize());
		blockCounter++;

	}
	LDATrainPerplexityReduce1 << <GridDim, BlockDim >> > (argDoc.devicePerplexity, argDoc.devicePerplexityMid, argDoc.TLLengthVec[argDT.chunkId]);

	H_ERR(hipDeviceSynchronize());


}








