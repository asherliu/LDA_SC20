#include "hip/hip_runtime.h"
#include "utility.cuh"

#define	BUFF_SIZE_LONG	100000


__device__ void prefix_sum(volatile int* p) {

	int tid = threadIdx.x;
	volatile __shared__ int p_temp[ShaMemSize + 1];
	int pTmp = 0;

	if (tid == blockDim.x - 1)
	{
		p_temp[tid + 1] = 0;
	}
	__syncthreads();
	p_temp[tid] = p[tid];
	__syncthreads();
	for (int i = 0; i <__ffs(blockDim.x) - 1; i++)
	{
		int step = 1 << (i + 1);
		int index = (tid + 1)*step - 1;
		pTmp = 0;
		__syncthreads();
		if (index < blockDim.x) {
			pTmp = p_temp[index - step / 2];
		}
		__syncthreads();
		if (index < blockDim.x) {
			p_temp[index] += pTmp;
		}
		__syncthreads();


	}
	__syncthreads();
	if (tid == blockDim.x - 1)
	{
		pTmp = p_temp[tid];
		p_temp[tid + 1] = pTmp;
		p_temp[tid] = 0;
	}
	__syncthreads();
	for (int j = 0; j <__ffs(blockDim.x) - 1; j++)
	{
		int step = blockDim.x >> j;
		int index = (tid + 1)*step - 1;
		pTmp = 0;
		__syncthreads();
		if (index < blockDim.x) {
			pTmp = p_temp[index - step / 2];
		}
		__syncthreads();
		if (index < blockDim.x) {
			p_temp[index] += pTmp;
		}
		__syncthreads();
		pTmp = 0;
		__syncthreads();
		if (index < blockDim.x) {
			//p_temp[index] += p_temp[index - step / 2];
			pTmp = p_temp[index];
		}
		__syncthreads();

		if (index < blockDim.x) {
			//p_temp[index] += p_temp[index - step / 2];
			p_temp[index - step / 2] = pTmp - p_temp[index - step / 2];
		}
		__syncthreads();

	}
	__syncthreads();
	p[tid] = p_temp[tid + 1];
	__syncthreads();
	//p_temp[tid] = 0;
	/*if (tid == blockDim.x - 1)
	{
	p_temp[tid + 1] = 0;

	}
	__syncthreads();*/

}


__device__ void radix_sort(volatile int* p) {
	volatile __shared__ int p_input[ShaMemSize];
	volatile __shared__ int p_split[ShaMemSize];
	volatile __shared__ int p_split_inverse[ShaMemSize];
	volatile __shared__ int p_t[ShaMemSize];
	volatile __shared__ int p_index[ShaMemSize];
	int pTmp = 0;
	int tid = threadIdx.x;

	p_input[tid] = 0;
	p_split[tid] = 0;
	p_split_inverse[tid] = 0;
	p_t[tid] = 0;
	p_index[tid] = 0;
	p_input[tid] = p[tid];
	__syncthreads();
	for (int i = 0; i < 32; i++) {
		int p_totalFalses = 0;
		p_split[tid] = (p_input[tid] >> i) & 1;
		__syncthreads();
		p_split_inverse[tid] = (p_split[tid] + 1) & 1;
		__syncthreads();
		p_totalFalses = p_split_inverse[blockDim.x - 1];
		__syncthreads();
		for (int j = 0; j <__ffs(blockDim.x) - 1; j++)
		{
			int step = 1 << (j + 1);
			int index = (tid + 1)*step - 1;
			pTmp = 0;
			__syncthreads();
			if (index < blockDim.x) {
				pTmp = p_split_inverse[index - step / 2];
			}
			__syncthreads();
			if (index < blockDim.x) {
				p_split_inverse[index] += pTmp;
			}
			__syncthreads();


		}
		__syncthreads();
		if (tid == blockDim.x - 1)
		{
			p_split_inverse[tid] = 0;
		}
		__syncthreads();
		for (int j = 0; j <__ffs(blockDim.x) - 1; j++)
		{
			int step = blockDim.x >> j;
			int index = (tid + 1)*step - 1;
			pTmp = 0;
			__syncthreads();
			if (index < blockDim.x) {
				pTmp = p_split_inverse[index - step / 2];
				//p_split_inverse[index - step / 2] = p_split_inverse[index] - p_split_inverse[index - step / 2];
			}
			__syncthreads();
			if (index < blockDim.x) {
				p_split_inverse[index] += pTmp;
				//p_split_inverse[index - step / 2] = p_split_inverse[index] - p_split_inverse[index - step / 2];
			}
			__syncthreads();

			pTmp = 0;
			if (index < blockDim.x) {
				//p_split_inverse[index] += p_split_inverse[index - step / 2];
				pTmp = p_split_inverse[index];
			}
			__syncthreads();

			if (index < blockDim.x) {
				//p_split_inverse[index] += p_split_inverse[index - step / 2];
				p_split_inverse[index - step / 2] = pTmp - p_split_inverse[index - step / 2];
			}
			__syncthreads();

		}
		__syncthreads();
		p_totalFalses += p_split_inverse[blockDim.x - 1];
		__syncthreads();
		p_t[tid] = tid - p_split_inverse[tid] + p_totalFalses;
		__syncthreads();
		p_index[tid] = (p_split[tid]>0) ? p_t[tid] : p_split_inverse[tid];
		__syncthreads();
		pTmp = p_input[tid];
		__syncthreads();
		p_input[p_index[tid]] = pTmp;
		__syncthreads();
	}
	p[tid] = p_input[tid];
	__syncthreads();

	/*p_input[tid] = 0;
	p_split[tid] = 0;
	p_split_inverse[tid] = 0;
	p_t[tid] = 0;
	p_index[tid] = 0;
	__syncthreads();*/

}

__device__ void index_value_count(volatile int* p, volatile int *index, volatile int *value) {

	volatile __shared__ int p_input[ShaMemSize];
	volatile __shared__ int adj_diff[ShaMemSize];

	volatile __shared__ int adj_diff_tmp[ShaMemSize];
	volatile __shared__ int nonezero_index[ShaMemSize];
	volatile __shared__ int nonezero_index_move[ShaMemSize];
	volatile __shared__ int p_input_tmp[ShaMemSize];
	int tid = threadIdx.x;
	int pTmp = 0;
	adj_diff[tid] = 0;
	nonezero_index[tid] = 0;
	nonezero_index_move[tid] = 0;
	p_input[tid] = p[tid];
	p_input_tmp[tid] = 0;
	adj_diff_tmp[tid] = 0;
	__syncthreads();
	if (tid <blockDim.x - 1)
	{
		p_input_tmp[tid] = p[tid + 1];
	}

	__syncthreads();
	if (tid <blockDim.x - 1)
	{
		adj_diff[tid] = p_input_tmp[tid] - p_input[tid];
	}
	__syncthreads();
	if (tid <blockDim.x - 1)
	{
		pTmp = (adj_diff[tid] != 0);
	}
	__syncthreads();
	if (tid <blockDim.x - 1)
	{
		adj_diff[tid] = pTmp;
	}
	__syncthreads();

	if (tid == (blockDim.x - 1))
	{
		adj_diff[tid] = 1;
	}
	__syncthreads();
	adj_diff_tmp[tid] = adj_diff[tid];
	__syncthreads();
	prefix_sum(adj_diff);
	//d_aout[tid] = adj_diff[tid];
	__syncthreads();
	if (adj_diff_tmp[tid])
		index[adj_diff[tid] - 1] = p_input[tid];
	__syncthreads();
	/*if (tid < blockDim.x - 1)
	{
	nonezero_index[tid] = (p_input_tmp[tid] - p_input[tid] != 0);
	}
	__syncthreads();
	if (tid == blockDim.x - 1) {
	nonezero_index[tid] = 1;
	}*/
	nonezero_index[tid] = adj_diff_tmp[tid];
	__syncthreads();
	if (nonezero_index[tid] != 0) {
		nonezero_index[tid] = tid + 1;
		//value[adj_diff[tid] - 1] = nonezero_index[tid];
	}
	__syncthreads();
	if (nonezero_index[tid] != 0) {
		//nonezero_index[tid] = tid + 1;
		value[adj_diff[tid] - 1] = nonezero_index[tid];
	}
	__syncthreads();

	if (tid < blockDim.x - 1)
		nonezero_index_move[tid + 1] = value[tid];
	__syncthreads();
	if ((value[tid] > 0) && (tid>0))
	{
		value[tid] = value[tid] - nonezero_index_move[tid];
	}
	__syncthreads();
	/*adj_diff[tid] = 0;
	nonezero_index[tid] = 0;
	nonezero_index_move[tid] = 0;
	p_input[tid] = 0;
	adj_diff_tmp[tid] = 0;
	p_input_tmp[tid] = 0;
	__syncthreads();*/


}

__device__ void dense_sparse_kernel(volatile int *p, int* index, int* value, int* count, int *d_sparse_slotcount, int *d_slotoffset, int *d_counter_0)
{
	volatile __shared__ int p_input[ShaMemSize];
	volatile __shared__ int p_nonezero[ShaMemSize];
	volatile __shared__ int p_temp[ShaMemSize];
	int p_blocksum = 0;
	int p_count = 0;
	int m = 0;
	int f = 0;
	m = d_slotoffset[blockIdx.x + (*d_counter_0)*gridDim.x];
	//int p_nonezero_tmp = 0;
	//__shared__ int p_temp[K];
	int tid = threadIdx.x;
	int blockId = blockIdx.x;
	p_input[tid] = 0;
	p_nonezero[tid] = 0;
	p_temp[tid] = 0;
	__syncthreads();
	for (int k = 0; k < ((K - 1) / blockDim.x + 1); k++)
	{
		if ((k*blockDim.x + tid) < K) {
			p_input[tid] = p[k*blockDim.x + tid + blockId*K];
			//p_nonezero[tid] = p_input[tid] && 1;
			//p[k] = 0;
		}
		__syncthreads();
		if ((k*blockDim.x + tid) < K) {
			//p_input[tid] = p[k];
			p_nonezero[tid] = p_input[tid] && 1;
			//p[k] = 0;
		}
		__syncthreads();
		/*if (p_nonezero[tid]) {
		p_temp[tid] = p_nonezero[tid] + p_blocksum;
		}*/
		//p_nonezero_tmp= p_nonezero[tid];
		p_temp[tid] = p_nonezero[tid];
		__syncthreads();
		p_count += __syncthreads_count(p_nonezero[tid]);
		__syncthreads();
		prefix_sum(p_temp);
		__syncthreads();
		p_temp[tid] += p_blocksum;
		__syncthreads();
		f = p_temp[tid] - 1;
		__syncthreads();
		f += m;
		__syncthreads();
		if (p_input[tid]) {
			index[f] = (k*blockDim.x + tid) + 1;
			value[f] = p_input[tid];
		}
		__syncthreads();
		p_blocksum = p_temp[blockDim.x - 1];
		__syncthreads();
		p_temp[tid] = 0;
		p_input[tid] = 0;
		p_nonezero[tid] = 0;
		f = 0;

		//m = 0;
		__syncthreads();
	}
	if (tid == 0) count[blockIdx.x + (*d_counter_0)*gridDim.x] = p_count;
	__syncthreads();
	/*p_input[tid] = 0;
	p_nonezero[tid] = 0;
	p_temp[tid] = 0;
	p_blocksum = 0;
	p_count = 0;
	__syncthreads();*/
}

__device__ void dense_sparse_kernel2(volatile int *p, int* index, int* value, int* count, int *d_sparse_slotcount, int *d_slotoffset, int *d_counter_0, int numOfwordD)
{
	volatile __shared__ int p_input[ShaMemSize];
	volatile __shared__ int p_nonezero[ShaMemSize];
	volatile __shared__ int p_temp[ShaMemSize];
	int p_blocksum = 0;
	int p_count = 0;
	int m = 0;
	int f = 0;
	m = d_slotoffset[blockIdx.x + (*d_counter_0)*gridDim.x+ numOfwordD]- K*numOfwordD;
	//int p_nonezero_tmp = 0;
	//__shared__ int p_temp[K];
	int tid = threadIdx.x;
	int blockId = blockIdx.x;
	p_input[tid] = 0;
	p_nonezero[tid] = 0;
	p_temp[tid] = 0;
	__syncthreads();
	for (int k = 0; k < ((K - 1) / blockDim.x + 1); k++)
	{
		if ((k*blockDim.x + tid) < K) {
			p_input[tid] = p[k*blockDim.x + tid + blockId*K];
			//p_nonezero[tid] = p_input[tid] && 1;
			//p[k] = 0;
		}
		__syncthreads();
		if ((k*blockDim.x + tid) < K) {
			//p_input[tid] = p[k];
			p_nonezero[tid] = p_input[tid] && 1;
			//p[k] = 0;
		}
		__syncthreads();
		/*if (p_nonezero[tid]) {
		p_temp[tid] = p_nonezero[tid] + p_blocksum;
		}*/
		//p_nonezero_tmp= p_nonezero[tid];
		p_temp[tid] = p_nonezero[tid];
		__syncthreads();
		p_count += __syncthreads_count(p_nonezero[tid]);
		__syncthreads();
		prefix_sum(p_temp);
		__syncthreads();
		p_temp[tid] += p_blocksum;
		__syncthreads();
		f = p_temp[tid] - 1;
		__syncthreads();
		f += m;
		__syncthreads();
		if (p_input[tid]) {
			index[f] = (k*blockDim.x + tid) + 1;
			value[f] = p_input[tid];
		}
		__syncthreads();
		p_blocksum = p_temp[blockDim.x - 1];
		__syncthreads();
		p_temp[tid] = 0;
		p_input[tid] = 0;
		p_nonezero[tid] = 0;
		f = 0;

		//m = 0;
		__syncthreads();
	}
	if (tid == 0) count[blockIdx.x + (*d_counter_0)*gridDim.x] = p_count;
	__syncthreads();
	/*p_input[tid] = 0;
	p_nonezero[tid] = 0;
	p_temp[tid] = 0;
	p_blocksum = 0;
	p_count = 0;
	__syncthreads();*/
}


__device__ void warp_prefix_sum(int& p) {

	int laneId = threadIdx.x % warpSize;

	int warpId = threadIdx.x / warpSize;
	int tid = threadIdx.x %warpSize;
	int p_temp;
	p_temp = p;
	for (int iter = 1; iter < 2 * warpSize; iter *= 2)
	{
		p_temp = __shfl_up(p, iter);
		p = (laneId >(iter - 1)) ? (p + p_temp) : p;
	}
}

__device__ void warp_radix_sort(volatile int* p) {
	volatile __shared__ int p_input[ShaMemSize];
	volatile __shared__ int p_split[ShaMemSize];
	volatile __shared__ int p_t[ShaMemSize];
	volatile __shared__ int p_index[ShaMemSize];
	int blockId = blockIdx.x;
	int tid = threadIdx.x;
	int laneId = threadIdx.x % warpSize;
	int warpId = threadIdx.x / warpSize;

	p_split[tid] = 0;
	p_t[tid] = 0;
	p_index[tid] = 0;
	p_input[tid] = p[tid];
	__syncthreads();
	for (int i = 0; i < 32; i++) {
		int p_totalFalses = 0;
		int p_split_inverse = 0;
		int pTmp = 0;
		p_split[tid] = (p_input[tid] >> i) & 1;
		__syncthreads();
		p_split_inverse = (p_split[tid] + 1) & 1;
		__syncthreads();
		warp_prefix_sum(p_split_inverse);
		__syncthreads();
		pTmp = __shfl(p_split_inverse, warpSize - 1, warpSize);
		__syncthreads();
		p_totalFalses = pTmp;
		__syncthreads();
		pTmp = __shfl_up(p_split_inverse, 1);
		__syncthreads();
		p_split_inverse = pTmp;
		__syncthreads();
		if (laneId == 0) {
			p_split_inverse = 0;
		}
		__syncthreads();
		p_t[tid] = laneId - p_split_inverse + p_totalFalses;
		__syncthreads();
		p_index[tid] = (p_split[tid]>0) ? p_t[tid] : p_split_inverse;
		__syncthreads();
		pTmp = p_input[tid];
		__syncthreads();
		p_input[p_index[tid] + warpId*warpSize] = pTmp;
		__syncthreads();
	}
	p[tid] = p_input[tid];
}

//scan a token list to generate the sparse WT/DT matrix
__device__ void warp_index_value_count(volatile int* p, volatile int *index, volatile int *value) {

	__shared__ int p_input[ShaMemSize];
	int adj_diff = 0;
	int nonezero_index = 0;
	int nonezero_index_move = 0;
	int blockId = blockIdx.x;
	int tid = threadIdx.x;
	int laneId = threadIdx.x % warpSize;
	int warpId = threadIdx.x / warpSize;
	int pTmp = 0;
	p_input[tid] = p[tid];
	__syncthreads();



	if (laneId  > 0)
	{
		pTmp = p_input[tid - 1];
	}
	__syncthreads();
	if (laneId  > 0)
	{
		adj_diff = (p_input[tid] != pTmp);
	}
	__syncthreads();

	if (laneId == 0) {
		adj_diff = 0;
	}
	__syncthreads();
	warp_prefix_sum(adj_diff);
	__syncthreads();
	//if (laneId == warpSize - 1)
	//{
	//	count[warpId+ blockId *blockDim.x/ warpSize] = adj_diff + 1;
	//}
	index[adj_diff + warpId*warpSize] = p_input[tid];
	__syncthreads();
	if (laneId < warpSize - 1)
	{
		pTmp = p_input[tid + 1];
	}
	__syncthreads();
	if (laneId < warpSize - 1)
	{
		nonezero_index = (p_input[tid] != pTmp);
	}
	__syncthreads();
	if (laneId == warpSize - 1) {
		nonezero_index = 1;
	}
	__syncthreads();
	if (nonezero_index != 0) {
		nonezero_index = laneId + 1;
		//value[adj_diff + warpId*warpSize] = nonezero_index;
	}
	__syncthreads();
	if (nonezero_index != 0) {
		//nonezero_index = laneId + 1;
		value[adj_diff + warpId*warpSize] = nonezero_index;
	}
	__syncthreads();
	nonezero_index_move = __shfl_up(value[laneId + warpId*warpSize], 1);
	__syncthreads();
	if ((value[laneId + warpId*warpSize] > 0) && (laneId>0))
	{
		value[laneId + warpId*warpSize] -= nonezero_index_move;
	}
	__syncthreads();

}


//reconstruct sparse matrix from token list
__global__ void tokenlist_to_matrix_warp(int *d_a, int *d_count, int *d_index, int *d_value, int *d_slotcount, int *d_slotoffset,  int *d_row_sum, int *d_counter_1, int* d_token_amount_0, int* d_token_amount_1,  int numOfTokenD)
{
	volatile __shared__ int p_input[ShaMemSize];
	volatile __shared__ int p_index[ShaMemSize];
	volatile __shared__ int p_value[ShaMemSize];
	int tid = threadIdx.x;
	int globalId = threadIdx.x + blockIdx.x * blockDim.x;
	int laneId = threadIdx.x % warpSize;
	int warpId = threadIdx.x / warpSize;
	int blockId = blockIdx.x;
	int GridWarpDim = gridDim.x*blockDim.x / 32;
	int GridWarpId = globalId / warpSize;
	int p_indexTmp;
	int p_valueTmp;
	if ((GridWarpId > (*d_token_amount_1 - 1 - *d_counter_1*GridWarpDim))|| (d_slotcount[GridWarpId + *d_counter_1*GridWarpDim + *d_token_amount_0] == 0))
	{
		return;
	}
	__syncthreads();
	p_input[tid] = 0;
	p_index[tid] = 0;
	p_value[tid] = 0;
	p_indexTmp = 0;
	p_valueTmp = 0;
	if (laneId < d_slotcount[GridWarpId + *d_counter_1*GridWarpDim + *d_token_amount_0])
	{
		p_input[tid] = d_a[d_slotoffset[GridWarpId + *d_counter_1*GridWarpDim + *d_token_amount_0] + numOfTokenD+laneId];
		//atomicAdd(&d_row_sum[p_input[tid] - 1], 1);
		//p_index[tid] = d_index[d_slotoffset[GridWarpId + *d_token_amount_0] + laneId];
		//p_value[tid] = d_value[d_slotoffset[GridWarpId + *d_token_amount_0] + laneId];
	}
	__syncthreads();
	warp_radix_sort(p_input);
	__syncthreads();

	warp_index_value_count(p_input, p_index, p_value);
	__syncthreads();
	p_indexTmp = p_index[tid];
	p_valueTmp = p_value[tid];
	__syncthreads();
	if (d_slotcount[GridWarpId + *d_counter_1*GridWarpDim + *d_token_amount_0] != warpSize)
	{
		p_index[tid] = __shfl_down(p_indexTmp, 1);
		p_value[tid] = __shfl_down(p_valueTmp, 1);
		if (laneId == warpSize - 1)
		{
			p_index[tid] = 0;
			p_value[tid] = 0;
		}
	}
	__syncthreads();
	d_count[GridWarpId + *d_counter_1*GridWarpDim + *d_token_amount_0] = __popc(__ballot(p_value[tid]));
	__syncthreads();
	//d_tmp[globalId] = p_value[tid];
	if (laneId < d_slotcount[GridWarpId + *d_counter_1*GridWarpDim + *d_token_amount_0])
	{
		d_index[d_slotoffset[GridWarpId + *d_counter_1*GridWarpDim + *d_token_amount_0] + laneId] = p_index[tid];
		d_value[d_slotoffset[GridWarpId + *d_counter_1*GridWarpDim + *d_token_amount_0] + laneId] = p_value[tid];
	}
	__syncthreads();

}
//reconstruct sparse matrix from token list
__global__ void tokenlist_to_matrix(int *d_a, int *d_count, int *d_index, int *d_value, int *d_slotcount, int *d_slotoffset, int *d_row_sum, int *d_counter_0, int *d_token_amount_0, int *d_dense,  int numOfTokenD)
{
	volatile __shared__ int p_input[ShaMemSize];
	volatile __shared__ int p_index[ShaMemSize];
	volatile __shared__ int p_value[ShaMemSize];
	volatile __shared__ int p_index_tmp[ShaMemSize];
	volatile __shared__ int p_value_tmp[ShaMemSize];
	//volatile __shared__ int p_dense[K];
	int tid = threadIdx.x;
	int globalId = threadIdx.x + blockIdx.x * blockDim.x;
	int blockId = blockIdx.x;
	int indicator = 0;
	int GridDim = gridDim.x;
	
	/*int wordIdWT = blockId + (*d_counter_0)*GridDim ;*/
	/*long long tokenStart = d_TokenOffset[wordId];
	long long tokenEnd = d_TokenOffset[wordId] + d_TokenCount[wordId];*/



	if ((blockId > (*d_token_amount_0 - 1 - *d_counter_0*gridDim.x))|| (d_slotcount[blockId + (*d_counter_0)*GridDim]==0))
	{
		return;
	}
	int wordId = blockId + (*d_counter_0)*GridDim;
	p_input[tid] = 0;
	p_index[tid] = 0;
	p_value[tid] = 0;
	p_index_tmp[tid] = 0;
	p_value_tmp[tid] = 0;
	for (int k = tid; k < K; k += blockDim.x)
	{
		d_dense[k + K*blockId] = 0;
	}

	__syncthreads();

	for (int i = tid; i < ((d_slotcount[wordId] - 1) / blockDim.x + 1)*blockDim.x; i += blockDim.x) {
		if (i < d_slotcount[wordId]) {
			int tmpIndex = d_slotoffset[wordId] + i + numOfTokenD;
			p_input[tid] = d_a[tmpIndex];
			//atomicAdd(&d_row_sum[p_input[tid] - 1], 1);
		}

		__syncthreads();
		radix_sort(p_input);
		__syncthreads();
		index_value_count(p_input, p_index, p_value);
		__syncthreads();
		if (((d_slotcount[wordId] - indicator*blockDim.x) < blockDim.x) && (tid<(blockDim.x - 1)))
		{
			p_index_tmp[tid] = p_index[tid + 1];
			p_value_tmp[tid] = p_value[tid + 1];
		}
		__syncthreads();

		if (((d_slotcount[wordId] - indicator*blockDim.x) < blockDim.x) && (tid<(blockDim.x - 1)))
		{
			p_index[tid] = p_index_tmp[tid];
			p_value[tid] = p_value_tmp[tid];
		}
		__syncthreads();

		if (((d_slotcount[wordId] - indicator*blockDim.x) < blockDim.x) && (tid == (blockDim.x - 1)))
		{
			p_index[tid] = 0;
			p_value[tid] = 0;
		}
		__syncthreads();
		if (p_index[tid])
		{
			//atomicAdd(&p_dense[p_index[tid] - 1], 1);
			d_dense[p_index[tid] - 1 + K*blockId] += p_value[tid];
		}
		__syncthreads();
		p_index[tid] = 0;
		p_value[tid] = 0;
		p_input[tid] = 0;
		p_index_tmp[tid] = 0;
		p_index_tmp[tid] = 0;
		indicator++;
		__syncthreads();
	}
	__syncthreads();
	/*if (globalId == 0) printf("%d mark\n", *d_counter_0);
	__syncthreads();*/
	dense_sparse_kernel(d_dense, d_index, d_value, d_count, d_slotcount, d_slotoffset, d_counter_0);
	__syncthreads();

}

__global__ void DT_Update_Kernel(int *d_Index, int *d_a, int *d_count, int *d_index, int *d_value, int *d_slotcount, int *d_slotoffset, int *d_sparse_slotcount, int *d_sparse_slotoffset, int *d_counter_0, int argD, int *d_dense)
{
	volatile __shared__ int p_input[ShaMemSize];
	volatile __shared__ int p_index[ShaMemSize];
	volatile __shared__ int p_value[ShaMemSize];
	volatile __shared__ int p_index_tmp[ShaMemSize];
	volatile __shared__ int p_value_tmp[ShaMemSize];
	//volatile __shared__ int p_dense[K];
	int tid = threadIdx.x;
	int globalId = threadIdx.x + blockIdx.x * blockDim.x;
	int blockId = blockIdx.x;
	int indicator = 0;
	int GridDim = gridDim.x;
	if (blockId > (argD - 1 - *d_counter_0*gridDim.x))
	{
		return;
	}
	p_input[tid] = 0;
	p_index[tid] = 0;
	p_value[tid] = 0;
	p_index_tmp[tid] = 0;
	p_value_tmp[tid] = 0;
	for (int k = tid; k < K; k += blockDim.x)
	{
		d_dense[k + K*blockId] = 0;
	}

	__syncthreads();

	for (int i = tid; i < ((d_slotcount[blockId + (*d_counter_0)*GridDim] - 1) / blockDim.x + 1)*blockDim.x; i += blockDim.x) {
		if (i < d_slotcount[blockId + (*d_counter_0)*GridDim]) {
			p_input[tid] = d_a[d_Index[d_slotoffset[blockId + (*d_counter_0)*GridDim] + i]];
			//atomicAdd(&d_row_sum[p_input[tid] - 1], 1);
		}
		else
		{
			p_input[tid] = 0;
		}
		__syncthreads();
		radix_sort(p_input);
		__syncthreads();
		index_value_count(p_input, p_index, p_value);
		__syncthreads();
		if (((d_slotcount[blockId + (*d_counter_0)*GridDim] - indicator*blockDim.x) < blockDim.x) && (tid<(blockDim.x - 1)))
		{
			p_index_tmp[tid] = p_index[tid + 1];
			p_value_tmp[tid] = p_value[tid + 1];
		}
		__syncthreads();
		if (((d_slotcount[blockId + (*d_counter_0)*GridDim] - indicator*blockDim.x) < blockDim.x) && (tid<(blockDim.x - 1)))
		{
			p_index[tid] = p_index_tmp[tid];
			p_value[tid] = p_value_tmp[tid];
		}
		__syncthreads();

		if (((d_slotcount[blockId + (*d_counter_0)*GridDim] - indicator*blockDim.x) < blockDim.x) && (tid == (blockDim.x - 1)))
		{
			p_index[tid] = 0;
			p_value[tid] = 0;
		}
		__syncthreads();
		if (p_index[tid])
		{
			//atomicAdd(&p_dense[p_index[tid] - 1], 1);
			d_dense[p_index[tid] - 1 + K*blockId] += p_value[tid];
		}
		__syncthreads();
		p_index[tid] = 0;
		p_value[tid] = 0;
		p_input[tid] = 0;
		p_index_tmp[tid] = 0;
		p_index_tmp[tid] = 0;
		indicator++;
		__syncthreads();
	}
	__syncthreads();

	dense_sparse_kernel(d_dense, d_index, d_value, d_count, d_sparse_slotcount, d_sparse_slotoffset, d_counter_0);
	__syncthreads();

}





__global__ void WTDen_Update_Kernel(int *deviceTopic, int *deviceWTDense, int *deviceTLCount, int *deviceTLOffset, int *deviceWTOffset, int numOfWordD, int counter)
{

	int input;
	
	int tid = threadIdx.x;
	int globalId = threadIdx.x + blockIdx.x * blockDim.x;
	int blockId = blockIdx.x;
	int wordId=blockId+counter*gridDim.x;
	int tmpTopic;
	int tmpWTOffset= deviceWTOffset[wordId];
	int tmpTLOffset= deviceTLOffset[wordId];

	if (wordId > numOfWordD - 1)
	{
		return;
	}

	for (int k = tid; k < deviceTLCount[wordId]; k += blockDim.x)
	{
		tmpTopic = deviceTopic[tmpTLOffset + k];
		atomicAdd(&deviceWTDense[tmpWTOffset + tmpTopic - 1], 1);
	}
	__syncthreads();

}

__global__ void WTDen_Sum_Update_Kernel(int *deviceWTDense, int *deviceWTRowSum, int *deviceWTOffset, int numOfWordD)
{

	int input;
	int tid = threadIdx.x;
	int globalId = threadIdx.x + blockIdx.x * blockDim.x;
	int blockId = blockIdx.x;
	int tmpIndex;

	for (int k = globalId; k < K; k += GridDim*BlockDim)
	{
		for (int i = 0; i < numOfWordD; i ++)
		{
			tmpIndex = deviceWTOffset[i]  + k;
			deviceWTRowSum[k] += deviceWTDense[tmpIndex];

		}
	}
	__syncthreads();

}





__global__ void sparseMatrixAdd(int* argCount0, int* argOffset0, int* argNZCount0, int* argIndex0, int* argValue0, int* argCount1, int* argOffset1, int* argNZCount1, int* argIndex1, int* argValue1, int* argDense, int argNumRows, int* argBlockCounter, int* argWTRowSum, int numOfWordD)
{

	int tid = threadIdx.x;
	int globalId = threadIdx.x + blockIdx.x * blockDim.x;
	int blockId = blockIdx.x;
	int row = *argBlockCounter*gridDim.x + blockId;

	if (blockId > (argNumRows - 1 - *argBlockCounter*gridDim.x))
	{
		return;
	}


	for (int k = tid; k < K; k += blockDim.x)
	{
		argDense[k + K*blockId] = 0;
	}

	for (int k = tid; k < argNZCount0[row]; k += blockDim.x)
	{
		argDense[K*blockId + argIndex0[argOffset0[row+ numOfWordD]-K*numOfWordD + k] - 1] += argValue0[argOffset0[row+ numOfWordD] - K*numOfWordD + k];
	}
	__syncthreads();

	for (int k = tid; k < argNZCount1[row]; k += blockDim.x)
	{
		atomicAdd(&argWTRowSum[argIndex1[argOffset1[row] + k] - 1], argValue1[argOffset1[row] + k]);
		argDense[K*blockId + argIndex1[argOffset1[row] + k] - 1] += argValue1[argOffset1[row] + k];
	}

	__syncthreads();

	dense_sparse_kernel2(argDense, argIndex0, argValue0, argNZCount0, argCount0, argOffset0, argBlockCounter,numOfWordD);

	__syncthreads();

}




__global__ void initRandState(hiprandState *state)
{
	int tid = blockIdx.x*blockDim.x + threadIdx.x;
	hiprand_init(clock() + tid, tid, 0, &state[tid]);
}



__global__ void LDAKernelTrain(double alpha, double beta, int* d_Index, int* d_TopicIndex, int* d_SparseDTCount, int* d_SparseDTIndex, int* d_SparseDTValue, int* d_TokenCountDT, int* d_TokenOffsetDT, int* d_DocListCount, int* d_DocListOffset, int* d_SparseWTCount, int* d_SparseWTIndex, int* d_SparseWTValue, int* d_TokenCount, int* d_TokenOffset, int* d_WordListCount, int* d_WordListOffset, int* d_WTRowSum, int* d_blockCounter, int*d_DocIndex, int D, int W, double* d_Perplexity, hiprandState *randState, double *WTHeadDense, int numOfWordD, int numOfWordS)

{

	int tid = threadIdx.x;

	int laneId = threadIdx.x % 32;
	int localId = threadIdx.x / 32;
	int wordId = *d_blockCounter*gridDim.x + blockIdx.x + numOfWordD;

	int blockId = blockIdx.x;

	volatile __shared__ double STree[ShaMemSize / 32][32];
	volatile __shared__ double pTemp[ShaMemSize];
	volatile __shared__ double prefixSumSample[ShaMemSize / 32][32];
	volatile __shared__ double pTmp[ShaMemSize / 32][32];

	volatile __shared__ double QTreeL1[K / 32];
	volatile __shared__ double QTreeL2[32];

	if (blockId > (numOfWordS - 1 - *d_blockCounter*gridDim.x))
	{
		return;
	}
	if (localId == 0) {
		QTreeL2[laneId] = 0;
	}

	double p_temp1 = 0.0;
	//double pTemp = 0.0;
	pTmp[localId][laneId] = 0.0;
	prefixSumSample[localId][laneId] = 0.0;

	__syncthreads();
	long long tokenStart = d_TokenOffset[wordId];
	long long tokenEnd = d_TokenOffset[wordId] + d_TokenCount[wordId];
	long long WTStart = d_WordListOffset[wordId] - d_WordListOffset[numOfWordD];
	long long WTEnd = d_WordListOffset[wordId] - d_WordListOffset[numOfWordD] + d_SparseWTCount[wordId- numOfWordD];
	//double WTHeadDenom;
	__syncthreads();

	pTemp[tid] = 0.0;
	//__syncthreads();
	// Reconstruct dense WT vector from sparse WT matrix
	//for (int i = tid; i < K; i += blockDim.x)
	//{
	//	WTHead[i] = beta / (d_WTRowSum[i] + W*beta);
	//	//__syncthreads();
	//}
	//__syncthreads();

	for (int i = tid; i < K; i += blockDim.x)
	{
		WTHeadDense[i + K*blockId] = beta / (d_WTRowSum[i] + W*beta);

	}
	__syncthreads();

	for (int i = tid + WTStart; i < WTEnd; i += blockDim.x)
	{
		if ((d_SparseWTIndex[i] < 1) || (d_SparseWTIndex[i] > K)) printf("wrong WTIndex:%d", d_SparseWTIndex[i]);
		WTHeadDense[d_SparseWTIndex[i] - 1 + K*blockId] = (d_SparseWTValue[i] + beta) / (d_WTRowSum[d_SparseWTIndex[i] - 1] + W*beta);
	}
	__syncthreads();

	// Construct Q tree from WTHead
	for (int i = localId; i < K / 32; i += blockDim.x / 32) {
		int   tmpK = i * 32 + laneId;
		//__syncthreads();
		double tmpVal = 0;
		tmpVal = WTHeadDense[tmpK + K*blockId];
		tmpVal += __shfl_down(tmpVal, 16);
		tmpVal += __shfl_down(tmpVal, 8);
		tmpVal += __shfl_down(tmpVal, 4);
		tmpVal += __shfl_down(tmpVal, 2);
		tmpVal += __shfl_down(tmpVal, 1);
		tmpVal = __shfl(tmpVal, 0);
		QTreeL1[i] = tmpVal;

	}
	__syncthreads();


	for (int i = localId; i < K / 32 / 32; i += blockDim.x / 32) {
		int   tmpK = i * 32 + laneId;
		//__syncthreads();
		double tmpVal = 0;
		tmpVal = QTreeL1[tmpK];
		tmpVal += __shfl_down(tmpVal, 16);
		tmpVal += __shfl_down(tmpVal, 8);
		tmpVal += __shfl_down(tmpVal, 4);
		tmpVal += __shfl_down(tmpVal, 2);
		tmpVal += __shfl_down(tmpVal, 1);
		tmpVal = __shfl(tmpVal, 0);
		QTreeL2[i] = tmpVal;
	}
	__syncthreads();



	if (localId == 0) {

		double value = alpha*QTreeL2[laneId];
		value += __shfl_up(value, 1, 32)*(laneId >= 1);
		value += __shfl_up(value, 2, 32)*(laneId >= 2);
		value += __shfl_up(value, 4, 32)*(laneId >= 4);
		value += __shfl_up(value, 8, 32)*(laneId >= 8);
		value += __shfl_up(value, 16, 32)*(laneId >= 16);
		QTreeL2[laneId] = value;
	}
	__syncthreads();

	double Q = QTreeL2[31];
	//__syncthreads();
	double sumPerplexity = 0.0;


	for (int tokenIdx = tokenStart + localId; tokenIdx < tokenEnd; tokenIdx += blockDim.x / 32) //iterate over tokens
	{
		//int docId = __ldg(&d_Index[d_TopicIndex[tokenIdx]]);
		int docId = d_DocIndex[tokenIdx];
		if ((docId < 1) || (docId > D)) printf("wrong docId:%d", docId);
		//computing S.
		double S = 0;
		double STmp = 0;
		double uTmp = 0;
		long long DTStart = d_DocListOffset[docId - 1];
		long long DTEnd = d_DocListOffset[docId - 1] + ((d_SparseDTCount[docId - 1] - 1) / 32 + 1) * 32;


		STree[localId][laneId] = 0;
		__syncthreads();
		for (int tmpIdx = DTStart + laneId, SIdx = 0; tmpIdx < DTEnd; tmpIdx += 32) {


			int   colVal = d_SparseDTValue[tmpIdx];
			int   colK = d_SparseDTIndex[tmpIdx];

			//if ((colK < 1) || (colK> K)) printf("wrong docIndex:%d", colK);

			double tmpP1k = 0.0;
			double ptmpP1k = 0.0;
			if (colK != 0) {
				tmpP1k = colVal*WTHeadDense[colK - 1 + K*blockId];
			}
			tmpP1k += __shfl_down(tmpP1k, 16);
			tmpP1k += __shfl_down(tmpP1k, 8);
			tmpP1k += __shfl_down(tmpP1k, 4);
			tmpP1k += __shfl_down(tmpP1k, 2);
			tmpP1k += __shfl_down(tmpP1k, 1);
			tmpP1k = __shfl(tmpP1k, 0);



			S += tmpP1k;
			//__syncthreads();
			STree[localId][SIdx] = S;
			//__syncthreads();
			SIdx++;
			//__syncthreads();
		}
		__syncthreads();
		/*STmp = S;

		S = __shfl(STmp, 0);*/
		S = __shfl(S, 0);
		//__syncthreads();
		//randomly generate u.
		double u;
		if (laneId == 0)u = hiprand_uniform(&(randState[threadIdx.x + blockDim.x*blockIdx.x])) / 1.00001;

		//if ((u == 1.0))printf("what's this");
		//if (laneId == 0)u = d_randu[tokenIdx];
		uTmp = u;
		//__syncthreads();
		u = __shfl(uTmp, 0);
		int newZ = 0;
		//__syncthreads();
		double tmpU = 0;
		double tmpU1 = 0;
		//__syncthreads();
		__syncthreads();


		if (u < S / (S + Q))
		{

			//totalS ++;
			//tmpClock = clock64();

			double transU = u*(S + Q);

			double tmpSumHigh, tmpSumLow = 0.0;
			tmpSumHigh = STree[localId][laneId];
			tmpSumLow = __shfl_up(tmpSumHigh, 1, 32);
			if (laneId == 0)tmpSumLow = 0;
			int voteFlag = 0;
			if ((transU < tmpSumHigh)) voteFlag = 1;
			int lvl1Idx = __ffs(__ballot(voteFlag)) - 1;
			if (lvl1Idx < 0) lvl1Idx = (DTEnd - DTStart) / 32 - 1;

			tmpU1 = transU;

			transU = tmpU1 - tmpSumLow;

			tmpU = transU;

			transU = __shfl(tmpU, lvl1Idx);


			int tmpIdx = DTStart + lvl1Idx * 32 + laneId;

			int tmpNewZ = d_SparseDTIndex[tmpIdx];
			int colVal = d_SparseDTValue[tmpIdx];

			double p1k = 0.0;
			if (tmpNewZ != 0)
			{
				p1k = colVal*WTHeadDense[tmpNewZ - 1 + K*blockId];
			}

			//__syncthreads();
			prefixSumSample[localId][laneId] = p1k;
			//__syncthreads();

			double value = prefixSumSample[localId][laneId];
			value += __shfl_up(value, 1, 32)*(laneId >= 1);
			value += __shfl_up(value, 2, 32)*(laneId >= 2);
			value += __shfl_up(value, 4, 32)*(laneId >= 4);
			value += __shfl_up(value, 8, 32)*(laneId >= 8);
			value += __shfl_up(value, 16, 32)*(laneId >= 16);

			prefixSumSample[localId][laneId] = value;

			double tmpSum = prefixSumSample[localId][laneId];

			voteFlag = 0;
			if (transU < tmpSum) voteFlag = 1;

			int offset = __ffs(__ballot(voteFlag)) - 1;
			//if (offset < 0) printf("bugs!");
			newZ = __shfl(tmpNewZ, offset);
			if ((newZ > K) || (newZ < 1)) {
				//printf("part1: u=%f, %d,%d,%d\n", u, newZ, lvl1Idx, offset);
				newZ = 5;
			}

		}


		//if (u > S / (S + Q))
		else //bucket Q
		{

			double transU = (u - S / (S + Q))*(S + Q);
			/*double tmpU;
			double tmpU1;*/
			//totalQ ++;
			//double originalU = transU;

			//level 1: decide position
			double tmpSumHigh, tmpSumLow = 0.0;
			tmpSumHigh = QTreeL2[laneId];
			//__syncthreads();
			tmpSumLow = __shfl_up(tmpSumHigh, 1, 32);
			//__syncthreads();
			if (laneId == 0)tmpSumLow = 0;

			//voting for lvl1Idx
			int voteFlag = 0;
			if (transU < tmpSumHigh) voteFlag = 1; //voteFlag = transU < tmpSumHigh;	
			int lvl1Idx = __ffs(__ballot(voteFlag)) - 1;
			//if (lvl1Idx == 10) printf("bugs! position1");
			if (lvl1Idx < 0) lvl1Idx = K / 1024 - 1;
			//if (lvl1Idx == 10) printf("lvl1Idx bugs! position2");
			//if (lvl1Idx == 31) printf("bugs!");
			tmpU1 = transU;
			transU = tmpU1 - tmpSumLow;
			tmpU = transU;
			transU = __shfl(tmpU, lvl1Idx);
			prefixSumSample[localId][laneId] = alpha*QTreeL1[32 * lvl1Idx + laneId];

			//accumulation

			double value = prefixSumSample[localId][laneId];
			value += __shfl_up(value, 1, 32)*(laneId >= 1);
			value += __shfl_up(value, 2, 32)*(laneId >= 2);
			value += __shfl_up(value, 4, 32)*(laneId >= 4);
			value += __shfl_up(value, 8, 32)*(laneId >= 8);
			value += __shfl_up(value, 16, 32)*(laneId >= 16);

			prefixSumSample[localId][laneId] = value;

			voteFlag = 0;
			tmpSumLow = 0;
			tmpSumHigh = prefixSumSample[localId][laneId];
			tmpSumLow = __shfl_up(tmpSumHigh, 1, 32);
			if (laneId == 0)tmpSumLow = 0;
			if (transU < tmpSumHigh)voteFlag = 1; //voteFlag = transU < tmpSumHigh;		
			int lvl2Idx = __ffs(__ballot(voteFlag)) - 1;
			if (lvl2Idx < 0)lvl2Idx = 31;

			tmpU1 = transU;
			transU = tmpU1 - tmpSumLow;
			tmpU = transU;
			transU = __shfl(tmpU, lvl2Idx);
			/*	transU = transU - tmpSumLow;
			transU = __shfl(transU, lvl2Idx)*/;

			prefixSumSample[localId][laneId] = alpha*WTHeadDense[1024 * lvl1Idx + 32 * lvl2Idx + laneId + K*blockId];

			value = prefixSumSample[localId][laneId];

			value += __shfl_up(value, 1, 32)*(laneId >= 1);
			value += __shfl_up(value, 2, 32)*(laneId >= 2);
			value += __shfl_up(value, 4, 32)*(laneId >= 4);
			value += __shfl_up(value, 8, 32)*(laneId >= 8);
			value += __shfl_up(value, 16, 32)*(laneId >= 16);
			prefixSumSample[localId][laneId] = value;

			voteFlag = 0;
			tmpSumLow = 0;
			tmpSumHigh = prefixSumSample[localId][laneId];
			tmpSumLow = __shfl_up(tmpSumHigh, 1, 32);
			if (laneId == 0)tmpSumLow = 0;
			if (transU < tmpSumHigh)voteFlag = 1; //voteFlag = transU < tmpSumHigh;		
			int lvl3Idx = __ffs(__ballot(voteFlag)) - 1;
			if (lvl3Idx < 0)lvl3Idx = 31;
			newZ = lvl1Idx * 1024 + 32 * lvl2Idx + lvl3Idx + 1;
			if ((newZ > K) || (newZ < 1)) {
				printf("part2: u=%f, %d,%d,%d,%d\n", u, newZ, lvl1Idx, lvl2Idx, lvl3Idx);
				newZ = 5;
			}
			//__syncthreads();
			//if(tmpFlag == 1)return;
		}


		if (laneId == 0) {
			d_TopicIndex[tokenIdx] = newZ;
			/*if (newZ > K) {
			printf("u=%f, %d,%d,%d,%d", u, newZ, lvl1Idx, lvl2Idx, lvl3Idx);
			}*/
			//p_temp = S + Q;
			//d_S[tokenIdx] = Q;
			d_Perplexity[tokenIdx] = log((S + Q) / (d_TokenCountDT[docId - 1] + K*alpha));
			//sumPerplexity += log((S + Q) / (d_TokenCountDT[docId - 1] + K*alpha));

		}
	}
	__syncthreads();
	//if (threadIdx.x % 32 == 0)
	//	d_Perplexity[(threadIdx.x + blockDim.x*blockIdx.x) / 32] = sumPerplexity;
	////wordPerplexity[(threadIdx.x + blockDim.x*blockIdx.x) / 32] = sumPerplexity;
	//__syncthreads();

}

__global__ void LDAKernelTrainD(double alpha, double beta, int* d_Index, int* d_TopicIndex, int* d_SparseDTCount, int* d_SparseDTIndex, int* d_SparseDTValue, int* d_TokenCountDT, int* d_TokenOffsetDT, int* d_DocListCount, int* d_DocListOffset, int* d_WTDense, int* d_WTDenseCopy, int* d_TokenCount, int* d_TokenOffset, int* d_WordListCount, int* d_WordListOffset, int* d_WTRowSum, int* d_blockCounter, int*d_DocIndex, int D, int W, double* d_Perplexity, hiprandState *randState, double *WTHeadDense, int numOfWordD)

{

	int tid = threadIdx.x;

	int laneId = threadIdx.x % 32;
	int localId = threadIdx.x / 32;
	int wordId = *d_blockCounter*gridDim.x + blockIdx.x ;

	int blockId = blockIdx.x;

	volatile __shared__ double STree[ShaMemSize / 32][32];
	volatile __shared__ double pTemp[ShaMemSize];
	volatile __shared__ double prefixSumSample[ShaMemSize / 32][32];
	volatile __shared__ double pTmp[ShaMemSize / 32][32];

	volatile __shared__ double QTreeL1[K / 32];
	volatile __shared__ double QTreeL2[32];

	if (blockId > (numOfWordD - 1 - *d_blockCounter*gridDim.x))
	{
		return;
	}
	if (localId == 0) {
		QTreeL2[laneId] = 0;
	}

	double p_temp1 = 0.0;
	//double pTemp = 0.0;
	pTmp[localId][laneId] = 0.0;
	prefixSumSample[localId][laneId] = 0.0;

	__syncthreads();
	long long tokenStart = d_TokenOffset[wordId];
	long long tokenEnd = d_TokenOffset[wordId] + d_TokenCount[wordId];
	long long WTStart = d_WordListOffset[wordId];
	/*long long WTEnd = d_WordListOffset[wordId] + d_SparseWTCount[wordId];*/
	/*double WTHeadDenom;*/
	__syncthreads();

	pTemp[tid] = 0.0;
	//__syncthreads();
	// Reconstruct dense WT vector from sparse WT matrix
	//for (int i = tid; i < K; i += blockDim.x)
	//{
	//	WTHead[i] = beta / (d_WTRowSum[i] + W*beta);
	//	//__syncthreads();
	//}
	//__syncthreads();

	for (int i = tid; i < K; i += blockDim.x)
	{
		WTHeadDense[i + K*blockId] = (d_WTDense[WTStart+i]+beta) / (d_WTRowSum[i] + W*beta);
	}
	__syncthreads();

	//for (int i = tid + WTStart; i < WTEnd; i += blockDim.x)
	//{
	//	if ((d_SparseWTIndex[i] < 1) || (d_SparseWTIndex[i] > K)) printf("wrong WTIndex:%d", d_SparseWTIndex[i]);
	//	WTHeadDense[d_SparseWTIndex[i] - 1 + K*blockId] = (d_SparseWTValue[i] + beta) / (d_WTRowSum[d_SparseWTIndex[i] - 1] + W*beta);
	//}
	//__syncthreads();

	// Construct Q tree from WTHead
	for (int i = localId; i < K / 32; i += blockDim.x / 32) {
		int   tmpK = i * 32 + laneId;
		//__syncthreads();
		double tmpVal = 0;
		tmpVal = WTHeadDense[tmpK + K*blockId];
		tmpVal += __shfl_down(tmpVal, 16);
		tmpVal += __shfl_down(tmpVal, 8);
		tmpVal += __shfl_down(tmpVal, 4);
		tmpVal += __shfl_down(tmpVal, 2);
		tmpVal += __shfl_down(tmpVal, 1);
		tmpVal = __shfl(tmpVal, 0);
		QTreeL1[i] = tmpVal;

	}
	__syncthreads();


	for (int i = localId; i < K / 32 / 32; i += blockDim.x / 32) {
		int   tmpK = i * 32 + laneId;
		//__syncthreads();
		double tmpVal = 0;
		tmpVal = QTreeL1[tmpK];
		tmpVal += __shfl_down(tmpVal, 16);
		tmpVal += __shfl_down(tmpVal, 8);
		tmpVal += __shfl_down(tmpVal, 4);
		tmpVal += __shfl_down(tmpVal, 2);
		tmpVal += __shfl_down(tmpVal, 1);
		tmpVal = __shfl(tmpVal, 0);
		QTreeL2[i] = tmpVal;
	}
	__syncthreads();



	if (localId == 0) {

		double value = alpha*QTreeL2[laneId];
		value += __shfl_up(value, 1, 32)*(laneId >= 1);
		value += __shfl_up(value, 2, 32)*(laneId >= 2);
		value += __shfl_up(value, 4, 32)*(laneId >= 4);
		value += __shfl_up(value, 8, 32)*(laneId >= 8);
		value += __shfl_up(value, 16, 32)*(laneId >= 16);
		QTreeL2[laneId] = value;
	}
	__syncthreads();

	double Q = QTreeL2[31];
	//__syncthreads();
	double sumPerplexity = 0.0;


	for (int tokenIdx = tokenStart + localId; tokenIdx < tokenEnd; tokenIdx += blockDim.x / 32) //iterate over tokens
	{
		//int docId = __ldg(&d_Index[d_TopicIndex[tokenIdx]]);
		int docId = d_DocIndex[tokenIdx];
		if ((docId < 1) || (docId > D)) printf("wrong docId:%d", docId);
		//computing S.
		double S = 0;
		double STmp = 0;
		double uTmp = 0;
		long long DTStart = d_DocListOffset[docId - 1];
		long long DTEnd = d_DocListOffset[docId - 1] + ((d_SparseDTCount[docId - 1] - 1) / 32 + 1) * 32;


		STree[localId][laneId] = 0;
		__syncthreads();
		for (int tmpIdx = DTStart + laneId, SIdx = 0; tmpIdx < DTEnd; tmpIdx += 32) {


			int   colVal = d_SparseDTValue[tmpIdx];
			int   colK = d_SparseDTIndex[tmpIdx];

			//if ((colK < 1) || (colK> K)) printf("wrong docIndex:%d", colK);

			double tmpP1k = 0.0;
			double ptmpP1k = 0.0;
			if (colK != 0) {
				tmpP1k = colVal*WTHeadDense[colK - 1 + K*blockId];
			}
			tmpP1k += __shfl_down(tmpP1k, 16);
			tmpP1k += __shfl_down(tmpP1k, 8);
			tmpP1k += __shfl_down(tmpP1k, 4);
			tmpP1k += __shfl_down(tmpP1k, 2);
			tmpP1k += __shfl_down(tmpP1k, 1);
			tmpP1k = __shfl(tmpP1k, 0);



			S += tmpP1k;
			//__syncthreads();
			STree[localId][SIdx] = S;
			//__syncthreads();
			SIdx++;
			//__syncthreads();
		}
		__syncthreads();
		/*STmp = S;

		S = __shfl(STmp, 0);*/
		S = __shfl(S, 0);
		//__syncthreads();
		//randomly generate u.
		double u;
		if (laneId == 0)u = hiprand_uniform(&(randState[threadIdx.x + blockDim.x*blockIdx.x])) / 1.00001;

		//if ((u == 1.0))printf("what's this");
		//if (laneId == 0)u = d_randu[tokenIdx];
		uTmp = u;
		//__syncthreads();
		u = __shfl(uTmp, 0);
		int newZ = 0;
		//__syncthreads();
		double tmpU = 0;
		double tmpU1 = 0;
		//__syncthreads();
		__syncthreads();


		if (u < S / (S + Q))
		{

			//totalS ++;
			//tmpClock = clock64();

			double transU = u*(S + Q);

			double tmpSumHigh, tmpSumLow = 0.0;
			tmpSumHigh = STree[localId][laneId];
			tmpSumLow = __shfl_up(tmpSumHigh, 1, 32);
			if (laneId == 0)tmpSumLow = 0;
			int voteFlag = 0;
			if ((transU < tmpSumHigh)) voteFlag = 1;
			int lvl1Idx = __ffs(__ballot(voteFlag)) - 1;
			if (lvl1Idx < 0) lvl1Idx = (DTEnd - DTStart) / 32 - 1;

			tmpU1 = transU;

			transU = tmpU1 - tmpSumLow;

			tmpU = transU;

			transU = __shfl(tmpU, lvl1Idx);


			int tmpIdx = DTStart + lvl1Idx * 32 + laneId;

			int tmpNewZ = d_SparseDTIndex[tmpIdx];
			int colVal = d_SparseDTValue[tmpIdx];

			double p1k = 0.0;
			if (tmpNewZ != 0)
			{
				p1k = colVal*WTHeadDense[tmpNewZ - 1 + K*blockId];
			}

			//__syncthreads();
			prefixSumSample[localId][laneId] = p1k;
			//__syncthreads();

			double value = prefixSumSample[localId][laneId];
			value += __shfl_up(value, 1, 32)*(laneId >= 1);
			value += __shfl_up(value, 2, 32)*(laneId >= 2);
			value += __shfl_up(value, 4, 32)*(laneId >= 4);
			value += __shfl_up(value, 8, 32)*(laneId >= 8);
			value += __shfl_up(value, 16, 32)*(laneId >= 16);

			prefixSumSample[localId][laneId] = value;

			double tmpSum = prefixSumSample[localId][laneId];

			voteFlag = 0;
			if (transU < tmpSum) voteFlag = 1;

			int offset = __ffs(__ballot(voteFlag)) - 1;
			//if (offset < 0) printf("bugs!");
			newZ = __shfl(tmpNewZ, offset);
			if ((newZ > K) || (newZ < 1)) {
				//printf("part1: u=%f, %d,%d,%d\n", u, newZ, lvl1Idx, offset);
				newZ = 5;
			}

		}


		//if (u > S / (S + Q))
		else //bucket Q
		{

			double transU = (u - S / (S + Q))*(S + Q);
			/*double tmpU;
			double tmpU1;*/
			//totalQ ++;
			//double originalU = transU;

			//level 1: decide position
			double tmpSumHigh, tmpSumLow = 0.0;
			tmpSumHigh = QTreeL2[laneId];
			//__syncthreads();
			tmpSumLow = __shfl_up(tmpSumHigh, 1, 32);
			//__syncthreads();
			if (laneId == 0)tmpSumLow = 0;

			//voting for lvl1Idx
			int voteFlag = 0;
			if (transU < tmpSumHigh) voteFlag = 1; //voteFlag = transU < tmpSumHigh;	
			int lvl1Idx = __ffs(__ballot(voteFlag)) - 1;
			//if (lvl1Idx == 10) printf("bugs! position1");
			if (lvl1Idx < 0) lvl1Idx = K / 1024 - 1;
			//if (lvl1Idx == 10) printf("lvl1Idx bugs! position2");
			//if (lvl1Idx == 31) printf("bugs!");
			tmpU1 = transU;
			transU = tmpU1 - tmpSumLow;
			tmpU = transU;
			transU = __shfl(tmpU, lvl1Idx);
			prefixSumSample[localId][laneId] = alpha*QTreeL1[32 * lvl1Idx + laneId];

			//accumulation

			double value = prefixSumSample[localId][laneId];
			value += __shfl_up(value, 1, 32)*(laneId >= 1);
			value += __shfl_up(value, 2, 32)*(laneId >= 2);
			value += __shfl_up(value, 4, 32)*(laneId >= 4);
			value += __shfl_up(value, 8, 32)*(laneId >= 8);
			value += __shfl_up(value, 16, 32)*(laneId >= 16);

			prefixSumSample[localId][laneId] = value;

			voteFlag = 0;
			tmpSumLow = 0;
			tmpSumHigh = prefixSumSample[localId][laneId];
			tmpSumLow = __shfl_up(tmpSumHigh, 1, 32);
			if (laneId == 0)tmpSumLow = 0;
			if (transU < tmpSumHigh)voteFlag = 1; //voteFlag = transU < tmpSumHigh;		
			int lvl2Idx = __ffs(__ballot(voteFlag)) - 1;
			if (lvl2Idx < 0)lvl2Idx = 31;

			tmpU1 = transU;
			transU = tmpU1 - tmpSumLow;
			tmpU = transU;
			transU = __shfl(tmpU, lvl2Idx);
			/*	transU = transU - tmpSumLow;
			transU = __shfl(transU, lvl2Idx)*/;

			prefixSumSample[localId][laneId] = alpha*WTHeadDense[1024 * lvl1Idx + 32 * lvl2Idx + laneId + K*blockId];

			value = prefixSumSample[localId][laneId];

			value += __shfl_up(value, 1, 32)*(laneId >= 1);
			value += __shfl_up(value, 2, 32)*(laneId >= 2);
			value += __shfl_up(value, 4, 32)*(laneId >= 4);
			value += __shfl_up(value, 8, 32)*(laneId >= 8);
			value += __shfl_up(value, 16, 32)*(laneId >= 16);
			prefixSumSample[localId][laneId] = value;

			voteFlag = 0;
			tmpSumLow = 0;
			tmpSumHigh = prefixSumSample[localId][laneId];
			tmpSumLow = __shfl_up(tmpSumHigh, 1, 32);
			if (laneId == 0)tmpSumLow = 0;
			if (transU < tmpSumHigh)voteFlag = 1; //voteFlag = transU < tmpSumHigh;		
			int lvl3Idx = __ffs(__ballot(voteFlag)) - 1;
			if (lvl3Idx < 0)lvl3Idx = 31;
			newZ = lvl1Idx * 1024 + 32 * lvl2Idx + lvl3Idx + 1;
			if ((newZ > K) || (newZ < 1)) {
				printf("part2: u=%f, %d,%d,%d,%d\n", u, newZ, lvl1Idx, lvl2Idx, lvl3Idx);
				newZ = 5;
			}
			//__syncthreads();
			//if(tmpFlag == 1)return;
		}


		if (laneId == 0) {
			d_TopicIndex[tokenIdx] = newZ;
			atomicAdd(&d_WTDenseCopy[WTStart + newZ - 1], 1);

			d_Perplexity[tokenIdx] = log((S + Q) / (d_TokenCountDT[docId - 1] + K*alpha));
			//sumPerplexity += log((S + Q) / (d_TokenCountDT[docId - 1] + K*alpha));

		}
	}
	__syncthreads();
	//if (threadIdx.x % 32 == 0)
	//	d_Perplexity[(threadIdx.x + blockDim.x*blockIdx.x) / 32] = sumPerplexity;
	////wordPerplexity[(threadIdx.x + blockDim.x*blockIdx.x) / 32] = sumPerplexity;
	//__syncthreads();

}






__global__ void LDATrainPerplexityReduce1(double *perplexity, double *perplexityMid, int numVals) {


	int numWarps = gridDim.x*blockDim.x / 32;
	int tid = threadIdx.x + blockIdx.x*blockDim.x;
	int warpId = tid / 32;
	int laneId = tid % 32;

	int perWarpSize = ((numVals + numWarps - 1) / numWarps + 31) / 32 * 32;
	int perWarpSizeMax = (numVals + numWarps - 1) / numWarps;
	int startIdx = perWarpSizeMax*warpId;
	int endIdx = perWarpSizeMax*warpId + perWarpSize;
	int endMax = perWarpSizeMax*warpId + perWarpSizeMax;

	double totalProd = 0.0;
	for (long long i = startIdx + laneId; i < endIdx; i += 32) {

		double tmpProd = 0.0;
		if ((i < numVals) && (i < endMax))tmpProd = perplexity[i];

		tmpProd += __shfl_down(tmpProd, 16);
		tmpProd += __shfl_down(tmpProd, 8);
		tmpProd += __shfl_down(tmpProd, 4);
		tmpProd += __shfl_down(tmpProd, 2);
		tmpProd += __shfl_down(tmpProd, 1);
		tmpProd = __shfl(tmpProd, 0);
		totalProd += tmpProd;
		//__syncthreads();
	}
	__syncthreads();
	if (laneId == 0) perplexityMid[warpId] += totalProd;

}









