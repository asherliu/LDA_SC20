#include "WTDense.cuh"

WTD::WTD(int argNumOfWordD, int argWordLength) {
	numOfWordD = argNumOfWordD;
	wordLength = argWordLength;
	WTDenseLength = argNumOfWordD*K;
	WTDense = new int[WTDenseLength];
	WTDenseCopy = new int[WTDenseLength];
	/*WTRowSumDense = new int[K];*/
}

void WTD::CPUMemSet() {

	memset(WTDense, 0, WTDenseLength * sizeof(int));
	memset(WTDenseCopy, 0, WTDenseLength * sizeof(int));
	//memset(WTRowSumDense, 0, K * sizeof(int));

}


void WTD::GPUMemAllocate() {

	hipMalloc((void**)&deviceWTDense, (WTDenseLength) * sizeof(int));
	hipMalloc((void**)&deviceWTDenseCopy, (WTDenseLength) * sizeof(int));
	//hipMalloc((void**)&deviceWTRowSumDense, (K) * sizeof(int));

	WTMemory = (2*WTDenseLength + K ) / 1000000000.0 * sizeof(int);
	printf("WT memory usage(Sparse Part):%f GB\n", WTMemory);
	
}

void WTD::GPUMemInit()
{
	hipMemset(deviceWTDense, 0, (WTDenseLength) * sizeof(int));
	hipMemset(deviceWTDenseCopy, 0, (WTDenseLength) * sizeof(int));
	//hipMemset(deviceWTRowSumDense, 0, (K) * sizeof(int));
}

void WTD::GPUMemCopy()
{
	hipMemcpy(deviceWTDense, deviceWTDenseCopy, (WTDenseLength) * sizeof(int), hipMemcpyDeviceToDevice);
	
}

void WTD::GPUMemset()
{
	hipMemset(deviceWTDenseCopy, 0, (WTDenseLength) * sizeof(int));
	//hipMemset(deviceWTRowSumDense, 0, (K) * sizeof(int));
}


void WTD::WTDenGPU2CPU()
{

	hipMemcpy(WTDense, deviceWTDense, (WTDenseLength) * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(WTDenseCopy, deviceWTDenseCopy, (WTDenseLength) * sizeof(int), hipMemcpyDeviceToHost);
	

}
void WTD::WTDenCPU2Disk(string argFilePrefix) {

	ofstream WTDen((argFilePrefix + string("/WTDen.txt")).c_str(), ios::binary);
	for (int i = 0; i < WTDenseLength; i++) {
		WTDen << WTDense[i] << "\n";
	}
	WTDen.close();
	ofstream WTDenCopy((argFilePrefix + string("/WTDenCopy.txt")).c_str(), ios::binary);
	for (int i = 0; i < WTDenseLength; i++) {
		WTDenCopy << WTDenseCopy[i] << "\n";
	}
	WTDen.close();

}