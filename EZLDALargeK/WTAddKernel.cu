
#include "WTAddKernel.cuh"
void WTAdditionKernel(WTAll &argWT, Document &argDoc) {


	int blockCounter = 0;
	int iterBlock = (argWT.numOfWordS - 1) / GridDim + 1;
	int* deviceWordLength;
	int numOfWordD = argWT.wordLength-argWT.numOfWordS;
	/*hipMalloc((void**)&deviceWordLength, (1) * sizeof(int));
	
	hipMemcpy(deviceWordLength, &argWT.numOfWordS, sizeof(int),hipMemcpyHostToDevice);*/
	for (int i = 0; i < iterBlock; i++) {
		hipMemcpy(argDoc.d_blockCounter, &blockCounter, (1) * sizeof(int), hipMemcpyHostToDevice);
		sparseMatrixAdd << <GridDim, BlockDim >> >(argWT.deviceWTCount, argWT.deviceWTOffset, argWT.deviceNZWTCount, argWT.deviceWTIndex, argWT.deviceWTValue, argWT.deviceChunkWTCount, argWT.deviceChunkWTOffset, argWT.deviceChunkNZWTCount, argWT.deviceChunkWTIndex, argWT.deviceChunkWTValue, argDoc.d_dense, argWT.numOfWordS, argDoc.d_blockCounter,argWT.deviceWTRowSum, numOfWordD);
		hipDeviceSynchronize();
		blockCounter++;
	}


}