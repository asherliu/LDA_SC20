#include "hip/hip_runtime.h"

#ifndef _MAINKERNEL_H_
#define _MAINKERNEL_H_

#include "DTUpdateKernel.cuh"
#include "WTUpdateKernel.cuh"
#include "WTDenUpdateKernel.cuh"
#include "WTAddKernel.cuh"
#include "SamplingKernel.cuh"
#include "WTDense.cuh"
using namespace std;

void fileErrorCheck(ifstream& argFileStream, string folderName)
{
	if (!argFileStream.is_open())
	{
		cout << "File " << folderName << " open failed" << endl;
		exit(0);
	}
}

int main(int argc, char *argv[]) {

	clock_t startTime, startTime1,endTime;
	double WTTime=0.0;
	double samplingTime=0.0;
	double DTTime=0.0;
	double totalTime=0.0;

	int maxTLLength;
	int maxDTLength;
	int maxWTLength;
	int maxDocLength;
	int wordLength;
	int maxChunkWTLength;
	int numOfWordD;
	int numOfWordS;
	int numChunks = 4;
	int numIters = 300;

	string chunkFilePrefix = "/gpfs/alpine/proj-shared/csc289/lda/datasets/nytimesLargeK";// folder that store preprocessed chunks

	ifstream lengthVec((chunkFilePrefix + string("/lengthVec.txt")).c_str(), ios::binary);//store max Doc and DT length

	ofstream timeRecord((chunkFilePrefix + string("/timeRecord.txt")).c_str(), ios::binary);

	fileErrorCheck(lengthVec, "/lengthVec.txt");

	lengthVec >> maxTLLength >> maxDTLength >> maxWTLength >> maxDocLength >> wordLength>>maxChunkWTLength>> numOfWordD>> numOfWordS;
	lengthVec.close();

	Document document(chunkFilePrefix,numChunks,maxTLLength,maxDocLength,wordLength);

	document.loadDocument();
	document.GPUMemAllocate();
	

	DTChunk chunkDT(maxDTLength,maxDocLength,numChunks);
	chunkDT.loadDocDTLength(chunkFilePrefix);
	chunkDT.CPUMemSet();
	chunkDT.GPUMemAllocate();	
	chunkDT.loadDTCountOffset(chunkFilePrefix);
	WTD WTDen(numOfWordD, wordLength);
	WTDen.GPUMemAllocate();
	WTDen.GPUMemInit();


	
	WTAll WT(maxWTLength, wordLength, numChunks, maxChunkWTLength,numOfWordS);




	WT.CPUMemSet();
	WT.GPUMemAllocate();
	WT.GPUMemset();
	WT.loadWTLength(chunkFilePrefix);
	WT.loadWTCountOffset(chunkFilePrefix);
	WT.blockWarpCountCPU2GPU();
	WT.CPU2GPUCountOffset();

	printf("Total memory usage : %f GB\n", document.TLMemory + WT.WTMemory + chunkDT.DTMemory);

	for (int chunkId = 0; chunkId < numChunks; chunkId++) {
		document.CPU2GPU(chunkId);
		WT.chunkCPU2GPUCountOffset(chunkId);
		WT.chunkGPUMemset();
		//--------------update WTDen matrix ---------
		UpdateWTDenKernel(WTDen, WT, document, chunkId);
		//--------------update WTDen matrix-----------

		//--------------update WT matrix--------
		
		//WT.chunkCPU2GPUCountOffset(chunkId);
		//WT.chunkGPUMemset();
		UpdateWTKernel(WT, document,chunkId);
		//WT.chunkWTGPU2CPU(chunkId);// marker
		//
		//WT.CPU2DiskChunk(chunkFilePrefix, chunkId);// marker
		/*printf("\n what's this %d\n", chunkId);*/
		//--------------update WT matrix-----------

	}
	/*WTDen.WTDenGPU2CPU();
	WTDen.WTDenCPU2Disk(chunkFilePrefix);*/
	printf("WT ended!\n");

	//WT.CPU2GPUCountOffset();
	startTime = clock();
	for (int iter = 0; iter < numIters; iter++) {


		startTime1=clock();
		//printf("chunk WT updated!\n");
		WT.GPUMemset();
		//--------------update WTDenSum -----------
		UpdateWTDenRowSumKernel(WTDen,WT);
		//--------------update WTDenSum -----------

		//--------------update WTSum -----------
		for (int chunkId = 0; chunkId < numChunks; chunkId++) {
			WT.chunkCPU2GPUCountOffset(chunkId);
			WT.chunkGPUMemset();
			WT.chunkWTCPU2GPU(chunkId);
			WTAdditionKernel(WT, document);
		}
		//--------------update WTSum -----------
		//WT.WTGPU2CPU();// marker
		//WT.CPU2Disk(chunkFilePrefix);// marker






		printf("WT updated!\n");
		endTime = clock();
		WTTime+=(double)(endTime-startTime1)/CLOCKS_PER_SEC;


		document.CPU2GPUPerplexity();
		
		for (int chunkId = 0; chunkId < numChunks; chunkId++) {
			
			startTime1=clock();
			/*printf("step: %d\n",chunkId);*/
			//--------------update DT matrix-----------
			document.CPU2GPU(chunkId);
			/*printf("%d\n", 1);*/
			chunkDT.GPUMemSet(chunkId);
			/*printf("%d\n", 2);*/
			chunkDT.CPU2GPUDTCountOffset(chunkId);
			/*printf("%d\n", 3);*/
			//chunkDT.CPU2GPU(chunkId, document.docLengthVec[chunkId]);
			UpdateDTKernel(chunkDT, document);
			/*printf("%d\n", 4);*/
			//chunkDT.GPU2CPU(chunkId);
			//chunkDT.CPU2Disk(chunkFilePrefix, chunkId);// marker
			
			//--------------update DT matrix-----------
			endTime = clock();
			/*printf("%d\n", 5);*/
			DTTime+=(double)(endTime-startTime1)/CLOCKS_PER_SEC;

			

			startTime1=clock();
			//--------------sampling-----------
			/*printf("%d\n", 6);*/
			SampleKernelD(WTDen, WT, chunkDT, document);
			/*printf("%d\n", 7);*/
			//WTDen.WTDenGPU2CPU();// marker
			//WTDen.WTDenCPU2Disk(chunkFilePrefix);// marker
			SampleKernel(WT, chunkDT, document);
			/*printf("%d\n", 8);*/
			//WT.WTGPU2CPU();// marker
			//WT.CPU2Disk(chunkFilePrefix);// marker
			document.GPU2CPU(chunkId);
			//--------------sampling-----------

			endTime = clock();
			samplingTime+=(double)(endTime-startTime1)/CLOCKS_PER_SEC;

			startTime1=clock();
			//--------------update chunkWT matrix-----------
			WT.chunkCPU2GPUCountOffset(chunkId);
			WT.chunkGPUMemset();
			UpdateWTKernel(WT, document, chunkId);
			WT.chunkWTGPU2CPU(chunkId);
			//WT.CPU2DiskChunk(chunkFilePrefix, chunkId);
			//--------------update chunkWT matrix-----------
			endTime = clock();
			WTTime+=(double)(endTime-startTime1)/CLOCKS_PER_SEC;
		}
		WTDen.GPUMemCopy();
		WTDen.GPUMemset();

		printf("done!!!!!");
		document.GPU2CPUPerplexity();



		endTime = clock();
		totalTime=(double)(endTime-startTime)/CLOCKS_PER_SEC;
		timeRecord << WTTime << " " << DTTime << " " << samplingTime << " " << totalTime << " " << document.sumPerplexity<< "\n";
		printf("WTTime: %f, DTTime: %f, samplingTime:%f, totalTime:%f, perplexity:%f\n",WTTime,DTTime,samplingTime,totalTime,document.sumPerplexity);

	}
	
	timeRecord.close();
}
#endif