#include "hip/hip_runtime.h"
#include "utility.cuh"

#define	BUFF_SIZE_LONG	100000

__device__ struct maxStruct {
	float maxProb=0.0;
	unsigned short int maxK=0;

};
__global__ void WT_Update_Kernel(unsigned short int *d_a, int *d_count, unsigned short int *d_index, unsigned short int *d_value, int *d_slotcount, int *d_slotoffset, int *d_row_sum, unsigned int *d_counter_0, int d_token_amount_0, int *d_dense, int numOfTokenD) {

	int globalId = threadIdx.x + blockIdx.x * blockDim.x;
	int laneId = threadIdx.x % 32;
	int warpId = globalId / 32;
	int iterCounter = 0;
	unsigned int Counter;


	if (laneId == 0) {

		Counter = atomicAdd(&d_counter_0[0], 1);
	}
	Counter = __shfl(Counter, 0);

	while (Counter < d_token_amount_0)
		//while (warpId + iterCounter * gridDim.x*blockDim.x / 32< argD)
	{
		int wordId = Counter;
	
		for (int k = laneId; k < K; k += 32)
		{
			d_dense[k + K*warpId] = 0;
		}

		for (int i = d_slotoffset[wordId] + laneId; i < d_slotoffset[wordId] + d_slotcount[wordId]; i += 32)
		{

			unsigned short int topic = d_a[i+numOfTokenD];
			if ((topic < 1) || (topic > K)) printf("wrong Index:%d", topic);
			atomicAdd(&d_dense[K*warpId + topic - 1], 1);
		}

		int noneZeroCount = 0;
		for (int k = laneId; k < K; k += 32) {
			int value = d_dense[K*warpId + k];
			int flag = value > 0;
			int tmpNoneZeroCount = __popc(__ballot(value));

			if (tmpNoneZeroCount == 0) continue;

			flag += __shfl_up(flag, 1, 32)*(laneId >= 1);
			flag += __shfl_up(flag, 2, 32)*(laneId >= 2);
			flag += __shfl_up(flag, 4, 32)*(laneId >= 4);
			flag += __shfl_up(flag, 8, 32)*(laneId >= 8);
			flag += __shfl_up(flag, 16, 32)*(laneId >= 16);

			if (value) {
				int idx = d_slotoffset[wordId] + noneZeroCount + flag - 1;
				d_index[idx] = k + 1;
				d_value[idx] = value;
			}
			noneZeroCount += tmpNoneZeroCount;

		}
		/*if(laneId==0) d_count[docId] = noneZeroCount;*/
		if (laneId == 0) {
			d_count[wordId] = noneZeroCount;
			Counter = atomicAdd(&d_counter_0[0], 1);
		}
		Counter = __shfl(Counter, 0);

		/*iterCounter ++;*/

	}
	
}


__global__ void DT_Update_Kernel(int *d_Index, unsigned short int *d_a, int *d_count, int *d_slotcount, int *d_slotoffset, int *d_sparse_slotcount, int *d_sparse_slotoffset, unsigned int *d_counter_0, int argD, int *d_dense, long long int* deviceMaxSecTopic, int* deviceDTIndexValue)
{

	int globalId = threadIdx.x + blockIdx.x * blockDim.x;
	int laneId = threadIdx.x % 32;
	int warpId = threadIdx.x / 32;
	int iterCounter = 0;
	unsigned int Counter;
	__shared__ unsigned int DT[K*BlockDim/32];

	if (laneId == 0) {

		Counter = atomicAdd(&d_counter_0[0], 1);
	}
	Counter = __shfl(Counter, 0);

	while (Counter < argD)
	//while (warpId + iterCounter * gridDim.x*blockDim.x / 32< argD)
	{
		/*warpId = Counter;*/
		int docId = Counter;
		int docId_new = docId + 1;

		for (int k = laneId; k < K; k += 32)
		{
			DT[k + K*warpId] = 0;
		}
		int numIter = d_slotcount[docId]/64;
		
		for (int i = d_slotoffset[docId] + laneId; i+32 < d_slotoffset[docId] + numIter*64; i += 64)
		{
			unsigned short int topic1 = d_a[d_Index[i]];
			unsigned short int topic2 = d_a[d_Index[i+32]];
			if ((topic1 < 1) || (topic1 > K)) printf("wrong Index:%d", topic1);
			if ((topic2 < 1) || (topic2 > K)) printf("wrong Index:%d", topic2);
			atomicAdd(&DT[topic1+ K*warpId-1], 1);
			atomicAdd(&DT[topic2+ K*warpId-1], 1);
		}

		for (int i = d_slotoffset[docId] + numIter*64+ laneId; i < d_slotoffset[docId] + d_slotcount[docId]; i += 32)
		{
			unsigned short int topic = d_a[d_Index[i]];
			if ((topic < 1) || (topic > K)) printf("wrong Index:%d", topic);
			atomicAdd(&DT[topic+ K*warpId-1], 1);
		}
		
		for (int i = d_slotoffset[docId] + laneId; i+32 < d_slotoffset[docId] + numIter*64; i += 64)
		{

			int topic1 = deviceMaxSecTopic[d_Index[i]]&0x00000000ffffffff;
			int topic2 = deviceMaxSecTopic[d_Index[i+32]]&0x00000000ffffffff;
			unsigned short int maxTopic1 = topic1&0xffff;
			unsigned short int maxTopic2 = topic2&0xffff;
			unsigned short int secondTopic1 = (topic1 >> 16)&0xffff;
			unsigned short int secondTopic2 = (topic2 >> 16)&0xffff;
			unsigned short int maxCount1 = DT[maxTopic1-1 + K*warpId];
			unsigned short int maxCount2 = DT[maxTopic2-1 + K*warpId];
			unsigned short int secondCount1 = DT[secondTopic1-1 + K*warpId];
			unsigned short int secondCount2 = DT[secondTopic2-1 + K*warpId];
			int tokenCount1 = ((secondCount1 | int(0)) << 16) | maxCount1;
			int tokenCount2 = ((secondCount2 | int(0)) << 16) | maxCount2;
			
			deviceMaxSecTopic[d_Index[i]] = ((long long)(docId_new) << 32)|tokenCount1;
			deviceMaxSecTopic[d_Index[i+32]] = ((long long)(docId_new) << 32)|tokenCount2;

//			deviceDocIndex[d_Index[i]]=docId+1;
//			deviceDocIndex[d_Index[i+32]]=docId+1;

		}

		for (int i = d_slotoffset[docId] + numIter*64+ laneId; i < d_slotoffset[docId] + d_slotcount[docId]; i += 32)
		{

			int topic = deviceMaxSecTopic[d_Index[i]]&0x00000000ffffffff;
			unsigned short int maxTopic = topic&0xffff;
			unsigned short int secondTopic = (topic >> 16)&0xffff;
			unsigned short int maxCount = DT[maxTopic-1 + K*warpId];
			unsigned short int secondCount = DT[secondTopic-1 + K*warpId];
			int tokenCount = ((secondCount | int(0)) << 16) | maxCount;
			deviceMaxSecTopic[d_Index[i]] = ((long long) (docId_new) << 32)| tokenCount;
//			deviceDocIndex[d_Index[i]]=docId+1;

		}

//		for (int i = d_slotoffset[docId] + laneId; i < d_slotoffset[docId] + d_slotcount[docId]; i += 32)
//		{
//			unsigned short int topic = d_a[d_Index[i]];
//			if ((topic < 1) || (topic > K)) printf("wrong Index:%d", topic);
//			atomicAdd(&DT[topic+ K*warpId-1], 1);
//		}
//

	
//		for (int i = d_slotoffset[docId] + laneId; i < d_slotoffset[docId] + d_slotcount[docId]; i += 32)
//		{
//
//			int topic = deviceMaxSecTopic[d_Index[i]];
//			unsigned short int maxTopic = topic&0xffff;
//			unsigned short int secondTopic = (topic >> 16)&0xffff;
//			unsigned short int maxCount = DT[maxTopic-1 + K*warpId];
//			unsigned short int secondCount = DT[secondTopic-1 + K*warpId];
//			deviceMaxSecTopic[d_Index[i]] = ((secondCount | int(0)) << 16) | maxCount;
//
//		}
		int noneZeroCount = 0;

		for (int k = laneId; k+32 < K; k += 64) {
			int value1 = DT[k + K*warpId];
			int value2 = DT[k + 32 + K*warpId];
			int flag1 = value1 > 0;
			int flag2 = value2 > 0;
			int tmpNoneZeroCount1 = __popc(__ballot(flag1));
			int tmpNoneZeroCount2 = __popc(__ballot(flag2));
			int tmpNoneZeroCount = tmpNoneZeroCount1 + tmpNoneZeroCount2;
			/*long int m=1;*/
			if (tmpNoneZeroCount==0) continue;
//			if (tmpNoneZeroCount == 0) continue;
			
			int idx1 = __popc((__ballot(flag1))&((long(1)<<(laneId+1))-1));
			int idx2 = __popc((__ballot(flag2))&((long(1)<<(laneId+1))-1));
	
			
			/*flag += __shfl_up(flag, 1, 32)*(laneId >= 1);
			flag += __shfl_up(flag, 2, 32)*(laneId >= 2);
			flag += __shfl_up(flag, 4, 32)*(laneId >= 4);
			flag += __shfl_up(flag, 8, 32)*(laneId >= 8);
			flag += __shfl_up(flag, 16, 32)*(laneId >= 16);*/

		
			idx1 += (d_sparse_slotoffset[docId]+ noneZeroCount - 1);
			idx2 += (d_sparse_slotoffset[docId]+ noneZeroCount + tmpNoneZeroCount1- 1);

			if (value1) {
				//int idx = d_sparse_slotoffset[docId] + noneZeroCount+ flag-1;
				deviceDTIndexValue[idx1] = (((k+1) | int(0)) << 16) | value1;
				/*d_index[idx] = k+1;
				d_value[idx] = value;*/

			}
			if (value2) {
				//int idx = d_sparse_slotoffset[docId] + noneZeroCount+ flag-1;
				deviceDTIndexValue[idx2] = (((k+1+32) | int(0)) << 16) | value2;
				/*d_index[idx] = k+1;
				d_value[idx] = value;*/

			}
			noneZeroCount += tmpNoneZeroCount;

		}





		/*if(laneId==0) d_count[docId] = noneZeroCount;*/
		if (laneId == 0) {
			d_count[docId] = noneZeroCount;
			Counter = atomicAdd(&d_counter_0[0], 1);
		}
		Counter = __shfl(Counter, 0);

//
//		int noneZeroCount = 0;
//		for (int k = laneId; k < K; k += 32) {
//			int value = DT[k + K*warpId];
//			int flag = value > 0;
//
//			int tmpNoneZeroCount = __popc(__ballot(flag));
//			/*long int m=1;*/
//			if (tmpNoneZeroCount == 0) continue;
//
//			int idx = __popc((__ballot(flag))&((long(1)<<(laneId+1))-1));
//
//
//			/*flag += __shfl_up(flag, 1, 32)*(laneId >= 1);
//			flag += __shfl_up(flag, 2, 32)*(laneId >= 2);
//			flag += __shfl_up(flag, 4, 32)*(laneId >= 4);
//			flag += __shfl_up(flag, 8, 32)*(laneId >= 8);
//			flag += __shfl_up(flag, 16, 32)*(laneId >= 16);*/
//
//
//			idx += (d_sparse_slotoffset[docId]+ noneZeroCount - 1);
//
//			if (value) {
//				//int idx = d_sparse_slotoffset[docId] + noneZeroCount+ flag-1;
//				deviceDTIndexValue[idx] = (((k+1) | int(0)) << 16) | value;
//				/*d_index[idx] = k+1;
//				d_value[idx] = value;*/
//
//			}
//			noneZeroCount += tmpNoneZeroCount;
//
//		}
//		/*if(laneId==0) d_count[docId] = noneZeroCount;*/
//		if (laneId == 0) {
//			d_count[docId] = noneZeroCount;
//			Counter = atomicAdd(&d_counter_0[0], 1);
//		}
//		Counter = __shfl(Counter, 0);

		/*iterCounter ++;*/

	}


}




















__global__ void MaxTopicDense_Update_Kernel(unsigned short int* deviceWordMaxTopic, unsigned short int* deviceWordSecondMaxTopic, int *deviceWTDense, int *deviceTLCount, int *deviceTLOffset, int *deviceWTOffset, int numOfWordD, unsigned int* deviceCounter, int *deviceWTRowSum,int wordLength, float beta, unsigned short int* deviceWordThirdMaxTopic, long long int* deviceMaxSecTopic, float* deviceQArray, float* deviceWordMaxProb, float* deviceWordSecondMaxProb, float* deviceWordThirdMaxProb) {

	int laneId = threadIdx.x % 32;
	int localId = threadIdx.x / 32;
	volatile __shared__ float WTHead[K];
	volatile __shared__ float MaxTree[32];
	volatile __shared__ float MaxWT[3];
	volatile __shared__ unsigned short int MaxKTree[32];
	volatile __shared__ unsigned short int MaxK[3];
	volatile __shared__ unsigned int Counter[1];
	volatile __shared__ float QTree[32];
	if (threadIdx.x == 0) {
		Counter[0] = atomicAdd(&deviceCounter[0], 1);
	}
	__syncthreads();

	while (Counter[0] < numOfWordD)
	{
		int wordId = Counter[0];
		if (localId == 0) {
			MaxTree[laneId] = 0;
			MaxKTree[laneId] = 0;
			QTree[laneId] = 0;
		}

		int tokenStart = deviceTLOffset[wordId];
		int tokenEnd = deviceTLOffset[wordId] + deviceTLCount[wordId];
		int WTStart = deviceWTOffset[wordId];
		// Reconstruct dense WT vector from sparse WT matrix
		for (int i = threadIdx.x; i < K; i += blockDim.x)
		{
			WTHead[i] = (deviceWTDense[WTStart + i] + beta) / (deviceWTRowSum[i] + wordLength*beta);
		}
		__syncthreads();

		for (int i = localId; i < K / 32; i += blockDim.x / 32) {
			short int   tmpK = i * 32 + laneId;
			float tmpVal = 0.0;
			tmpVal = WTHead[tmpK];
			tmpVal += __shfl_down(tmpVal, 16);
			tmpVal += __shfl_down(tmpVal, 8);
			tmpVal += __shfl_down(tmpVal, 4);
			tmpVal += __shfl_down(tmpVal, 2);
			tmpVal += __shfl_down(tmpVal, 1);
			tmpVal = __shfl(tmpVal, 0);
			QTree[i] = tmpVal;

		}
		__syncthreads();

		if (localId == 0) {

			float value = QTree[laneId];
			value += __shfl_down(value, 16);
			value += __shfl_down(value, 8);
			value += __shfl_down(value, 4);
			value += __shfl_down(value, 2);
			value += __shfl_down(value, 1);
			value = __shfl(value, 0);
			deviceQArray[wordId] = value;

		}



		




		// Find maxK
		for (int i = localId; i < K / 32; i += blockDim.x / 32) {
			unsigned short int   tmpK = i * 32 + laneId;
			float tmpMax = 0.0;
			float tmpMax1 = 0.0;
			unsigned short int tmpK1 = 0;
			tmpMax = WTHead[tmpK];
			tmpMax1 = __shfl_down(tmpMax, 16);
			tmpK1 = __shfl_down(tmpK, 16);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 8);
			tmpK1 = __shfl_down(tmpK, 8);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 4);
			tmpK1 = __shfl_down(tmpK, 4);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 2);
			tmpK1 = __shfl_down(tmpK, 2);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 1);
			tmpK1 = __shfl_down(tmpK, 1);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax = __shfl(tmpMax, 0);
			tmpK = __shfl(tmpK, 0);
			MaxTree[i] = tmpMax;
			MaxKTree[i] = tmpK;
		}
		__syncthreads();

		if (localId == 0) {

			float tmpMax = 0.0;
			float tmpMax1 = 0.0;
			unsigned short int tmpMaxK = 0;
			unsigned short int tmpMaxK1 = 0;
			tmpMax = MaxTree[laneId];
			tmpMaxK = MaxKTree[laneId];
			tmpMax1 = __shfl_down(tmpMax, 16);
			tmpMaxK1 = __shfl_down(tmpMaxK, 16);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpMaxK = tmpMaxK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 8);
			tmpMaxK1 = __shfl_down(tmpMaxK, 8);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpMaxK = tmpMaxK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 4);
			tmpMaxK1 = __shfl_down(tmpMaxK, 4);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpMaxK = tmpMaxK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 2);
			tmpMaxK1 = __shfl_down(tmpMaxK, 2);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpMaxK = tmpMaxK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 1);
			tmpMaxK1 = __shfl_down(tmpMaxK, 1);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpMaxK = tmpMaxK1;
			}
			tmpMaxK = __shfl(tmpMaxK, 0);
			if (laneId == 0) {
				MaxWT[laneId] = tmpMax;
				MaxK[laneId] = tmpMaxK;
			}

		}
		__syncthreads();
		if (localId == 0) {

			float tmpMax = 0.0;
			float tmpMax1 = 0.0;
			unsigned short int tmpPosition = MaxK[0] / 32;
			unsigned short int tmpK = (tmpPosition) * 32 + laneId;
			unsigned short int tmpK1 = 0;
			

			tmpMax = WTHead[tmpK];
			if (tmpK == MaxK[0]) tmpMax = 0.0;
			tmpMax1 = __shfl_down(tmpMax, 16);
			tmpK1 = __shfl_down(tmpK, 16);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 8);
			tmpK1 = __shfl_down(tmpK, 8);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 4);
			tmpK1 = __shfl_down(tmpK, 4);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 2);
			tmpK1 = __shfl_down(tmpK, 2);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 1);
			tmpK1 = __shfl_down(tmpK, 1);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax = __shfl(tmpMax, 0);
			tmpK = __shfl(tmpK, 0);
			MaxTree[tmpPosition] = tmpMax;
			MaxKTree[tmpPosition] = tmpK;

			tmpMax = 0.0;
			tmpMax1 = 0.0;
			tmpMax = MaxTree[laneId];
			tmpMax1 = __shfl_down(tmpMax, 16);
			tmpK = MaxKTree[laneId];
			tmpK1 = __shfl_down(tmpK, 16);

			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 8);
			tmpK1 = __shfl_down(tmpK, 8);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 4);
			tmpK1 = __shfl_down(tmpK, 4);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 2);
			tmpK1 = __shfl_down(tmpK, 2);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 1);
			tmpK1 = __shfl_down(tmpK, 1);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			if (laneId == 0) {
				MaxWT[1] = tmpMax;
				MaxK[1] = tmpK;
			}
		}
		__syncthreads();

		if (localId == 0) {

			float tmpMax = 0.0;
			float tmpMax1 = 0.0;
			unsigned short int tmpPosition = MaxK[1] / 32;
			unsigned short int tmpK = (tmpPosition) * 32 + laneId;
			unsigned short int tmpK1 = 0;


			tmpMax = WTHead[tmpK];
			if (tmpK == MaxK[0]) tmpMax = 0.0;
			if (tmpK == MaxK[1]) tmpMax = 0.0;
			tmpMax1 = __shfl_down(tmpMax, 16);
			tmpK1 = __shfl_down(tmpK, 16);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 8);
			tmpK1 = __shfl_down(tmpK, 8);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 4);
			tmpK1 = __shfl_down(tmpK, 4);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 2);
			tmpK1 = __shfl_down(tmpK, 2);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 1);
			tmpK1 = __shfl_down(tmpK, 1);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax = __shfl(tmpMax, 0);
			tmpK = __shfl(tmpK, 0);
			MaxTree[tmpPosition] = tmpMax;
			MaxKTree[tmpPosition] = tmpK;

			tmpMax = 0.0;
			tmpMax1 = 0.0;
			tmpMax = MaxTree[laneId];
			tmpMax1 = __shfl_down(tmpMax, 16);
			tmpK = MaxKTree[laneId];
			tmpK1 = __shfl_down(tmpK, 16);

			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 8);
			tmpK1 = __shfl_down(tmpK, 8);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 4);
			tmpK1 = __shfl_down(tmpK, 4);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 2);
			tmpK1 = __shfl_down(tmpK, 2);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 1);
			tmpK1 = __shfl_down(tmpK, 1);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			if (laneId == 0) {
				MaxWT[2] = tmpMax;
				MaxK[2] = tmpK;
			}
		}

		__syncthreads();


		//float WTMax = MaxWT[0];
		//float WTMax2 = MaxWT[1];
		//float WTMax3 = MaxWT[2];
		unsigned short int WTMaxK = MaxK[0]+1;
		unsigned short int WTSecondMaxK = MaxK[1]+1;
		unsigned short int WTThirdMaxK = MaxK[2]+1;
		if (threadIdx.x == 0) {
			deviceWordMaxTopic[wordId] = WTMaxK;
			deviceWordSecondMaxTopic[wordId] = WTSecondMaxK;
			deviceWordThirdMaxTopic[wordId] = WTThirdMaxK;
			
			
		}
		if (threadIdx.x == 32) {
			deviceWordMaxProb[wordId]= WTHead[WTMaxK - 1];
			deviceWordSecondMaxProb[wordId]= WTHead[WTSecondMaxK - 1];
			deviceWordThirdMaxProb[wordId]= WTHead[WTThirdMaxK - 1];
			deviceQArray[wordId] = deviceQArray[wordId]- deviceWordMaxProb[wordId];
		}
		for (int i = tokenStart+ threadIdx.x; i < tokenEnd; i += blockDim.x) {
			//int maxSecTopic = 0;
			//maxSecTopic = maxSecTopic | WTSecondMaxK;
			//maxSecTopic = maxSecTopic << 16;
			///*deviceMaxTopic[i] = WTMaxK;
			//deviceSecondMaxTopic[i] = WTSecondMaxK;*/
			//deviceMaxSecTopic[i] = maxSecTopic|WTMaxK;
			deviceMaxSecTopic[i] = ((WTSecondMaxK | int(0)) << 16) | WTMaxK;

		}

		if (threadIdx.x == 0) Counter[0] = atomicAdd(&deviceCounter[0], 1);
		__syncthreads();
	}

}
__global__ void MaxTopicSparse_Update_Kernel(unsigned short int* deviceWordMaxTopic, unsigned short int* deviceWordSecondMaxTopic,  int *deviceTLCount, int *deviceTLOffset, int *deviceWTOffset, int numOfWordD, unsigned int* deviceCounter, int *deviceWTRowSum, int wordLength, int numOfWordS, int* d_WordListOffset, int* d_SparseWTCount, unsigned short int* d_SparseWTIndex, unsigned short int* d_SparseWTValue, float beta, unsigned short int* deviceWordThirdMaxTopic, long long int* deviceMaxSecTopic, float* deviceQArray, float* deviceWordMaxProb, float* deviceWordSecondMaxProb, float* deviceWordThirdMaxProb) {


	int laneId = threadIdx.x % 32;
	int localId = threadIdx.x / 32;
	volatile __shared__ float WTHead[K];
	volatile __shared__ float MaxTree[32];
	volatile __shared__ float MaxWT[3];
	volatile __shared__ unsigned short int MaxKTree[32];
	volatile __shared__ unsigned short int MaxK[3];
	volatile __shared__ unsigned int Counter[1];
	volatile __shared__ float QTree[32];
	if (threadIdx.x == 0) {
		Counter[0] = atomicAdd(&deviceCounter[0], 1);
	}
	__syncthreads();

	while (Counter[0] < numOfWordS)
	{
		int wordId = Counter[0] + numOfWordD;
		if (localId == 0) {
			MaxTree[laneId] = 0;
			MaxKTree[laneId] = 0;
			QTree[laneId] = 0;
		}

		long long tokenStart = deviceTLOffset[wordId];
		long long tokenEnd = deviceTLOffset[wordId] + deviceTLCount[wordId];
		long long WTStart = d_WordListOffset[wordId] - d_WordListOffset[numOfWordD];
		long long WTEnd = d_WordListOffset[wordId] - d_WordListOffset[numOfWordD] + d_SparseWTCount[wordId - numOfWordD];
		// Reconstruct dense WT vector from sparse WT matrix
		for (int i = threadIdx.x; i < K; i += blockDim.x)
		{
			WTHead[i] = beta / (deviceWTRowSum[i] + wordLength*beta);

		}
		__syncthreads();

		for (int i = threadIdx.x + WTStart; i < WTEnd; i += blockDim.x)
		{
			WTHead[d_SparseWTIndex[i] - 1] = (d_SparseWTValue[i] + beta) / (deviceWTRowSum[d_SparseWTIndex[i] - 1] + wordLength*beta);

		}
		__syncthreads();




		for (int i = localId; i < K / 32; i += blockDim.x / 32) {
			short int   tmpK = i * 32 + laneId;
			float tmpVal = 0.0;
			tmpVal = WTHead[tmpK];
			tmpVal += __shfl_down(tmpVal, 16);
			tmpVal += __shfl_down(tmpVal, 8);
			tmpVal += __shfl_down(tmpVal, 4);
			tmpVal += __shfl_down(tmpVal, 2);
			tmpVal += __shfl_down(tmpVal, 1);
			tmpVal = __shfl(tmpVal, 0);
			QTree[i] = tmpVal;

		}
		__syncthreads();

		if (localId == 0) {

			float value = QTree[laneId];
			value += __shfl_down(value, 16);
			value += __shfl_down(value, 8);
			value += __shfl_down(value, 4);
			value += __shfl_down(value, 2);
			value += __shfl_down(value, 1);
			value = __shfl(value, 0);
			deviceQArray[wordId] = value;

		}





		// Find maxK
		for (int i = localId; i < K / 32; i += blockDim.x / 32) {
			unsigned short int   tmpK = i * 32 + laneId;
			float tmpMax = 0.0;
			float tmpMax1 = 0.0;
			unsigned short int tmpK1 = 0;
			tmpMax = WTHead[tmpK];
			tmpMax1 = __shfl_down(tmpMax, 16);
			tmpK1 = __shfl_down(tmpK, 16);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 8);
			tmpK1 = __shfl_down(tmpK, 8);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 4);
			tmpK1 = __shfl_down(tmpK, 4);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 2);
			tmpK1 = __shfl_down(tmpK, 2);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 1);
			tmpK1 = __shfl_down(tmpK, 1);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax = __shfl(tmpMax, 0);
			tmpK = __shfl(tmpK, 0);
			MaxTree[i] = tmpMax;
			MaxKTree[i] = tmpK;
		}
		__syncthreads();

		if (localId == 0) {

			float tmpMax = 0.0;
			float tmpMax1 = 0.0;
			unsigned short int tmpMaxK = 0;
			unsigned short int tmpMaxK1 = 0;
			tmpMax = MaxTree[laneId];
			tmpMaxK = MaxKTree[laneId];
			tmpMax1 = __shfl_down(tmpMax, 16);
			tmpMaxK1 = __shfl_down(tmpMaxK, 16);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpMaxK = tmpMaxK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 8);
			tmpMaxK1 = __shfl_down(tmpMaxK, 8);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpMaxK = tmpMaxK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 4);
			tmpMaxK1 = __shfl_down(tmpMaxK, 4);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpMaxK = tmpMaxK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 2);
			tmpMaxK1 = __shfl_down(tmpMaxK, 2);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpMaxK = tmpMaxK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 1);
			tmpMaxK1 = __shfl_down(tmpMaxK, 1);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpMaxK = tmpMaxK1;
			}
			tmpMaxK = __shfl(tmpMaxK, 0);
			if (laneId == 0) {
				MaxWT[laneId] = tmpMax;
				MaxK[laneId] = tmpMaxK;
			}

		}
		__syncthreads();
		if (localId == 0) {

			float tmpMax = 0.0;
			float tmpMax1 = 0.0;
			unsigned short int tmpPosition = MaxK[0] / 32;
			unsigned short int tmpK = (tmpPosition) * 32 + laneId;
			unsigned short int tmpK1 = 0;


			tmpMax = WTHead[tmpK];
			if (tmpK == MaxK[0]) tmpMax = 0.0;
			tmpMax1 = __shfl_down(tmpMax, 16);
			tmpK1 = __shfl_down(tmpK, 16);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 8);
			tmpK1 = __shfl_down(tmpK, 8);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 4);
			tmpK1 = __shfl_down(tmpK, 4);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 2);
			tmpK1 = __shfl_down(tmpK, 2);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 1);
			tmpK1 = __shfl_down(tmpK, 1);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax = __shfl(tmpMax, 0);
			tmpK = __shfl(tmpK, 0);
			MaxTree[tmpPosition] = tmpMax;
			MaxKTree[tmpPosition] = tmpK;

			tmpMax = 0.0;
			tmpMax1 = 0.0;
			tmpMax = MaxTree[laneId];
			tmpMax1 = __shfl_down(tmpMax, 16);
			tmpK = MaxKTree[laneId];
			tmpK1 = __shfl_down(tmpK, 16);

			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 8);
			tmpK1 = __shfl_down(tmpK, 8);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 4);
			tmpK1 = __shfl_down(tmpK, 4);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 2);
			tmpK1 = __shfl_down(tmpK, 2);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 1);
			tmpK1 = __shfl_down(tmpK, 1);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			if (laneId == 0) {
				MaxWT[1] = tmpMax;
				MaxK[1] = tmpK;
			}
		}
		__syncthreads();


		if (localId == 0) {

			float tmpMax = 0.0;
			float tmpMax1 = 0.0;
			unsigned short int tmpPosition = MaxK[1] / 32;
			unsigned short int tmpK = (tmpPosition) * 32 + laneId;
			unsigned short int tmpK1 = 0;


			tmpMax = WTHead[tmpK];
			if (tmpK == MaxK[0]) tmpMax = 0.0;
			if (tmpK == MaxK[1]) tmpMax = 0.0;
			tmpMax1 = __shfl_down(tmpMax, 16);
			tmpK1 = __shfl_down(tmpK, 16);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 8);
			tmpK1 = __shfl_down(tmpK, 8);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 4);
			tmpK1 = __shfl_down(tmpK, 4);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 2);
			tmpK1 = __shfl_down(tmpK, 2);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 1);
			tmpK1 = __shfl_down(tmpK, 1);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax = __shfl(tmpMax, 0);
			tmpK = __shfl(tmpK, 0);
			MaxTree[tmpPosition] = tmpMax;
			MaxKTree[tmpPosition] = tmpK;

			tmpMax = 0.0;
			tmpMax1 = 0.0;
			tmpMax = MaxTree[laneId];
			tmpMax1 = __shfl_down(tmpMax, 16);
			tmpK = MaxKTree[laneId];
			tmpK1 = __shfl_down(tmpK, 16);

			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 8);
			tmpK1 = __shfl_down(tmpK, 8);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 4);
			tmpK1 = __shfl_down(tmpK, 4);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 2);
			tmpK1 = __shfl_down(tmpK, 2);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			tmpMax1 = __shfl_down(tmpMax, 1);
			tmpK1 = __shfl_down(tmpK, 1);
			if (tmpMax < tmpMax1) {
				tmpMax = tmpMax1;
				tmpK = tmpK1;
			}
			if (laneId == 0) {
				MaxWT[2] = tmpMax;
				MaxK[2] = tmpK;
			}
		}

		__syncthreads();

	/*	float WTMax = MaxWT[0];
		float WTMax2 = MaxWT[1];
		float WTMax3 = MaxWT[2];*/
		unsigned short int WTMaxK = MaxK[0]+1;
		unsigned short int WTSecondMaxK = MaxK[1]+1;
		unsigned short int WTThirdMaxK = MaxK[2]+1;
		if (threadIdx.x == 0) {
			deviceWordMaxTopic[wordId] = WTMaxK;
			deviceWordSecondMaxTopic[wordId] = WTSecondMaxK;
			deviceWordThirdMaxTopic[wordId] = WTThirdMaxK;
		}
		if (threadIdx.x == 32) {
			deviceWordMaxProb[wordId] = WTHead[WTMaxK - 1];
			deviceWordSecondMaxProb[wordId] = WTHead[WTSecondMaxK - 1];
			deviceWordThirdMaxProb[wordId] = WTHead[WTThirdMaxK - 1];
			deviceQArray[wordId] = deviceQArray[wordId]- deviceWordMaxProb[wordId];
		}

		for (int i = tokenStart + threadIdx.x; i < tokenEnd; i += blockDim.x) {
			/*deviceMaxTopic[i] = WTMaxK + 1;
			deviceSecondMaxTopic[i] = WTSecondMaxK + 1;*/
			//int maxSecTopic = 0;
			//maxSecTopic = maxSecTopic | WTSecondMaxK;
			//maxSecTopic = maxSecTopic << 16;
			///*deviceMaxTopic[i] = WTMaxK;
			//deviceSecondMaxTopic[i] = WTSecondMaxK;*/
			//deviceMaxSecTopic[i] = maxSecTopic | WTMaxK;
			deviceMaxSecTopic[i] = ((WTSecondMaxK | int(0)) << 16) | WTMaxK;
		}
		if (threadIdx.x == 0) Counter[0] = atomicAdd(&deviceCounter[0], 1);
		__syncthreads();

	}

}




__global__ void WTDen_Update_Kernel(unsigned short int *deviceTopic, int *deviceWTDense, int *deviceTLCount, int *deviceTLOffset, int *deviceWTOffset, int numOfWordD, unsigned int* deviceCounter)
{
	int globalId = threadIdx.x + blockIdx.x * blockDim.x;
	int laneId = threadIdx.x % 32;
	int warpId = globalId / 32;
	unsigned int Counter;


	if (laneId == 0) {

		Counter = atomicAdd(&deviceCounter[0], 1);
	}
	Counter = __shfl(Counter, 0);

	while (Counter < numOfWordD)
		
	{
		int wordId = Counter;
		unsigned short int tmpTopic;
		int tmpWTOffset = deviceWTOffset[wordId];
		int tmpTLOffset = deviceTLOffset[wordId];

		for (int k = laneId; k < deviceTLCount[wordId]; k += 32)
		{
			tmpTopic = deviceTopic[tmpTLOffset + k];
			atomicAdd(&deviceWTDense[tmpWTOffset + tmpTopic - 1], 1);
		}

		if (laneId == 0)  Counter = atomicAdd(&deviceCounter[0], 1);
		Counter = __shfl(Counter, 0);

	}



}

__global__ void WTDen_Sum_Update_Kernel(int *deviceWTDense, int *deviceWTRowSum, int *deviceWTOffset, int numOfWordD)
{

	int input;
	int tid = threadIdx.x;
	int globalId = threadIdx.x + blockIdx.x * blockDim.x;
	int blockId = blockIdx.x;
	int tmpIndex;

	for (int k = globalId; k < K; k += GridDim*BlockDim)
	{
		for (int i = 0; i < numOfWordD; i ++)
		{
			tmpIndex = deviceWTOffset[i]  + k;
			deviceWTRowSum[k] += deviceWTDense[tmpIndex];

		}
	}
	__syncthreads();



}





__global__ void sparseMatrixAdd(int* argCount0, int* argOffset0, int* argNZCount0, unsigned short int* argIndex0, unsigned short int* argValue0, int* argCount1, int* argOffset1, int* argNZCount1, unsigned short int* argIndex1, unsigned short int* argValue1, int* argDense, int argNumRows, unsigned int* deviceCounter, int* argWTRowSum, int numOfWordD)
{

	int globalId = threadIdx.x + blockIdx.x * blockDim.x;
	int laneId = threadIdx.x % 32;
	int warpId = globalId / 32;
	int iterCounter = 0;
	unsigned int Counter;

	if (laneId == 0) {

		Counter = atomicAdd(&deviceCounter[0], 1);
	}
	Counter = __shfl(Counter, 0);

	while (Counter < argNumRows)
		//while (warpId + iterCounter * gridDim.x*blockDim.x / 32< argD)
	{
		int wordId = Counter;

		for (int k = laneId; k < K; k += 32)
		{
			argDense[k + K*warpId] = 0;
		}

		for (int k = laneId; k < argNZCount0[wordId]; k += 32)
		{
			int tmpIdx = argOffset0[wordId + numOfWordD] - K*numOfWordD + k;
			argDense[K*warpId + argIndex0[tmpIdx] - 1] += argValue0[tmpIdx];
		}

		for (int k = laneId; k < argNZCount1[wordId]; k += 32)
		{

			int tmpIdx = argOffset1[wordId] + k;
			atomicAdd(&argWTRowSum[argIndex1[tmpIdx] - 1], argValue1[tmpIdx]);
			argDense[K*warpId + argIndex1[tmpIdx] - 1] += argValue1[tmpIdx];
		}
		int noneZeroCount = 0;
		for (int k = laneId; k < K; k += 32) {
			int value = argDense[K*warpId + k];
			int flag = value > 0;
			int tmpNoneZeroCount = __popc(__ballot(value));

			if (tmpNoneZeroCount == 0) continue;

			flag += __shfl_up(flag, 1, 32)*(laneId >= 1);
			flag += __shfl_up(flag, 2, 32)*(laneId >= 2);
			flag += __shfl_up(flag, 4, 32)*(laneId >= 4);
			flag += __shfl_up(flag, 8, 32)*(laneId >= 8);
			flag += __shfl_up(flag, 16, 32)*(laneId >= 16);

			if (value) {
				int idx = argOffset0[wordId + numOfWordD] - K*numOfWordD + noneZeroCount + flag - 1;
				argIndex0[idx] = k + 1;
				argValue0[idx] = value;
			}
			noneZeroCount += tmpNoneZeroCount;

		}

		if (laneId == 0) {
			argNZCount0[wordId] = noneZeroCount;
			Counter = atomicAdd(&deviceCounter[0], 1);
		}
		Counter = __shfl(Counter, 0);


	}



}




__global__ void initRandState(hiprandState *state)
{
	int tid = blockIdx.x*blockDim.x + threadIdx.x;
	hiprand_init(clock() + tid, tid, 0, &state[tid]);
}



__global__ void LDAKernelTrain(float alpha, float beta, int* d_Index, unsigned short int* d_TopicIndex, int* d_SparseDTCount, int* d_TokenCountDT, int* d_TokenOffsetDT, int* d_DocListCount, int* d_DocListOffset, int* d_SparseWTCount, unsigned short int* d_SparseWTIndex, unsigned short int* d_SparseWTValue, int* d_TokenCount, int* d_TokenOffset, int* d_WordListCount, int* d_WordListOffset, int* d_WTRowSum, unsigned int* d_blockCounter, int*d_DocIndex, int D, int W, float* d_Perplexity, hiprandState *randState, float *WTHeadDense, int numOfWordD, int numOfWordS,  unsigned short int* deviceWordMaxTopic, unsigned short int* deviceWordSecondMaxTopic, long long int* deviceMaxSecTopic, int* deviceDTIndexValue)

{
	int tid = threadIdx.x;
	int laneId = threadIdx.x % 32;
	int localId = threadIdx.x / 32;
	int blockId = blockIdx.x;
	volatile __shared__ float WTHead[K];
	volatile __shared__ float QTree[32];
	volatile __shared__ float WTMax[2];
	volatile __shared__ float STree[ShaMemSize / 32][K / 32];
	volatile __shared__ float prefixSumSample[ShaMemSize / 32][32];
	volatile __shared__ unsigned int Counter[1];
	__shared__ unsigned int WarpCounter[1];

	if (tid == 0) {
		Counter[0] = atomicAdd(&d_blockCounter[0], 1);	
	}
	__syncthreads();

	float sumPerplexity = 0.0;

	while (Counter[0]<numOfWordS)
	{
		int wordId = Counter[0]+ numOfWordD;
		if (localId == 0) {
			QTree[laneId] = 0;

		}
		float p_temp1 = 0.0;
		prefixSumSample[localId][laneId] = 0.0;
		long long tokenStart = d_TokenOffset[wordId];
		long long tokenEnd = d_TokenOffset[wordId] + d_TokenCount[wordId];
		long long WTStart = d_WordListOffset[wordId] - d_WordListOffset[numOfWordD];
		long long WTEnd = d_WordListOffset[wordId] - d_WordListOffset[numOfWordD] + d_SparseWTCount[wordId - numOfWordD];

		unsigned short int maxK = deviceWordMaxTopic[wordId];
		unsigned short int secondMaxK = deviceWordSecondMaxTopic[wordId];

		
		for (int i = tid; i < K; i += blockDim.x)
		{
			WTHead[i] = beta / (d_WTRowSum[i] + W*beta);
			
		}

		__syncthreads();

		for (int i = tid + WTStart; i < WTEnd; i += blockDim.x)
		{
			WTHead[d_SparseWTIndex[i] - 1] = (d_SparseWTValue[i] + beta) / (d_WTRowSum[d_SparseWTIndex[i] - 1] + W*beta);
			
		}
		
		__syncthreads();
		if (threadIdx.x == 0) {
			WTMax[0] = WTHead[maxK - 1];
			WTMax[1] = WTHead[secondMaxK - 1];
			WTHead[maxK - 1] = 0.0;
		}
		__syncthreads();
		

		for (int i = localId; i < K / 32; i += blockDim.x / 32) {
			unsigned short int   tmpK = i * 32 + laneId;
			//__syncthreads();
			float tmpVal = 0.0;
			tmpVal = alpha*WTHead[tmpK];
			tmpVal += __shfl_down(tmpVal, 16);
			tmpVal += __shfl_down(tmpVal, 8);
			tmpVal += __shfl_down(tmpVal, 4);
			tmpVal += __shfl_down(tmpVal, 2);
			tmpVal += __shfl_down(tmpVal, 1);
			tmpVal = __shfl(tmpVal, 0);
			QTree[i] = tmpVal;

			
		}
		__syncthreads();


		if (localId == 0) {

			float value = QTree[laneId];
			value += __shfl_up(value, 1, 32)*(laneId >= 1);
			value += __shfl_up(value, 2, 32)*(laneId >= 2);
			value += __shfl_up(value, 4, 32)*(laneId >= 4);
			value += __shfl_up(value, 8, 32)*(laneId >= 8);
			value += __shfl_up(value, 16, 32)*(laneId >= 16);

			QTree[laneId] = value;



		}



		if (tid == 0) WarpCounter[0] = tokenStart;
		
		__syncthreads();


		float Q = QTree[31];
		int tokenIdx;

		if (laneId == 0)
		{
			tokenIdx = atomicAdd(&WarpCounter[0], 1);

		}
		tokenIdx = __shfl(tokenIdx, 0);

		//unsigned short int maxTopic = deviceMaxTopic[tokenIdx];
		float WTMaxProb = WTMax[0];
		float WTSecondMaxProb = WTMax[1];

		while (tokenIdx<tokenEnd)
		{

			//int docId = __ldg(&d_Index[d_TopicIndex[tokenIdx]]);
			int oldZ=d_TopicIndex[tokenIdx];
			//int docId = d_DocIndex[tokenIdx];
			/*int docId = (deviceMaxSecTopic[tokenIdx]&0xffffffff00000000)>>32;*/
			int docId = (int)(deviceMaxSecTopic[tokenIdx] >> 32);
			
			
			//unsigned short int maxTokenCount = deviceMaxTokenCount[tokenIdx];
			unsigned short int maxTokenCount = ((int) deviceMaxSecTopic[tokenIdx])&(0xffff);
			int totalTokenCount = d_TokenCountDT[docId - 1];
			float maxProbability = (maxTokenCount+alpha)*WTMaxProb;
			float maxS = (totalTokenCount - maxTokenCount)*WTSecondMaxProb;
			float thresProb = maxProbability / (maxProbability + maxS + Q);

			if (maxS < 0) printf("wrong maxS\n");
			float u;
			if (laneId == 0)u = hiprand_uniform(&(randState[threadIdx.x + blockDim.x*blockIdx.x])) / 1.00001;
			u = __shfl(u, 0);

			int newZ = 1;
			unsigned short int sFlag = 1;

			if (u < thresProb) {
		
				newZ = maxK;
				//if (laneId == 0) {
				//	//newZ = maxTopic;
				//	/*atomicAdd(&d_WTDenseCopy[WTStart + newZ - 1], 1);*/
				//	//d_TopicIndex[tokenIdx] = newZ;
				//	tokenIdx = atomicAdd(&WarpCounter[0], 1);
				//}
				//tokenIdx = __shfl(tokenIdx, 0);
				/*continue;*/
				sFlag = 0;
				goto endloop;

			}
			else{
			//computing S.
				float S = 0;
				long long DTStart = d_DocListOffset[docId - 1];
				long long DTEnd = d_DocListOffset[docId - 1] + ((d_SparseDTCount[docId - 1] - 1) / 32 + 1) * 32;

				//long long DTEnd = d_DocListOffset[docId - 1] + d_SparseDTCount[docId - 1];


				STree[localId][laneId] = 0;
				// //__syncthreads();
				int SIdx = 0;
				float  tmpP1k = 0.0;
				short int  colVal;
				short int  colK;
				
				//maxStruct maxValue;
				for (int tmpIdx = DTStart + laneId; tmpIdx < DTEnd - 32; tmpIdx += 32) {

					int DTIndexValue = deviceDTIndexValue[tmpIdx];
					colVal = DTIndexValue & 0xffff;
					colK = (DTIndexValue >> 16) & 0xffff;
					/*colVal = d_SparseDTValue[tmpIdx];
					colK = d_SparseDTIndex[tmpIdx];*/
					/*colVal = d_SparseDTValue[tmpIdx];
					colK = d_SparseDTIndex[tmpIdx];*/
					tmpP1k = colVal*WTHead[colK - 1];	
					//if (colK == maxTopic) tmpP1k = 0.0;


					tmpP1k += __shfl_down(tmpP1k, 16);
					tmpP1k += __shfl_down(tmpP1k, 8);
					tmpP1k += __shfl_down(tmpP1k, 4);
					tmpP1k += __shfl_down(tmpP1k, 2);
					tmpP1k += __shfl_down(tmpP1k, 1);
					tmpP1k = __shfl(tmpP1k, 0);

					S += tmpP1k;
					STree[localId][SIdx] = S;

					SIdx++;
				}

				tmpP1k = 0.0;
				int DTIndexValue = deviceDTIndexValue[DTEnd - 32 + laneId];
				colVal = DTIndexValue & 0xffff;
				colK = (DTIndexValue >> 16) & 0xffff;
				/*colVal = d_SparseDTValue[DTEnd - 32 + laneId];
				colK = d_SparseDTIndex[DTEnd - 32 + laneId];*/
				if (colK != 0) tmpP1k = colVal*WTHead[colK - 1];
				//if (colK == maxTopic) tmpP1k = 0.0;

				tmpP1k += __shfl_down(tmpP1k, 16);
				tmpP1k += __shfl_down(tmpP1k, 8);
				tmpP1k += __shfl_down(tmpP1k, 4);
				tmpP1k += __shfl_down(tmpP1k, 2);
				tmpP1k += __shfl_down(tmpP1k, 1);
				tmpP1k = __shfl(tmpP1k, 0);
				S += tmpP1k;
				STree[localId][SIdx] = S;


				//__syncthreads();
				/*STmp = S;

				S = __shfl(STmp, 0);*/
				S = __shfl(S, 0);
				//__syncthreads();
				//randomly generate u.


				/*if (maxProbability / (maxProbability + S + Q) < thresProb) printf("Wrong Prob!!!!%f,%f,%d\n", maxProbability / (maxProbability + S + Q), thresProb, maxK);*/

				if (u < maxProbability / (maxProbability + S + Q)) {

					newZ = maxK;

				}

				else if ((u >= maxProbability / (maxProbability + S + Q)) && (u< (maxProbability + S) / (maxProbability + S + Q)))
				{
					//float transU = u*(S + Q);
					float transU = u*(maxProbability + S + Q) - maxProbability;

					float tmpSumHigh, tmpSumLow = 0.0;
					tmpSumHigh = STree[localId][laneId];
					tmpSumLow = __shfl_up(tmpSumHigh, 1, 32);
					if (laneId == 0)tmpSumLow = 0;
					int voteFlag = 0;
					if ((transU < tmpSumHigh)) voteFlag = 1;
					int lvl1Idx = __ffs(__ballot(voteFlag)) - 1;
					//int overflowFlag = 0;

					if (lvl1Idx < 0) lvl1Idx = (DTEnd - DTStart) / 32 - 1;
					
					transU = transU - tmpSumLow;

					transU = __shfl(transU, lvl1Idx);
					int tmpIdx = DTStart + lvl1Idx * 32 + laneId;
					/*int tmpNewZ = d_SparseDTIndex[tmpIdx];
					int colVal = d_SparseDTValue[tmpIdx];*/
					int  DTIndexValue = deviceDTIndexValue[tmpIdx];
					short int colVal = DTIndexValue & 0xffff;
					short int tmpNewZ = (DTIndexValue >> 16) & 0xffff;
					float p1k = 0.0;
					if (tmpNewZ != 0)
					{
						p1k = colVal*WTHead[tmpNewZ - 1];
					}
					prefixSumSample[localId][laneId] = p1k;
					float value = prefixSumSample[localId][laneId];
					value += __shfl_up(value, 1, 32)*(laneId >= 1);
					value += __shfl_up(value, 2, 32)*(laneId >= 2);
					value += __shfl_up(value, 4, 32)*(laneId >= 4);
					value += __shfl_up(value, 8, 32)*(laneId >= 8);
					value += __shfl_up(value, 16, 32)*(laneId >= 16);
					prefixSumSample[localId][laneId] = value;
					float tmpSum = prefixSumSample[localId][laneId];
					voteFlag = 0;
					if (transU < tmpSum) voteFlag = 1;
					int offset = __ffs(__ballot(voteFlag)) - 1;
					// int tmpoffset=0;
					if (offset<0) offset = 0;

					// tmpoffset=__ldg(&d_SparseDTCount[docId - 1])-lvl1Idx*32-1;
					newZ = __shfl(tmpNewZ, offset);
					// if ((newZ < 1) || (newZ > K)) {
					// 	printf("wrong Index from sampling Dense:%d,%f,%f,%f,%f\n", newZ, u - S / (S + Q),u,S,Q);
					// 	printf("TmpNewZ and offset: %d,%d\n",tmpNewZ,offset);
					// 	printf("transU and tmpSum and voteFlag: %.10f,%.10f,%d\n",transU,tmpSum,voteFlag);
					// }
					if ((newZ == 0) || (newZ > K)) {
						int tmpoffset = d_SparseDTCount[docId - 1] - lvl1Idx * 32 - 1;
						newZ = __shfl(tmpNewZ, tmpoffset);
						// printf("Dense part:NewZ , tmpNewZ and tmpoffset: %d,%d,%d\n",newZ,tmpNewZ,tmpoffset);
					}

				}

				else //bucket Q
				{

					//float transU = (u - S / (S + Q))*(S + Q);

					float transU = (u - (maxProbability + S) / (maxProbability + S + Q))*(maxProbability + S + Q);
					//level 1: decide position
					float tmpSumHigh, tmpSumLow = 0.0;
					tmpSumHigh = QTree[laneId];
					tmpSumLow = __shfl_up(tmpSumHigh, 1, 32);
					if (laneId == 0)tmpSumLow = 0;
					//voting for lvl1Idx
					int voteFlag = 0;
					if (transU < tmpSumHigh) voteFlag = 1;
					int lvl1Idx = __ffs(__ballot(voteFlag)) - 1;
					if (lvl1Idx < 0) lvl1Idx = 31;
					transU = transU - tmpSumLow;
					transU = __shfl(transU, lvl1Idx);
					prefixSumSample[localId][laneId] = alpha*WTHead[32 * lvl1Idx + laneId];
					//accumulation

					float value = prefixSumSample[localId][laneId];
					value += __shfl_up(value, 1, 32)*(laneId >= 1);
					value += __shfl_up(value, 2, 32)*(laneId >= 2);
					value += __shfl_up(value, 4, 32)*(laneId >= 4);
					value += __shfl_up(value, 8, 32)*(laneId >= 8);
					value += __shfl_up(value, 16, 32)*(laneId >= 16);

					prefixSumSample[localId][laneId] = value;

					voteFlag = 0;
					tmpSumLow = 0;
					tmpSumHigh = prefixSumSample[localId][laneId];
					tmpSumLow = __shfl_up(tmpSumHigh, 1, 32);
					if (laneId == 0)tmpSumLow = 0;

					if (transU < tmpSumHigh)voteFlag = 1;
					int lvl2Idx = __ffs(__ballot(voteFlag)) - 1;
					if (lvl2Idx < 0)lvl2Idx = 31;
					newZ = lvl1Idx * 32 + lvl2Idx + 1;

					if ((newZ < 1) || (newZ > K)) {
						printf("wrong Index from sampling Dense else :%d,%f,%f,%f,%f\n", newZ, u - S / (S + Q), u, S, Q);
					}


				}
			}

		endloop:

			if (laneId == 0) {
				d_TopicIndex[tokenIdx] = newZ;
				//deviceMaxTopic[tokenIdx] = newZ;

				/*sumPerplexity += log((S + maxProbability + Q) / (totalTokenCount + K*alpha));*/

				sumPerplexity += 1.0;
				/*if(oldZ==newZ) sumPerplexity += 1.0;*/
				//d_Perplexity[tokenIdx] = log((S + Q) / (d_TokenCountDT[docId - 1] + K*alpha));

				//d_Perplexity[tokenIdx] = 1.0;
				// printf("Perplexity:%f, %d, %d, %d, %d\n",d_Perplexity[tokenIdx],tokenStart,tokenIdx,newZ,wordId);
				// printf("Perplexity: %d\n",tokenStart);

				tokenIdx = atomicAdd(&WarpCounter[0], 1);

				// sumPerplexity += log((S + Q) / (d_TokenCountDT[docId - 1] + K*alpha));

			}


			tokenIdx = __shfl(tokenIdx, 0);

		}

		if (tid == 0) Counter[0] = atomicAdd(&d_blockCounter[0], 1);

		__syncthreads();

	}

	if (laneId == 0) QTree[localId] = sumPerplexity;

	__syncthreads();

	if (localId == 0) {
		float perplexity = 0.0;
		perplexity = QTree[laneId] * (laneId < BlockDim / 32);
		perplexity += __shfl_down(perplexity, 16);
		perplexity += __shfl_down(perplexity, 8);
		perplexity += __shfl_down(perplexity, 4);
		perplexity += __shfl_down(perplexity, 2);
		perplexity += __shfl_down(perplexity, 1);
		if (laneId == 0) d_Perplexity[blockId] += perplexity;
	}


}

__device__ volatile int sem = 0;
__device__ unsigned int subCount = 0;

__device__ void acquire_semaphore(volatile int *lock) {
	while (atomicCAS((int *)lock, 0, 1) != 0);
}

__device__ void release_semaphore(volatile int *lock) {
	*lock = 0;
	__threadfence();
}

__global__ void LDAKernelTrainD(float alpha, float beta, int* d_Index, unsigned short int* d_TopicIndex, int* d_SparseDTCount, int* d_TokenCountDT, int* d_TokenOffsetDT, int* d_DocListCount, int* d_DocListOffset, int* d_WTDense, int* d_WTDenseCopy, int* d_TokenCount, int* d_TokenOffset, int* d_WordListCount, int* d_WordListOffset, int* d_WTRowSum, unsigned int* d_blockCounter, int*d_DocIndex, int D, int W, float* d_Perplexity, hiprandState *randState, float *WTHeadDense, int numOfWordD, unsigned short int* deviceWordMaxTopic, unsigned short int* deviceWordSecondMaxTopic, float* deviceMaxProb, float* deviceThresProb,float* deviceTimeRecord, int tokenSegment, float* deviceRandomfloat, int* deviceEffectiveTokenIndex, int* deviceNewTokenCount, int* deviceDTIndexValue,long long int* deviceMaxSecTopic)

{
	
	int laneId = threadIdx.x % 32;
	int localId = threadIdx.x / 32;

	volatile __shared__ float WTHead[K];
	volatile __shared__ float QTree[32];



	volatile __shared__ float WTMax[2];

	volatile __shared__ float STree[ShaMemSize / 32][K / 32];
	volatile __shared__ float prefixSumSample[ShaMemSize / 32][32];
	volatile __shared__ unsigned int Counter[1];
	__shared__ unsigned int tokenRegionStart[1];
	volatile __shared__ unsigned int tokenEndFlag[1];
	__shared__ unsigned int WarpCounter[1];

	/*
	clock_t start0, finish0, finish1, finish2, finish3, finish4;
	double costtime0 = 0.0, costtime1 = 0.0, costtime2 = 0.0, costtime3 = 0.0, costtime4 = 0.0, total=0.0;*/
	
	clock_t start0, finish0, finish1;
	double costtime0 = 0.0, costtime1 = 0.0;

	

//	if (threadIdx.x == 0)
//	{
//		acquire_semaphore(&sem);
//		tokenEndFlag[0] = 0;
//		Counter[0] = d_blockCounter[0];
//		unsigned int numRegions = (deviceNewTokenCount[Counter[0]] == 0) ? 0 : ((deviceNewTokenCount[Counter[0]] - 1) / tokenSegment);
//		tokenRegionStart[0] = atomicInc(&subCount, numRegions);
//		if (subCount == 0) {
//			d_blockCounter[0] = d_blockCounter[0] + 1;
//			tokenEndFlag[0] = 1;
//		}
//		release_semaphore(&sem);
//	}
//	__syncthreads();

	/*if(threadIdx.x==0){
		Counter[0]=atomicAdd(&d_blockCounter[0],1);
	}
	__syncthreads();*/

	float sumPerplexity = 0.0;

	//start0 = clock64();

	for(int wordId = blockIdx.x; wordId<numOfWordD; wordId+=gridDim.x)
	//while (Counter[0]<numOfWordD)
	{
		/*start0 = clock64();*/

//		int wordId = Counter[0] ;
		if (localId == 0) {
			QTree[laneId] = 0;
		}

		prefixSumSample[localId][laneId] = 0.0;
		int tokenStart = d_TokenOffset[wordId];
		int tokenEnd = d_TokenOffset[wordId] + deviceNewTokenCount[wordId];

		int WTStart = d_WordListOffset[wordId];
		unsigned short int maxK = deviceWordMaxTopic[wordId];
		unsigned short int secondMaxK = deviceWordSecondMaxTopic[wordId];

		// Reconstruct dense WT vector from sparse WT matrix
		for (int i = threadIdx.x; i < K; i += blockDim.x)
		{
			WTHead[i] = (d_WTDense[WTStart + i] + beta) / (d_WTRowSum[i] + W*beta);
			//__syncthreads();
		}
		__syncthreads();

		if (threadIdx.x == 0) {
			WTMax[0] = WTHead[maxK - 1];
			WTMax[1] = WTHead[secondMaxK - 1];
			WTHead[maxK - 1] = 0.0;
		}
		__syncthreads();


		for (int i = localId; i < K / 32; i += blockDim.x / 32) {
			short int   tmpK = i * 32 + laneId;
			float tmpVal = 0.0;
			tmpVal = alpha*WTHead[tmpK];
			tmpVal += __shfl_down(tmpVal, 16);
			tmpVal += __shfl_down(tmpVal, 8);
			tmpVal += __shfl_down(tmpVal, 4);
			tmpVal += __shfl_down(tmpVal, 2);
			tmpVal += __shfl_down(tmpVal, 1);
			tmpVal = __shfl(tmpVal, 0);
			QTree[i] = tmpVal;

		}
		__syncthreads();

		if (localId == 0) {

			float value = QTree[laneId];
			value += __shfl_up(value, 1, 32)*(laneId >= 1);
			value += __shfl_up(value, 2, 32)*(laneId >= 2);
			value += __shfl_up(value, 4, 32)*(laneId >= 4);
			value += __shfl_up(value, 8, 32)*(laneId >= 8);
			value += __shfl_up(value, 16, 32)*(laneId >= 16);

			QTree[laneId] = value;

		}

		if (threadIdx.x == 0) WarpCounter[0] = tokenStart;
		__syncthreads();

		//float WTMax = MaxWT[0];
		//float WTMax2 = MaxWT[1];
		//unsigned short int WTMaxK = MaxK[0];

		float Q = QTree[31];
		int tokenIdx;

		if (laneId == 0)
		{
			tokenIdx = atomicAdd(&WarpCounter[0], 1);

		}
		tokenIdx = __shfl(tokenIdx, 0);

		
		// float WTMaxProb = WTMax[0];
		// float WTSecondMaxProb = WTMax[1];

		// for (int tokenIdx = tokenStart + localId; tokenIdx < tokenEnd; tokenIdx += blockDim.x / 32) //iterate over tokens
		// {

		/*finish0 = clock64();
		costtime0 += (double)(finish0 - start0);*/

		while (tokenIdx<tokenEnd)
		{
			////int oldZ = d_TopicIndex[tokenIdx];
		 //   
			////unsigned short int sFlag = 1;
			//start0 = clock64();
			//
			//if (deviceMflag[tokenIdx]) {
			//	if (laneId==0) tokenIdx = atomicAdd(&WarpCounter[0], 1);
			//	tokenIdx = __shfl(tokenIdx, 0);
			//	finish0 = clock64();
			//	costtime0 += (double)(finish0 - start0);
			//	continue;
			//	//newZ = maxK;
			//	//sFlag = 0;
			//	//goto endloop;

			//}


			int tokenNewIdx = deviceEffectiveTokenIndex[tokenIdx];

			unsigned short int newZ = 1;
			//int docId = d_DocIndex[tokenNewIdx]-1;
			int docId = ((int)(deviceMaxSecTopic[tokenNewIdx]>>32)-1);
			float maxProbability = deviceMaxProb[tokenNewIdx];
			float thresProb = deviceThresProb[tokenNewIdx];
			float u = deviceRandomfloat[tokenNewIdx];

		/*	if (laneId == 0)u = hiprand_uniform(&(randState[threadIdx.x + blockDim.x*blockIdx.x])) / 1.00001;
			u = __shfl(u, 0);*/

			//computing S.
			float S = 0;

			int DTStart = d_DocListOffset[docId];
			int DTEnd = d_DocListOffset[docId] + ((d_SparseDTCount[docId] - 1) / 32 + 1) * 32;

			STree[localId][laneId] = 0;
			short int SIdx = 0;
			float  tmpP1k = 0.0;
			short int  colVal;
			short int  colK;
			/*finish1 = clock64();
			costtime1 += (double)(finish1 - finish0);*/
			//maxStruct maxValue;
			for (int tmpIdx = DTStart + laneId; tmpIdx < DTEnd - 32; tmpIdx += 32) {
				int DTIndexValue = deviceDTIndexValue[tmpIdx];
				colVal = DTIndexValue & 0xffff;
				colK = (DTIndexValue >> 16) & 0xffff;

				/*colVal = d_SparseDTValue[tmpIdx];
				colK = d_SparseDTIndex[tmpIdx];*/
				tmpP1k = colVal*WTHead[colK - 1];
				//tmpP1k = tmpP1k*(colK != maxTopic);
				//if (colK == maxTopic) tmpP1k = 0.0;
				tmpP1k += __shfl_down(tmpP1k, 16);
				tmpP1k += __shfl_down(tmpP1k, 8);
				tmpP1k += __shfl_down(tmpP1k, 4);
				tmpP1k += __shfl_down(tmpP1k, 2);
				tmpP1k += __shfl_down(tmpP1k, 1);
				tmpP1k = __shfl(tmpP1k, 0);
				S += tmpP1k;
				STree[localId][SIdx] = S;
				SIdx++;
			}

			tmpP1k = 0.0;
			int DTIndexValue = deviceDTIndexValue[DTEnd - 32 + laneId];
			colVal = DTIndexValue & 0xffff;
			colK = (DTIndexValue >> 16) & 0xffff;
			/*colVal = d_SparseDTValue[DTEnd - 32 + laneId];
			colK = d_SparseDTIndex[DTEnd - 32 + laneId];*/
			if (colK != 0) tmpP1k = colVal*WTHead[colK - 1];
			//if (colK == maxTopic) tmpP1k = 0.0;
			//tmpP1k = tmpP1k*(colK != maxTopic);

			tmpP1k += __shfl_down(tmpP1k, 16);
			tmpP1k += __shfl_down(tmpP1k, 8);
			tmpP1k += __shfl_down(tmpP1k, 4);
			tmpP1k += __shfl_down(tmpP1k, 2);
			tmpP1k += __shfl_down(tmpP1k, 1);
			tmpP1k = __shfl(tmpP1k, 0);
			S += tmpP1k;
			STree[localId][SIdx] = S;


			//__syncthreads();
			/*STmp = S;

			S = __shfl(STmp, 0);*/
			S = __shfl(S, 0);
			//__syncthreads();
			//randomly generate u.

			float totalProb = maxProbability+S+Q;

			//if (maxProbability / totalProb <thresProb) printf("What!!!!%f,%f,%d,%d\n", maxProbability / totalProb, thresProb, maxK - 1,  wordId);

			/*finish2 = clock64();
			costtime2 += (double)(finish2 - finish1);*/

			/*if ((wordId == 40) && (laneId == 0) && (tokenIdx - tokenStart<50)) printf("thresProb: %f,%f,%d,%d,%f,%f\n", thresProb, (maxProbability + alpha*(WTHead[maxTopic - 1])) / (maxProbability + S + Q), maxTokenCount, totalTokenCount, WTHead[maxTopic - 1] / Q, WTMax / Q);*/

			if (maxProbability / (maxProbability + S + Q) < thresProb) printf("Wrong Prob!!!!%f,%f\n", maxProbability / (maxProbability + S + Q), thresProb);

				

			//if (maxProbability / (maxProbability + S + Q) < thresProb) printf("Wrong Prob!!!!");

			if (u < maxProbability / totalProb) {

				newZ = maxK;

			}
				

			else if ((u>= maxProbability / totalProb) && (u< (maxProbability + S) / totalProb))
			{
				//float transU = u*(S + Q);
				float transU = u*totalProb- maxProbability;
				float tmpSumHigh, tmpSumLow = 0.0;
				tmpSumHigh = STree[localId][laneId];
				tmpSumLow = __shfl_up(tmpSumHigh, 1, 32);
				if (laneId == 0) tmpSumLow = 0;
				int voteFlag = 0;
				if ((transU < tmpSumHigh)) voteFlag = 1;
				int lvl1Idx = __ffs(__ballot(voteFlag)) - 1;

				if (lvl1Idx < 0) lvl1Idx = (DTEnd - DTStart) / 32 - 1;
				//tmpU1 = transU;
				transU = transU - tmpSumLow;
				/*tmpU = transU;*/
				transU = __shfl(transU, lvl1Idx);
				int tmpIdx = DTStart + lvl1Idx * 32 + laneId;
				int  DTIndexValue = deviceDTIndexValue[tmpIdx];
				short int colVal = DTIndexValue & 0xffff;
				short int tmpNewZ = (DTIndexValue >>16) & 0xffff;
				
				float p1k = 0.0;
				if (tmpNewZ != 0)
				{
					p1k = colVal*WTHead[tmpNewZ - 1];
				}

				//p1k = p1k*(colK != maxTopic);

				prefixSumSample[localId][laneId] = p1k;
				float value = prefixSumSample[localId][laneId];
				value += __shfl_up(value, 1, 32)*(laneId >= 1);
				value += __shfl_up(value, 2, 32)*(laneId >= 2);
				value += __shfl_up(value, 4, 32)*(laneId >= 4);
				value += __shfl_up(value, 8, 32)*(laneId >= 8);
				value += __shfl_up(value, 16, 32)*(laneId >= 16);
				prefixSumSample[localId][laneId] = value;
				float tmpSum = prefixSumSample[localId][laneId];
				voteFlag = 0;
				if (transU < tmpSum) voteFlag = 1;
				int offset = __ffs(__ballot(voteFlag)) - 1;
				// int tmpoffset=0;
				if(offset<0) offset=0;

				// tmpoffset=__ldg(&d_SparseDTCount[docId - 1])-lvl1Idx*32-1;
				newZ = __shfl(tmpNewZ, offset);
				// if ((newZ < 1) || (newZ > K)) {
				// 	printf("wrong Index from sampling Dense:%d,%f,%f,%f,%f\n", newZ, u - S / (S + Q),u,S,Q);
				// 	printf("TmpNewZ and offset: %d,%d\n",tmpNewZ,offset);
				// 	printf("transU and tmpSum and voteFlag: %.10f,%.10f,%d\n",transU,tmpSum,voteFlag);
				// }
				if ((newZ == 0) || (newZ > K)){
					int tmpoffset=d_SparseDTCount[docId]-lvl1Idx*32-1;
					newZ=__shfl(tmpNewZ, tmpoffset);
					// printf("Dense part:NewZ , tmpNewZ and tmpoffset: %d,%d,%d\n",newZ,tmpNewZ,tmpoffset);
				}

			}

			else //bucket Q
			{

				//float transU = (u - S / (S + Q))*(S + Q);

				float transU = (u - (maxProbability + S) / totalProb)*totalProb;
				//level 1: decide position
				float tmpSumHigh, tmpSumLow = 0.0;
				tmpSumHigh = QTree[laneId];
				tmpSumLow = __shfl_up(tmpSumHigh, 1, 32);
				if (laneId == 0)tmpSumLow = 0;
				//voting for lvl1Idx
				int voteFlag = 0;
				if (transU < tmpSumHigh) voteFlag = 1;
				int lvl1Idx = __ffs(__ballot(voteFlag)) - 1;
				if (lvl1Idx < 0) lvl1Idx = 31;
				/*tmpU1 = transU;*/
				transU = transU - tmpSumLow;
				/*tmpU = transU;*/
				transU = __shfl(transU, lvl1Idx);
				prefixSumSample[localId][laneId] = alpha*WTHead[32 * lvl1Idx + laneId];
				//accumulation

				float value = prefixSumSample[localId][laneId];
				value += __shfl_up(value, 1, 32)*(laneId >= 1);
				value += __shfl_up(value, 2, 32)*(laneId >= 2);
				value += __shfl_up(value, 4, 32)*(laneId >= 4);
				value += __shfl_up(value, 8, 32)*(laneId >= 8);
				value += __shfl_up(value, 16, 32)*(laneId >= 16);

				prefixSumSample[localId][laneId] = value;

				voteFlag = 0;
				tmpSumLow = 0;
				tmpSumHigh = prefixSumSample[localId][laneId];
				tmpSumLow = __shfl_up(tmpSumHigh, 1, 32);
				if (laneId == 0)tmpSumLow = 0;

				if (transU < tmpSumHigh)voteFlag = 1;
				int lvl2Idx = __ffs(__ballot(voteFlag)) - 1;
				if (lvl2Idx < 0)lvl2Idx = 31;
				newZ = lvl1Idx * 32 + lvl2Idx + 1;

				if ((newZ < 1) || (newZ > K)) {
					printf("wrong Index from sampling Dense else :%d,%f,%f,%f,%f\n", newZ, u - S / (S + Q),u,S,Q);
				}

			}

			if (laneId == 0) {
				d_TopicIndex[tokenNewIdx] = newZ;
				//deviceMaxTopic[tokenIdx] = newZ;

				atomicAdd(&d_WTDenseCopy[WTStart + newZ - 1], 1);

				//p_temp = S + Q;
				// d_S[tokenIdx] = Q;

				/*sumPerplexity+= log(totalProb / (d_TokenCountDT[docId] + K*alpha));*/
				sumPerplexity += 1.0;

				//if(oldZ==newZ) sumPerplexity += 1.0;

				//d_Perplexity[tokenIdx] = log((S + Q) / (d_TokenCountDT[docId - 1] + K*alpha));

				//d_Perplexity[tokenIdx] = 1.0;
				// printf("Perplexity:%f, %d, %d, %d, %d\n",d_Perplexity[tokenIdx],tokenStart,tokenIdx,newZ,wordId);
				// printf("Perplexity: %d\n",tokenStart);

				tokenIdx = atomicAdd(&WarpCounter[0], 1);

				// sumPerplexity += log((S + Q) / (d_TokenCountDT[docId - 1] + K*alpha));

			}

		//	// if(laneId==0) 
		//	// {

		//	// 	__threadfence_block();
		//	// }
		tokenIdx = __shfl(tokenIdx, 0);
		//	finish1 = clock64();
		//	costtime1 += (double)(finish1 - start0);


		//	/*finish4 = clock64();
		//	costtime4 += (double)(finish4 - finish3);*/
	
		//		/*finish3 = clock64();
		//		costtime3 += (double)(finish3 - finish2);*/

		///*endloop:*/
		//	/*finish3 = clock64();*/
		//	


	        
		}

		/*if(threadIdx.x==0) Counter[0]=atomicAdd(&d_blockCounter[0],1);

		__syncthreads();*/
		//__syncthreads();

		__syncthreads();
		//if (localId == 0) {
		//	unsigned short int topic = 0;
		//	topic = maxTopicVec[laneId] * (laneId < BlockDim / 32);
		//	topic += __shfl_down(topic, 16);
		//	topic += __shfl_down(topic, 8);
		//	topic += __shfl_down(topic, 4);
		//	topic += __shfl_down(topic, 2);
		//	topic += __shfl_down(topic, 1);
		//	if (laneId == 0) atomicAdd(&d_WTDenseCopy[WTStart + MaxK[0]], topic);
		//}
		//__syncthreads();

	}

	if (laneId == 0) QTree[localId] = sumPerplexity;
	__syncthreads();
	
	if (localId == 0) {
		float perplexity = 0.0;
		perplexity = QTree[laneId] * (laneId < BlockDim / 32);
		perplexity += __shfl_down(perplexity, 16);
		perplexity += __shfl_down(perplexity, 8);
		perplexity += __shfl_down(perplexity, 4);
		perplexity += __shfl_down(perplexity, 2);
		perplexity += __shfl_down(perplexity, 1);
		if (laneId == 0) d_Perplexity[blockIdx.x] += perplexity;
	}

	/*finish0 = clock64();
	costtime0 = (double)(finish0 - start0);*/

	//if (threadIdx.x + blockDim.x*blockIdx.x == 0) printf("costtime0,costtime1:%f,%f", costtime0 / (158200000 * 1.0), costtime1 / (158200000 * 1.0)); 

	/*deviceTimeRecord[(threadIdx.x + blockDim.x*blockIdx.x)/32] = costtime0 / (158200000 * 1.0);

	if (threadIdx.x+blockDim.x*blockIdx.x == 0) printf("costtime0,costtime1,costtime2,costtime3,costtime4,total:%f,%f,%f,%f,%f,%f", costtime0/(158200000*1.0), costtime1 / (158200000 * 1.0), costtime2 / (158200000 * 1.0), costtime3 / (158200000 * 1.0), costtime4 / (158200000 * 1.0), (costtime0 + costtime1+ costtime2+ costtime3+ costtime4)/ (158200000 * 1.0));*/
	//if (threadIdx.x % 32 == 0)
	//	d_Perplexity[(threadIdx.x + blockDim.x*blockIdx.x) / 32] = sumPerplexity;
	////wordPerplexity[(threadIdx.x + blockDim.x*blockIdx.x) / 32] = sumPerplexity;
	//__syncthreads();

}


__global__ void LDATrainPerplexityReduce(float *perplexity,float numOfTokens,float* devicePerplexityAve) {

	int tid = threadIdx.x;
	int laneId = threadIdx.x % 32;
	int localId = threadIdx.x / 32;
	
	float S = 0.0;
	volatile __shared__ float perplexityMid[32];
	for (int i = tid; i < GridDim; i += BlockDim) {

		float tmpPerplexity = 0.0;
		tmpPerplexity = perplexity[i];
		tmpPerplexity += __shfl_down(tmpPerplexity, 16);
		tmpPerplexity += __shfl_down(tmpPerplexity, 8);
		tmpPerplexity += __shfl_down(tmpPerplexity, 4);
		tmpPerplexity += __shfl_down(tmpPerplexity, 2);
		tmpPerplexity += __shfl_down(tmpPerplexity, 1);
		S += tmpPerplexity;
	}
	if (laneId == 0) perplexityMid[localId] = S;
	__syncthreads();
	if (localId == 0) {
		float AveragePerplexity = 0.0;
		S = 0.0;
		S = perplexityMid[laneId] * (laneId < BlockDim / 32);
		//printf("\nS=:%f\n", S);
		S += __shfl_down(S, 16);
		S += __shfl_down(S, 8);
		S += __shfl_down(S, 4);
		S += __shfl_down(S, 2);
		S += __shfl_down(S, 1);

		
		if (laneId == 0)
		{
			AveragePerplexity = S / numOfTokens; 
			devicePerplexityAve[0]=AveragePerplexity;
			printf("\nAverage Perplexity:%f\n", AveragePerplexity);
		}
		
		
	}



}





__global__ void LDATrainPerplexityReduce1(float *perplexity, float *perplexityMid, int numVals) {


	int numWarps = gridDim.x*blockDim.x / 32;
	int tid = threadIdx.x + blockIdx.x*blockDim.x;
	int warpId = tid / 32;
	int laneId = tid % 32;

	int perWarpSize = ((numVals + numWarps - 1) / numWarps + 31) / 32 * 32;
	int perWarpSizeMax = (numVals + numWarps - 1) / numWarps;
	int startIdx = perWarpSizeMax*warpId;
	int endIdx = perWarpSizeMax*warpId + perWarpSize;
	int endMax = perWarpSizeMax*warpId + perWarpSizeMax;
	
	float totalProd = 0.0;
	for (long long i = startIdx + laneId; i < endIdx; i += 32) {

		float tmpProd = 0.0;
		if ((i < numVals) && (i < endMax))tmpProd = perplexity[i];

		tmpProd += __shfl_down(tmpProd, 16);
		tmpProd += __shfl_down(tmpProd, 8);
		tmpProd += __shfl_down(tmpProd, 4);
		tmpProd += __shfl_down(tmpProd, 2);
		tmpProd += __shfl_down(tmpProd, 1);
		tmpProd = __shfl(tmpProd, 0);
		totalProd += tmpProd;
		//__syncthreads();
	}
	__syncthreads();
	if (laneId == 0) perplexityMid[warpId] += totalProd;

}

//__device__ volatile int sem1 = 0;
//__device__ unsigned int subCount1 = 0;



__global__ void UpdateProbKernelTrainD(float alpha, float beta, int* d_Index, unsigned short int* d_TopicIndex, int* d_SparseDTCount, int* d_TokenCountDT, int* d_TokenOffsetDT, int* d_DocListCount, int* d_DocListOffset, int* d_WTDense, int* d_WTDenseCopy, int* d_TokenCount, int* d_TokenOffset, int* d_WordListCount, int* d_WordListOffset, int* d_WTRowSum, unsigned int* d_blockCounter, int*d_DocIndex, int D, int W, float* d_Perplexity, hiprandState *randState, float *WTHeadDense, int numOfWordD, unsigned short int* deviceWordMaxTopic, unsigned short int* deviceWordSecondMaxTopic,float* deviceMaxProb, float* deviceThresProb, unsigned short int* deviceWordThirdMaxTopic, float* deviceRandomfloat,  int* deviceEffectiveTokenIndex, int* deviceNewTokenCount, int* deviceMaxSecTopic, float* deviceQArray, float* deviceWordMaxProb, float* deviceWordSecondMaxProb, float* deviceWordThirdMaxProb, int tokenSegment)

{


	/*volatile __shared__ float WTHead[K];*/
	volatile __shared__ float QTree[32];
	volatile __shared__ float WTMax[3];
	volatile __shared__ unsigned int Counter[1];
	//__shared__ unsigned int WarpCounter[1];
//	volatile unsigned int tokenRegionStart;
//	volatile unsigned int tokenEndFlag;
//	__shared__ unsigned int tokenRegionStart[1];
//	volatile __shared__ unsigned int tokenEndFlag[1];
	__shared__ int newTokenCount[1];

//	clock_t start0, finish0, finish1, finish2, finish3;
//	double costtime0 = 0.0, costtime1 = 0.0, costtime2 = 0.0, costtime3 = 0.0;

//	volatile unsigned int counter = 0;
//	if (threadIdx.x== 0)
//	{
//		acquire_semaphore(&sem1);
//		tokenEndFlag[0] = 0;
//		Counter[0] = d_blockCounter[0];
//		unsigned int numRegions = (d_TokenCount[Counter[0]] == 0) ? 0 : ((d_TokenCount[Counter[0]] - 1) / tokenSegment);
//		tokenRegionStart[0] = atomicInc(&subCount1, numRegions);
//		if (subCount1 == 0) {
//			d_blockCounter[0] = d_blockCounter[0] + 1;
//			tokenEndFlag[0] = 1;
//		}
//		release_semaphore(&sem1);
//	}
//
//	tokenRegionStart = __shfl(tokenRegionStart, 0);
//	tokenEndFlag = __shfl(tokenEndFlag, 0);
//
//	counter = __shfl(counter, 0);




//	if (threadIdx.x == 0) Counter[0] = atomicAdd(&d_blockCounter[0], 1);
//	__syncthreads();

	float sumPerplexity = 0.0;

	//while (Counter[0]<numOfWordD)
//	while (Counter[0]<numOfWordD)
	for(int wordId = blockIdx.x; wordId<numOfWordD; wordId+=gridDim.x)
	{
//		start0 = clock64();

//		int wordId =Counter[0];

		int tokenStart = d_TokenOffset[wordId];
		int tokenEnd = d_TokenOffset[wordId] + d_TokenCount[wordId];
//		int tokenStart = d_TokenOffset[wordId] + tokenRegionStart[0] * tokenSegment;
//		int tokenStartNew = d_TokenOffset[wordId];
//		int tokenEnd = d_TokenOffset[wordId] + (tokenRegionStart[0] + 1) * tokenSegment;
//		if (tokenEndFlag[0]) tokenEnd = d_TokenOffset[wordId] + d_TokenCount[wordId];

		int WTStart = d_WordListOffset[wordId];
		unsigned short int maxK = deviceWordMaxTopic[wordId];
		/*unsigned short int secondMaxK = deviceWordSecondMaxTopic[wordId];
		unsigned short int thirdMaxK = deviceWordThirdMaxTopic[wordId];*/
		// Reconstruct dense WT vector from sparse WT matrix
		//for (int i = threadIdx.x; i < K; i += blockDim.x)
		//{
		//	WTHead[i] = (d_WTDense[WTStart + i] + beta) / (d_WTRowSum[i] + W*beta);
		//	//__syncthreads();
		//}
		//__syncthreads();

		//if (threadIdx.x == 0) {
		//	WTMax[0] = WTHead[maxK - 1];
		//	WTMax[1] = WTHead[secondMaxK - 1];
		//	WTMax[2] = WTHead[thirdMaxK - 1];
		//	WTHead[maxK - 1] = 0.0;
		//	//WTHead[secondMaxK - 1] = 0.0;
		//}
		//__syncthreads();


		//for (int i = localId; i < K / 32; i += blockDim.x / 32) {
		//	unsigned short int   tmpK = i * 32 + laneId;
		//	float tmpVal = 0.0;
		//	tmpVal = alpha*WTHead[tmpK];
		//	tmpVal += __shfl_down(tmpVal, 16);
		//	tmpVal += __shfl_down(tmpVal, 8);
		//	tmpVal += __shfl_down(tmpVal, 4);
		//	tmpVal += __shfl_down(tmpVal, 2);
		//	tmpVal += __shfl_down(tmpVal, 1);
		//	tmpVal = __shfl(tmpVal, 0);
		//	QTree[i] = tmpVal;

		//}
		//__syncthreads();

		//if (localId == 0) {

		//	float value = QTree[laneId];
		//	value += __shfl_up(value, 1, 32)*(laneId >= 1);
		//	value += __shfl_up(value, 2, 32)*(laneId >= 2);
		//	value += __shfl_up(value, 4, 32)*(laneId >= 4);
		//	value += __shfl_up(value, 8, 32)*(laneId >= 8);
		//	value += __shfl_up(value, 16, 32)*(laneId >= 16);

		//	QTree[laneId] = value;

		//}
		//if (threadIdx.x == 0) WarpCounter[0] = 0;
		//__syncthreads();
		//float Q = QTree[31];
		//int tokenIdx;
		/*float WTMaxProb = WTMax[0];
		float WTSecondMaxProb = WTMax[1];
		float WTThirdMaxProb = WTMax[2];*/
		float WTMaxProb = deviceWordMaxProb[wordId];
		float WTSecondMaxProb = deviceWordSecondMaxProb[wordId];
		float WTThirdMaxProb = deviceWordThirdMaxProb[wordId];
		float Q = alpha* deviceQArray[wordId];

//		finish0 = clock64();
//		costtime0 += (double)(finish0 - start0);

		for (int tokenIdx = tokenStart + threadIdx.x; tokenIdx < tokenEnd; tokenIdx += blockDim.x)
		{

			int docId = d_DocIndex[tokenIdx]-1;

			int totalTokenCount = d_TokenCountDT[docId];


			int nonSkipTokenIdx;

			float u = hiprand_uniform(&(randState[threadIdx.x + blockDim.x*blockIdx.x])) / 1.00001;
			deviceRandomfloat[tokenIdx] = u;
			/*unsigned short int maxTokenCount = deviceMaxTokenCount[tokenIdx];
			unsigned short int maxSecondTokenCount = deviceSecondMaxTokenCount[tokenIdx];*/

			unsigned short int maxTokenCount = deviceMaxSecTopic[tokenIdx]&(0x0000ffff);
			unsigned short int maxSecondTokenCount = (deviceMaxSecTopic[tokenIdx]&(0xffff0000))>>16;

			


			//float maxS = (totalTokenCount - maxTokenCount)*WTSecondMaxProb;

			float maxS = (totalTokenCount - maxTokenCount - maxSecondTokenCount)*WTThirdMaxProb + maxSecondTokenCount*WTSecondMaxProb;
			float maxProb = (maxTokenCount + alpha)*WTMaxProb;
			float thresProb= maxProb/(maxProb+maxS+Q);
			deviceMaxProb[tokenIdx] = maxProb;

			deviceThresProb[tokenIdx] = thresProb;
			if(u > thresProb) {
//				finish0 = clock64();
				//nonSkipTokenIdx = atomicAdd(&deviceNewTokenCount[wordId], 1);
				nonSkipTokenIdx = atomicAdd(&newTokenCount[0], 1);
//				finish1 = clock64();
//				costtime1 += (double)(finish1 - finish0);
//				finish1 = clock64();
				deviceEffectiveTokenIndex[nonSkipTokenIdx + tokenStart] = tokenIdx;
//				finish2 = clock64();
//				costtime2 += (double)(finish2 - finish1);
			}
			else {


				d_TopicIndex[tokenIdx] = maxK;
				atomicAdd(&d_WTDenseCopy[WTStart + maxK-1], 1);
				sumPerplexity += 1.0;

			}



//			short int flag = (u > deviceThresProb[tokenIdx]);
//			short int warpNonZeroCount=__popc(__ballot(flag));
//			if (laneId==0) nonSkipTokenIdx = atomicAdd(&deviceNewTokenCount[wordId], warpNonZeroCount);
//			nonSkipTokenIdx=__shfl(nonSkipTokenIdx,0);
//			flag += __shfl_up(flag, 1, 32)*(laneId >= 1);
//			flag += __shfl_up(flag, 2, 32)*(laneId >= 2);
//			flag += __shfl_up(flag, 4, 32)*(laneId >= 4);
//			flag += __shfl_up(flag, 8, 32)*(laneId >= 8);
//			flag += __shfl_up(flag, 16, 32)*(laneId >= 16);
//			if (u > deviceThresProb[tokenIdx]) deviceEffectiveTokenIndex[nonSkipTokenIdx + tokenStart+flag-1] = tokenIdx;


			//__syncthreads();
		}
		__syncthreads();

//		if (laneId == 0)
//		{
//			acquire_semaphore(&sem1);
//			tokenEndFlag = 0;
//			counter = d_blockCounter[0];
//			unsigned int numRegions = (d_TokenCount[counter] == 0) ? 0 : ((d_TokenCount[counter] - 1) / tokenSegment);
//			tokenRegionStart = atomicInc(&subCount1, numRegions);
//			if (subCount1 == 0) {
//				d_blockCounter[0] = d_blockCounter[0] + 1;
//				tokenEndFlag = 1;
//			}
//			release_semaphore(&sem1);
//		}
//
//		tokenRegionStart = __shfl(tokenRegionStart, 0);
//		tokenEndFlag = __shfl(tokenEndFlag, 0);
//		counter = __shfl(counter, 0);
//		finish2 = clock64();
//		if (threadIdx.x== 0)
//		{
//			acquire_semaphore(&sem1);
//			tokenEndFlag[0] = 0;
//			Counter[0] = d_blockCounter[0];
//			unsigned int numRegions = (d_TokenCount[Counter[0]] == 0) ? 0 : ((d_TokenCount[Counter[0]] - 1) / tokenSegment);
//			tokenRegionStart[0] = atomicInc(&subCount1, numRegions);
//			if (subCount1 == 0) {
//				d_blockCounter[0] = d_blockCounter[0] + 1;
//				tokenEndFlag[0] = 1;
//			}
//			release_semaphore(&sem1);
//		}
//		finish3 = clock64();
//		costtime3 += (double)(finish3 - finish2);
//	//
//	//	tokenRegionStart = __shfl(tokenRegionStart, 0);
//	//	tokenEndFlag = __shfl(tokenEndFlag, 0);
//	//
//	//	counter = __shfl(counter, 0);
//		__syncthreads();


//		finish2 = clock64();
//		if (threadIdx.x == 0) Counter[0] = atomicAdd(&d_blockCounter[0], 1);
		if (threadIdx.x == 0)
		{
			deviceNewTokenCount[wordId]=newTokenCount[0];
			newTokenCount[0]=0;
			//Counter[0] = atomicAdd(&d_blockCounter[0], 1);
		}



		__syncthreads();
//		finish3 = clock64();
//		costtime3 += (double)(finish3 - finish2);

		/*if (threadIdx.x == 0) deviceNewTokenCount[wordId] = WarpCounter[0];
		__syncthreads();
*/
	}

//	if (threadIdx.x + blockDim.x*blockIdx.x == 0) printf("costtime0,costtime1,costtime2,costtime3:%f,%f,%f,%f\n", costtime0 / (158200000 * 1.0), costtime1 / (158200000 * 1.0),costtime2 / (158200000 * 1.0), costtime3 / (158200000 * 1.0));
	sumPerplexity += __shfl_down(sumPerplexity, 16);
	sumPerplexity += __shfl_down(sumPerplexity, 8);
	sumPerplexity += __shfl_down(sumPerplexity, 4);
	sumPerplexity += __shfl_down(sumPerplexity, 2);
	sumPerplexity += __shfl_down(sumPerplexity, 1);
	sumPerplexity = __shfl(sumPerplexity, 0);
	int laneId = threadIdx.x % 32;
	int localId = threadIdx.x / 32;
	if (localId == 0) {
		QTree[laneId] = 0;
	}
	__syncthreads();
	if (laneId == 0) QTree[localId] = sumPerplexity;
	__syncthreads();

	if (localId == 0) {
		float perplexity = 0.0;
		perplexity = QTree[laneId] * (laneId < blockDim.x / 32);
		perplexity += __shfl_down(perplexity, 16);
		perplexity += __shfl_down(perplexity, 8);
		perplexity += __shfl_down(perplexity, 4);
		perplexity += __shfl_down(perplexity, 2);
		perplexity += __shfl_down(perplexity, 1);
		if (laneId == 0) d_Perplexity[blockIdx.x] += perplexity;
	}

}
__global__ void UpdateProbKernelTrainD0(float alpha, float beta, int* d_Index, unsigned short int* d_TopicIndex, int* d_SparseDTCount, int* d_TokenCountDT, int* d_TokenOffsetDT, int* d_DocListCount, int* d_DocListOffset, int* d_WTDense, int* d_WTDenseCopy, int* d_TokenCount, int* d_TokenOffset, int* d_WordListCount, int* d_WordListOffset, int* d_WTRowSum, unsigned int* d_blockCounter, int*d_DocIndex, int D, int W, float* d_Perplexity, hiprandState *randState, float *WTHeadDense, int numOfWordD, unsigned short int* deviceWordMaxTopic, unsigned short int* deviceWordSecondMaxTopic,float* deviceMaxProb, float* deviceThresProb, unsigned short int* deviceWordThirdMaxTopic, float* deviceRandomfloat,  int* deviceEffectiveTokenIndex, int* deviceNewTokenCount, int* deviceMaxSecTopic, float* deviceQArray, float* deviceWordMaxProb, float* deviceWordSecondMaxProb, float* deviceWordThirdMaxProb, int tokenSegment,unsigned short int* deviceTotalTokenCount)

{


	/*volatile __shared__ float WTHead[K];*/
	volatile __shared__ float QTree[32];
	volatile __shared__ float WTMax[3];
	volatile __shared__ unsigned int Counter[1];
	//__shared__ unsigned int WarpCounter[1];
//	volatile unsigned int tokenRegionStart;
//	volatile unsigned int tokenEndFlag;
//	__shared__ unsigned int tokenRegionStart[1];
//	volatile __shared__ unsigned int tokenEndFlag[1];


	clock_t start0, finish0, finish1, finish2, finish3;
	double costtime0 = 0.0, costtime1 = 0.0, costtime2 = 0.0, costtime3 = 0.0;

//	volatile unsigned int counter = 0;
//	if (threadIdx.x== 0)
//	{
//		acquire_semaphore(&sem1);
//		tokenEndFlag[0] = 0;
//		Counter[0] = d_blockCounter[0];
//		unsigned int numRegions = (d_TokenCount[Counter[0]] == 0) ? 0 : ((d_TokenCount[Counter[0]] - 1) / tokenSegment);
//		tokenRegionStart[0] = atomicInc(&subCount1, numRegions);
//		if (subCount1 == 0) {
//			d_blockCounter[0] = d_blockCounter[0] + 1;
//			tokenEndFlag[0] = 1;
//		}
//		release_semaphore(&sem1);
//	}
//
//	tokenRegionStart = __shfl(tokenRegionStart, 0);
//	tokenEndFlag = __shfl(tokenEndFlag, 0);
//
//	counter = __shfl(counter, 0);




	if (threadIdx.x == 0) Counter[0] = atomicAdd(&d_blockCounter[0], 1);
	__syncthreads();

	float sumPerplexity = 0.0;

	//while (Counter[0]<numOfWordD)
	while (Counter[0]<numOfWordD)
	{
//		start0 = clock64();

		int wordId =Counter[0];

		int tokenStart = d_TokenOffset[wordId];
		int tokenEnd = d_TokenOffset[wordId] + d_TokenCount[wordId];
//		int tokenStart = d_TokenOffset[wordId] + tokenRegionStart[0] * tokenSegment;
//		int tokenStartNew = d_TokenOffset[wordId];
//		int tokenEnd = d_TokenOffset[wordId] + (tokenRegionStart[0] + 1) * tokenSegment;
//		if (tokenEndFlag[0]) tokenEnd = d_TokenOffset[wordId] + d_TokenCount[wordId];

//		int WTStart = d_WordListOffset[wordId];
//		unsigned short int maxK = deviceWordMaxTopic[wordId];
		/*unsigned short int secondMaxK = deviceWordSecondMaxTopic[wordId];
		unsigned short int thirdMaxK = deviceWordThirdMaxTopic[wordId];*/
		// Reconstruct dense WT vector from sparse WT matrix
		//for (int i = threadIdx.x; i < K; i += blockDim.x)
		//{
		//	WTHead[i] = (d_WTDense[WTStart + i] + beta) / (d_WTRowSum[i] + W*beta);
		//	//__syncthreads();
		//}
		//__syncthreads();

		//if (threadIdx.x == 0) {
		//	WTMax[0] = WTHead[maxK - 1];
		//	WTMax[1] = WTHead[secondMaxK - 1];
		//	WTMax[2] = WTHead[thirdMaxK - 1];
		//	WTHead[maxK - 1] = 0.0;
		//	//WTHead[secondMaxK - 1] = 0.0;
		//}
		//__syncthreads();


		//for (int i = localId; i < K / 32; i += blockDim.x / 32) {
		//	unsigned short int   tmpK = i * 32 + laneId;
		//	float tmpVal = 0.0;
		//	tmpVal = alpha*WTHead[tmpK];
		//	tmpVal += __shfl_down(tmpVal, 16);
		//	tmpVal += __shfl_down(tmpVal, 8);
		//	tmpVal += __shfl_down(tmpVal, 4);
		//	tmpVal += __shfl_down(tmpVal, 2);
		//	tmpVal += __shfl_down(tmpVal, 1);
		//	tmpVal = __shfl(tmpVal, 0);
		//	QTree[i] = tmpVal;

		//}
		//__syncthreads();

		//if (localId == 0) {

		//	float value = QTree[laneId];
		//	value += __shfl_up(value, 1, 32)*(laneId >= 1);
		//	value += __shfl_up(value, 2, 32)*(laneId >= 2);
		//	value += __shfl_up(value, 4, 32)*(laneId >= 4);
		//	value += __shfl_up(value, 8, 32)*(laneId >= 8);
		//	value += __shfl_up(value, 16, 32)*(laneId >= 16);

		//	QTree[laneId] = value;

		//}
		//if (threadIdx.x == 0) WarpCounter[0] = 0;
		//__syncthreads();
		//float Q = QTree[31];
		//int tokenIdx;
		/*float WTMaxProb = WTMax[0];
		float WTSecondMaxProb = WTMax[1];
		float WTThirdMaxProb = WTMax[2];*/
		float WTMaxProb = deviceWordMaxProb[wordId];
		float WTSecondMaxProb = deviceWordSecondMaxProb[wordId];
		float WTThirdMaxProb = deviceWordThirdMaxProb[wordId];
		float Q = alpha* deviceQArray[wordId];

//		finish0 = clock64();
//		costtime0 += (double)(finish0 - start0);

		for (int tokenIdx = tokenStart + threadIdx.x; tokenIdx < tokenEnd; tokenIdx += blockDim.x)
		{


			int docId = d_DocIndex[tokenIdx]-1;

			int totalTokenCount = d_TokenCountDT[docId];

			deviceTotalTokenCount[tokenIdx] = totalTokenCount;


		}


//		if (laneId == 0)
//		{
//			acquire_semaphore(&sem1);
//			tokenEndFlag = 0;
//			counter = d_blockCounter[0];
//			unsigned int numRegions = (d_TokenCount[counter] == 0) ? 0 : ((d_TokenCount[counter] - 1) / tokenSegment);
//			tokenRegionStart = atomicInc(&subCount1, numRegions);
//			if (subCount1 == 0) {
//				d_blockCounter[0] = d_blockCounter[0] + 1;
//				tokenEndFlag = 1;
//			}
//			release_semaphore(&sem1);
//		}
//
//		tokenRegionStart = __shfl(tokenRegionStart, 0);
//		tokenEndFlag = __shfl(tokenEndFlag, 0);
//		counter = __shfl(counter, 0);
//		finish2 = clock64();
//		if (threadIdx.x== 0)
//		{
//			acquire_semaphore(&sem1);
//			tokenEndFlag[0] = 0;
//			Counter[0] = d_blockCounter[0];
//			unsigned int numRegions = (d_TokenCount[Counter[0]] == 0) ? 0 : ((d_TokenCount[Counter[0]] - 1) / tokenSegment);
//			tokenRegionStart[0] = atomicInc(&subCount1, numRegions);
//			if (subCount1 == 0) {
//				d_blockCounter[0] = d_blockCounter[0] + 1;
//				tokenEndFlag[0] = 1;
//			}
//			release_semaphore(&sem1);
//		}
//		finish3 = clock64();
//		costtime3 += (double)(finish3 - finish2);
//	//
//	//	tokenRegionStart = __shfl(tokenRegionStart, 0);
//	//	tokenEndFlag = __shfl(tokenEndFlag, 0);
//	//
//	//	counter = __shfl(counter, 0);
//		__syncthreads();


//		finish2 = clock64();
//		if (threadIdx.x == 0) Counter[0] = atomicAdd(&d_blockCounter[0], 1);
		if (threadIdx.x == 0)
		{

			Counter[0] = atomicAdd(&d_blockCounter[0], 1);
		}



		__syncthreads();
//		finish3 = clock64();
//		costtime3 += (double)(finish3 - finish2);

		/*if (threadIdx.x == 0) deviceNewTokenCount[wordId] = WarpCounter[0];
		__syncthreads();
*/
	}

	//if (threadIdx.x + blockDim.x*blockIdx.x == 0) printf("costtime0,costtime1,costtime2,costtime3:%f,%f,%f,%f\n", costtime0 / (158200000 * 1.0), costtime1 / (158200000 * 1.0),costtime2 / (158200000 * 1.0), costtime3 / (158200000 * 1.0));


}
__global__ void UpdateProbKernelTrainD1(float alpha, float beta, int* d_Index, unsigned short int* d_TopicIndex, int* d_SparseDTCount, int* d_TokenCountDT, int* d_TokenOffsetDT, int* d_DocListCount, int* d_DocListOffset, int* d_WTDense, int* d_WTDenseCopy, int* d_TokenCount, int* d_TokenOffset, int* d_WordListCount, int* d_WordListOffset, int* d_WTRowSum, unsigned int* d_blockCounter, int*d_DocIndex, int D, int W, float* d_Perplexity, hiprandState *randState, float *WTHeadDense, int numOfWordD, unsigned short int* deviceWordMaxTopic, unsigned short int* deviceWordSecondMaxTopic,float* deviceMaxProb, float* deviceThresProb, unsigned short int* deviceWordThirdMaxTopic, float* deviceRandomfloat,  int* deviceEffectiveTokenIndex, int* deviceNewTokenCount, long long int* deviceMaxSecTopic, float* deviceQArray, float* deviceWordMaxProb, float* deviceWordSecondMaxProb, float* deviceWordThirdMaxProb, int tokenSegment)

{


	/*volatile __shared__ float WTHead[K];*/
	volatile __shared__ float QTree[32];
	volatile __shared__ float WTMax[3];
	volatile __shared__ unsigned int Counter[1];
	//__shared__ unsigned int WarpCounter[1];
//	volatile unsigned int tokenRegionStart;
//	volatile unsigned int tokenEndFlag;
//	__shared__ unsigned int tokenRegionStart[1];
//	volatile __shared__ unsigned int tokenEndFlag[1];


//	clock_t start0, finish0, finish1, finish2, finish3;
//	double costtime0 = 0.0, costtime1 = 0.0, costtime2 = 0.0, costtime3 = 0.0;

//	volatile unsigned int counter = 0;
//	if (threadIdx.x== 0)
//	{
//		acquire_semaphore(&sem1);
//		tokenEndFlag[0] = 0;
//		Counter[0] = d_blockCounter[0];
//		unsigned int numRegions = (d_TokenCount[Counter[0]] == 0) ? 0 : ((d_TokenCount[Counter[0]] - 1) / tokenSegment);
//		tokenRegionStart[0] = atomicInc(&subCount1, numRegions);
//		if (subCount1 == 0) {
//			d_blockCounter[0] = d_blockCounter[0] + 1;
//			tokenEndFlag[0] = 1;
//		}
//		release_semaphore(&sem1);
//	}
//
//	tokenRegionStart = __shfl(tokenRegionStart, 0);
//	tokenEndFlag = __shfl(tokenEndFlag, 0);
//
//	counter = __shfl(counter, 0);



//
//	if (threadIdx.x == 0) Counter[0] = atomicAdd(&d_blockCounter[0], 1);
//	__syncthreads();

	float sumPerplexity = 0.0;

	//while (Counter[0]<numOfWordD)
//	while (Counter[0]<numOfWordD)
	for(int wordId = blockIdx.x; wordId<numOfWordD; wordId+=gridDim.x)
	{
//		start0 = clock64();

//		int wordId =Counter[0];

		int tokenStart = d_TokenOffset[wordId];
		int tokenEnd = d_TokenOffset[wordId] + d_TokenCount[wordId];
//		int tokenStart = d_TokenOffset[wordId] + tokenRegionStart[0] * tokenSegment;
//		int tokenStartNew = d_TokenOffset[wordId];
//		int tokenEnd = d_TokenOffset[wordId] + (tokenRegionStart[0] + 1) * tokenSegment;
//		if (tokenEndFlag[0]) tokenEnd = d_TokenOffset[wordId] + d_TokenCount[wordId];

		int WTStart = d_WordListOffset[wordId];
		unsigned short int maxK = deviceWordMaxTopic[wordId];
		/*unsigned short int secondMaxK = deviceWordSecondMaxTopic[wordId];
		unsigned short int thirdMaxK = deviceWordThirdMaxTopic[wordId];*/
		// Reconstruct dense WT vector from sparse WT matrix
		//for (int i = threadIdx.x; i < K; i += blockDim.x)
		//{
		//	WTHead[i] = (d_WTDense[WTStart + i] + beta) / (d_WTRowSum[i] + W*beta);
		//	//__syncthreads();
		//}
		//__syncthreads();

		//if (threadIdx.x == 0) {
		//	WTMax[0] = WTHead[maxK - 1];
		//	WTMax[1] = WTHead[secondMaxK - 1];
		//	WTMax[2] = WTHead[thirdMaxK - 1];
		//	WTHead[maxK - 1] = 0.0;
		//	//WTHead[secondMaxK - 1] = 0.0;
		//}
		//__syncthreads();


		//for (int i = localId; i < K / 32; i += blockDim.x / 32) {
		//	unsigned short int   tmpK = i * 32 + laneId;
		//	float tmpVal = 0.0;
		//	tmpVal = alpha*WTHead[tmpK];
		//	tmpVal += __shfl_down(tmpVal, 16);
		//	tmpVal += __shfl_down(tmpVal, 8);
		//	tmpVal += __shfl_down(tmpVal, 4);
		//	tmpVal += __shfl_down(tmpVal, 2);
		//	tmpVal += __shfl_down(tmpVal, 1);
		//	tmpVal = __shfl(tmpVal, 0);
		//	QTree[i] = tmpVal;

		//}
		//__syncthreads();

		//if (localId == 0) {

		//	float value = QTree[laneId];
		//	value += __shfl_up(value, 1, 32)*(laneId >= 1);
		//	value += __shfl_up(value, 2, 32)*(laneId >= 2);
		//	value += __shfl_up(value, 4, 32)*(laneId >= 4);
		//	value += __shfl_up(value, 8, 32)*(laneId >= 8);
		//	value += __shfl_up(value, 16, 32)*(laneId >= 16);

		//	QTree[laneId] = value;

		//}
		//if (threadIdx.x == 0) WarpCounter[0] = 0;
		//__syncthreads();
		//float Q = QTree[31];
		//int tokenIdx;
		/*float WTMaxProb = WTMax[0];
		float WTSecondMaxProb = WTMax[1];
		float WTThirdMaxProb = WTMax[2];*/
		float WTMaxProb = deviceWordMaxProb[wordId];
		float WTSecondMaxProb = deviceWordSecondMaxProb[wordId];
		float WTThirdMaxProb = deviceWordThirdMaxProb[wordId];
		float Q = alpha* deviceQArray[wordId];

//		finish0 = clock64();
//		costtime0 += (double)(finish0 - start0);

		int numIter= d_TokenCount[wordId]/(4*blockDim.x);
		int numIter1 = (d_TokenCount[wordId] - numIter*4*blockDim.x)/(2*blockDim.x);

		for (int tokenIdx = tokenStart + threadIdx.x; (tokenIdx + 3 * blockDim.x) < (tokenStart + numIter*(4 * blockDim.x)); tokenIdx += 4 * blockDim.x)
		{
			//start0= clock64();

//			int docId1 = __ldg(&d_DocIndex[tokenIdx])-1;
//			int docId2 = __ldg(&d_DocIndex[tokenIdx+blockDim.x])-1;
//			int docId3 = __ldg(&d_DocIndex[tokenIdx+2*blockDim.x])-1;
//			int docId4 = __ldg(&d_DocIndex[tokenIdx+3*blockDim.x])-1;
			/*int docId1 = (deviceMaxSecTopic[tokenIdx] >> 32) & 0xffffffff - 1;
			
			int docId2 = (deviceMaxSecTopic[tokenIdx + blockDim.x] >> 32)&0xffffffff - 1;
			int docId3=(deviceMaxSecTopic[tokenIdx+2*blockDim.x] >> 32) & 0xffffffff - 1;
			int docId4=(deviceMaxSecTopic[tokenIdx+3*blockDim.x] >> 32) & 0xffffffff - 1;*/
			int docId1 = ((int)(deviceMaxSecTopic[tokenIdx] >> 32))-1;
			int docId2 = ((int)(deviceMaxSecTopic[tokenIdx + blockDim.x] >> 32))-1;
			int docId3 = ((int)(deviceMaxSecTopic[tokenIdx + 2*blockDim.x] >> 32))-1;
			int docId4 = ((int)(deviceMaxSecTopic[tokenIdx + 3*blockDim.x] >> 32))-1;
//			docId1 = docId1 - 1;
//			docId2 = docId2 - 1;
//			docId3 = docId3 - 1;
//			docId4 = docId4 - 1;
			unsigned short int totalTokenCount1 = d_TokenCountDT[docId1];
			unsigned short int totalTokenCount2 = d_TokenCountDT[docId2];
			unsigned short int totalTokenCount3 = d_TokenCountDT[docId3];
			unsigned short int totalTokenCount4 = d_TokenCountDT[docId4];

//			int docId = __ldg(&d_DocIndex[tokenIdx])-1;


//			unsigned short int totalTokenCount1 = deviceTotalTokenCount[tokenIdx];
//			unsigned short int totalTokenCount2 = deviceTotalTokenCount[tokenIdx+blockDim.x];
//			unsigned short int totalTokenCount3 = deviceTotalTokenCount[tokenIdx+2*blockDim.x];
//			unsigned short int totalTokenCount4 = deviceTotalTokenCount[tokenIdx+3*blockDim.x];

//			unsigned short int totalTokenCount5 = deviceTotalTokenCount[tokenIdx+4*blockDim.x];
//			unsigned short int totalTokenCount6 = deviceTotalTokenCount[tokenIdx+5*blockDim.x];
//			finish0 = clock64();
//			costtime0 += (double)(finish0 - start0);
			//int nonSkipTokenIdx;

			float u1 = hiprand_uniform(&(randState[threadIdx.x + blockDim.x*blockIdx.x])) / 1.00001;
			float u2 = hiprand_uniform(&(randState[threadIdx.x + blockDim.x*blockIdx.x+blockDim.x])) / 1.00001;
			float u3 = hiprand_uniform(&(randState[threadIdx.x + blockDim.x*blockIdx.x+2*blockDim.x])) / 1.00001;
			float u4 = hiprand_uniform(&(randState[threadIdx.x + blockDim.x*blockIdx.x+3*blockDim.x])) / 1.00001;

//			float u5 = hiprand_uniform(&(randState[threadIdx.x + blockDim.x*blockIdx.x+4*blockDim.x])) / 1.00001;
//			float u6 = hiprand_uniform(&(randState[threadIdx.x + blockDim.x*blockIdx.x+5*blockDim.x])) / 1.00001;


//			finish1 = clock64();
//			costtime1 += (double)(finish1 - finish0);
			deviceRandomfloat[tokenIdx] = u1;
			deviceRandomfloat[tokenIdx+blockDim.x] = u2;
			deviceRandomfloat[tokenIdx+2*blockDim.x] = u3;
			deviceRandomfloat[tokenIdx+3*blockDim.x] = u4;
//			deviceRandomfloat[tokenIdx+4*blockDim.x] = u5;
//			deviceRandomfloat[tokenIdx+5*blockDim.x] = u6;
			/*unsigned short int maxTokenCount = deviceMaxTokenCount[tokenIdx];
			unsigned short int maxSecondTokenCount = deviceSecondMaxTokenCount[tokenIdx];*/

			unsigned short int maxTokenCount1 = ((int) deviceMaxSecTopic[tokenIdx])&0xffff;
			unsigned short int maxSecondTokenCount1 = (((int) deviceMaxSecTopic[tokenIdx])&0xffff0000)>>16;

			unsigned short int maxTokenCount2 = ((int)deviceMaxSecTopic[tokenIdx+blockDim.x])&(0xffff);
			unsigned short int maxSecondTokenCount2 = (((int) deviceMaxSecTopic[tokenIdx+blockDim.x])&(0xffff0000))>>16;

			unsigned short int maxTokenCount3 = ((int)deviceMaxSecTopic[tokenIdx+2*blockDim.x])&(0xffff);
			unsigned short int maxSecondTokenCount3 = (((int)deviceMaxSecTopic[tokenIdx+2*blockDim.x])&(0xffff0000))>>16;

			unsigned short int maxTokenCount4 = ((int)deviceMaxSecTopic[tokenIdx+3*blockDim.x])&(0xffff);
			unsigned short int maxSecondTokenCount4 = (((int)deviceMaxSecTopic[tokenIdx+3*blockDim.x])&(0x00000000ffff0000))>>16;

//
//			unsigned short int maxTokenCount5 = deviceMaxSecTopic[tokenIdx+4*blockDim.x]&(0x0000ffff);
//			unsigned short int maxSecondTokenCount5 = (deviceMaxSecTopic[tokenIdx+4*blockDim.x]&(0xffff0000))>>16;
//
//			unsigned short int maxTokenCount6 = deviceMaxSecTopic[tokenIdx+5*blockDim.x]&(0x0000ffff);
//			unsigned short int maxSecondTokenCount6 = (deviceMaxSecTopic[tokenIdx+5*blockDim.x]&(0xffff0000))>>16;




//			finish2 = clock64();
//			costtime2 += (double)(finish2 - finish1);


			//float maxS = (totalTokenCount - maxTokenCount)*WTSecondMaxProb;

			float maxS1 = (totalTokenCount1 - maxTokenCount1 - maxSecondTokenCount1)*WTThirdMaxProb + maxSecondTokenCount1*WTSecondMaxProb;
			float maxProb1 = (maxTokenCount1 + alpha)*WTMaxProb;
			float thresProb1= maxProb1/(maxProb1+maxS1+Q);
			deviceMaxProb[tokenIdx] = maxProb1;
			deviceThresProb[tokenIdx] = thresProb1;

			float maxS2 = (totalTokenCount2 - maxTokenCount2 - maxSecondTokenCount2)*WTThirdMaxProb + maxSecondTokenCount2*WTSecondMaxProb;
			float maxProb2 = (maxTokenCount2 + alpha)*WTMaxProb;
			float thresProb2= maxProb2/(maxProb2+maxS2+Q);
			deviceMaxProb[tokenIdx+blockDim.x] = maxProb2;
			deviceThresProb[tokenIdx+blockDim.x] = thresProb2;


			float maxS3 = (totalTokenCount3 - maxTokenCount3 - maxSecondTokenCount3)*WTThirdMaxProb + maxSecondTokenCount3*WTSecondMaxProb;
			float maxProb3 = (maxTokenCount3 + alpha)*WTMaxProb;
			float thresProb3= maxProb3/(maxProb3+maxS3+Q);
			deviceMaxProb[tokenIdx+2*blockDim.x] = maxProb3;
			deviceThresProb[tokenIdx+2*blockDim.x] = thresProb3;

			float maxS4 = (totalTokenCount4 - maxTokenCount4 - maxSecondTokenCount4)*WTThirdMaxProb + maxSecondTokenCount4*WTSecondMaxProb;
			float maxProb4 = (maxTokenCount4 + alpha)*WTMaxProb;
			float thresProb4= maxProb4/(maxProb4+maxS4+Q);
			deviceMaxProb[tokenIdx+3*blockDim.x] = maxProb4;
			deviceThresProb[tokenIdx+3*blockDim.x] = thresProb4;
			int nonSkipTokenIdx1;
			int nonSkipTokenIdx2;
			int nonSkipTokenIdx3;
			int nonSkipTokenIdx4;


			if(u1 > thresProb1) {

				nonSkipTokenIdx1 = atomicAdd(&deviceNewTokenCount[wordId], 1);

				deviceEffectiveTokenIndex[nonSkipTokenIdx1 + tokenStart] = tokenIdx;

			}
			else {


				d_TopicIndex[tokenIdx] = maxK;
				atomicAdd(&d_WTDenseCopy[WTStart + maxK-1], 1);
				//sumPerplexity += 1.0;

			}
			if(u2 > thresProb2) {

				nonSkipTokenIdx2 = atomicAdd(&deviceNewTokenCount[wordId], 1);

				deviceEffectiveTokenIndex[nonSkipTokenIdx2 + tokenStart] = tokenIdx+blockDim.x;

			}
			else {


				d_TopicIndex[tokenIdx+blockDim.x] = maxK;
				atomicAdd(&d_WTDenseCopy[WTStart + maxK-1], 1);
				//sumPerplexity += 1.0;

			}
			if(u3 > thresProb3) {

				nonSkipTokenIdx3 = atomicAdd(&deviceNewTokenCount[wordId], 1);

				deviceEffectiveTokenIndex[nonSkipTokenIdx3 + tokenStart] = tokenIdx+2*blockDim.x;

			}
			else {


				d_TopicIndex[tokenIdx+2*blockDim.x] = maxK;
				atomicAdd(&d_WTDenseCopy[WTStart + maxK-1], 1);
				//sumPerplexity += 1.0;

			}

			if(u4 > thresProb4) {

				nonSkipTokenIdx4 = atomicAdd(&deviceNewTokenCount[wordId], 1);

				deviceEffectiveTokenIndex[nonSkipTokenIdx4 + tokenStart] = tokenIdx+3*blockDim.x;

			}
			else {


				d_TopicIndex[tokenIdx+3*blockDim.x] = maxK;
				atomicAdd(&d_WTDenseCopy[WTStart + maxK-1], 1);
				//sumPerplexity += 1.0;

			}




//			float maxS5 = (totalTokenCount5 - maxTokenCount5 - maxSecondTokenCount5)*WTThirdMaxProb + maxSecondTokenCount5*WTSecondMaxProb;
//			float maxProb5 = (maxTokenCount5 + alpha)*WTMaxProb;
//			float thresProb5= maxProb5/(maxProb5+maxS5+Q);
//			deviceMaxProb[tokenIdx+4*blockDim.x] = maxProb5;
//			deviceThresProb[tokenIdx+4*blockDim.x] = thresProb5;
//
//			float maxS6 = (totalTokenCount6 - maxTokenCount6 - maxSecondTokenCount6)*WTThirdMaxProb + maxSecondTokenCount6*WTSecondMaxProb;
//			float maxProb6 = (maxTokenCount6 + alpha)*WTMaxProb;
//			float thresProb6= maxProb6/(maxProb6+maxS6+Q);
//			deviceMaxProb[tokenIdx+5*blockDim.x] = maxProb6;
//			deviceThresProb[tokenIdx+5*blockDim.x] = thresProb6;





//			finish3 = clock64();
//			costtime3 += (double)(finish3 - finish2);

//			short int flag = (u > deviceThresProb[tokenIdx]);
//			short int warpNonZeroCount=__popc(__ballot(flag));
//			if (laneId==0) nonSkipTokenIdx = atomicAdd(&deviceNewTokenCount[wordId], warpNonZeroCount);
//			nonSkipTokenIdx=__shfl(nonSkipTokenIdx,0);
//			flag += __shfl_up(flag, 1, 32)*(laneId >= 1);
//			flag += __shfl_up(flag, 2, 32)*(laneId >= 2);
//			flag += __shfl_up(flag, 4, 32)*(laneId >= 4);
//			flag += __shfl_up(flag, 8, 32)*(laneId >= 8);
//			flag += __shfl_up(flag, 16, 32)*(laneId >= 16);
//			if (u > deviceThresProb[tokenIdx]) deviceEffectiveTokenIndex[nonSkipTokenIdx + tokenStart+flag-1] = tokenIdx;


			//__syncthreads();
		}

		for (int tokenIdx = tokenStart+numIter*(4*blockDim.x) + threadIdx.x; (tokenIdx+blockDim.x) < tokenStart+numIter*4*blockDim.x+numIter1*2*blockDim.x; tokenIdx += 2*blockDim.x)
		{
			//start0= clock64();

//			int docId = __ldg(&d_DocIndex[tokenIdx])-1;
//
//			int totalTokenCount = d_TokenCountDT[docId];
//			int docId = __ldg(&d_DocIndex[tokenIdx])-1;
//			int docId1 = __ldg(&d_DocIndex[tokenIdx])-1;
//			int docId2 = __ldg(&d_DocIndex[tokenIdx+blockDim.x])-1;
			int docId1 = ((int)(deviceMaxSecTopic[tokenIdx] >> 32))-1;
			int docId2 = ((int)(deviceMaxSecTopic[tokenIdx + blockDim.x] >> 32))-1;
			
//			docId1 = docId1 - 1;
//			docId2 = docId2 - 1;
//



			//int docId1=(deviceMaxSecTopic[tokenIdx] >> 32) & 0xffffffff - 1;
			//int docId2=(deviceMaxSecTopic[tokenIdx+blockDim.x] >> 32) & 0xffffffff - 1;

			unsigned short int totalTokenCount1 = d_TokenCountDT[docId1];
			unsigned short int totalTokenCount2 = d_TokenCountDT[docId2];


//			unsigned short int totalTokenCount1 = deviceTotalTokenCount[tokenIdx];
//			unsigned short int totalTokenCount2 = deviceTotalTokenCount[tokenIdx+blockDim.x];


//			unsigned short int totalTokenCount5 = deviceTotalTokenCount[tokenIdx+4*blockDim.x];
//			unsigned short int totalTokenCount6 = deviceTotalTokenCount[tokenIdx+5*blockDim.x];
//			finish0 = clock64();
//			costtime0 += (double)(finish0 - start0);
			//int nonSkipTokenIdx;

			float u1 = hiprand_uniform(&(randState[threadIdx.x + blockDim.x*blockIdx.x])) / 1.00001;
			float u2 = hiprand_uniform(&(randState[threadIdx.x + blockDim.x*blockIdx.x+blockDim.x])) / 1.00001;


//			float u5 = hiprand_uniform(&(randState[threadIdx.x + blockDim.x*blockIdx.x+4*blockDim.x])) / 1.00001;
//			float u6 = hiprand_uniform(&(randState[threadIdx.x + blockDim.x*blockIdx.x+5*blockDim.x])) / 1.00001;


//			finish1 = clock64();
//			costtime1 += (double)(finish1 - finish0);
			deviceRandomfloat[tokenIdx] = u1;
			deviceRandomfloat[tokenIdx+blockDim.x] = u2;

//			deviceRandomfloat[tokenIdx+4*blockDim.x] = u5;
//			deviceRandomfloat[tokenIdx+5*blockDim.x] = u6;
			/*unsigned short int maxTokenCount = deviceMaxTokenCount[tokenIdx];
			unsigned short int maxSecondTokenCount = deviceSecondMaxTokenCount[tokenIdx];*/

			unsigned short int maxTokenCount1 = ((int)deviceMaxSecTopic[tokenIdx]) & 0xffff;
			unsigned short int maxSecondTokenCount1 = (((int)deviceMaxSecTopic[tokenIdx]) & 0xffff0000) >> 16;

			unsigned short int maxTokenCount2 = ((int)deviceMaxSecTopic[tokenIdx + blockDim.x])&(0xffff);
			unsigned short int maxSecondTokenCount2 = (((int)deviceMaxSecTopic[tokenIdx + blockDim.x])&(0xffff0000)) >> 16;



//
//			unsigned short int maxTokenCount5 = deviceMaxSecTopic[tokenIdx+4*blockDim.x]&(0x0000ffff);
//			unsigned short int maxSecondTokenCount5 = (deviceMaxSecTopic[tokenIdx+4*blockDim.x]&(0xffff0000))>>16;
//
//			unsigned short int maxTokenCount6 = deviceMaxSecTopic[tokenIdx+5*blockDim.x]&(0x0000ffff);
//			unsigned short int maxSecondTokenCount6 = (deviceMaxSecTopic[tokenIdx+5*blockDim.x]&(0xffff0000))>>16;




//			finish2 = clock64();
//			costtime2 += (double)(finish2 - finish1);


			//float maxS = (totalTokenCount - maxTokenCount)*WTSecondMaxProb;

			float maxS1 = (totalTokenCount1 - maxTokenCount1 - maxSecondTokenCount1)*WTThirdMaxProb + maxSecondTokenCount1*WTSecondMaxProb;
			float maxProb1 = (maxTokenCount1 + alpha)*WTMaxProb;
			float thresProb1= maxProb1/(maxProb1+maxS1+Q);
			deviceMaxProb[tokenIdx] = maxProb1;
			deviceThresProb[tokenIdx] = thresProb1;

			float maxS2 = (totalTokenCount2 - maxTokenCount2 - maxSecondTokenCount2)*WTThirdMaxProb + maxSecondTokenCount2*WTSecondMaxProb;
			float maxProb2 = (maxTokenCount2 + alpha)*WTMaxProb;
			float thresProb2= maxProb2/(maxProb2+maxS2+Q);
			deviceMaxProb[tokenIdx+blockDim.x] = maxProb2;
			deviceThresProb[tokenIdx+blockDim.x] = thresProb2;
			int nonSkipTokenIdx1;
			int nonSkipTokenIdx2;

			if(u1 > thresProb1) {

				nonSkipTokenIdx1 = atomicAdd(&deviceNewTokenCount[wordId], 1);

				deviceEffectiveTokenIndex[nonSkipTokenIdx1 + tokenStart] = tokenIdx;

			}
			else {


				d_TopicIndex[tokenIdx] = maxK;
				atomicAdd(&d_WTDenseCopy[WTStart + maxK-1], 1);
				//sumPerplexity += 1.0;

			}
			if(u2 > thresProb2) {

				nonSkipTokenIdx2 = atomicAdd(&deviceNewTokenCount[wordId], 1);

				deviceEffectiveTokenIndex[nonSkipTokenIdx2 + tokenStart] = tokenIdx+blockDim.x;

			}
			else {


				d_TopicIndex[tokenIdx+blockDim.x] = maxK;
				atomicAdd(&d_WTDenseCopy[WTStart + maxK-1], 1);
				//sumPerplexity += 1.0;

			}




		}








		for (int tokenIdx = tokenStart+numIter*(4*blockDim.x)+ numIter1*2*blockDim.x+ threadIdx.x; tokenIdx < tokenEnd; tokenIdx += blockDim.x)
		{

//			int docId = __ldg(&d_DocIndex[tokenIdx])-1;
//
//			int totalTokenCount = d_TokenCountDT[docId];
//
			//unsigned short int totalTokenCount = deviceTotalTokenCount[tokenIdx];

			//int docId = __ldg(&d_DocIndex[tokenIdx])-1;
			int docId= ((int) (deviceMaxSecTopic[tokenIdx] >> 32))-1;

			//docId = docId - 1;
			unsigned short int totalTokenCount = d_TokenCountDT[docId];



			//int nonSkipTokenIdx;
//
			float u = hiprand_uniform(&(randState[threadIdx.x + blockDim.x*blockIdx.x])) / 1.00001;
			deviceRandomfloat[tokenIdx] = u;
			/*unsigned short int maxTokenCount = deviceMaxTokenCount[tokenIdx];
			unsigned short int maxSecondTokenCount = deviceSecondMaxTokenCount[tokenIdx];*/

			unsigned short int maxTokenCount = ((int)deviceMaxSecTopic[tokenIdx])&(0xffff);
			unsigned short int maxSecondTokenCount = (((int)deviceMaxSecTopic[tokenIdx])&(0xffff0000))>>16;




			//float maxS = (totalTokenCount - maxTokenCount)*WTSecondMaxProb;

			float maxS = (totalTokenCount - maxTokenCount - maxSecondTokenCount)*WTThirdMaxProb + maxSecondTokenCount*WTSecondMaxProb;
			float maxProb = (maxTokenCount + alpha)*WTMaxProb;
			float thresProb= maxProb/(maxProb+maxS+Q);
			deviceMaxProb[tokenIdx] = maxProb;
			deviceThresProb[tokenIdx] = thresProb;
			int nonSkipTokenIdx;

			if(u > thresProb) {

				nonSkipTokenIdx = atomicAdd(&deviceNewTokenCount[wordId], 1);
				deviceEffectiveTokenIndex[nonSkipTokenIdx + tokenStart] = tokenIdx;

			}
			else {

				d_TopicIndex[tokenIdx] = maxK;
				atomicAdd(&d_WTDenseCopy[WTStart + maxK-1], 1);
				//sumPerplexity += 1.0;

			}

		}










//		if (laneId == 0)
//		{
//			acquire_semaphore(&sem1);
//			tokenEndFlag = 0;
//			counter = d_blockCounter[0];
//			unsigned int numRegions = (d_TokenCount[counter] == 0) ? 0 : ((d_TokenCount[counter] - 1) / tokenSegment);
//			tokenRegionStart = atomicInc(&subCount1, numRegions);
//			if (subCount1 == 0) {
//				d_blockCounter[0] = d_blockCounter[0] + 1;
//				tokenEndFlag = 1;
//			}
//			release_semaphore(&sem1);
//		}
//
//		tokenRegionStart = __shfl(tokenRegionStart, 0);
//		tokenEndFlag = __shfl(tokenEndFlag, 0);
//		counter = __shfl(counter, 0);
//		finish2 = clock64();
//		if (threadIdx.x== 0)
//		{
//			acquire_semaphore(&sem1);
//			tokenEndFlag[0] = 0;
//			Counter[0] = d_blockCounter[0];
//			unsigned int numRegions = (d_TokenCount[Counter[0]] == 0) ? 0 : ((d_TokenCount[Counter[0]] - 1) / tokenSegment);
//			tokenRegionStart[0] = atomicInc(&subCount1, numRegions);
//			if (subCount1 == 0) {
//				d_blockCounter[0] = d_blockCounter[0] + 1;
//				tokenEndFlag[0] = 1;
//			}
//			release_semaphore(&sem1);
//		}
//		finish3 = clock64();
//		costtime3 += (double)(finish3 - finish2);
//	//
//	//	tokenRegionStart = __shfl(tokenRegionStart, 0);
//	//	tokenEndFlag = __shfl(tokenEndFlag, 0);
//	//
//	//	counter = __shfl(counter, 0);
//		__syncthreads();


//		finish2 = clock64();
//		if (threadIdx.x == 0) Counter[0] = atomicAdd(&d_blockCounter[0], 1);
//		if (threadIdx.x == 0)
//		{
//
//			Counter[0] = atomicAdd(&d_blockCounter[0], 1);
//		}



		__syncthreads();
//		finish3 = clock64();
//		costtime3 += (double)(finish3 - finish2);

		/*if (threadIdx.x == 0) deviceNewTokenCount[wordId] = WarpCounter[0];
		__syncthreads();
*/
	}
	sumPerplexity += __shfl_down(sumPerplexity, 16);
	sumPerplexity += __shfl_down(sumPerplexity, 8);
	sumPerplexity += __shfl_down(sumPerplexity, 4);
	sumPerplexity += __shfl_down(sumPerplexity, 2);
	sumPerplexity += __shfl_down(sumPerplexity, 1);
	sumPerplexity = __shfl(sumPerplexity, 0);
	int laneId = threadIdx.x % 32;
	int localId = threadIdx.x / 32;
	if (localId == 0) {
		QTree[laneId] = 0;
	}
	__syncthreads();
	if (laneId == 0) QTree[localId] = sumPerplexity;
	__syncthreads();

	if (localId == 0) {
		float perplexity = 0.0;
		perplexity = QTree[laneId] * (laneId < blockDim.x / 32);
		perplexity += __shfl_down(perplexity, 16);
		perplexity += __shfl_down(perplexity, 8);
		perplexity += __shfl_down(perplexity, 4);
		perplexity += __shfl_down(perplexity, 2);
		perplexity += __shfl_down(perplexity, 1);
		if (laneId == 0) d_Perplexity[blockIdx.x] += perplexity;
	}

	//if (threadIdx.x + blockDim.x*blockIdx.x == 0) printf("costtime0,costtime1,costtime2,costtime3:%f,%f,%f,%f\n", costtime0 / (158200000 * 1.0), costtime1 / (158200000 * 1.0),costtime2 / (158200000 * 1.0), costtime3 / (158200000 * 1.0));


}
__global__ void UpdateProbKernelTrainD2(float alpha, float beta, int* d_Index, unsigned short int* d_TopicIndex, int* d_SparseDTCount, int* d_TokenCountDT, int* d_TokenOffsetDT, int* d_DocListCount, int* d_DocListOffset, int* d_WTDense, int* d_WTDenseCopy, int* d_TokenCount, int* d_TokenOffset, int* d_WordListCount, int* d_WordListOffset, int* d_WTRowSum, unsigned int* d_blockCounter, int*d_DocIndex, int D, int W, float* d_Perplexity, hiprandState *randState, float *WTHeadDense, int numOfWordD, unsigned short int* deviceWordMaxTopic, unsigned short int* deviceWordSecondMaxTopic,float* deviceMaxProb, float* deviceThresProb, unsigned short int* deviceWordThirdMaxTopic, float* deviceRandomfloat,  int* deviceEffectiveTokenIndex, int* deviceNewTokenCount, int* deviceMaxSecTopic, float* deviceQArray, float* deviceWordMaxProb, float* deviceWordSecondMaxProb, float* deviceWordThirdMaxProb, int tokenSegment)

{


	/*volatile __shared__ float WTHead[K];*/
	volatile __shared__ float QTree[32];
	volatile __shared__ float WTMax[3];
	volatile __shared__ unsigned int Counter[1];
	//__shared__ unsigned int WarpCounter[1];
//	volatile unsigned int tokenRegionStart;
//	volatile unsigned int tokenEndFlag;
//	__shared__ unsigned int tokenRegionStart[1];
//	volatile __shared__ unsigned int tokenEndFlag[1];
	//__shared__ int newTokenCount[1];

//	clock_t start0, finish0, finish1, finish2, finish3;
//	double costtime0 = 0.0, costtime1 = 0.0, costtime2 = 0.0, costtime3 = 0.0;

//	volatile unsigned int counter = 0;
//	if (threadIdx.x== 0)
//	{
//		acquire_semaphore(&sem1);
//		tokenEndFlag[0] = 0;
//		Counter[0] = d_blockCounter[0];
//		unsigned int numRegions = (d_TokenCount[Counter[0]] == 0) ? 0 : ((d_TokenCount[Counter[0]] - 1) / tokenSegment);
//		tokenRegionStart[0] = atomicInc(&subCount1, numRegions);
//		if (subCount1 == 0) {
//			d_blockCounter[0] = d_blockCounter[0] + 1;
//			tokenEndFlag[0] = 1;
//		}
//		release_semaphore(&sem1);
//	}
//
//	tokenRegionStart = __shfl(tokenRegionStart, 0);
//	tokenEndFlag = __shfl(tokenEndFlag, 0);
//
//	counter = __shfl(counter, 0);




	if (threadIdx.x == 0) Counter[0] = atomicAdd(&d_blockCounter[0], 1);
	__syncthreads();

	float sumPerplexity = 0.0;

	//while (Counter[0]<numOfWordD)
	while (Counter[0]<numOfWordD)
	{
//		start0 = clock64();

		int wordId =Counter[0];

		int tokenStart = d_TokenOffset[wordId];
		int tokenEnd = d_TokenOffset[wordId] + d_TokenCount[wordId];
//		int tokenStart = d_TokenOffset[wordId] + tokenRegionStart[0] * tokenSegment;
//		int tokenStartNew = d_TokenOffset[wordId];
//		int tokenEnd = d_TokenOffset[wordId] + (tokenRegionStart[0] + 1) * tokenSegment;
//		if (tokenEndFlag[0]) tokenEnd = d_TokenOffset[wordId] + d_TokenCount[wordId];

		int WTStart = d_WordListOffset[wordId];
		unsigned short int maxK = deviceWordMaxTopic[wordId];
		/*unsigned short int secondMaxK = deviceWordSecondMaxTopic[wordId];
		unsigned short int thirdMaxK = deviceWordThirdMaxTopic[wordId];*/
		// Reconstruct dense WT vector from sparse WT matrix
		//for (int i = threadIdx.x; i < K; i += blockDim.x)
		//{
		//	WTHead[i] = (d_WTDense[WTStart + i] + beta) / (d_WTRowSum[i] + W*beta);
		//	//__syncthreads();
		//}
		//__syncthreads();

		//if (threadIdx.x == 0) {
		//	WTMax[0] = WTHead[maxK - 1];
		//	WTMax[1] = WTHead[secondMaxK - 1];
		//	WTMax[2] = WTHead[thirdMaxK - 1];
		//	WTHead[maxK - 1] = 0.0;
		//	//WTHead[secondMaxK - 1] = 0.0;
		//}
		//__syncthreads();


		//for (int i = localId; i < K / 32; i += blockDim.x / 32) {
		//	unsigned short int   tmpK = i * 32 + laneId;
		//	float tmpVal = 0.0;
		//	tmpVal = alpha*WTHead[tmpK];
		//	tmpVal += __shfl_down(tmpVal, 16);
		//	tmpVal += __shfl_down(tmpVal, 8);
		//	tmpVal += __shfl_down(tmpVal, 4);
		//	tmpVal += __shfl_down(tmpVal, 2);
		//	tmpVal += __shfl_down(tmpVal, 1);
		//	tmpVal = __shfl(tmpVal, 0);
		//	QTree[i] = tmpVal;

		//}
		//__syncthreads();

		//if (localId == 0) {

		//	float value = QTree[laneId];
		//	value += __shfl_up(value, 1, 32)*(laneId >= 1);
		//	value += __shfl_up(value, 2, 32)*(laneId >= 2);
		//	value += __shfl_up(value, 4, 32)*(laneId >= 4);
		//	value += __shfl_up(value, 8, 32)*(laneId >= 8);
		//	value += __shfl_up(value, 16, 32)*(laneId >= 16);

		//	QTree[laneId] = value;

		//}
		//if (threadIdx.x == 0) WarpCounter[0] = 0;
		//__syncthreads();
		//float Q = QTree[31];
		//int tokenIdx;
		/*float WTMaxProb = WTMax[0];
		float WTSecondMaxProb = WTMax[1];
		float WTThirdMaxProb = WTMax[2];*/
//		float WTMaxProb = deviceWordMaxProb[wordId];
//		float WTSecondMaxProb = deviceWordSecondMaxProb[wordId];
//		float WTThirdMaxProb = deviceWordThirdMaxProb[wordId];
//		float Q = alpha* deviceQArray[wordId];

//		finish0 = clock64();
//		costtime0 += (double)(finish0 - start0);


		int numIter= d_TokenCount[wordId]/(4*blockDim.x);
		//int numIter1 = (d_TokenCount[wordId] - numIter*4*blockDim.x)/(2*blockDim.x);
		for (int tokenIdx = tokenStart + threadIdx.x; (tokenIdx+3*blockDim.x) < (tokenStart+numIter*(4*blockDim.x)); tokenIdx += 4*blockDim.x){
			int nonSkipTokenIdx1;
			int nonSkipTokenIdx2;
			int nonSkipTokenIdx3;
			int nonSkipTokenIdx4;
			float u1 = deviceRandomfloat[tokenIdx];
			float u2 = deviceRandomfloat[tokenIdx+blockDim.x];
			float u3 = deviceRandomfloat[tokenIdx+2*blockDim.x];
			float u4 = deviceRandomfloat[tokenIdx+3*blockDim.x];
			float thresProb1 = deviceThresProb[tokenIdx];
			float thresProb2 = deviceThresProb[tokenIdx+blockDim.x];
			float thresProb3 = deviceThresProb[tokenIdx+2*blockDim.x];
			float thresProb4 = deviceThresProb[tokenIdx+3*blockDim.x];
			if(u1 > thresProb1) {

				nonSkipTokenIdx1 = atomicAdd(&deviceNewTokenCount[wordId], 1);

				deviceEffectiveTokenIndex[nonSkipTokenIdx1 + tokenStart] = tokenIdx;

			}
			else {


				d_TopicIndex[tokenIdx] = maxK;
				atomicAdd(&d_WTDenseCopy[WTStart + maxK-1], 1);
				//sumPerplexity += 1.0;

			}
			if(u2 > thresProb2) {

				nonSkipTokenIdx2 = atomicAdd(&deviceNewTokenCount[wordId], 1);

				deviceEffectiveTokenIndex[nonSkipTokenIdx2 + tokenStart] = tokenIdx+blockDim.x;

			}
			else {


				d_TopicIndex[tokenIdx] = maxK;
				atomicAdd(&d_WTDenseCopy[WTStart + maxK-1], 1);
				//sumPerplexity += 1.0;

			}
			if(u3 > thresProb3) {

				nonSkipTokenIdx3 = atomicAdd(&deviceNewTokenCount[wordId], 1);

				deviceEffectiveTokenIndex[nonSkipTokenIdx3 + tokenStart] = tokenIdx+2*blockDim.x;

			}
			else {


				d_TopicIndex[tokenIdx] = maxK;
				atomicAdd(&d_WTDenseCopy[WTStart + maxK-1], 1);
				//sumPerplexity += 1.0;

			}

			if(u4 > thresProb4) {

				nonSkipTokenIdx4 = atomicAdd(&deviceNewTokenCount[wordId], 1);

				deviceEffectiveTokenIndex[nonSkipTokenIdx4 + tokenStart] = tokenIdx+3*blockDim.x;

			}
			else {


				d_TopicIndex[tokenIdx] = maxK;
				atomicAdd(&d_WTDenseCopy[WTStart + maxK-1], 1);
				//sumPerplexity += 1.0;

			}






		}

		for (int tokenIdx = (tokenStart+numIter*(4*blockDim.x)) + threadIdx.x; tokenIdx < tokenEnd; tokenIdx += blockDim.x)
		{

//			int docId = __ldg(&d_DocIndex[tokenIdx])-1;
//
//			int totalTokenCount = d_TokenCountDT[docId];
//
//
			int nonSkipTokenIdx;
//
//			float u = hiprand_uniform(&(randState[threadIdx.x + blockDim.x*blockIdx.x])) / 1.00001;
//			deviceRandomfloat[tokenIdx] = u;
//			/*unsigned short int maxTokenCount = deviceMaxTokenCount[tokenIdx];
//			unsigned short int maxSecondTokenCount = deviceSecondMaxTokenCount[tokenIdx];*/
//
//			unsigned short int maxTokenCount = deviceMaxSecTopic[tokenIdx]&(0x0000ffff);
//			unsigned short int maxSecondTokenCount = (deviceMaxSecTopic[tokenIdx]&(0xffff0000))>>16;
//
//
//
//
//			//float maxS = (totalTokenCount - maxTokenCount)*WTSecondMaxProb;
//
//			float maxS = (totalTokenCount - maxTokenCount - maxSecondTokenCount)*WTThirdMaxProb + maxSecondTokenCount*WTSecondMaxProb;
//			float maxProb = (maxTokenCount + alpha)*WTMaxProb;
//			float thresProb= maxProb/(maxProb+maxS+Q);
//			deviceMaxProb[tokenIdx] = maxProb;
			float u = deviceRandomfloat[tokenIdx];
			float thresProb = deviceThresProb[tokenIdx];
			if(u > thresProb) {
//				finish0 = clock64();
				//nonSkipTokenIdx = atomicAdd(&deviceNewTokenCount[wordId], 1);
				nonSkipTokenIdx = atomicAdd(&deviceNewTokenCount[wordId], 1);
//				finish1 = clock64();
//				costtime1 += (double)(finish1 - finish0);
//				finish1 = clock64();
				deviceEffectiveTokenIndex[nonSkipTokenIdx + tokenStart] = tokenIdx;
//				finish2 = clock64();
//				costtime2 += (double)(finish2 - finish1);
			}
			else {


				d_TopicIndex[tokenIdx] = maxK;
				atomicAdd(&d_WTDenseCopy[WTStart + maxK-1], 1);
				//sumPerplexity += 1.0;

			}



//			short int flag = (u > deviceThresProb[tokenIdx]);
//			short int warpNonZeroCount=__popc(__ballot(flag));
//			if (laneId==0) nonSkipTokenIdx = atomicAdd(&deviceNewTokenCount[wordId], warpNonZeroCount);
//			nonSkipTokenIdx=__shfl(nonSkipTokenIdx,0);
//			flag += __shfl_up(flag, 1, 32)*(laneId >= 1);
//			flag += __shfl_up(flag, 2, 32)*(laneId >= 2);
//			flag += __shfl_up(flag, 4, 32)*(laneId >= 4);
//			flag += __shfl_up(flag, 8, 32)*(laneId >= 8);
//			flag += __shfl_up(flag, 16, 32)*(laneId >= 16);
//			if (u > deviceThresProb[tokenIdx]) deviceEffectiveTokenIndex[nonSkipTokenIdx + tokenStart+flag-1] = tokenIdx;


			//__syncthreads();
		}
		//__syncthreads();

//		if (laneId == 0)
//		{
//			acquire_semaphore(&sem1);
//			tokenEndFlag = 0;
//			counter = d_blockCounter[0];
//			unsigned int numRegions = (d_TokenCount[counter] == 0) ? 0 : ((d_TokenCount[counter] - 1) / tokenSegment);
//			tokenRegionStart = atomicInc(&subCount1, numRegions);
//			if (subCount1 == 0) {
//				d_blockCounter[0] = d_blockCounter[0] + 1;
//				tokenEndFlag = 1;
//			}
//			release_semaphore(&sem1);
//		}
//
//		tokenRegionStart = __shfl(tokenRegionStart, 0);
//		tokenEndFlag = __shfl(tokenEndFlag, 0);
//		counter = __shfl(counter, 0);
//		finish2 = clock64();
//		if (threadIdx.x== 0)
//		{
//			acquire_semaphore(&sem1);
//			tokenEndFlag[0] = 0;
//			Counter[0] = d_blockCounter[0];
//			unsigned int numRegions = (d_TokenCount[Counter[0]] == 0) ? 0 : ((d_TokenCount[Counter[0]] - 1) / tokenSegment);
//			tokenRegionStart[0] = atomicInc(&subCount1, numRegions);
//			if (subCount1 == 0) {
//				d_blockCounter[0] = d_blockCounter[0] + 1;
//				tokenEndFlag[0] = 1;
//			}
//			release_semaphore(&sem1);
//		}
//		finish3 = clock64();
//		costtime3 += (double)(finish3 - finish2);
//	//
//	//	tokenRegionStart = __shfl(tokenRegionStart, 0);
//	//	tokenEndFlag = __shfl(tokenEndFlag, 0);
//	//
//	//	counter = __shfl(counter, 0);
//		__syncthreads();


//		finish2 = clock64();
//		if (threadIdx.x == 0) Counter[0] = atomicAdd(&d_blockCounter[0], 1);
		if (threadIdx.x == 0)
		{
//			deviceNewTokenCount[wordId]=newTokenCount[0];
//			newTokenCount[0]=0;
			Counter[0] = atomicAdd(&d_blockCounter[0], 1);
		}



		__syncthreads();
//		finish3 = clock64();
//		costtime3 += (double)(finish3 - finish2);

		/*if (threadIdx.x == 0) deviceNewTokenCount[wordId] = WarpCounter[0];
		__syncthreads();
*/
	}

//	if (threadIdx.x + blockDim.x*blockIdx.x == 0) printf("costtime0,costtime1,costtime2,costtime3:%f,%f,%f,%f\n", costtime0 / (158200000 * 1.0), costtime1 / (158200000 * 1.0),costtime2 / (158200000 * 1.0), costtime3 / (158200000 * 1.0));
	sumPerplexity += __shfl_down(sumPerplexity, 16);
	sumPerplexity += __shfl_down(sumPerplexity, 8);
	sumPerplexity += __shfl_down(sumPerplexity, 4);
	sumPerplexity += __shfl_down(sumPerplexity, 2);
	sumPerplexity += __shfl_down(sumPerplexity, 1);
	sumPerplexity = __shfl(sumPerplexity, 0);
	int laneId = threadIdx.x % 32;
	int localId = threadIdx.x / 32;
	if (localId == 0) {
		QTree[laneId] = 0;
	}
	__syncthreads();
	if (laneId == 0) QTree[localId] = sumPerplexity;
	__syncthreads();

	if (localId == 0) {
		float perplexity = 0.0;
		perplexity = QTree[laneId] * (laneId < blockDim.x / 32);
		perplexity += __shfl_down(perplexity, 16);
		perplexity += __shfl_down(perplexity, 8);
		perplexity += __shfl_down(perplexity, 4);
		perplexity += __shfl_down(perplexity, 2);
		perplexity += __shfl_down(perplexity, 1);
		if (laneId == 0) d_Perplexity[blockIdx.x] += perplexity;
	}

}





