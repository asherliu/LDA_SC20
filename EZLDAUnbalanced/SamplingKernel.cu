


#include "SamplingKernel.cuh"


#define gpuErr(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


void SampleKernelD(WTD &argWTDen, WTAll &argWT, DTChunk &argDT, Document &argDoc, hiprandState* randState, int argStreamId, hipStream_t& stream)
{

	////unsigned int blockCounter = 0;
	//unsigned int* deviceCounter;
	//hipMalloc(&deviceCounter, sizeof(unsigned int));
	hipMemsetAsync(argDoc.deviceCounterSampleKernelD[argStreamId], 0, sizeof(unsigned int), stream);

	/*hipMemcpyAsync(argDoc.deviceCounterSampleKernelD[argStreamId], &argDoc.counterSampleKernelD, sizeof(unsigned int), hipMemcpyHostToDevice, stream);*/



	// srand(time(NULL));

	// hiprandState* randState;
	// hipMalloc(&randState, sizeof(hiprandState)*GridDim*BlockDim);
	// H_ERR(hipDeviceSynchronize());
   
 //    gpuErr(hipPeekAtLastError());

	//initRandState << <GridDim, BlockDim, 0, stream >> >(randState);

	/*H_ERR(hipDeviceSynchronize());*/

	// for (int i = 0; i < iterWT; i++) {

	//hipMemcpy(deviceCounter, &blockCounter, sizeof(unsigned int), hipMemcpyHostToDevice);

	LDAKernelTrainD << <GridDim, BlockDim, 0, stream >> > (alpha, beta, argDoc.deviceMapWord2Doc[argStreamId], argDoc.deviceTLTopic[argStreamId], argDT.deviceNZDTCount[argStreamId], argDoc.deviceTLDocCount[argStreamId], argDoc.deviceTLDocOffset[argStreamId], argDT.deviceDTCount[argStreamId], argDT.deviceDTOffset[argStreamId], argWTDen.deviceWTDense, argWTDen.deviceWTDenseCopy, argDoc.deviceTLWordCount[argStreamId], argDoc.deviceTLWordOffset[argStreamId], argWT.deviceWTCount, argWT.deviceWTOffset, argWT.deviceWTRowSum, argDoc.deviceCounterSampleKernelD[argStreamId], argDoc.deviceMapDoc2Word[argStreamId], argDoc.docLengthVec[argDT.chunkId], argWT.wordLength, argDoc.devicePerplexityMid[argStreamId], randState, argDoc.deviceWTHeadDense[argStreamId], argWTDen.numOfWordD, argDoc.deviceWordMaxTopic[argStreamId], argDoc.deviceWordSecondMaxTopic[argStreamId], argDoc.deviceMaxProb[argStreamId], argDoc.deviceThresProb[argStreamId], argDoc.deviceTimeRecord[argStreamId], argDoc.tokenSegment, argDoc.deviceRandomfloat[argStreamId],  argDoc.deviceEffectiveTokenIndex[argStreamId], argDoc.deviceNewTokenCount[argStreamId], argDT.deviceDTIndexValue[argStreamId],argDoc.deviceMaxSecTopic[argStreamId]);

	
	/*H_ERR(hipDeviceSynchronize());*/

}
//(double alpha, double beta, int* d_Index, int* d_TopicIndex, int* d_SparseDTCount, int* d_SparseDTIndex, int* d_SparseDTValue, int* d_TokenCountDT, int* d_TokenOffsetDT, int* d_DocListCount, int* d_DocListOffset, int* d_WTDense, int* d_WTDenseCopy, int* d_TokenCount, int* d_TokenOffset, int* d_WordListCount, int* d_WordListOffset, int* d_WTRowSum, int* d_blockCounter, int*d_DocIndex, int D, int W, double* d_Perplexity, hiprandState *randState, double *WTHeadDense, int numOfWordD);


void SampleKernel(WTAll &argWT, DTChunk &argDT, Document &argDoc, hiprandState* randState, int argStreamId, hipStream_t& stream) {

	int numOfWordD = argWT.wordLength - argWT.numOfWordS;
	/*unsigned int* deviceCounter;
	hipMalloc(&deviceCounter, sizeof(unsigned int));
	hipMemset(deviceCounter, 0, sizeof(unsigned int));*/
	hipMemsetAsync(argDoc.deviceCounterSampleKernelS[argStreamId], 0, sizeof(unsigned int), stream);

	/*hipMemcpyAsync(argDoc.deviceCounterSampleKernelS[argStreamId], &argDoc.counterSampleKernelS, sizeof(unsigned int), hipMemcpyHostToDevice, stream);*/

	//initRandState << <GridDim, BlockDim, 0, stream>> >(randState);
	/*H_ERR(hipDeviceSynchronize());*/

	LDAKernelTrain << <GridDim, BlockDim, 0, stream >> > (alpha, beta, argDoc.deviceMapWord2Doc[argStreamId], argDoc.deviceTLTopic[argStreamId], argDT.deviceNZDTCount[argStreamId], argDoc.deviceTLDocCount[argStreamId], argDoc.deviceTLDocOffset[argStreamId], argDT.deviceDTCount[argStreamId], argDT.deviceDTOffset[argStreamId], argWT.deviceNZWTCount, argWT.deviceWTIndex, argWT.deviceWTValue, argDoc.deviceTLWordCount[argStreamId], argDoc.deviceTLWordOffset[argStreamId], argWT.deviceWTCount, argWT.deviceWTOffset, argWT.deviceWTRowSum, argDoc.deviceCounterSampleKernelS[argStreamId], argDoc.deviceMapDoc2Word[argStreamId], argDoc.docLengthVec[argDT.chunkId], argWT.wordLength, argDoc.devicePerplexityMid[argStreamId], randState, argDoc.deviceWTHeadDense[argStreamId], numOfWordD, argWT.numOfWordS, argDoc.deviceWordMaxTopic[argStreamId], argDoc.deviceWordSecondMaxTopic[argStreamId], argDoc.deviceMaxSecTopic[argStreamId], argDT.deviceDTIndexValue[argStreamId]);

	//H_ERR(hipDeviceSynchronize());

	


}

void MaxTopicKernel(WTAll &argWT, Document &argDoc, WTD &argWTDen, int argStreamId, hipStream_t& stream) {

	int numOfWordD = argWT.wordLength - argWT.numOfWordS;
	/*unsigned int* deviceCounter;
	hipMalloc(&deviceCounter, sizeof(unsigned int));*/
	hipMemsetAsync(argDoc.deviceCounterMaxTopicKernel[argStreamId], 0, sizeof(unsigned int),stream);
	/*hipMemcpyAsync(argDoc.deviceCounterMaxTopicKernel[argStreamId], &argDoc.counterMaxTopicKernel, sizeof(unsigned int), hipMemcpyHostToDevice, stream);*/
	/*hipMemcpyAsync(argDoc.deviceCounterMaxTopicKernel[argStreamId], &argDoc.counterMaxTopicKernel, sizeof(unsigned int), hipMemcpyHostToDevice, stream);*/

	MaxTopicDense_Update_Kernel << <GridDim, BlockDim, 0, stream >> >(argDoc.deviceWordMaxTopic[argStreamId], argDoc.deviceWordSecondMaxTopic[argStreamId],  argWTDen.deviceWTDense, argDoc.deviceTLWordCount[argStreamId], argDoc.deviceTLWordOffset[argStreamId], argWT.deviceWTOffset, numOfWordD, argDoc.deviceCounterMaxTopicKernel[argStreamId], argWT.deviceWTRowSum, argWT.wordLength, beta, argDoc.deviceWordThirdMaxTopic[argStreamId],  argDoc.deviceMaxSecTopic[argStreamId], argDoc.deviceQArray[argStreamId], argDoc.deviceWordMaxProb[argStreamId], argDoc.deviceWordSecondMaxProb[argStreamId], argDoc.deviceWordThirdMaxProb[argStreamId]);
	//H_ERR(hipDeviceSynchronize());
	/*hipMemcpyAsync(argDoc.deviceCounterMaxTopicKernel[argStreamId], &argDoc.counterMaxTopicKernel, sizeof(unsigned int), hipMemcpyHostToDevice, stream);*/

	hipMemsetAsync(argDoc.deviceCounterMaxTopicKernel[argStreamId], 0, sizeof(unsigned int),stream);

	MaxTopicSparse_Update_Kernel << <GridDim, BlockDim, 0, stream>> >(argDoc.deviceWordMaxTopic[argStreamId], argDoc.deviceWordSecondMaxTopic[argStreamId],  argDoc.deviceTLWordCount[argStreamId], argDoc.deviceTLWordOffset[argStreamId], argWT.deviceWTOffset, numOfWordD, argDoc.deviceCounterMaxTopicKernel[argStreamId], argWT.deviceWTRowSum, argWT.wordLength, argWT.numOfWordS, argWT.deviceWTOffset, argWT.deviceNZWTCount, argWT.deviceWTIndex, argWT.deviceWTValue, beta, argDoc.deviceWordThirdMaxTopic[argStreamId], argDoc.deviceMaxSecTopic[argStreamId], argDoc.deviceQArray[argStreamId], argDoc.deviceWordMaxProb[argStreamId], argDoc.deviceWordSecondMaxProb[argStreamId], argDoc.deviceWordThirdMaxProb[argStreamId]);
	/*H_ERR(hipDeviceSynchronize());*/

}


//void UpdateProbKernelD(WTD &argWTDen, WTAll &argWT, DTChunk &argDT, Document &argDoc, hiprandState* randState, int argStreamId, hipStream_t& stream)
//{
//
//	//unsigned int blockCounter = 0;
//	//unsigned int* deviceCounter;
//	//hipMalloc(&deviceCounter, sizeof(unsigned int));
//	hipMemsetAsync(argDoc.deviceCounterUpdateProbKernel[argStreamId], 0, sizeof(unsigned int),stream);
//	/*hipMemcpyAsync(argDoc.deviceCounterUpdateProbKernel[argStreamId], &argDoc.counterUpdateProbKernel, sizeof(unsigned int), hipMemcpyHostToDevice, stream);*/
//
//	initRandState << <GridDim, BlockDim, 0, stream >> >(randState);
//	/*H_ERR(hipDeviceSynchronize());*/
//
//	// for (int i = 0; i < iterWT; i++) {
//
//	//hipMemcpy(deviceCounter, &blockCounter, sizeof(unsigned int), hipMemcpyHostToDevice);
//
//	UpdateProbKernelTrainD << <256, 256, 0, stream >> > (alpha, beta, argDoc.deviceMapWord2Doc[argStreamId], argDoc.deviceTLTopic[argStreamId], argDT.deviceNZDTCount[argStreamId],  argDoc.deviceTLDocCount[argStreamId], argDoc.deviceTLDocOffset[argStreamId], argDT.deviceDTCount[argStreamId], argDT.deviceDTOffset[argStreamId], argWTDen.deviceWTDense, argWTDen.deviceWTDenseCopy, argDoc.deviceTLWordCount[argStreamId], argDoc.deviceTLWordOffset[argStreamId], argWT.deviceWTCount, argWT.deviceWTOffset, argWT.deviceWTRowSum, argDoc.deviceCounterUpdateProbKernel[argStreamId], argDoc.deviceMapDoc2Word[argStreamId], argDoc.docLengthVec[argDT.chunkId], argWT.wordLength, argDoc.devicePerplexityMid[argStreamId], randState, argDoc.deviceWTHeadDense[argStreamId], argWTDen.numOfWordD, argDoc.deviceWordMaxTopic[argStreamId], argDoc.deviceWordSecondMaxTopic[argStreamId], argDoc.deviceMaxProb[argStreamId], argDoc.deviceThresProb[argStreamId], argDoc.deviceWordThirdMaxTopic[argStreamId], argDoc.deviceRandomfloat[argStreamId], argDoc.deviceEffectiveTokenIndex[argStreamId], argDoc.deviceNewTokenCount[argStreamId], argDoc.deviceMaxSecTopic[argStreamId], argDoc.deviceQArray[argStreamId], argDoc.deviceWordMaxProb[argStreamId], argDoc.deviceWordSecondMaxProb[argStreamId], argDoc.deviceWordThirdMaxProb[argStreamId], argDoc.tokenSegment);
//
//	/*H_ERR(hipDeviceSynchronize());
//*/
//}


void UpdateProbKernelD(WTD &argWTDen, WTAll &argWT, DTChunk &argDT, Document &argDoc, hiprandState* randState, int argStreamId, hipStream_t& stream)
{
	//hipMemsetAsync(argDoc.deviceCounterUpdateProbKernel[argStreamId], 0, sizeof(unsigned int),stream);
	/*hipMemcpyAsync(argDoc.deviceCounterUpdateProbKernel[argStreamId], &argDoc.counterUpdateProbKernel, sizeof(unsigned int), hipMemcpyHostToDevice, stream);*/


	/*H_ERR(hipDeviceSynchronize());*/

	// for (int i = 0; i < iterWT; i++) {

	//hipMemcpy(deviceCounter, &blockCounter, sizeof(unsigned int), hipMemcpyHostToDevice);

	//UpdateProbKernelTrainD0 << <256, 256, 0, stream >> > (alpha, beta, argDoc.deviceMapWord2Doc[argStreamId], argDoc.deviceTLTopic[argStreamId], argDT.deviceNZDTCount[argStreamId],  argDoc.deviceTLDocCount[argStreamId], argDoc.deviceTLDocOffset[argStreamId], argDT.deviceDTCount[argStreamId], argDT.deviceDTOffset[argStreamId], argWTDen.deviceWTDense, argWTDen.deviceWTDenseCopy, argDoc.deviceTLWordCount[argStreamId], argDoc.deviceTLWordOffset[argStreamId], argWT.deviceWTCount, argWT.deviceWTOffset, argWT.deviceWTRowSum, argDoc.deviceCounterUpdateProbKernel[argStreamId], argDoc.deviceMapDoc2Word[argStreamId], argDoc.docLengthVec[argDT.chunkId], argWT.wordLength, argDoc.devicePerplexityMid[argStreamId], randState, argDoc.deviceWTHeadDense[argStreamId], argWTDen.numOfWordD, argDoc.deviceWordMaxTopic[argStreamId], argDoc.deviceWordSecondMaxTopic[argStreamId], argDoc.deviceMaxProb[argStreamId], argDoc.deviceThresProb[argStreamId], argDoc.deviceWordThirdMaxTopic[argStreamId], argDoc.deviceRandomfloat[argStreamId], argDoc.deviceEffectiveTokenIndex[argStreamId], argDoc.deviceNewTokenCount[argStreamId], argDoc.deviceMaxSecTopic[argStreamId], argDoc.deviceQArray[argStreamId], argDoc.deviceWordMaxProb[argStreamId], argDoc.deviceWordSecondMaxProb[argStreamId], argDoc.deviceWordThirdMaxProb[argStreamId], argDoc.tokenSegment, argDoc.deviceTotalTokenCount[argStreamId]);


	hipMemsetAsync(argDoc.deviceCounterUpdateProbKernel[argStreamId], 0, sizeof(unsigned int),stream);
	/*hipMemcpyAsync(argDoc.deviceCounterUpdateProbKernel[argStreamId], &argDoc.counterUpdateProbKernel, sizeof(unsigned int), hipMemcpyHostToDevice, stream);*/

	initRandState << <GridDim, BlockDim, 0, stream >> >(randState);
	/*H_ERR(hipDeviceSynchronize());*/

	// for (int i = 0; i < iterWT; i++) {

	//hipMemcpy(deviceCounter, &blockCounter, sizeof(unsigned int), hipMemcpyHostToDevice);

	UpdateProbKernelTrainD1 << <256, 256, 0, stream >> > (alpha, beta, argDoc.deviceMapWord2Doc[argStreamId], argDoc.deviceTLTopic[argStreamId], argDT.deviceNZDTCount[argStreamId],  argDoc.deviceTLDocCount[argStreamId], argDoc.deviceTLDocOffset[argStreamId], argDT.deviceDTCount[argStreamId], argDT.deviceDTOffset[argStreamId], argWTDen.deviceWTDense, argWTDen.deviceWTDenseCopy, argDoc.deviceTLWordCount[argStreamId], argDoc.deviceTLWordOffset[argStreamId], argWT.deviceWTCount, argWT.deviceWTOffset, argWT.deviceWTRowSum, argDoc.deviceCounterUpdateProbKernel[argStreamId], argDoc.deviceMapDoc2Word[argStreamId], argDoc.docLengthVec[argDT.chunkId], argWT.wordLength, argDoc.devicePerplexityMid[argStreamId], randState, argDoc.deviceWTHeadDense[argStreamId], argWTDen.numOfWordD, argDoc.deviceWordMaxTopic[argStreamId], argDoc.deviceWordSecondMaxTopic[argStreamId], argDoc.deviceMaxProb[argStreamId], argDoc.deviceThresProb[argStreamId], argDoc.deviceWordThirdMaxTopic[argStreamId], argDoc.deviceRandomfloat[argStreamId], argDoc.deviceEffectiveTokenIndex[argStreamId], argDoc.deviceNewTokenCount[argStreamId], argDoc.deviceMaxSecTopic[argStreamId], argDoc.deviceQArray[argStreamId], argDoc.deviceWordMaxProb[argStreamId], argDoc.deviceWordSecondMaxProb[argStreamId], argDoc.deviceWordThirdMaxProb[argStreamId], argDoc.tokenSegment);

	/*H_ERR(hipDeviceSynchronize());
	 *
*/
	//unsigned int blockCounter = 0;
	//unsigned int* deviceCounter;
	//hipMalloc(&deviceCounter, sizeof(unsigned int));
	//hipMemsetAsync(argDoc.deviceCounterUpdateProbKernel[argStreamId], 0, sizeof(unsigned int),stream);
	/*hipMemcpyAsync(argDoc.deviceCounterUpdateProbKernel[argStreamId], &argDoc.counterUpdateProbKernel, sizeof(unsigned int), hipMemcpyHostToDevice, stream);*/

	//initRandState << <GridDim, BlockDim, 0, stream >> >(randState);
	/*H_ERR(hipDeviceSynchronize());*/

	// for (int i = 0; i < iterWT; i++) {

	//hipMemcpy(deviceCounter, &blockCounter, sizeof(unsigned int), hipMemcpyHostToDevice);

	//UpdateProbKernelTrainD2 << <256, 256, 0, stream >> > (alpha, beta, argDoc.deviceMapWord2Doc[argStreamId], argDoc.deviceTLTopic[argStreamId], argDT.deviceNZDTCount[argStreamId],  argDoc.deviceTLDocCount[argStreamId], argDoc.deviceTLDocOffset[argStreamId], argDT.deviceDTCount[argStreamId], argDT.deviceDTOffset[argStreamId], argWTDen.deviceWTDense, argWTDen.deviceWTDenseCopy, argDoc.deviceTLWordCount[argStreamId], argDoc.deviceTLWordOffset[argStreamId], argWT.deviceWTCount, argWT.deviceWTOffset, argWT.deviceWTRowSum, argDoc.deviceCounterUpdateProbKernel[argStreamId], argDoc.deviceMapDoc2Word[argStreamId], argDoc.docLengthVec[argDT.chunkId], argWT.wordLength, argDoc.devicePerplexityMid[argStreamId], randState, argDoc.deviceWTHeadDense[argStreamId], argWTDen.numOfWordD, argDoc.deviceWordMaxTopic[argStreamId], argDoc.deviceWordSecondMaxTopic[argStreamId], argDoc.deviceMaxProb[argStreamId], argDoc.deviceThresProb[argStreamId], argDoc.deviceWordThirdMaxTopic[argStreamId], argDoc.deviceRandomfloat[argStreamId], argDoc.deviceEffectiveTokenIndex[argStreamId], argDoc.deviceNewTokenCount[argStreamId], argDoc.deviceMaxSecTopic[argStreamId], argDoc.deviceQArray[argStreamId], argDoc.deviceWordMaxProb[argStreamId], argDoc.deviceWordSecondMaxProb[argStreamId], argDoc.deviceWordThirdMaxProb[argStreamId], argDoc.tokenSegment);

	/*H_ERR(hipDeviceSynchronize());*/
}



void PerplexityKernel(Document &argDoc, int argStreamId, hipStream_t& stream) {


	LDATrainPerplexityReduce << <1, BlockDim, 0, stream >> > (argDoc.devicePerplexityMid[argStreamId], argDoc.totalNumOfTokens, argDoc.devicePerplexityAve[argStreamId]);

	hipMemcpyAsync(argDoc.perplexityAve, argDoc.devicePerplexityAve[argStreamId],sizeof(float), hipMemcpyDeviceToHost, stream);

	/*H_ERR(hipDeviceSynchronize());*/
}













//
//void SampleKernel(WTAll &argWT, DTChunk &argDT, Document &argDoc, hiprandState* randState) {
//
//	int blockCounter = 0;
//	int iterWT = (argWT.numOfWordS - 1) / GridDim + 1;
//	float Perplexity = 0.0;
//	int numOfWordD = argWT.wordLength - argWT.numOfWordS;
//	// srand(time(NULL));
//
//	// hiprandState* randState;
//	// hipMalloc(&randState, sizeof(hiprandState)*GridDim*BlockDim);
//	// H_ERR(hipDeviceSynchronize());
//	//    gpuErr(hipPeekAtLastError());
//
//	initRandState << <GridDim, BlockDim >> >(randState);
//	H_ERR(hipDeviceSynchronize());
//
//	for (int i = 0; i < iterWT; i++) {
//
//		hipMemcpy(argDoc.d_blockCounter, &blockCounter, sizeof(int), hipMemcpyHostToDevice);
//
//		LDAKernelTrain << <GridDim, BlockDim >> > (alpha, beta, argDoc.deviceMapWord2Doc, argDoc.deviceTLTopic, argDT.deviceNZDTCount, argDT.deviceDTIndex, argDT.deviceDTValue, argDoc.deviceTLDocCount, argDoc.deviceTLDocOffset, argDT.deviceDTCount, argDT.deviceDTOffset, argWT.deviceNZWTCount, argWT.deviceWTIndex, argWT.deviceWTValue, argDoc.deviceTLWordCount, argDoc.deviceTLWordOffset, argWT.deviceWTCount, argWT.deviceWTOffset, argWT.deviceWTRowSum, argDoc.d_blockCounter, argDoc.deviceMapDoc2Word, argDoc.docLengthVec[argDT.chunkId], argWT.wordLength, argDoc.devicePerplexity, randState, argDoc.deviceWTHeadDense, numOfWordD, argWT.numOfWordS);
//		H_ERR(hipDeviceSynchronize());
//		blockCounter++;
//
//	}
//	LDATrainPerplexityReduce1 << <GridDim, BlockDim >> > (argDoc.devicePerplexity, argDoc.devicePerplexityMid, argDoc.TLLengthVec[argDT.chunkId]);
//
//	H_ERR(hipDeviceSynchronize());
//
//
//}
//
//
//
















