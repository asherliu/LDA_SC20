
#include "DTUpdateKernel.cuh"
void UpdateDTKernel(DTChunk &argDT,Document &argDoc) {


	unsigned int* deviceCounter;
	hipMalloc(&deviceCounter, sizeof(unsigned int));
	hipMemset(deviceCounter, 0, sizeof(unsigned int));

	DT_Update_Kernel << <GridDim, BlockDim >> > (argDoc.deviceMapWord2Doc, argDoc.deviceTLTopic,  argDT.deviceNZDTCount, argDT.deviceDTIndex, argDT.deviceDTValue, argDoc.deviceTLDocCount, argDoc.deviceTLDocOffset, argDT.deviceDTCount, argDT.deviceDTOffset, deviceCounter, argDT.docLengthVec[argDT.chunkId], argDoc.d_dense);
	H_ERR(hipDeviceSynchronize());
	
}

