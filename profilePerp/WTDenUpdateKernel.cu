#include "WTDenUpdateKernel.cuh"
void UpdateWTDenKernel(WTD &argWTDen, WTAll &argWT, Document &argDoc, int argChunkId) {



	/*int numOfTokenD = argWTDen.numOfWordD;*/

		unsigned int* deviceCounter;
		hipMalloc(&deviceCounter, sizeof(unsigned int));
		hipMemset(deviceCounter, 0, sizeof(unsigned int));

		WTDen_Update_Kernel << <GridDim, BlockDim >> >(argDoc.deviceTLTopic, argWTDen.deviceWTDense, argDoc.deviceTLWordCount, argDoc.deviceTLWordOffset, argWT.deviceWTOffset, argWTDen.numOfWordD, deviceCounter);

		H_ERR(hipDeviceSynchronize());
		


}

void UpdateWTDenRowSumKernel(WTD &argWTDen, WTAll &argWT)
{
	WTDen_Sum_Update_Kernel << <GridDim, BlockDim >> >(argWTDen.deviceWTDense, argWT.deviceWTRowSum, argWT.deviceWTOffset, argWTDen.numOfWordD);
	H_ERR(hipDeviceSynchronize());
}



