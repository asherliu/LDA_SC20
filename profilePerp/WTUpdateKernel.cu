#include "WTUpdateKernel.cuh"
void UpdateWTKernel(WTAll &argWT, Document &argDoc, int argChunkId) {

	
	unsigned int* deviceCounter;
	hipMalloc(&deviceCounter, sizeof(unsigned int));
	hipMemset(deviceCounter, 0, sizeof(unsigned int));

	int numOfTokenD = argDoc.numOfTokenVecD[argChunkId];
	int numOfWordS = argWT.blockCount + argWT.warpCount;

	WT_Update_Kernel << <GridDim, BlockDim >> > (argDoc.deviceTLTopic, argWT.deviceChunkNZWTCount, argWT.deviceChunkWTIndex, argWT.deviceChunkWTValue, argWT.deviceChunkWTCount, argWT.deviceChunkWTOffset, argWT.deviceWTRowSum, deviceCounter, numOfWordS, argDoc.d_dense, numOfTokenD);

	H_ERR(hipDeviceSynchronize());
	
}


//
//
//void UpdateWTKernel(WTAll &argWT, Document &argDoc, int argChunkId) {
//
//	int iterBlock = (argWT.blockCount - 1) / GridDim + 1;// number of iterations for block.
//														 //int iterBlock = 9;// number of iterations for block.
//	int GridWarpDim = GridDim*BlockDim / 32;
//	int iterAll = (argWT.blockCount - 1) / GridDim + 1 + (argWT.warpCount - 1) / GridWarpDim + 1; // number of total iterations.
//
//	int blockCounter = 0;
//	int warpCounter = 0;
//	int numOfTokenD = argDoc.numOfTokenVecD[argChunkId];
//	for (int i = 0; i < iterAll; i++)
//	{
//		if (i < iterBlock)
//		{
//			H_ERR(hipMemcpy(argDoc.d_blockCounter, &blockCounter, sizeof(int), hipMemcpyHostToDevice));
//			tokenlist_to_matrix << <GridDim, BlockDim >> > (argDoc.deviceTLTopic, argWT.deviceChunkNZWTCount, argWT.deviceChunkWTIndex, argWT.deviceChunkWTValue, argWT.deviceChunkWTCount, argWT.deviceChunkWTOffset, argWT.deviceWTRowSum, argDoc.d_blockCounter, argWT.deviceBlockCount, argDoc.d_dense, numOfTokenD);
//			H_ERR(hipDeviceSynchronize());
//			blockCounter++;
//
//
//		}
//		else
//		{
//			hipMemcpy(argDoc.d_warpCounter, &warpCounter, sizeof(int), hipMemcpyHostToDevice);
//			tokenlist_to_matrix_warp << <GridDim, BlockDim >> > (argDoc.deviceTLTopic, argWT.deviceChunkNZWTCount, argWT.deviceChunkWTIndex, argWT.deviceChunkWTValue, argWT.deviceChunkWTCount, argWT.deviceChunkWTOffset, argWT.deviceWTRowSum, argDoc.d_warpCounter, argWT.deviceBlockCount, argWT.deviceWarpCount, numOfTokenD);
//			/*printf("abc %d", warpCounter);*/
//			H_ERR(hipDeviceSynchronize());
//			warpCounter++;
//		}
//		H_ERR(hipDeviceSynchronize());
//
//
//	}
//
//}
//




