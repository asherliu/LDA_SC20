


#include "SamplingKernel.cuh"


#define gpuErr(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


void SampleKernelD(WTD &argWTDen, WTAll &argWT, DTChunk &argDT, Document &argDoc, hiprandState* randState)
{

	//unsigned int blockCounter = 0;
	unsigned int* deviceCounter;
	hipMalloc(&deviceCounter, sizeof(unsigned int));
	hipMemset(deviceCounter, 0, sizeof(unsigned int));
	// srand(time(NULL));

	// hiprandState* randState;
	// hipMalloc(&randState, sizeof(hiprandState)*GridDim*BlockDim);
	// H_ERR(hipDeviceSynchronize());
   
 //    gpuErr(hipPeekAtLastError());

	initRandState << <GridDim, BlockDim >> >(randState);
	H_ERR(hipDeviceSynchronize());

	// for (int i = 0; i < iterWT; i++) {

	//hipMemcpy(deviceCounter, &blockCounter, sizeof(unsigned int), hipMemcpyHostToDevice);

	LDAKernelTrainD << <GridDim, BlockDim >> > (alpha, beta, argDoc.deviceMapWord2Doc, argDoc.deviceTLTopic, argDT.deviceNZDTCount, argDT.deviceDTIndex, argDT.deviceDTValue, argDoc.deviceTLDocCount, argDoc.deviceTLDocOffset, argDT.deviceDTCount, argDT.deviceDTOffset, argWTDen.deviceWTDense, argWTDen.deviceWTDenseCopy, argDoc.deviceTLWordCount, argDoc.deviceTLWordOffset, argWT.deviceWTCount, argWT.deviceWTOffset, argWT.deviceWTRowSum, deviceCounter, argDoc.deviceMapDoc2Word, argDoc.docLengthVec[argDT.chunkId], argWT.wordLength, argDoc.devicePerplexityMid, randState, argDoc.deviceWTHeadDense, argWTDen.numOfWordD, argDoc.tokenSegment);

	
	H_ERR(hipDeviceSynchronize());

}
//(double alpha, double beta, int* d_Index, int* d_TopicIndex, int* d_SparseDTCount, int* d_SparseDTIndex, int* d_SparseDTValue, int* d_TokenCountDT, int* d_TokenOffsetDT, int* d_DocListCount, int* d_DocListOffset, int* d_WTDense, int* d_WTDenseCopy, int* d_TokenCount, int* d_TokenOffset, int* d_WordListCount, int* d_WordListOffset, int* d_WTRowSum, int* d_blockCounter, int*d_DocIndex, int D, int W, double* d_Perplexity, hiprandState *randState, double *WTHeadDense, int numOfWordD);


void SampleKernel(WTAll &argWT, DTChunk &argDT, Document &argDoc, hiprandState* randState) {

	int numOfWordD = argWT.wordLength - argWT.numOfWordS;
	unsigned int* deviceCounter;
	hipMalloc(&deviceCounter, sizeof(unsigned int));
	hipMemset(deviceCounter, 0, sizeof(unsigned int));

	initRandState << <GridDim, BlockDim >> >(randState);
	H_ERR(hipDeviceSynchronize());

	LDAKernelTrain << <GridDim, BlockDim >> > (alpha, beta, argDoc.deviceMapWord2Doc, argDoc.deviceTLTopic, argDT.deviceNZDTCount, argDT.deviceDTIndex, argDT.deviceDTValue, argDoc.deviceTLDocCount, argDoc.deviceTLDocOffset, argDT.deviceDTCount, argDT.deviceDTOffset, argWT.deviceNZWTCount, argWT.deviceWTIndex, argWT.deviceWTValue, argDoc.deviceTLWordCount, argDoc.deviceTLWordOffset, argWT.deviceWTCount, argWT.deviceWTOffset, argWT.deviceWTRowSum, deviceCounter, argDoc.deviceMapDoc2Word, argDoc.docLengthVec[argDT.chunkId], argWT.wordLength, argDoc.devicePerplexityMid, randState, argDoc.deviceWTHeadDense, numOfWordD, argWT.numOfWordS);

	H_ERR(hipDeviceSynchronize());

	


}



void PerplexityKernel(Document &argDoc) {

	float* sumPerplexity;

	hipMalloc(&sumPerplexity, sizeof(float));
	LDATrainPerplexityReduce << <1, BlockDim >> > (argDoc.devicePerplexityMid, argDoc.totalNumOfTokens, sumPerplexity);
	hipMemcpy(&argDoc.sumPerplexity, sumPerplexity, sizeof(float), hipMemcpyDeviceToHost);
	H_ERR(hipDeviceSynchronize());
}













//
//void SampleKernel(WTAll &argWT, DTChunk &argDT, Document &argDoc, hiprandState* randState) {
//
//	int blockCounter = 0;
//	int iterWT = (argWT.numOfWordS - 1) / GridDim + 1;
//	float Perplexity = 0.0;
//	int numOfWordD = argWT.wordLength - argWT.numOfWordS;
//	// srand(time(NULL));
//
//	// hiprandState* randState;
//	// hipMalloc(&randState, sizeof(hiprandState)*GridDim*BlockDim);
//	// H_ERR(hipDeviceSynchronize());
//	//    gpuErr(hipPeekAtLastError());
//
//	initRandState << <GridDim, BlockDim >> >(randState);
//	H_ERR(hipDeviceSynchronize());
//
//	for (int i = 0; i < iterWT; i++) {
//
//		hipMemcpy(argDoc.d_blockCounter, &blockCounter, sizeof(int), hipMemcpyHostToDevice);
//
//		LDAKernelTrain << <GridDim, BlockDim >> > (alpha, beta, argDoc.deviceMapWord2Doc, argDoc.deviceTLTopic, argDT.deviceNZDTCount, argDT.deviceDTIndex, argDT.deviceDTValue, argDoc.deviceTLDocCount, argDoc.deviceTLDocOffset, argDT.deviceDTCount, argDT.deviceDTOffset, argWT.deviceNZWTCount, argWT.deviceWTIndex, argWT.deviceWTValue, argDoc.deviceTLWordCount, argDoc.deviceTLWordOffset, argWT.deviceWTCount, argWT.deviceWTOffset, argWT.deviceWTRowSum, argDoc.d_blockCounter, argDoc.deviceMapDoc2Word, argDoc.docLengthVec[argDT.chunkId], argWT.wordLength, argDoc.devicePerplexity, randState, argDoc.deviceWTHeadDense, numOfWordD, argWT.numOfWordS);
//		H_ERR(hipDeviceSynchronize());
//		blockCounter++;
//
//	}
//	LDATrainPerplexityReduce1 << <GridDim, BlockDim >> > (argDoc.devicePerplexity, argDoc.devicePerplexityMid, argDoc.TLLengthVec[argDT.chunkId]);
//
//	H_ERR(hipDeviceSynchronize());
//
//
//}
//
//
//
















