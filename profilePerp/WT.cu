#include "hip/hip_runtime.h"

#include "WT.cuh"
WTAll::WTAll(int argmaxWTLength, int argWordLength, int argNumChunks, int argMaxChunkWTLength, int argNumOfWordS) {
	maxWTLength = argmaxWTLength;
	wordLength = argWordLength;
	numChunks = argNumChunks;
	maxChunkWTLength = argMaxChunkWTLength;
	numOfWordS = argNumOfWordS;
	WTLengthVec = new int[numChunks];
	WTRowSum = new int[K];

	NZWTCount = new int[numOfWordS];
	WTIndex = new unsigned short int[maxWTLength];
	WTValue = new unsigned short int[maxWTLength];
	WTCount = new int[wordLength];
	WTOffset = new int[wordLength];
	


	////-----chunkWT-----for test--------
	//chunkNZWTCount = new int[wordLength];
	//chunkWTIndex = new int[maxChunkWTLength];
	//chunkWTValue = new int[maxChunkWTLength];
	////-----chunkWT-----for test--------


}




void WTAll::CPUMemSet() {

	memset(NZWTCount, 0, numOfWordS * sizeof(int));
	memset(WTIndex, 0, maxWTLength * sizeof(unsigned short int));
	memset(WTValue, 0, maxWTLength * sizeof(unsigned short int));
	memset(WTCount, 0, wordLength * sizeof(int));
	memset(WTOffset, 0, wordLength * sizeof(int));
	memset(WTRowSum, 0, K * sizeof(int));

}
void WTAll::GPUMemAllocate() {

	hipMalloc((void**)&deviceNZWTCount, (numOfWordS) * sizeof(int));
	hipMalloc((void**)&deviceWTIndex, (maxWTLength) * sizeof(unsigned short int));
	hipMalloc((void**)&deviceWTValue, (maxWTLength) * sizeof(unsigned short int));
	hipMalloc((void**)&deviceWTCount, (wordLength) * sizeof(int));
	hipMalloc((void**)&deviceWTOffset, (wordLength) * sizeof(int));

	hipMalloc((void**)&deviceWTRowSum, (K) * sizeof(int));
	hipMalloc((void**)&deviceBlockCount, (1) * sizeof(int));
	hipMalloc((void**)&deviceWarpCount, (1) * sizeof(int));

	hipMalloc((void**)&deviceChunkWTCount, (numOfWordS) * sizeof(int));
	hipMalloc((void**)&deviceChunkWTOffset, (numOfWordS) * sizeof(int));
	hipMalloc((void**)&deviceChunkNZWTCount, (numOfWordS) * sizeof(int));
	hipMalloc((void**)&deviceChunkWTIndex, (maxChunkWTLength) * sizeof(unsigned short int));
	hipMalloc((void**)&deviceChunkWTValue, (maxChunkWTLength) * sizeof(unsigned short int));


	WTMemory = (6 * wordLength + 2 * maxWTLength + K + 2 * maxChunkWTLength) /1000000000.0 * sizeof(int);
	printf("WT memory usage(Sparse):%f GB\n", WTMemory);
	WTMemory = K /1000000000.0 * wordLength * sizeof(int);
	printf("WT memory usage(Dense):%f GB\n", WTMemory);

}
void WTAll::GPUMemset()
{
	hipMemset(deviceNZWTCount, 0, (numOfWordS) * sizeof(int));
	hipMemset(deviceWTIndex, 0, (maxWTLength) * sizeof(unsigned short int));
	hipMemset(deviceWTValue, 0, (maxWTLength) * sizeof(unsigned short int));
	hipMemset(deviceWTRowSum, 0, (K) * sizeof(int));

}

void WTAll::chunkGPUMemset()
{
	hipMemset(deviceChunkNZWTCount, 0, (numOfWordS) * sizeof(int));
	hipMemset(deviceChunkWTIndex, 0, (maxChunkWTLength) * sizeof(unsigned short int));
	hipMemset(deviceChunkWTValue, 0, (maxChunkWTLength) * sizeof(unsigned short int));
	//hipMemset(deviceWTRowSum, 0, (K) * sizeof(int));

}


void WTAll::loadWTLength(string argFilePrefix) {

	ifstream WTLength((argFilePrefix + string("/WTLength.txt")).c_str(), ios::binary);//store max Doc and DT length	
	for (int chunkId = 0; chunkId < numChunks; chunkId++) {
		WTLength >> WTLengthVec[chunkId];

	}
	WTLength.close();
}

void WTAll::loadWTCountOffset(string argFilePrefix) {

	//--------load chunkWTCountOffset--------------
	for (int chunkId = 0; chunkId < numChunks; chunkId++) {

		WTChunkData chunkWTData(chunkId, wordLength, maxChunkWTLength, WTLengthVec[chunkId], numOfWordS);
		chunkWTData.CPUMemSet();
		chunkWTData.loadWTCountOffset(argFilePrefix);
		WTChunkVec.push_back(chunkWTData);
	}
	//--------load chunkWTCountOffset--------------



	//--------load WTCountOffset--------------

	ifstream WTCountOffset((argFilePrefix + string("/WTCountOffset.txt")).c_str(), ios::binary);//store Word offset of TL
	blockCount = 0;
	for (int i = 0; i < wordLength; i++)
	{
		WTCountOffset >> WTCount[i] >> WTOffset[i];

		if (i >= wordLength - numOfWordS) {
			if (WTCount[i] > 32) {
				blockCount++;
			}
		}
		
	}
	WTCountOffset.close();
	warpCount = numOfWordS - blockCount;
	printf("WT Count and Offset loaded!...\n");

	//--------load WTCountOffset--------------

}


void WTAll::blockWarpCountCPU2GPU() {

	hipMemcpy(deviceBlockCount, &blockCount, (1) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(deviceWarpCount, &warpCount, (1) * sizeof(int), hipMemcpyHostToDevice);

}

void WTAll::CPU2GPUCountOffset() {

	hipMemcpy(deviceWTCount, WTCount, (wordLength) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(deviceWTOffset, WTOffset, (wordLength) * sizeof(int), hipMemcpyHostToDevice);

}

void WTAll::WTCPU2GPU() {

	hipMemcpy(deviceNZWTCount, NZWTCount, (numOfWordS) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(deviceWTIndex, WTIndex, (maxWTLength) * sizeof(unsigned short int), hipMemcpyHostToDevice);
	hipMemcpy(deviceWTValue, WTValue, (maxWTLength) * sizeof(unsigned short int), hipMemcpyHostToDevice);
	
}

void WTAll::WTGPU2CPU() {

	hipMemcpy(NZWTCount, deviceNZWTCount, (numOfWordS) * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(WTIndex, deviceWTIndex, (maxWTLength) * sizeof(unsigned short int), hipMemcpyDeviceToHost);
	hipMemcpy(WTValue, deviceWTValue, (maxWTLength) * sizeof(unsigned short int), hipMemcpyDeviceToHost);
	hipMemcpy(WTRowSum, deviceWTRowSum, (K) * sizeof(int), hipMemcpyDeviceToHost);
}


void WTAll::chunkCPU2GPUCountOffset(int argChunkId) {

	int chunkId = argChunkId;
	
	hipMemcpy(deviceChunkWTCount, WTChunkVec[chunkId].WTCount, (numOfWordS) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(deviceChunkWTOffset, WTChunkVec[chunkId].WTOffset, (numOfWordS) * sizeof(int), hipMemcpyHostToDevice);

}

void WTAll::chunkWTCPU2GPU(int argChunkId) {

	int chunkId = argChunkId;

	hipMemcpy(deviceChunkNZWTCount, WTChunkVec[chunkId].NZWTCount, (numOfWordS) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(deviceChunkWTIndex, WTChunkVec[chunkId].WTIndex, (WTLengthVec[chunkId]) * sizeof(unsigned short int), hipMemcpyHostToDevice);
	hipMemcpy(deviceChunkWTValue, WTChunkVec[chunkId].WTValue, (WTLengthVec[chunkId]) * sizeof(unsigned short int), hipMemcpyHostToDevice);

}

void WTAll::chunkWTGPU2CPU(int argChunkId) {

	int chunkId = argChunkId;

	hipMemcpy(WTChunkVec[chunkId].NZWTCount, deviceChunkNZWTCount, (numOfWordS) * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(WTChunkVec[chunkId].WTIndex, deviceChunkWTIndex, (WTLengthVec[chunkId]) * sizeof(unsigned short int), hipMemcpyDeviceToHost);
	hipMemcpy(WTChunkVec[chunkId].WTValue, deviceChunkWTValue, (WTLengthVec[chunkId]) * sizeof(unsigned short int), hipMemcpyDeviceToHost);

}


void WTAll::CPU2Disk(string argFilePrefix) {

	ofstream OutputNZWTCount((argFilePrefix + string("/NZWTCount.txt")).c_str(), ios::binary);
	for (int i = 0; i < numOfWordS; i++) {
		OutputNZWTCount << NZWTCount[i] << "\n";
	}
	OutputNZWTCount.close();
	ofstream OutputWTIndexValue((argFilePrefix + string("/WTIndexValue.txt")).c_str(), ios::binary);
	for (int i = 0; i < maxWTLength; i++) {
		OutputWTIndexValue << WTIndex[i] << " " << WTValue[i] << "\n";
	}
	OutputWTIndexValue.close();

	ofstream OutputWTRowSum((argFilePrefix + string("/WTRowSum.txt")).c_str(), ios::binary);

	for (int i = 0; i < K; i++) {
		OutputWTRowSum << WTRowSum[i]<< "\n";
	}
	OutputWTRowSum.close();

}


void WTAll::CPU2DiskChunk(string argFilePrefix, int argChunkId) {

	int chunkId = argChunkId;
	string chunkFolderName = argFilePrefix + "/chunk" + to_string(chunkId);

	ofstream OutputNZWTCount((chunkFolderName + string("/NZWTCount.txt")).c_str(), ios::binary);
	for (int i = 0; i < numOfWordS; i++) {
		OutputNZWTCount << WTChunkVec[chunkId].NZWTCount[i] << "\n";
	}
	OutputNZWTCount.close();
	ofstream OutputWTIndexValue((chunkFolderName + string("/WTIndexValue.txt")).c_str(), ios::binary);
	for (int i = 0; i < WTLengthVec[chunkId]; i++) {
		OutputWTIndexValue << WTChunkVec[chunkId].WTIndex[i] << " " << WTChunkVec[chunkId].WTValue[i] << "\n";
	}
	OutputWTIndexValue.close();
}







