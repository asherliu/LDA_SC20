#include "hip/hip_runtime.h"

#ifndef _MAINKERNEL_H_
#define _MAINKERNEL_H_

#include "DTUpdateKernel.cuh"
#include "WTUpdateKernel.cuh"
#include "WTDenUpdateKernel.cuh"
#include "WTAddKernel.cuh"
#include "SamplingKernel.cuh"
#include "WTDense.cuh"
using namespace std;

void fileErrorCheck(ifstream& argFileStream, string folderName)
{
	if (!argFileStream.is_open())
	{
		cout << "File " << folderName << " open failed" << endl;
		exit(0);
	}
}

int main(int argc, char *argv[]) {

	clock_t startTime, startTime1,endTime;
	double transferTimeCPU2GPU=0.0;
	double transferTimeGPU2CPU=0.0;
	double WTTime=0.0;
	double samplingTimeD=0.0;
	double samplingTimeS=0.0;
	double DTTime=0.0;
	double totalTime=0.0;

	int maxTLLength;
	int maxDTLength;
	int maxWTLength;
	int maxDocLength;
	int wordLength;
	int maxChunkWTLength;
	int numOfWordD;
	int numOfWordS;
	int numChunks = 4;
	int numIters = 200;

	
	string chunkFilePrefix ="/gpfs/alpine/proj-shared/csc289/lda/datasets/nytimes";

	ifstream lengthVec((chunkFilePrefix + string("/lengthVec.txt")).c_str(), ios::binary);//store max Doc and DT length

	ofstream timeRecord((chunkFilePrefix + string("/timeRecord.txt")).c_str(), ios::binary);
	ofstream SamplingDRecord((chunkFilePrefix + string("/SamplingDRecord.txt")).c_str(), ios::binary);

	fileErrorCheck(lengthVec, "/lengthVec.txt");

	lengthVec >> maxTLLength >> maxDTLength >> maxWTLength >> maxDocLength >> wordLength>>maxChunkWTLength>> numOfWordD>> numOfWordS;
	lengthVec.close();

	Document document(chunkFilePrefix,numChunks,maxTLLength,maxDocLength,wordLength);

	document.loadDocument();
	document.GPUMemAllocate();
	

	DTChunk chunkDT(maxDTLength,maxDocLength,numChunks);
	chunkDT.loadDocDTLength(chunkFilePrefix);
	chunkDT.CPUMemSet();
	chunkDT.GPUMemAllocate();	
	chunkDT.loadDTCountOffset(chunkFilePrefix);
	WTD WTDen(numOfWordD, wordLength);
	WTDen.GPUMemAllocate();
	WTDen.GPUMemInit();


	
	WTAll WT(maxWTLength, wordLength, numChunks, maxChunkWTLength,numOfWordS);




	WT.CPUMemSet();
	WT.GPUMemAllocate();
	WT.GPUMemset();
	WT.loadWTLength(chunkFilePrefix);
	WT.loadWTCountOffset(chunkFilePrefix);
	WT.blockWarpCountCPU2GPU();
	WT.CPU2GPUCountOffset();
	srand(time(NULL));

	//hiprandState* randState[2];
	//srand(time(NULL));
	//for (int i = 0; i < 2; i++) {
	//	hipSetDevice(i);
	//	hipMalloc(&randState[i], sizeof(hiprandState)*GridDim*BlockDim);//may have bugs
	//}
	//H_ERR(hipDeviceSynchronize());


	hiprandState* randState;

	hipMalloc(&randState, sizeof(hiprandState)*GridDim*BlockDim);
	H_ERR(hipDeviceSynchronize());

	printf("Total memory usage : %f GB\n", document.TLMemory + WT.WTMemory + chunkDT.DTMemory);

	for (int chunkId = 0; chunkId < numChunks; chunkId++) {
		document.CPU2GPU(chunkId);
		WT.chunkCPU2GPUCountOffset(chunkId);
		WT.chunkGPUMemset();
		//--------------update WTDen matrix ---------
		UpdateWTDenKernel(WTDen, WT, document, chunkId);
		//--------------update WTDen matrix-----------

		//--------------update WT matrix--------
		
		//WT.chunkCPU2GPUCountOffset(chunkId);
		//WT.chunkGPUMemset();
		UpdateWTKernel(WT, document,chunkId);
		WT.chunkWTGPU2CPU(chunkId);// marker
		//
		//WT.CPU2DiskChunk(chunkFilePrefix, chunkId);// marker
		/*printf("\n what's this %d\n", chunkId);*/
		//--------------update WT matrix-----------

	}
	/*WTDen.WTDenGPU2CPU();
	WTDen.WTDenCPU2Disk(chunkFilePrefix);*/
	printf("WT ended!\n");

	//WT.CPU2GPUCountOffset();
	startTime = clock();
	for (int iter = 0; iter < numIters; iter++) {


		startTime1=clock();
		//printf("chunk WT updated!\n");
		WT.GPUMemset();
		//--------------update WTDenSum -----------
		UpdateWTDenRowSumKernel(WTDen,WT);
		//--------------update WTDenSum -----------

		//--------------update WTSum -----------
		for (int chunkId = 0; chunkId < numChunks; chunkId++) {
			WT.chunkCPU2GPUCountOffset(chunkId);
			WT.chunkGPUMemset();
			WT.chunkWTCPU2GPU(chunkId);
			WTAdditionKernel(WT, document);
		}
		//--------------update WTSum -----------
		//WT.WTGPU2CPU();// marker
		//WT.CPU2Disk(chunkFilePrefix);// marker






		printf("WT updated!\n");
		endTime = clock();
		WTTime+=(double)(endTime-startTime1)/CLOCKS_PER_SEC;


		document.CPU2GPUPerplexity();
		samplingTimeD=0;
		
		for (int chunkId = 0; chunkId < numChunks; chunkId++) {
			
			// startTime1=clock();
			printf("step: %d\n",chunkId);
			//--------------update DT matrix-----------
			

			startTime1=clock();
			document.CPU2GPU(chunkId);
			endTime = clock();
			transferTimeCPU2GPU+=(double)(endTime-startTime1)/CLOCKS_PER_SEC;

			printf("%d\n", 1);
			chunkDT.GPUMemSet(chunkId);
			printf("%d\n", 2);
			chunkDT.CPU2GPUDTCountOffset(chunkId);
			printf("%d\n", 3);
			//chunkDT.CPU2GPU(chunkId, document.docLengthVec[chunkId]);
			startTime1=clock();
			UpdateDTKernel(chunkDT, document);
			endTime = clock();
			printf("%d\n", 4);
			//chunkDT.GPU2CPU(chunkId);
			//chunkDT.CPU2Disk(chunkFilePrefix, chunkId);// marker
			
			//--------------update DT matrix-----------
			// endTime = clock();
			printf("%d\n", 5);
			DTTime+=(double)(endTime-startTime1)/CLOCKS_PER_SEC;

			

			startTime1=clock();
			//--------------sampling-----------
			printf("%d\n", 6);
			SampleKernelD(WTDen, WT, chunkDT, document, randState);
			endTime = clock();
			samplingTimeD+=(double)(endTime-startTime1)/CLOCKS_PER_SEC;
			

            printf("%d\n", 7);
			//WTDen.WTDenGPU2CPU();// marker
			//WTDen.WTDenCPU2Disk(chunkFilePrefix);// marker
            startTime1=clock();

			SampleKernel(WT, chunkDT, document, randState);
			printf("%d\n", 8);
			endTime = clock();
			//WT.WTGPU2CPU();// marker
			//WT.CPU2Disk(chunkFilePrefix);// marker
		
			//--------------sampling-----------

		//	endTime = clock();
			samplingTimeS+=(double)(endTime-startTime1)/CLOCKS_PER_SEC;

			startTime1=clock();
            document.GPU2CPU(chunkId);
			endTime = clock();
			transferTimeGPU2CPU+=(double)(endTime-startTime1)/CLOCKS_PER_SEC;

			startTime1=clock();
			//--------------update chunkWT matrix-----------
			WT.chunkCPU2GPUCountOffset(chunkId);
			WT.chunkGPUMemset();
			UpdateWTKernel(WT, document, chunkId);
			WT.chunkWTGPU2CPU(chunkId);
			//WT.CPU2DiskChunk(chunkFilePrefix, chunkId);
			//--------------update chunkWT matrix-----------
			endTime = clock();
			WTTime+=(double)(endTime-startTime1)/CLOCKS_PER_SEC;


		}
		WTDen.GPUMemCopy();
		WTDen.GPUMemset();
		PerplexityKernel(document);
		printf("done!!!!!");
		/*document.GPU2CPUPerplexity();*/

		// document.CPU2DiskPerplexity(chunkFilePrefix);

		endTime = clock();
		totalTime=(double)(endTime-startTime)/CLOCKS_PER_SEC;
		timeRecord << WTTime << " " << DTTime << " " << samplingTimeD << " " << samplingTimeS << " " << transferTimeCPU2GPU << " " <<transferTimeGPU2CPU << " " <<totalTime << " " << document.sumPerplexity<< "\n";

		SamplingDRecord << samplingTimeD << "\n";

		printf("WTTime: %f, DTTime: %f, samplingTimeD:%f, samplingTimeS:%f,transferTimeCPU2GPU:%f,transferTimeGPU2CPU:%f,totalTime:%f，sumPerplexity：%f\n",WTTime,DTTime,samplingTimeD,samplingTimeS,transferTimeCPU2GPU,transferTimeGPU2CPU,totalTime, document.sumPerplexity);

	}
	
	timeRecord.close();
}
#endif




//
//
//volatile __shared__ int p_input[ShaMemSize];
//volatile __shared__ int p_index[ShaMemSize];
//volatile __shared__ int p_value[ShaMemSize];
//volatile __shared__ int p_index_tmp[ShaMemSize];
//volatile __shared__ int p_value_tmp[ShaMemSize];
////volatile __shared__ int p_dense[K];
//int tid = threadIdx.x;
//int globalId = threadIdx.x + blockIdx.x * blockDim.x;
//int blockId = blockIdx.x;
//int indicator = 0;
//int GridDim = gridDim.x;
//
///*int wordIdWT = blockId + (*d_counter_0)*GridDim ;*/
///*long long tokenStart = d_TokenOffset[wordId];
//long long tokenEnd = d_TokenOffset[wordId] + d_TokenCount[wordId];*/
//
//
//
//if ((blockId > (*d_token_amount_0 - 1 - *d_counter_0*gridDim.x)) || (d_slotcount[blockId + (*d_counter_0)*GridDim] == 0))
//{
//	return;
//}
//int wordId = blockId + (*d_counter_0)*GridDim;
//p_input[tid] = 0;
//p_index[tid] = 0;
//p_value[tid] = 0;
//p_index_tmp[tid] = 0;
//p_value_tmp[tid] = 0;
//for (int k = tid; k < K; k += blockDim.x)
//{
//	d_dense[k + K*blockId] = 0;
//}
//
//__syncthreads();
//
//for (int i = tid; i < ((d_slotcount[wordId] - 1) / blockDim.x + 1)*blockDim.x; i += blockDim.x) {
//	if (i < d_slotcount[wordId]) {
//		int tmpIndex = d_slotoffset[wordId] + i + numOfTokenD;
//		p_input[tid] = d_a[tmpIndex];
//		//atomicAdd(&d_row_sum[p_input[tid] - 1], 1);
//	}
//
//	__syncthreads();
//	radix_sort(p_input);
//	__syncthreads();
//	index_value_count(p_input, p_index, p_value);
//	__syncthreads();
//	if (((d_slotcount[wordId] - indicator*blockDim.x) < blockDim.x) && (tid<(blockDim.x - 1)))
//	{
//		p_index_tmp[tid] = p_index[tid + 1];
//		p_value_tmp[tid] = p_value[tid + 1];
//	}
//	__syncthreads();
//
//	if (((d_slotcount[wordId] - indicator*blockDim.x) < blockDim.x) && (tid<(blockDim.x - 1)))
//	{
//		p_index[tid] = p_index_tmp[tid];
//		p_value[tid] = p_value_tmp[tid];
//	}
//	__syncthreads();
//
//	if (((d_slotcount[wordId] - indicator*blockDim.x) < blockDim.x) && (tid == (blockDim.x - 1)))
//	{
//		p_index[tid] = 0;
//		p_value[tid] = 0;
//	}
//	__syncthreads();
//	if (p_index[tid])
//	{
//		//atomicAdd(&p_dense[p_index[tid] - 1], 1);
//		d_dense[p_index[tid] - 1 + K*blockId] += p_value[tid];
//	}
//	__syncthreads();
//	p_index[tid] = 0;
//	p_value[tid] = 0;
//	p_input[tid] = 0;
//	p_index_tmp[tid] = 0;
//	p_index_tmp[tid] = 0;
//	indicator++;
//	__syncthreads();
//}
//__syncthreads();
///*if (globalId == 0) printf("%d mark\n", *d_counter_0);
//__syncthreads();*/
//dense_sparse_kernel(d_dense, d_index, d_value, d_count, d_slotcount, d_slotoffset, d_counter_0);
//__syncthreads();
//
//
//








