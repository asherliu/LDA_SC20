#include "hip/hip_runtime.h"
#include "DT.cuh"


DTChunk::DTChunk(int argmaxDTLength, int argMaxDocLength, int argNumChunks) {

	maxDTLength = argmaxDTLength;
	maxDocLength = argMaxDocLength;
	numChunks = argNumChunks;
	NZDTCount = new int[maxDocLength];
	DTIndex = new unsigned short int[maxDTLength];
	DTValue = new int[maxDTLength];
	//DTCount = new int[maxDocLength];
	//DTOffset = new int[maxDocLength];
	DTLengthVec = new int[numChunks];
	docLengthVec = new int[numChunks];
}

void DTChunk::loadDocDTLength(string argFilePrefix) {
	ifstream DTLength((argFilePrefix + string("/DTLength.txt")).c_str(), ios::binary);//store max Doc and DT length
	ifstream docLength((argFilePrefix + string("/docLength.txt")).c_str(), ios::binary);//store max Doc and DT length
	for (int chunkId = 0; chunkId < numChunks; chunkId++) {
		DTLength >> DTLengthVec[chunkId];
		docLength >> docLengthVec[chunkId];
	}
	DTLength.close();
	docLength.close();
}



void DTChunk::CPUMemSet() {

	memset(NZDTCount, 0, maxDocLength * sizeof(int));
	memset(DTIndex, 0, maxDTLength * sizeof(unsigned short int));
	memset(DTValue, 0, maxDTLength * sizeof(int));
	//memset(DTCount, 0, maxDocLength * sizeof(int));
	//memset(DTOffset, 0, maxDocLength * sizeof(int));

}

void DTChunk::GPUMemAllocate() {
	hipMalloc((void**)&deviceNZDTCount, (maxDocLength) * sizeof(int));
	hipMalloc((void**)&deviceDTIndex, (maxDTLength) * sizeof(unsigned short int));
	hipMalloc((void**)&deviceDTValue, (maxDTLength) * sizeof(int));
	hipMalloc((void**)&deviceDTCount, (maxDocLength) * sizeof(int));
	hipMalloc((void**)&deviceDTOffset, (maxDocLength) * sizeof(int));

	DTMemory = (3 * maxDocLength + 2 * maxDTLength) * sizeof(int) / 1000000000.0;
	printf("DT memory usage:%f GB\n", DTMemory);

}

void DTChunk::loadDTCountOffset(string argFilePrefix) {

	/*chunkId = argChunkId;*/
	
	for (int chunkId = 0; chunkId < numChunks; chunkId++) {
		string chunkFolderName = argFilePrefix + "/chunk" + to_string(chunkId);
		ifstream DTCountOffset((chunkFolderName + string("/DTCountOffset.txt")).c_str(), ios::binary);//store Word offset of TL
		int* DTCount = new int[docLengthVec[chunkId]];
		int* DTOffset = new int[docLengthVec[chunkId]];
		memset(DTCount, 0, docLengthVec[chunkId] * sizeof(int));
		memset(DTOffset, 0, docLengthVec[chunkId] * sizeof(int));

		for (int i = 0; i < docLengthVec[chunkId]; i++)
		{
			DTCountOffset >> DTCount[i] >> DTOffset[i];
		}
		DTCountOffset.close();
		DTCountVec.push_back(DTCount);
		DTOffsetVec.push_back(DTOffset);


	}
	


}




void DTChunk::CPU2GPU(int argChunkId) {
	chunkId = argChunkId;
	//docLength = argDocLength;
	hipMemcpy(deviceNZDTCount, NZDTCount, (docLengthVec[chunkId]) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(deviceDTIndex, DTIndex, (DTLengthVec[chunkId]) * sizeof(unsigned short int), hipMemcpyHostToDevice);
	hipMemcpy(deviceDTValue, DTValue, (DTLengthVec[chunkId]) * sizeof(int), hipMemcpyHostToDevice);
	

}


void DTChunk::GPUMemSet(int argChunkId)
{
	chunkId = argChunkId;
	hipMemset(deviceNZDTCount, 0, (maxDocLength) * sizeof(int));
	hipMemset(deviceDTIndex, 0, (maxDTLength) * sizeof(unsigned short int));
	hipMemset(deviceDTValue, 0, (maxDTLength) * sizeof(int));

}






void DTChunk::CPU2GPUDTCountOffset(int argChunkId) {
	chunkId = argChunkId;
	//docLength = argDocLength;

	hipMemcpy(deviceDTCount, DTCountVec[chunkId], (docLengthVec[chunkId]) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(deviceDTOffset, DTOffsetVec[chunkId], (docLengthVec[chunkId]) * sizeof(int), hipMemcpyHostToDevice);

}


void DTChunk::GPU2CPU(int argChunkId) {
	chunkId = argChunkId;
	//docLength = argDocLength;
	hipMemcpy(NZDTCount, deviceNZDTCount, (docLengthVec[chunkId]) * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(DTIndex, deviceDTIndex, (DTLengthVec[chunkId]) * sizeof(unsigned short int), hipMemcpyDeviceToHost);
	hipMemcpy(DTValue, deviceDTValue, (DTLengthVec[chunkId]) * sizeof(int), hipMemcpyDeviceToHost);

}

void DTChunk::CPU2Disk(string argFilePrefix,int argChunkId) {
	chunkId = argChunkId;
	//docLength = argDocLength;
	string chunkFolderName = argFilePrefix + "/chunk" + to_string(chunkId);
	ofstream OutputNZDTCount((chunkFolderName + string("/NZDTCount.txt")).c_str(), ios::binary);
	for (int i = 0; i < docLengthVec[chunkId]; i++) {
		OutputNZDTCount << NZDTCount[i] << "\n";
	}
	OutputNZDTCount.close();
	ofstream OutputDTIndexValue((chunkFolderName + string("/DTIndexValue.txt")).c_str(), ios::binary);
	for (int i = 0; i < DTLengthVec[chunkId]; i++) {
		OutputDTIndexValue << DTIndex[i] <<" "<<DTValue[i]<< "\n";
	}
	OutputDTIndexValue.close();
}