#include "hip/hip_runtime.h"
#include "Doc.cuh"

Document::Document(string argFilePrefix, int argNumChunks, int argMaxTLLength, int argmaxDocLength, int argWordLength) {

	filePrefix = argFilePrefix;
	numChunks = argNumChunks;
	maxTLLength = argMaxTLLength;
	maxDocLength = argmaxDocLength;
	wordLength = argWordLength;
	perplexityMid = new float[GridDim];
	perplexity = new float[maxTLLength];

	
}

void Document::loadDocument() {


	TLLengthVec = new int[numChunks];
	docLengthVec = new int[numChunks];
	numOfTokenVecD = new int[numChunks];
	numOfTokenVecS = new int[numChunks];

	ifstream docLength((filePrefix + string("/docLength.txt")).c_str(), ios::binary);//store max Doc and DT length
	ifstream TLLength((filePrefix + string("/TLLength.txt")).c_str(), ios::binary);
	ifstream TLSplit((filePrefix + string("/TLSplit.txt")).c_str(), ios::binary);

	for (int chunkId = 0; chunkId < numChunks; chunkId++) {

		TLLength >> TLLengthVec[chunkId];
		docLength >> docLengthVec[chunkId];
		TLSplit >> numOfTokenVecD[chunkId] >> numOfTokenVecS[chunkId];
		totalNumOfTokens += TLLengthVec[chunkId];

		DocChunk tmpDocChunk(TLLengthVec[chunkId], docLengthVec[chunkId], wordLength);
		tmpDocChunk.CPUMemSet();
		tmpDocChunk.loadChunk(filePrefix, chunkId);
		docChunkVec.push_back(tmpDocChunk);

	}

	printf("total num of tokens:%f\n", totalNumOfTokens);
	printf("All chunks loaded!");
	docLength.close();
	TLLength.close();

}

void Document::CPU2GPUPerplexity() {

	
	memset(perplexityMid, 0, GridDim * sizeof(float));
	hipMemcpy(devicePerplexityMid, perplexityMid, GridDim* sizeof(float), hipMemcpyHostToDevice);
    /*hipMemset(devicePerplexity,0,maxTLLength*sizeof(float));*/


}


void Document::GPU2CPUPerplexity() {

	hipMemcpy(perplexityMid, devicePerplexityMid, (GridDim) * sizeof(float), hipMemcpyDeviceToHost);

	//hipMemcpy(perplexity, devicePerplexity, maxTLLength*sizeof(float), hipMemcpyDeviceToHost);
	sumPerplexity = 0.0;


	for (int i = 0; i < GridDim; i++) {
		//printf("Perplexity:%f \n", perplexityMid[i]);
		sumPerplexity += perplexityMid[i]/ 467723.0;
	}

	//printf("Parallel Perplexity:%f \n", sumPerplexity);

}

void Document::CPU2DiskPerplexity(string argFilePrefix) {

	ofstream OutPutPerplexity((argFilePrefix + string("/Perplexity.txt")).c_str(), ios::binary);
	for (int i = 0; i < maxTLLength; i++) {
		OutPutPerplexity << perplexity[i] << "\n";
	}
	OutPutPerplexity.close();
}

void Document::GPUMemAllocate() {

	hipMalloc((void**)&deviceTLTopic, (maxTLLength) * sizeof(unsigned short int));
	hipMalloc((void**)&deviceTLDocCount, (maxDocLength) * sizeof(int));
	hipMalloc((void**)&deviceTLDocOffset, (maxDocLength) * sizeof(int));
	hipMalloc((void**)&deviceTLWordCount, (wordLength) * sizeof(int));
	hipMalloc((void**)&deviceTLWordOffset, (wordLength) * sizeof(int));
	hipMalloc((void**)&deviceMapWord2Doc, (maxTLLength) * sizeof(int));
	hipMalloc((void**)&deviceMapDoc2Word, (maxTLLength) * sizeof(int));
	hipMalloc((void**)&devicePerplexity, (maxTLLength) * sizeof(float));
	hipMalloc((void**)&devicePerplexityMid, sizeof(float)*GridDim);
	
	hipMalloc((void **)&d_blockCounter, sizeof(int)*(1));
	hipMalloc((void **)&d_warpCounter, sizeof(int)*(1));
	hipMalloc((void **)&d_dense, sizeof(int)*(GridDim*BlockDim*K/32));
	hipMalloc((void **)&deviceWTHeadDense, sizeof(float)*(GridDim*K));


	TLMemory = ((3 * maxTLLength + 2 * maxDocLength + 2 * wordLength + GridDim*K) * sizeof(int) + (maxTLLength + GridDim*BlockDim / 32 + GridDim*K) * sizeof(float))/ 1000000000.0;

	printf("Token list memory usage:%f GB\n", TLMemory);


}


void Document::CPU2GPU(int argChunkId) {

	hipMemcpy(deviceTLTopic, docChunkVec[argChunkId].TLTopic, (TLLengthVec[argChunkId]) * sizeof(unsigned short int), hipMemcpyHostToDevice);
	hipMemcpy(deviceTLDocCount, docChunkVec[argChunkId].TLDocCount, (docLengthVec[argChunkId]) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(deviceTLDocOffset, docChunkVec[argChunkId].TLDocOffset, (docLengthVec[argChunkId]) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(deviceTLWordCount, docChunkVec[argChunkId].TLWordCount, (wordLength) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(deviceTLWordOffset, docChunkVec[argChunkId].TLWordOffset, (wordLength) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(deviceMapWord2Doc, docChunkVec[argChunkId].mapWord2Doc, (TLLengthVec[argChunkId]) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(deviceMapDoc2Word, docChunkVec[argChunkId].mapDoc2Word, (TLLengthVec[argChunkId]) * sizeof(int), hipMemcpyHostToDevice);


}

void Document::GPU2CPU(int argChunkId) {

	hipMemcpy(docChunkVec[argChunkId].TLTopic, deviceTLTopic, (TLLengthVec[argChunkId]) * sizeof(unsigned short int), hipMemcpyDeviceToHost);


}
