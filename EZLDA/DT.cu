#include "hip/hip_runtime.h"
#include "DT.cuh"


DTChunk::DTChunk(int argmaxDTLength, int argMaxDocLength, int argNumChunks) {

	maxDTLength = argmaxDTLength;
	maxDocLength = argMaxDocLength;
	numChunks = argNumChunks;

	//NZDTCount = new int[maxDocLength];
	//
	//DTIndex = new unsigned short int[maxDTLength];
	//DTValue = new int[maxDTLength];
	////DTCount = new int[maxDocLength];
	////DTOffset = new int[maxDocLength];
	//DTLengthVec = new int[numChunks];
	//docLengthVec = new int[numChunks];
	
//	hipHostMalloc((void**)&NZDTCount, maxDocLength * sizeof(int));
//	hipHostMalloc((void**)&DTIndex, maxDTLength * sizeof(unsigned short int));
//	hipHostMalloc((void**)&DTValue, maxDTLength * sizeof(int));
	hipHostMalloc((void**)&DTLengthVec, numChunks * sizeof(int));
	hipHostMalloc((void**)&docLengthVec, numChunks * sizeof(int));
}

void DTChunk::loadDocDTLength(string argFilePrefix) {
	ifstream DTLength((argFilePrefix + string("/DTLength.txt")).c_str(), ios::binary);//store max Doc and DT length
	ifstream docLength((argFilePrefix + string("/docLength.txt")).c_str(), ios::binary);//store max Doc and DT length
	for (int chunkId = 0; chunkId < numChunks; chunkId++) {
		DTLength >> DTLengthVec[chunkId];
		docLength >> docLengthVec[chunkId];
	}
	DTLength.close();
	docLength.close();
//	memset(NZDTCount, 0, maxDocLength * sizeof(int));
//	memset(DTIndex, 0, maxDTLength * sizeof(unsigned short int));
//	memset(DTValue, 0, maxDTLength * sizeof(int));
}



void DTChunk::CPUMemSet() {

//	memset(NZDTCount, 0, maxDocLength * sizeof(int));
//	memset(DTIndex, 0, maxDTLength * sizeof(unsigned short int));
//	memset(DTValue, 0, maxDTLength * sizeof(int));
	//memset(DTCount, 0, maxDocLength * sizeof(int));
	//memset(DTOffset, 0, maxDocLength * sizeof(int));

}

void DTChunk::GPUMemAllocate() {
	for (int i = 0; i < numStreams; i++) {

		hipMalloc((void**)&deviceNZDTCount[i], (maxDocLength) * sizeof(int));
		/*hipMalloc((void**)&deviceDTIndex[i], (maxDTLength) * sizeof(unsigned short int));
		hipMalloc((void**)&deviceDTValue[i], (maxDTLength) * sizeof(int));*/

		hipMalloc((void**)&deviceDTIndexValue[i], (maxDTLength) * sizeof(int));
		hipMalloc((void**)&deviceDTCount[i], (maxDocLength) * sizeof(int));
		hipMalloc((void**)&deviceDTOffset[i], (maxDocLength) * sizeof(int));

	}
	

	DTMemory = 2 * (3 * maxDocLength + maxDTLength) * sizeof(int) / 1000000000.0;
	printf("DT memory usage:%f GB\n", DTMemory);

}

void DTChunk::loadDTCountOffset(string argFilePrefix) {

	/*chunkId = argChunkId;*/
	
	for (int chunkId = 0; chunkId < numChunks; chunkId++) {
		string chunkFolderName = argFilePrefix + "/chunk" + to_string(chunkId);
		ifstream DTCountOffset((chunkFolderName + string("/DTCountOffset.txt")).c_str(), ios::binary);//store Word offset of TL
		/*int* DTCount = new int[docLengthVec[chunkId]];
		int* DTOffset = new int[docLengthVec[chunkId]];*/
		int* DTCount;
		int* DTOffset;
		hipHostMalloc((void**)&DTCount, docLengthVec[chunkId] * sizeof(int));
		hipHostMalloc((void**)&DTOffset, docLengthVec[chunkId] * sizeof(int));
		memset(DTCount, 0, docLengthVec[chunkId] * sizeof(int));
		memset(DTOffset, 0, docLengthVec[chunkId] * sizeof(int));

		for (int i = 0; i < docLengthVec[chunkId]; i++)
		{
			DTCountOffset >> DTCount[i] >> DTOffset[i];
		}
		DTCountOffset.close();
		DTCountVec.push_back(DTCount);
		DTOffsetVec.push_back(DTOffset);


	}
	


}




//void DTChunk::CPU2GPU(int argChunkId, int argStreamId, hipStream_t& stream) {
//	chunkId = argChunkId;
//	//docLength = argDocLength;
//	hipMemcpy(deviceNZDTCount, NZDTCount, (docLengthVec[chunkId]) * sizeof(int), hipMemcpyHostToDevice);
//	hipMemcpy(deviceDTIndex, DTIndex, (DTLengthVec[chunkId]) * sizeof(unsigned short int), hipMemcpyHostToDevice);
//	hipMemcpy(deviceDTValue, DTValue, (DTLengthVec[chunkId]) * sizeof(int), hipMemcpyHostToDevice);
//	
//
//}


void DTChunk::GPUMemSet(int argChunkId, int argStreamId, hipStream_t& stream)
{
	chunkId = argChunkId;
	hipMemsetAsync(deviceNZDTCount[argStreamId], 0, (maxDocLength) * sizeof(int),stream);
	/*hipMemsetAsync(deviceDTIndex[argStreamId], 0, (maxDTLength) * sizeof(unsigned short int),stream);
	hipMemsetAsync(deviceDTValue[argStreamId], 0, (maxDTLength) * sizeof(int),stream);*/
	hipMemsetAsync(deviceDTIndexValue[argStreamId], 0, (maxDTLength) * sizeof(int), stream);
	

	/*hipMemcpyAsync(deviceNZDTCount[argStreamId], NZDTCount, (maxDocLength) * sizeof(int), hipMemcpyHostToDevice,stream);
	hipMemcpyAsync(deviceDTIndex[argStreamId], DTIndex, (maxDTLength) * sizeof(unsigned short int), hipMemcpyHostToDevice, stream);
	hipMemcpyAsync(deviceDTValue[argStreamId], DTValue, (maxDTLength) * sizeof(int), hipMemcpyHostToDevice,stream);*/
}






void DTChunk::CPU2GPUDTCountOffset(int argChunkId, int argStreamId, hipStream_t& stream) {
	chunkId = argChunkId;
	//docLength = argDocLength;

	hipMemcpyAsync(deviceDTCount[argStreamId], DTCountVec[chunkId], (docLengthVec[chunkId]) * sizeof(int), hipMemcpyHostToDevice,stream);
	hipMemcpyAsync(deviceDTOffset[argStreamId], DTOffsetVec[chunkId], (docLengthVec[chunkId]) * sizeof(int), hipMemcpyHostToDevice,stream);

}


//void DTChunk::GPU2CPU(int argChunkId, int argStreamId, hipStream_t& stream) {
//	chunkId = argChunkId;
//	//docLength = argDocLength;
//	hipMemcpy(NZDTCount, deviceNZDTCount, (docLengthVec[chunkId]) * sizeof(int), hipMemcpyDeviceToHost);
//	hipMemcpy(DTIndex, deviceDTIndex, (DTLengthVec[chunkId]) * sizeof(unsigned short int), hipMemcpyDeviceToHost);
//	hipMemcpy(DTValue, deviceDTValue, (DTLengthVec[chunkId]) * sizeof(int), hipMemcpyDeviceToHost);
//
//}

void DTChunk::CPU2Disk(string argFilePrefix,int argChunkId) {
	chunkId = argChunkId;
	//docLength = argDocLength;
	string chunkFolderName = argFilePrefix + "/chunk" + to_string(chunkId);
	ofstream OutputNZDTCount((chunkFolderName + string("/NZDTCount.txt")).c_str(), ios::binary);
	for (int i = 0; i < docLengthVec[chunkId]; i++) {
		OutputNZDTCount << NZDTCount[i] << "\n";
	}
	OutputNZDTCount.close();
	ofstream OutputDTIndexValue((chunkFolderName + string("/DTIndexValue.txt")).c_str(), ios::binary);
	for (int i = 0; i < DTLengthVec[chunkId]; i++) {
		OutputDTIndexValue << DTIndex[i] <<" "<<DTValue[i]<< "\n";
	}
	OutputDTIndexValue.close();
}
