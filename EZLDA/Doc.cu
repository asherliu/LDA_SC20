#include "hip/hip_runtime.h"
#include "Doc.cuh"

Document::Document(string argFilePrefix, int argNumChunks, int argMaxTLLength, int argmaxDocLength, int argWordLength) {

	filePrefix = argFilePrefix;
	numChunks = argNumChunks;
	maxTLLength = argMaxTLLength;
	maxDocLength = argmaxDocLength;
	wordLength = argWordLength;

	chunksPerStream = numChunks / numStreams;




	//perplexityMid = new float[GridDim];
	hipHostMalloc((void**)&perplexityMid, GridDim * sizeof(float));

	/*perplexity = new float[maxTLLength];*/

	hipHostMalloc((void**)&perplexity, maxTLLength * sizeof(float));

	//perplexityAve = new float[1];

	hipHostMalloc((void**)&perplexityAve, 1 * sizeof(float));

	//effectiveTokenIndex = new int[maxTLLength];

	hipHostMalloc((void**)&effectiveTokenIndex, maxTLLength * sizeof(int));

	//newTokenCount = new int[wordLength];

	hipHostMalloc((void**)&newTokenCount, wordLength * sizeof(int));

	//maxTokenCount = new unsigned short int[maxTLLength];

	hipHostMalloc((void**)&maxTokenCount, maxTLLength * sizeof(unsigned short int));

	//Mflag = new unsigned short int[maxTLLength];

	hipHostMalloc((void**)&Mflag, maxTLLength * sizeof(unsigned short int));
	
}

void Document::loadDocument() {


	/*TLLengthVec = new int[numChunks];
	docLengthVec = new int[numChunks];
	numOfTokenVecD = new int[numChunks];
	numOfTokenVecS = new int[numChunks];
	timeRecord = new float[GridDim*BlockDim/32];*/
	hipHostMalloc((void**)&TLLengthVec, numChunks * sizeof(int));
	hipHostMalloc((void**)&docLengthVec, numChunks * sizeof(int));
	hipHostMalloc((void**)&numOfTokenVecD, numChunks * sizeof(int));
	hipHostMalloc((void**)&numOfTokenVecS, numChunks * sizeof(int));
	hipHostMalloc((void**)&timeRecord, GridDim*BlockDim / 32 * sizeof(float));
	

	ifstream docLength((filePrefix + string("/docLength.txt")).c_str(), ios::binary);//store max Doc and DT length
	ifstream TLLength((filePrefix + string("/TLLength.txt")).c_str(), ios::binary);
	ifstream TLSplit((filePrefix + string("/TLSplit.txt")).c_str(), ios::binary);

	for (int chunkId = 0; chunkId < numChunks; chunkId++) {

		TLLength >> TLLengthVec[chunkId];
		docLength >> docLengthVec[chunkId];
		TLSplit >> numOfTokenVecD[chunkId] >> numOfTokenVecS[chunkId];
		totalNumOfTokens += TLLengthVec[chunkId];


		DocChunk tmpDocChunk(TLLengthVec[chunkId], docLengthVec[chunkId], wordLength);
		tmpDocChunk.CPUMemSet();
		tmpDocChunk.loadChunk(filePrefix, chunkId);
		docChunkVec.push_back(tmpDocChunk);


		//float* tmpProbMaxChunk = new float[TLLengthVec[chunkId]];
		//memset(tmpProbMaxChunk, 0, TLLengthVec[chunkId] * sizeof(float));
		//unsigned short int* tmpProbMaxTopicChunk = new unsigned short int[TLLengthVec[chunkId]];
		//memset(tmpProbMaxTopicChunk, 0, TLLengthVec[chunkId] * sizeof(unsigned short int));

		//unsigned short int* tmpProbMaxFlagChunk = new unsigned short int[TLLengthVec[chunkId]];
		//memset(tmpProbMaxFlagChunk, 0, TLLengthVec[chunkId] * sizeof(unsigned short int));

		//unsigned short int* tmpProbMaxTopicFlagChunk = new unsigned short int[TLLengthVec[chunkId]];
		//memset(tmpProbMaxTopicFlagChunk, 0, TLLengthVec[chunkId] * sizeof(unsigned short int));

		unsigned short int* tmpMaxTokenCount = new unsigned short int[TLLengthVec[chunkId]];
		memset(tmpMaxTokenCount, 0, TLLengthVec[chunkId] * sizeof(unsigned short int));

		//probMaxChunkVec.push_back(tmpProbMaxChunk);
		//probMaxTopicChunkVec.push_back(tmpProbMaxTopicChunk);
		//probMaxFlagChunkVec.push_back(tmpProbMaxFlagChunk);
		//probMaxTopicFlagChunkVec.push_back(tmpProbMaxTopicFlagChunk);
		maxTokenCountVec.push_back(tmpMaxTokenCount);

	}
	memset(effectiveTokenIndex, 0, maxTLLength * sizeof(int));
	memset(newTokenCount, 0, wordLength * sizeof(int));
	memset(maxTokenCount, 0, maxTLLength * sizeof(unsigned short int));
	memset(Mflag, 0, maxTLLength * sizeof(unsigned short int));
	printf("total num of tokens:%f\n", totalNumOfTokens);
	printf("All chunks loaded!");
	docLength.close();
	TLLength.close();

}

void Document::CPU2GPUPerplexity(hipStream_t& stream) {

	
	//memset(perplexityMid, 0, GridDim * sizeof(float));
	for (int i = 0; i < numStreams; i++) {
		hipMemsetAsync(devicePerplexityMid[i], 0, GridDim * sizeof(float), stream);
		//hipMemcpyAsync(devicePerplexityMid[i], perplexityMid, GridDim * sizeof(float), hipMemcpyHostToDevice, stream);
	}
	
 //   /*hipMemset(devicePerplexity,0,maxTLLength*sizeof(float));*/

	/*hipMemsetAsync(devicePerplexityMid, 0, GridDim * sizeof(float), stream);*/

}


void Document::GPU2CPUPerplexity(hipStream_t& stream) {

	hipMemcpyAsync(perplexityMid, devicePerplexityMid, (GridDim) * sizeof(float), hipMemcpyDeviceToHost, stream);

	//hipMemcpy(perplexity, devicePerplexity, maxTLLength*sizeof(float), hipMemcpyDeviceToHost);
	sumPerplexity = 0.0;

	for (int i = 0; i < GridDim; i++) {
		//printf("Perplexity:%f \n", perplexityMid[i]);
		sumPerplexity += perplexityMid[i]/ 467723.0;
	}

	//printf("Parallel Perplexity:%f \n", sumPerplexity);

}

void Document::CPU2DiskPerplexity(string argFilePrefix) {

	ofstream OutPutPerplexity((argFilePrefix + string("/Perplexity.txt")).c_str(), ios::binary);
	for (int i = 0; i < maxTLLength; i++) {
		OutPutPerplexity << perplexity[i] << "\n";
	}
	OutPutPerplexity.close();


}


void Document::GPU2CPUEffectiveTokenIndex() {

	hipMemcpy(effectiveTokenIndex, deviceEffectiveTokenIndex, maxTLLength * sizeof(int), hipMemcpyDeviceToHost);

	hipMemcpy(newTokenCount, deviceNewTokenCount, wordLength * sizeof(int), hipMemcpyDeviceToHost);
}


void Document::CPU2DiskEffectiveTokenIndex(string argFilePrefix) {

	ofstream OutPutEffectiveTokenIndex((argFilePrefix + string("/EffectiveTokenIndex.txt")).c_str(), ios::binary);
	for (int i = 0; i < maxTLLength; i++) {
		OutPutEffectiveTokenIndex << effectiveTokenIndex[i] << "\n";
	}
	OutPutEffectiveTokenIndex.close();

	ofstream OutPutNewTokenCount((argFilePrefix + string("/NewTokenCount.txt")).c_str(), ios::binary);
	for (int i = 0; i < wordLength; i++) {
		OutPutNewTokenCount << newTokenCount[i] << "\n";
	}
	OutPutNewTokenCount.close();


}

void Document::GPUMemAllocate() {

	for (int i = 0; i < numStreams; i++) {
		hipMalloc((void**)&deviceTLTopic[i], (maxTLLength) * sizeof(unsigned short int));
		hipMalloc((void**)&deviceTLDocCount[i], (maxDocLength) * sizeof(int));
		hipMalloc((void**)&deviceTLDocOffset[i], (maxDocLength) * sizeof(int));
		hipMalloc((void**)&deviceTLWordCount[i], (wordLength) * sizeof(int));
		hipMalloc((void**)&deviceTLWordOffset[i], (wordLength) * sizeof(int));
		hipMalloc((void**)&deviceMapWord2Doc[i], (maxTLLength) * sizeof(int));
		hipMalloc((void**)&deviceMapDoc2Word[i], (maxTLLength) * sizeof(int));
		hipMalloc((void**)&deviceRandomfloat[i], (maxTLLength) * sizeof(float));
		/*hipMalloc((void**)&deviceMflag[i], (maxTLLength) * sizeof(unsigned short int));*/

		hipMalloc((void**)&deviceEffectiveTokenIndex[i], (maxTLLength) * sizeof(int));
		hipMalloc((void**)&deviceNewTokenCount[i], (wordLength) * sizeof(int));

		/*hipMalloc((void**)&devicePerplexity[i], (maxTLLength) * sizeof(float));*/
		
	
		hipMalloc((void **)&d_blockCounter[i], sizeof(int)*(1));
		hipMalloc((void **)&d_warpCounter[i], sizeof(int)*(1));
		hipMalloc((void **)&d_dense[i], sizeof(int)*(GridDim*BlockDim*K/32));
		hipMalloc((void **)&deviceWTHeadDense[i], sizeof(float)*(GridDim*K));

	/*
		hipMalloc((void**)&deviceProbMax, (maxTLLength) * sizeof(float));
		hipMalloc((void**)&deviceProbMaxTopic, (maxTLLength) * sizeof(unsigned short int));
		hipMalloc((void**)&deviceProbMaxFlag, (maxTLLength) * sizeof(unsigned short int));
		hipMalloc((void**)&deviceProbMaxTopicFlag, (maxTLLength) * sizeof(unsigned short int));*/

		/*hipMalloc((void**)&deviceMaxTokenCount[i], (maxTLLength) * sizeof(unsigned short int));
		hipMalloc((void**)&deviceMaxTopic[i], (maxTLLength) * sizeof(unsigned short int));

		hipMalloc((void**)&deviceSecondMaxTokenCount[i], (maxTLLength) * sizeof(unsigned short int));
		hipMalloc((void**)&deviceSecondMaxTopic[i], (maxTLLength) * sizeof(unsigned short int));*/

		hipMalloc((void**)&deviceMaxSecTopic[i], (maxTLLength) * sizeof(long long int));

		hipMalloc((void**)&deviceWordMaxTopic[i], (wordLength) * sizeof(unsigned short int));
		hipMalloc((void**)&deviceWordSecondMaxTopic[i], (wordLength) * sizeof(unsigned short int));
		hipMalloc((void**)&deviceWordThirdMaxTopic[i], (wordLength) * sizeof(unsigned short int));

		hipMalloc((void**)&deviceWordMaxProb[i], (wordLength) * sizeof(float));
		hipMalloc((void**)&deviceWordSecondMaxProb[i], (wordLength) * sizeof(float));
		hipMalloc((void**)&deviceWordThirdMaxProb[i], (wordLength) * sizeof(float));

		hipMalloc((void**)&deviceQArray[i], (wordLength) * sizeof(float));

		hipMalloc((void**)&deviceMaxProb[i], (maxTLLength) * sizeof(float));
		hipMalloc((void**)&deviceThresProb[i], (maxTLLength) * sizeof(float));
		hipMalloc((void**)&deviceTimeRecord[i], (GridDim*BlockDim/32) * sizeof(float));

		hipMalloc((void**)&devicePerplexityAve[i], 1 * sizeof(float));
		hipMalloc((void**)&devicePerplexityMid[i], sizeof(float)*GridDim);
		//hipMalloc((void**)&deviceTotalTokenCount[i], (maxTLLength) * sizeof(unsigned short int));


	}
	
	


	TLMemory = (2*(6 * maxTLLength + 2 * maxDocLength + 7 * wordLength) * sizeof(int))/ 1000000000.0;

	printf("Token list memory usage:%f GB\n", TLMemory);


}

void Document::GPU2CPUTime() {

	hipMemcpy(timeRecord, deviceTimeRecord, (GridDim*BlockDim / 32) * sizeof(float), hipMemcpyDeviceToHost);
	hipMemset(deviceTimeRecord, 0, (GridDim*BlockDim / 32) * sizeof(float));

}

//void Document::CPU2DiskTime(ofstream argOutPutTime) {
//
//	for (int i = 0; i < GridDim*BlockDim / 32; i++) {
//		argOutPutTime << timeRecord[i] << " ";
//	}
//	argOutPutTime << "\n";
//}




void Document::CPU2GPU(int argChunkId, int argStreamId, hipStream_t& stream) {

	hipMemcpyAsync(deviceTLTopic[argStreamId], docChunkVec[argChunkId].TLTopic, (TLLengthVec[argChunkId]) * sizeof(unsigned short int), hipMemcpyHostToDevice, stream);
	hipMemcpyAsync(deviceTLDocCount[argStreamId], docChunkVec[argChunkId].TLDocCount, (docLengthVec[argChunkId]) * sizeof(int), hipMemcpyHostToDevice, stream);
	hipMemcpyAsync(deviceTLDocOffset[argStreamId], docChunkVec[argChunkId].TLDocOffset, (docLengthVec[argChunkId]) * sizeof(int), hipMemcpyHostToDevice, stream);
	hipMemcpyAsync(deviceTLWordCount[argStreamId], docChunkVec[argChunkId].TLWordCount, (wordLength) * sizeof(int), hipMemcpyHostToDevice, stream);
	hipMemcpyAsync(deviceTLWordOffset[argStreamId], docChunkVec[argChunkId].TLWordOffset, (wordLength) * sizeof(int), hipMemcpyHostToDevice, stream);
	hipMemcpyAsync(deviceMapWord2Doc[argStreamId], docChunkVec[argChunkId].mapWord2Doc, (TLLengthVec[argChunkId]) * sizeof(int), hipMemcpyHostToDevice, stream);
	//hipMemcpyAsync(deviceMapDoc2Word[argStreamId], docChunkVec[argChunkId].mapDoc2Word, (TLLengthVec[argChunkId]) * sizeof(int), hipMemcpyHostToDevice, stream);
	//hipMemcpyAsync(deviceTotalTokenCount[argStreamId], docChunkVec[argChunkId].totalTokenCount, (TLLengthVec[argChunkId]) * sizeof(unsigned short int), hipMemcpyHostToDevice, stream);

	//hipMemcpy(deviceProbMax, probMaxChunkVec[argChunkId], (TLLengthVec[argChunkId]) * sizeof(float), hipMemcpyHostToDevice);
	//hipMemcpy(deviceProbMaxTopic, probMaxTopicChunkVec[argChunkId], (TLLengthVec[argChunkId]) * sizeof(unsigned short int), hipMemcpyHostToDevice);

	//hipMemcpy(deviceProbMaxTopicFlag, probMaxTopicFlagChunkVec[argChunkId], (TLLengthVec[argChunkId]) * sizeof(unsigned short int), hipMemcpyHostToDevice);
	//hipMemcpy(deviceProbMaxFlag, probMaxFlagChunkVec[argChunkId], (TLLengthVec[argChunkId]) * sizeof(unsigned short int), hipMemcpyHostToDevice);

	/*hipMemcpy(deviceMaxTopic, docChunkVec[argChunkId].TLMaxTopic, (TLLengthVec[argChunkId]) * sizeof(unsigned short int), hipMemcpyHostToDevice);*/

	//hipMemset(deviceProbMaxTopicFlag, 0, (maxTLLength) * sizeof(unsigned short int));
	//hipMemset(deviceProbMaxFlag, 0, (maxTLLength) * sizeof(unsigned short int));
	/*hipMemsetAsync(deviceMaxTokenCount[argStreamId], 0, (maxTLLength) * sizeof(unsigned short int), stream);*/
	hipMemsetAsync(deviceMaxSecTopic[argStreamId], 0, (maxTLLength) * sizeof(long long int), stream);
	/*hipMemsetAsync(deviceMflag[argStreamId], 0, (maxTLLength) * sizeof(unsigned short int), stream);*/


	hipMemsetAsync(deviceEffectiveTokenIndex[argStreamId], 0, (maxTLLength) * sizeof(int), stream);
	hipMemsetAsync(deviceNewTokenCount[argStreamId], 0, (wordLength) * sizeof(int), stream);
	//hipMemsetAsync(deviceTotalTokenCount[argStreamId], 0, (maxTLLength) * sizeof(unsigned short int), stream);

	/*hipMemcpyAsync(deviceMflag[argStreamId], Mflag, (maxTLLength) * sizeof(unsigned short int), hipMemcpyHostToDevice, stream);*/

	/*hipMemcpyAsync(deviceMaxTokenCount[argStreamId], maxTokenCount, (maxTLLength) * sizeof(unsigned short int), hipMemcpyHostToDevice, stream);
	hipMemcpyAsync(deviceEffectiveTokenIndex[argStreamId], effectiveTokenIndex, (maxTLLength) * sizeof(int), hipMemcpyHostToDevice, stream);
	hipMemcpyAsync(deviceNewTokenCount[argStreamId], newTokenCount, (wordLength) * sizeof(int), hipMemcpyHostToDevice, stream);*/



}

void Document::GPU2CPU(int argChunkId, int argStreamId, hipStream_t& stream) {

	hipMemcpyAsync(docChunkVec[argChunkId].TLTopic, deviceTLTopic[argStreamId], (TLLengthVec[argChunkId]) * sizeof(unsigned short int), hipMemcpyDeviceToHost, stream);
	/*hipMemcpy(probMaxTopicFlagChunkVec[argChunkId],deviceProbMaxTopicFlag,  (TLLengthVec[argChunkId]) * sizeof(unsigned short int), hipMemcpyDeviceToHost);
	hipMemcpy( probMaxFlagChunkVec[argChunkId],deviceProbMaxFlag, (TLLengthVec[argChunkId]) * sizeof(unsigned short int), hipMemcpyDeviceToHost);
	hipMemcpy(probMaxChunkVec[argChunkId],deviceProbMax,  (TLLengthVec[argChunkId]) * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(probMaxTopicChunkVec[argChunkId], deviceProbMaxTopic, (TLLengthVec[argChunkId]) * sizeof(unsigned short int), hipMemcpyDeviceToHost);*/
	//hipMemcpyAsync(docChunkVec[argChunkId].TLMaxTopic, deviceMaxTopic[argStreamId], (TLLengthVec[argChunkId]) * sizeof(unsigned short int), hipMemcpyDeviceToHost, stream);





}
//
//void Document::PercentageCalculate()
//{
//	increasePercent = 0.0;
//	topicUnchangedPercent = 0.0;
//	for (int chunkId = 0; chunkId < numChunks; chunkId++) {
//		for (int i = 0; i < TLLengthVec[chunkId]; i++) {
//			increasePercent += float(probMaxFlagChunkVec[chunkId][i]);
//			topicUnchangedPercent += float(probMaxTopicFlagChunkVec[chunkId][i]);
//		}		
//	}
//	printf("increasePercent:%f\n", increasePercent);
//	printf("topicUnchangedPercent:%f\n", topicUnchangedPercent);
//	printf("total num of tokens:%f\n", totalNumOfTokens);
//	increasePercent /= totalNumOfTokens;
//	topicUnchangedPercent /= totalNumOfTokens;
//	
//
//}
//
//
void Document::deviceCounterMemAllocate() {
	for (int i = 0; i < numStreams; i++) {
		hipMalloc((void**)&deviceCounterWTUpdateKernel[i], sizeof(unsigned int));
		hipMalloc((void**)&deviceCounterWTDenUpdateKernel[i], sizeof(unsigned int));
		hipMalloc((void**)&deviceCounterWTAdditionKernel[i], sizeof(unsigned int));
		hipMalloc((void**)&deviceCounterMaxTopicKernel[i], sizeof(unsigned int));
		hipMalloc((void**)&deviceCounterDTUpdateKernel[i], sizeof(unsigned int));
		hipMalloc((void**)&deviceCounterUpdateProbKernel[i], sizeof(unsigned int));
		hipMalloc((void**)&deviceCounterSampleKernelD[i], sizeof(unsigned int));
		hipMalloc((void**)&deviceCounterSampleKernelS[i], sizeof(unsigned int));
	}
	



}
