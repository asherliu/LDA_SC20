#include "WTChunk.cuh"
WTChunkData::WTChunkData(int argChunkId, int argWordLength, int argMaxChunkWTLength,int argWTLength, int argNumOfWordS) {

	chunkId=argChunkId;
	wordLength=argWordLength;
	maxChunkWTLength = argMaxChunkWTLength;
	WTLength = argWTLength;
	numOfWordS = argNumOfWordS;

	//NZWTCount=new int[numOfWordS];

	//WTIndex=new unsigned short int[WTLength];
	//WTValue=new unsigned short int[WTLength];

	//WTCount=new int[numOfWordS];
	//WTOffset= new int[numOfWordS];

	hipHostMalloc((void**)&NZWTCount, numOfWordS * sizeof(int));
	hipHostMalloc((void**)&WTIndex, WTLength * sizeof(unsigned short int));
	hipHostMalloc((void**)&WTValue, WTLength * sizeof(unsigned short int));
	hipHostMalloc((void**)&WTCount, numOfWordS * sizeof(int));
	hipHostMalloc((void**)&WTOffset, numOfWordS * sizeof(int));


}

void WTChunkData::CPUMemSet() {

	memset(NZWTCount, 0, numOfWordS * sizeof(int));
	memset(WTIndex, 0, WTLength * sizeof(unsigned short int));
	memset(WTValue, 0, WTLength * sizeof(unsigned short int));
	memset(WTCount, 0, numOfWordS * sizeof(int));
	memset(WTOffset, 0, numOfWordS * sizeof(int));

}


void WTChunkData::loadWTCountOffset(string argFilePrefix) {

	string chunkFolderName = argFilePrefix + "/chunk" + to_string(chunkId);

	ifstream WTCountOffset((chunkFolderName + string("/WTCountOffset.txt")).c_str(), ios::binary);//store Word offset of TL

	for (int i = 0; i < numOfWordS; i++)
	{
		WTCountOffset >> WTCount[i] >> WTOffset[i];
	}
	WTCountOffset.close();

	printf("WT Count and Offset loaded!...: chunkId=%d\n",chunkId);

}